#include "hip/hip_runtime.h"
#include "lbmInitialization.cuh"


__global__
void gpuInitialization_mom(
    Moments mom)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalarGlobal(x, y, z);
    //printf("tx % d ty % d tz % d  bix %d biy %d biz %d --  x: %d y: %d z: %d idx %d\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, x, y, z, index);
    
    // zeroth moment
    mom.rho[index] = RHO_0;

    //first moments
    dfloat ux,uy,uz;

    ux = 0.0;
    uy = 0.0;
    uz = 0.0;

    mom.ux[index] = ux;
    mom.uy[index] = uy;
    mom.uz[index] = uz;

    //second moments
    //define equilibrium populations
    dfloat feq[Q];
    dfloat meq[3][3] = {0,0,0,0,0,0,0,0,0};
    char c1, c2;
    for (int i = 0; i < Q; i++) {
        feq[i] = gpu_f_eq(w[i] * RHO_0,
            3 * (ux * cx[i] + uy * cy[i] + uz * cz[i]),
            1 - 1.5 * (ux * ux
                + uy * uy
                + uz * uz));
    }
    for (int i = 0; i < Q; i++) {
        for (int d1 = 0; d1 < 3; d1++) {
            if (d1 == 0) { //x
                c1 = cx[i];
            }
            if (d1 == 1) { //y
                c1 = cy[i];
            }
            if (d1 == 2) { //z
                c1 = cz[i];
            }
            if (c1 == 0) {
                continue;
            }
            for (int d2 = 0; d2 < 3; d2++) {
                if (d2 == 0) { //x
                    c2 = cx[i];
                }
                if (d2 == 1) { //y
                    c2 = cy[i];
                }
                if (d2 == 2) { //z
                    c2 = cz[i];
                }
                meq[d1][d2] = feq[i] * c1 * c2;
            }
        }
    }
    mom.pxx[index] = meq[1][1];
    mom.pxy[index] = meq[1][2];
    mom.pxz[index] = meq[1][3];
    mom.pyy[index] = meq[2][2];
    mom.pyz[index] = meq[2][3];
    mom.pzz[index] = meq[3][3];

}

__global__
void gpuInitialization_pop(
    Moments mom,
    Populations pop)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalarGlobal(x, y, z);
    // zeroth moment

    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    unsigned int bz = blockIdx.z;

    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int tz = threadIdx.z;

    dfloat rhoVar, uxVar, uyVar, uzVar;

    rhoVar = mom.rho[index];
    uxVar = mom.ux[index];
    uyVar = mom.uy[index];
    uzVar = mom.uz[index];

    dfloat fNode[Q];

    // Calculate temporary variables
    const dfloat p1_muu15 = 1 - 1.5 * (uxVar * uxVar + uyVar * uyVar + uzVar * uzVar);
    const dfloat rhoW0 = rhoVar * W0;
    const dfloat rhoW1 = rhoVar * W1;
    const dfloat rhoW2 = rhoVar * W2;
    const dfloat W1t3d2 = W1 * 3.0 / 2.0;
    const dfloat W2t3d2 = W2 * 3.0 / 2.0;
    const dfloat W1t9d2 = W1t3d2 * 3.0;
    const dfloat W2t9d2 = W2t3d2 * 3.0;

    #ifdef D3Q27
    const dfloat rhoW3 = rhoVar * W3;
    const dfloat W3t9d2 = W3 * 9 / 2;
    #endif
    const dfloat ux3 = 3 * uxVar;
    const dfloat uy3 = 3 * uyVar;
    const dfloat uz3 = 3 * uzVar;

    // Calculate equilibrium fNode
    fNode[0] = gpu_f_eq(rhoW0, 0, p1_muu15);
    fNode[1] = gpu_f_eq(rhoW1, ux3, p1_muu15);
    fNode[2] = gpu_f_eq(rhoW1, -ux3, p1_muu15);
    fNode[3] = gpu_f_eq(rhoW1, uy3, p1_muu15);
    fNode[4] = gpu_f_eq(rhoW1, -uy3, p1_muu15);
    fNode[5] = gpu_f_eq(rhoW1, uz3, p1_muu15);
    fNode[6] = gpu_f_eq(rhoW1, -uz3, p1_muu15);
    fNode[7] = gpu_f_eq(rhoW2, ux3 + uy3, p1_muu15);
    fNode[8] = gpu_f_eq(rhoW2, -ux3 - uy3, p1_muu15);
    fNode[9] = gpu_f_eq(rhoW2, ux3 + uz3, p1_muu15);
    fNode[10] = gpu_f_eq(rhoW2, -ux3 - uz3, p1_muu15);
    fNode[11] = gpu_f_eq(rhoW2, uy3 + uz3, p1_muu15);
    fNode[12] = gpu_f_eq(rhoW2, -uy3 - uz3, p1_muu15);
    fNode[13] = gpu_f_eq(rhoW2, ux3 - uy3, p1_muu15);
    fNode[14] = gpu_f_eq(rhoW2, -ux3 + uy3, p1_muu15);
    fNode[15] = gpu_f_eq(rhoW2, ux3 - uz3, p1_muu15);
    fNode[16] = gpu_f_eq(rhoW2, -ux3 + uz3, p1_muu15);
    fNode[17] = gpu_f_eq(rhoW2, uy3 - uz3, p1_muu15);
    fNode[18] = gpu_f_eq(rhoW2, -uy3 + uz3, p1_muu15);
    #ifdef D3Q27
    fNode[19] = gpu_f_eq(rhoW3, ux3 + uy3 + uz3, p1_muu15);
    fNode[20] = gpu_f_eq(rhoW3, -ux3 - uy3 - uz3, p1_muu15);
    fNode[21] = gpu_f_eq(rhoW3, ux3 + uy3 - uz3, p1_muu15);
    fNode[22] = gpu_f_eq(rhoW3, -ux3 - uy3 + uz3, p1_muu15);
    fNode[23] = gpu_f_eq(rhoW3, ux3 - uy3 + uz3, p1_muu15);
    fNode[24] = gpu_f_eq(rhoW3, -ux3 + uy3 - uz3, p1_muu15);
    fNode[25] = gpu_f_eq(rhoW3, -ux3 + uy3 + uz3, p1_muu15);
    fNode[26] = gpu_f_eq(rhoW3, ux3 - uy3 - uz3, p1_muu15);
    #endif
 
    if(ty == 0)             {//s
        pop.y[idxPopY(tx,tz,5,(bx+cx[ 4]+BLOCK_NX)%BLOCK_NX,(by+cy[ 4]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 4]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 4];
        pop.y[idxPopY(tx,tz,6,(bx+cx[ 8]+BLOCK_NX)%BLOCK_NX,(by+cy[ 8]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 8]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 8];
        pop.y[idxPopY(tx,tz,7,(bx+cx[12]+BLOCK_NX)%BLOCK_NX,(by+cy[12]+BLOCK_NY)%BLOCK_NY,(bz+cz[12]+BLOCK_NZ)%BLOCK_NZ)] = fNode[12];
        pop.y[idxPopY(tx,tz,8,(bx+cx[13]+BLOCK_NX)%BLOCK_NX,(by+cy[13]+BLOCK_NY)%BLOCK_NY,(bz+cz[13]+BLOCK_NZ)%BLOCK_NZ)] = fNode[13];
        pop.y[idxPopY(tx,tz,9,(bx+cx[18]+BLOCK_NX)%BLOCK_NX,(by+cy[18]+BLOCK_NY)%BLOCK_NY,(bz+cz[18]+BLOCK_NZ)%BLOCK_NZ)] = fNode[18];
    }else if(ty == (BLOCK_NY-1))  {//n
        pop.y[idxPopY(tx,tz,0,(bx+cx[ 3]+BLOCK_NX)%BLOCK_NX,(by+cy[ 3]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 3]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 3];
        pop.y[idxPopY(tx,tz,1,(bx+cx[ 7]+BLOCK_NX)%BLOCK_NX,(by+cy[ 7]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 7]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 7];
        pop.y[idxPopY(tx,tz,2,(bx+cx[11]+BLOCK_NX)%BLOCK_NX,(by+cy[11]+BLOCK_NY)%BLOCK_NY,(bz+cz[11]+BLOCK_NZ)%BLOCK_NZ)] = fNode[11];
        pop.y[idxPopY(tx,tz,3,(bx+cx[14]+BLOCK_NX)%BLOCK_NX,(by+cy[14]+BLOCK_NY)%BLOCK_NY,(bz+cz[14]+BLOCK_NZ)%BLOCK_NZ)] = fNode[14];
        pop.y[idxPopY(tx,tz,4,(bx+cx[17]+BLOCK_NX)%BLOCK_NX,(by+cy[17]+BLOCK_NY)%BLOCK_NY,(bz+cz[17]+BLOCK_NZ)%BLOCK_NZ)] = fNode[17];
    }

    
    if(tx == 0)             {//w
        pop.x[idxPopX(ty,tz,5,(bx+cx[ 2]+BLOCK_NX)%BLOCK_NX,(by+cy[ 2]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 2]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 2];
        pop.x[idxPopX(ty,tz,6,(bx+cx[ 8]+BLOCK_NX)%BLOCK_NX,(by+cy[ 8]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 8]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 8];
        pop.x[idxPopX(ty,tz,7,(bx+cx[10]+BLOCK_NX)%BLOCK_NX,(by+cy[10]+BLOCK_NY)%BLOCK_NY,(bz+cz[10]+BLOCK_NZ)%BLOCK_NZ)] = fNode[10];
        pop.x[idxPopX(ty,tz,8,(bx+cx[14]+BLOCK_NX)%BLOCK_NX,(by+cy[14]+BLOCK_NY)%BLOCK_NY,(bz+cz[14]+BLOCK_NZ)%BLOCK_NZ)] = fNode[14];
        pop.x[idxPopX(ty,tz,9,(bx+cx[16]+BLOCK_NX)%BLOCK_NX,(by+cy[16]+BLOCK_NY)%BLOCK_NY,(bz+cz[16]+BLOCK_NZ)%BLOCK_NZ)] = fNode[16];
    }else if(tx == (BLOCK_NX-1))  {//e
        pop.x[idxPopX(ty,tz,0,(bx+cx[ 1]+BLOCK_NX)%BLOCK_NX,(by+cy[ 1]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 1]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 1];
        pop.x[idxPopX(ty,tz,1,(bx+cx[ 7]+BLOCK_NX)%BLOCK_NX,(by+cy[ 7]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 7]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 7];
        pop.x[idxPopX(ty,tz,2,(bx+cx[ 9]+BLOCK_NX)%BLOCK_NX,(by+cy[ 9]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 9]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 9];
        pop.x[idxPopX(ty,tz,3,(bx+cx[13]+BLOCK_NX)%BLOCK_NX,(by+cy[13]+BLOCK_NY)%BLOCK_NY,(bz+cz[13]+BLOCK_NZ)%BLOCK_NZ)] = fNode[13];
        pop.x[idxPopX(ty,tz,4,(bx+cx[15]+BLOCK_NX)%BLOCK_NX,(by+cy[15]+BLOCK_NY)%BLOCK_NY,(bz+cz[15]+BLOCK_NZ)%BLOCK_NZ)] = fNode[15];
    } 


    if(tz == 0)             {//b
        pop.z[idxPopZ(tx,ty,5,(bx+cx[ 6]+BLOCK_NX)%BLOCK_NX,(by+cy[ 6]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 6]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 6];
        pop.z[idxPopZ(tx,ty,6,(bx+cx[10]+BLOCK_NX)%BLOCK_NX,(by+cy[10]+BLOCK_NY)%BLOCK_NY,(bz+cz[10]+BLOCK_NZ)%BLOCK_NZ)] = fNode[10];
        pop.z[idxPopZ(tx,ty,7,(bx+cx[12]+BLOCK_NX)%BLOCK_NX,(by+cy[12]+BLOCK_NY)%BLOCK_NY,(bz+cz[12]+BLOCK_NZ)%BLOCK_NZ)] = fNode[12];
        pop.z[idxPopZ(tx,ty,8,(bx+cx[15]+BLOCK_NX)%BLOCK_NX,(by+cy[15]+BLOCK_NY)%BLOCK_NY,(bz+cz[15]+BLOCK_NZ)%BLOCK_NZ)] = fNode[15];
        pop.z[idxPopZ(tx,ty,9,(bx+cx[17]+BLOCK_NX)%BLOCK_NX,(by+cy[17]+BLOCK_NY)%BLOCK_NY,(bz+cz[17]+BLOCK_NZ)%BLOCK_NZ)] = fNode[17];
    } else if(tz == (BLOCK_NZ-1))  {//f
        pop.z[idxPopZ(tx,ty,0,(bx+cx[ 5]+BLOCK_NX)%BLOCK_NX,(by+cy[ 5]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 5]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 5];
        pop.z[idxPopZ(tx,ty,1,(bx+cx[ 9]+BLOCK_NX)%BLOCK_NX,(by+cy[ 9]+BLOCK_NY)%BLOCK_NY,(bz+cz[ 9]+BLOCK_NZ)%BLOCK_NZ)] = fNode[ 9];
        pop.z[idxPopZ(tx,ty,2,(bx+cx[11]+BLOCK_NX)%BLOCK_NX,(by+cy[11]+BLOCK_NY)%BLOCK_NY,(bz+cz[11]+BLOCK_NZ)%BLOCK_NZ)] = fNode[11];
        pop.z[idxPopZ(tx,ty,3,(bx+cx[16]+BLOCK_NX)%BLOCK_NX,(by+cy[16]+BLOCK_NY)%BLOCK_NY,(bz+cz[16]+BLOCK_NZ)%BLOCK_NZ)] = fNode[16];
        pop.z[idxPopZ(tx,ty,4,(bx+cx[18]+BLOCK_NX)%BLOCK_NX,(by+cy[18]+BLOCK_NY)%BLOCK_NY,(bz+cz[18]+BLOCK_NZ)%BLOCK_NZ)] = fNode[18];
    }

        
        


}
