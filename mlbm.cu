#include "hip/hip_runtime.h"
#include "mlbm.cuh"

__global__ void gpuMomCollisionStream(
    dfloat *fMom,
    dfloat *fGhostX_0, dfloat *fGhostX_1,
    dfloat *fGhostY_0, dfloat *fGhostY_1,
    dfloat *fGhostZ_0, dfloat *fGhostZ_1,
    dfloat *gGhostX_0, dfloat *gGhostX_1,
    dfloat *gGhostY_0, dfloat *gGhostY_1,
    dfloat *gGhostZ_0, dfloat *gGhostZ_1)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;
    dfloat pop[Q];

    // Load moments from global memory

    //rho'
    dfloat rhoVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 0, blockIdx.x, blockIdx.y, blockIdx.z)];
    //u'
    dfloat uxVar  = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 1, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uyVar  = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 2, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uzVar  = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 3, blockIdx.x, blockIdx.y, blockIdx.z)];
    //m'
    dfloat pixx   = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 4, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixy   = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 5, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixz   = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 6, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat piyy   = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 7, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat piyz   = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 8, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pizz   = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 9, blockIdx.x, blockIdx.y, blockIdx.z)];

    //calculate post collision populations
    #pragma unroll //equation 10
    for (int i = 0; i < Q; i++)
    {
        pop[i] = rhoVar * w[i] * (1 
        + as2 * (uxVar * cx[i] + uyVar * cy[i] + uzVar * cz[i]) 
        + 0.5 * as2 * as2 * (
            pixx * (cx[i] * cx[i] - cs2) + 
            2.0*pixy * (cx[i] * cy[i]) + 
            2.0*pixz * (cx[i] * cz[i]) + 
            piyy * (cy[i] * cy[i] - cs2) + 
            2.0*piyz * (cy[i] * cz[i]) + 
            pizz * (cz[i] * cz[i] - cs2))
        );
    }

    __shared__ dfloat s_pop[BLOCK_LBM_SIZE * (Q - 1)];

    //save populations in shared memory
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  0)] = pop[ 1];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  1)] = pop[ 2];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  2)] = pop[ 3];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  3)] = pop[ 4];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  4)] = pop[ 5];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  5)] = pop[ 6];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  6)] = pop[ 7];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  7)] = pop[ 8];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  8)] = pop[ 9];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  9)] = pop[10];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 10)] = pop[11];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 11)] = pop[12];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 12)] = pop[13];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 13)] = pop[14];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 14)] = pop[15];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 15)] = pop[16];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 16)] = pop[17];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 17)] = pop[18];
    #ifdef D3Q27
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 18)] = pop[19];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 19)] = pop[20];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 20)] = pop[21];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 21)] = pop[22];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 22)] = pop[23];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 23)] = pop[24];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 24)] = pop[25];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 25)] = pop[26];
    #endif //D3Q27

    //sync threads of the block so all populations are saved
    __syncthreads();

    /* pull */

    const unsigned short int xp1 = (threadIdx.x + 1 + BLOCK_NX) % BLOCK_NX;
    const unsigned short int xm1 = (threadIdx.x - 1 + BLOCK_NX) % BLOCK_NX;

    const unsigned short int yp1 = (threadIdx.y + 1 + BLOCK_NY) % BLOCK_NY;
    const unsigned short int ym1 = (threadIdx.y - 1 + BLOCK_NY) % BLOCK_NY;

    const unsigned short int zp1 = (threadIdx.z + 1 + BLOCK_NZ) % BLOCK_NZ;
    const unsigned short int zm1 = (threadIdx.z - 1 + BLOCK_NZ) % BLOCK_NZ;

    pop[ 1] = s_pop[idxPopBlock(xm1, threadIdx.y, threadIdx.z, 0)];
    pop[ 2] = s_pop[idxPopBlock(xp1, threadIdx.y, threadIdx.z, 1)];
    pop[ 3] = s_pop[idxPopBlock(threadIdx.x, ym1, threadIdx.z, 2)];
    pop[ 4] = s_pop[idxPopBlock(threadIdx.x, yp1, threadIdx.z, 3)];
    pop[ 5] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zm1, 4)];
    pop[ 6] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zp1, 5)];
    pop[ 7] = s_pop[idxPopBlock(xm1, ym1, threadIdx.z, 6)];
    pop[ 8] = s_pop[idxPopBlock(xp1, yp1, threadIdx.z, 7)];
    pop[ 9] = s_pop[idxPopBlock(xm1, threadIdx.y, zm1, 8)];
    pop[10] = s_pop[idxPopBlock(xp1, threadIdx.y, zp1, 9)];
    pop[11] = s_pop[idxPopBlock(threadIdx.x, ym1, zm1, 10)];
    pop[12] = s_pop[idxPopBlock(threadIdx.x, yp1, zp1, 11)];
    pop[13] = s_pop[idxPopBlock(xm1, yp1, threadIdx.z, 12)];
    pop[14] = s_pop[idxPopBlock(xp1, ym1, threadIdx.z, 13)];
    pop[15] = s_pop[idxPopBlock(xm1, threadIdx.y, zp1, 14)];
    pop[16] = s_pop[idxPopBlock(xp1, threadIdx.y, zm1, 15)];
    pop[17] = s_pop[idxPopBlock(threadIdx.x, ym1, zp1, 16)];
    pop[18] = s_pop[idxPopBlock(threadIdx.x, yp1, zm1, 17)];
    #ifdef D3Q27
    pop[19] = s_pop[idxPopBlock(xm1, ym1, zm1, 18)];
    pop[20] = s_pop[idxPopBlock(xp1, yp1, zp1, 19)];
    pop[21] = s_pop[idxPopBlock(xm1, ym1, zp1, 20)];
    pop[22] = s_pop[idxPopBlock(xp1, yp1, zm1, 21)];
    pop[23] = s_pop[idxPopBlock(xm1, yp1, zm1, 22)];
    pop[24] = s_pop[idxPopBlock(xp1, ym1, zp1, 23)];
    pop[25] = s_pop[idxPopBlock(xp1, ym1, zm1, 24)];
    pop[26] = s_pop[idxPopBlock(xm1, yp1, zp1, 25)];
    #endif

    /* load pop from global in cover nodes */

    gpuInterfacePullOffset(threadIdx,blockIdx,pop,fGhostX_0, fGhostX_1, fGhostY_0, fGhostY_1, fGhostZ_0, fGhostZ_1);


//calculate streaming moments
    #ifdef D3Q19
        //equation3
        rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18];
        dfloat invRho = 1 / rhoVar;
        //equation4 + force correction
        uxVar = ((pop[ 1] + pop[7] + pop[ 9] + pop[13] + pop[15]) - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16]) + 0.5 * FX) * invRho;
        uyVar = ((pop[ 3] + pop[7] + pop[11] + pop[14] + pop[17]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18]) + 0.5 * FY) * invRho;
        uzVar = ((pop[ 5] + pop[9] + pop[11] + pop[16] + pop[18]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17]) + 0.5 * FZ) * invRho;

        //equation5
        pixx =  (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16] -  cs2) * invRho;
        pixy = ((pop[7] + pop[ 8]) - (pop[13] + pop[14])) * invRho;
        pixz = ((pop[9] + pop[10]) - (pop[15] + pop[16])) * invRho;
        piyy =  (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18] - cs2) * invRho;
        piyz = ((pop[11]+pop[12])-(pop[17]+pop[18])) * invRho;
        pizz =  (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18] - cs2) * invRho;


    #endif
    #ifdef D3Q27
        rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26];
        dfloat invRho = 1 / rhoVar;
        uxVar = ((pop[1] + pop[7] + pop[9] + pop[13] + pop[15] + pop[19] + pop[21] + pop[23] + pop[26])  - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16] + pop[20] + pop[22] + pop[24] + pop[25]) + 0.5 * FX) * invRho;
        uyVar = ((pop[3] + pop[7] + pop[11] + pop[14] + pop[17] + pop[19] + pop[21] + pop[24] + pop[25]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18] + pop[20] + pop[22] + pop[23] + pop[26]) + 0.5 * FY) * invRho;
        uzVar = ((pop[5] + pop[9] + pop[11] + pop[16] + pop[18] + pop[19] + pop[22] + pop[23] + pop[25]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17] + pop[20] + pop[21] + pop[24] + pop[26]) + 0.5 * FZ) * invRho;

        pixx =  (pop[ 1] + pop[ 2] + pop[ 7] + pop[ 8] + pop[ 9] + pop[10]  +  pop[13] + pop[14] + pop[15] + pop[16] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26] - cs2) * invRho;
        pixy = ((pop[ 7] + pop[ 8] + pop[19] + pop[20] + pop[21] + pop[22]) - (pop[13] + pop[14] + pop[23] + pop[24] + pop[25] + pop[26])) * invRho;
        pixz = ((pop[ 9] + pop[10] + pop[19] + pop[20] + pop[23] + pop[24]) - (pop[15] + pop[16] + pop[21] + pop[22] + pop[25] + pop[26])) * invRho;
        piyy =  (pop[ 3] + pop[ 4] + pop[ 7] + pop[ 8] + pop[11] + pop[12]  +  pop[13] + pop[14] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26] - cs2) * invRho;
        piyz = ((pop[11] + pop[12] + pop[19] + pop[20] + pop[25] + pop[26]) - (pop[17] + pop[18] + pop[21] + pop[22] + pop[23] + pop[24]))*invRho;
        pizz =  (pop[ 5] + pop[ 6] + pop[ 9] + pop[10] + pop[11] + pop[12]  +  pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26] - cs2) * invRho;
    #endif

    //NOTE : STREAMING DONE, NOW COLLIDE

    //Collide Moments
    //Equiblibrium momements
    
    //equation 90
    pixx = (T_OMEGA * rhoVar * (pixx) + OMEGA * rhoVar * (uxVar * uxVar) + TT_OMEGA * (FX * uxVar + FX * uxVar))*invRho;
    pixy = (T_OMEGA * rhoVar * (pixy) + OMEGA * rhoVar * (uxVar * uyVar) + TT_OMEGA * (FX * uyVar + FY * uxVar))*invRho;
    pixz = (T_OMEGA * rhoVar * (pixz) + OMEGA * rhoVar * (uxVar * uzVar) + TT_OMEGA * (FX * uzVar + FZ * uxVar))*invRho;
    piyy = (T_OMEGA * rhoVar * (piyy) + OMEGA * rhoVar * (uyVar * uyVar) + TT_OMEGA * (FY * uyVar + FY * uyVar))*invRho;
    piyz = (T_OMEGA * rhoVar * (piyz) + OMEGA * rhoVar * (uyVar * uzVar) + TT_OMEGA * (FY * uzVar + FZ * uyVar))*invRho;
    pizz = (T_OMEGA * rhoVar * (pizz) + OMEGA * rhoVar * (uzVar * uzVar) + TT_OMEGA * (FZ * uzVar + FZ * uzVar))*invRho;

    for (int i = 0; i < Q; i++)
    {
        pop[i] = rhoVar * w[i] * (1 
        + as2 * (uxVar * cx[i] + uyVar * cy[i] + uzVar * cz[i]) 
        + 0.5 * as2 * as2 * (
            pixx * (cx[i] * cx[i] - cs2) + 
            2.0*pixy * (cx[i] * cy[i]) + 
            2.0*pixz * (cx[i] * cz[i]) + 
            piyy * (cy[i] * cy[i] - cs2) + 
            2.0*piyz * (cy[i] * cz[i]) + 
            pizz * (cz[i] * cz[i] - cs2))
        );
    }

    /* write to global mom */

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 0, blockIdx.x, blockIdx.y, blockIdx.z)] = rhoVar;

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 1, blockIdx.x, blockIdx.y, blockIdx.z)] = uxVar;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 2, blockIdx.x, blockIdx.y, blockIdx.z)] = uyVar;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 3, blockIdx.x, blockIdx.y, blockIdx.z)] = uzVar;

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 4, blockIdx.x, blockIdx.y, blockIdx.z)] = pixx;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 5, blockIdx.x, blockIdx.y, blockIdx.z)] = pixy;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 6, blockIdx.x, blockIdx.y, blockIdx.z)] = pixz;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 7, blockIdx.x, blockIdx.y, blockIdx.z)] = piyy;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 8, blockIdx.x, blockIdx.y, blockIdx.z)] = piyz;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 9, blockIdx.x, blockIdx.y, blockIdx.z)] = pizz;

    /* write to global pop */
    gpuInterfacePushCentered(threadIdx, blockIdx, pop, gGhostX_0, gGhostX_1, gGhostY_0, gGhostY_1, gGhostZ_0, gGhostZ_1);

}
