#include "hip/hip_runtime.h"
#include "mlbm.cuh"

__global__ void gpuMomCollisionStream(
    Moments mom,
    Populations pop)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;



    size_t indexNodeLBM = idxScalarGlobal(x,y,z);
    size_t indexBlock = idxBlock(blockIdx.x,blockIdx.y,blockIdx.z);

    dfloat rhoVar = mom.rho[indexNodeLBM];

    dfloat uxVar = mom.ux[indexNodeLBM];
    dfloat uyVar = mom.uy[indexNodeLBM];
    dfloat uzVar = mom.uz[indexNodeLBM];

    dfloat pxxVar = mom.pxx[indexNodeLBM];
    dfloat pxyVar = mom.pxy[indexNodeLBM];
    dfloat pxzVar = mom.pxz[indexNodeLBM];
    dfloat pyyVar = mom.pyy[indexNodeLBM];
    dfloat pyzVar = mom.pyz[indexNodeLBM];
    dfloat pzzVar = mom.pzz[indexNodeLBM];

    dfloat fNodeEq[Q];
    dfloat fNodeNeq[Q];
    dfloat fPop[Q];
    dfloat fStream[Q];

    // CALCULATE EQUILIBRIUM

    // Moments

    dfloat pxx_eq = rhoVar * (uxVar * uxVar + cs2);
    dfloat pxy_eq = rhoVar * (uxVar * uyVar);
    dfloat pxz_eq = rhoVar * (uxVar * uzVar);
    dfloat pyy_eq = rhoVar * (uyVar * uyVar + cs2);
    dfloat pyz_eq = rhoVar * (uyVar * uzVar);
    dfloat pzz_eq = rhoVar * (uzVar * uzVar + cs2);

    // Calculate temporary variables
    const dfloat p1_muu15 = 1 - 1.5 * (uxVar * uxVar + uyVar * uyVar + uzVar * uzVar);
    const dfloat rhoW0 = rhoVar * W0;
    const dfloat rhoW1 = rhoVar * W1;
    const dfloat rhoW2 = rhoVar * W2;
    const dfloat W1t3d2 = W1 * 3.0 / 2.0;
    const dfloat W2t3d2 = W2 * 3.0 / 2.0;
    const dfloat W1t9d2 = W1t3d2 * 3.0;
    const dfloat W2t9d2 = W2t3d2 * 3.0;

    #ifdef D3Q27
    const dfloat rhoW3 = rhoVar * W3;
    const dfloat W3t9d2 = W3 * 9 / 2;
    #endif
    const dfloat ux3 = 3 * uxVar;
    const dfloat uy3 = 3 * uyVar;
    const dfloat uz3 = 3 * uzVar;

    // Calculate equilibrium fNodeEq
    fNodeEq[0 ] = gpu_f_eq(rhoW0, 0, p1_muu15);
    fNodeEq[1 ] = gpu_f_eq(rhoW1, ux3, p1_muu15);
    fNodeEq[2 ] = gpu_f_eq(rhoW1, -ux3, p1_muu15);
    fNodeEq[3 ] = gpu_f_eq(rhoW1, uy3, p1_muu15);
    fNodeEq[4 ] = gpu_f_eq(rhoW1, -uy3, p1_muu15);
    fNodeEq[5 ] = gpu_f_eq(rhoW1, uz3, p1_muu15);
    fNodeEq[6 ] = gpu_f_eq(rhoW1, -uz3, p1_muu15);
    fNodeEq[7 ] = gpu_f_eq(rhoW2, ux3 + uy3, p1_muu15);
    fNodeEq[8 ] = gpu_f_eq(rhoW2, -ux3 - uy3, p1_muu15);
    fNodeEq[9 ] = gpu_f_eq(rhoW2, ux3 + uz3, p1_muu15);
    fNodeEq[10] = gpu_f_eq(rhoW2, -ux3 - uz3, p1_muu15);
    fNodeEq[11] = gpu_f_eq(rhoW2, uy3 + uz3, p1_muu15);
    fNodeEq[12] = gpu_f_eq(rhoW2, -uy3 - uz3, p1_muu15);
    fNodeEq[13] = gpu_f_eq(rhoW2, ux3 - uy3, p1_muu15);
    fNodeEq[14] = gpu_f_eq(rhoW2, -ux3 + uy3, p1_muu15);
    fNodeEq[15] = gpu_f_eq(rhoW2, ux3 - uz3, p1_muu15);
    fNodeEq[16] = gpu_f_eq(rhoW2, -ux3 + uz3, p1_muu15);
    fNodeEq[17] = gpu_f_eq(rhoW2, uy3 - uz3, p1_muu15);
    fNodeEq[18] = gpu_f_eq(rhoW2, -uy3 + uz3, p1_muu15);
    #ifdef D3Q27
    fNodeEq[19] = gpu_f_eq(rhoW3, ux3 + uy3 + uz3, p1_muu15);
    fNodeEq[20] = gpu_f_eq(rhoW3, -ux3 - uy3 - uz3, p1_muu15);
    fNodeEq[21] = gpu_f_eq(rhoW3, ux3 + uy3 - uz3, p1_muu15);
    fNodeEq[22] = gpu_f_eq(rhoW3, -ux3 - uy3 + uz3, p1_muu15);
    fNodeEq[23] = gpu_f_eq(rhoW3, ux3 - uy3 + uz3, p1_muu15);
    fNodeEq[24] = gpu_f_eq(rhoW3, -ux3 + uy3 - uz3, p1_muu15);
    fNodeEq[25] = gpu_f_eq(rhoW3, -ux3 + uy3 + uz3, p1_muu15);
    fNodeEq[26] = gpu_f_eq(rhoW3, ux3 - uy3 - uz3, p1_muu15);
    #endif

    // CALCULATE NON-EQUILIBRIUM POPULATIONS
    #pragma unroll
    for (int i = 0; i < Q; i++){
        fNodeNeq[i] = rhoVar * 1.5 * w[i] * 
                    (((cx[i] * cx[i] - cs2) * (pxxVar - pxx_eq) + //Q-iab*(m_ab - m_ab^eq)
                        2 * (cx[i] * cy[i]) * (pxyVar - pxy_eq) + 
                        2 * (cx[i] * cz[i]) * (pxzVar - pxz_eq) + 
                        (cy[i] * cy[i] - cs2) * (pyyVar - pyy_eq) + 
                        2 * (cy[i] * cz[i]) * (pyzVar - pyz_eq) + 
                        (cz[i] * cz[i] - cs2) * (pzzVar - pzz_eq)) -
                        cs2*(cx[i] * FX + cy[i] * FY + cz[i] * FZ)); //force term
    }

    //CALCULATE COLLISION POPULATIONS
    fPop[0 ] = fNodeEq[0 ] + fNodeNeq[0 ];
    fPop[1 ] = fNodeEq[1 ] + fNodeNeq[1 ];
    fPop[2 ] = fNodeEq[2 ] + fNodeNeq[2 ];
    fPop[3 ] = fNodeEq[3 ] + fNodeNeq[3 ];
    fPop[4 ] = fNodeEq[4 ] + fNodeNeq[4 ];
    fPop[5 ] = fNodeEq[5 ] + fNodeNeq[5 ];
    fPop[6 ] = fNodeEq[6 ] + fNodeNeq[6 ];
    fPop[7 ] = fNodeEq[7 ] + fNodeNeq[7 ];
    fPop[8 ] = fNodeEq[8 ] + fNodeNeq[8 ];
    fPop[9 ] = fNodeEq[9 ] + fNodeNeq[9 ];
    fPop[10] = fNodeEq[10] + fNodeNeq[10];
    fPop[11] = fNodeEq[11] + fNodeNeq[11];
    fPop[12] = fNodeEq[12] + fNodeNeq[12];
    fPop[13] = fNodeEq[13] + fNodeNeq[13];
    fPop[14] = fNodeEq[14] + fNodeNeq[14];
    fPop[15] = fNodeEq[15] + fNodeNeq[15];
    fPop[16] = fNodeEq[16] + fNodeNeq[16];
    fPop[17] = fNodeEq[17] + fNodeNeq[17];
    fPop[18] = fNodeEq[18] + fNodeNeq[18];
    #ifdef D3Q27
    fPop[19] = fNodeEq[19] + fNodeNeq[19];
    fPop[20] = fNodeEq[20] + fNodeNeq[20];
    fPop[21] = fNodeEq[21] + fNodeNeq[21];
    fPop[22] = fNodeEq[22] + fNodeNeq[22];
    fPop[23] = fNodeEq[23] + fNodeNeq[23];
    fPop[24] = fNodeEq[24] + fNodeNeq[24];
    fPop[25] = fNodeEq[25] + fNodeNeq[25];
    fPop[26] = fNodeEq[26] + fNodeNeq[26];
    #endif


    __shared__ dfloat stream_population[BLOCK_LBM_SIZE*Q];

    //save populations in shared memory
    #pragma unroll
    for (int i = 0; i < Q; i++){
        stream_population[idxPopBlock(threadIdx.x,threadIdx.y,threadIdx.z,i)] = fPop[i];
    }


    //sync threads of the block so all populations are saved
    __syncthreads();

    //stream populations from other nodes towards the current node (pull)
    //define directions
    const unsigned short int tx = threadIdx.x;
    const unsigned short int ty = threadIdx.y;
    const unsigned short int tz = threadIdx.z;

    const unsigned short int bx = blockIdx.x;
    const unsigned short int by = blockIdx.y;
    const unsigned short int bz = blockIdx.z;

    // it is added the block size to get the populations from the other side, 
    //it will later be replaced with the populations from the interfarce
    const unsigned short int xp1 = (tx + 1 + BLOCK_NX)%BLOCK_NX;
    const unsigned short int xm1 = (tx - 1 + BLOCK_NX)%BLOCK_NX;

    const unsigned short int yp1 = (ty + 1 + BLOCK_NY)%BLOCK_NY;
    const unsigned short int ym1 = (ty - 1 + BLOCK_NY)%BLOCK_NY;

    const unsigned short int zp1 = (tz + 1 + BLOCK_NZ)%BLOCK_NZ;
    const unsigned short int zm1 = (tz - 1 + BLOCK_NZ)%BLOCK_NZ;

    //fStream[0 ] = stream_population[idxPopBlock(tx ,  ty,  tz,  1)]; // [idxPopBlock(tx ,  ty,  tz,  1)];
    fStream[1 ] = stream_population[idxPopBlock(xm1,  ty,  tz,  1)]; // [idxPopBlock(xp1,  ty,  tz,  1)];
    fStream[2 ] = stream_population[idxPopBlock(xp1,  ty,  tz,  2)]; // [idxPopBlock(xm1,  ty,  tz,  2)];
    fStream[3 ] = stream_population[idxPopBlock(tx,  ym1,  tz,  3)]; // [idxPopBlock(tx,  yp1,  tz,  3)];
    fStream[4 ] = stream_population[idxPopBlock(tx,  yp1,  tz,  4)]; // [idxPopBlock(tx,  ym1,  tz,  4)];
    fStream[5 ] = stream_population[idxPopBlock(tx,   ty, zm1,  5)]; // [idxPopBlock(tx,   ty, zp1,  5)];
    fStream[6 ] = stream_population[idxPopBlock(tx,   ty, zp1,  6)]; // [idxPopBlock(tx,   ty, zm1,  6)];
    fStream[7 ] = stream_population[idxPopBlock(xm1, ym1,  tz,  7)]; // [idxPopBlock(xp1, yp1,  tz,  7)];
    fStream[8 ] = stream_population[idxPopBlock(xp1, yp1,  tz,  8)]; // [idxPopBlock(xm1, ym1,  tz,  8)];
    fStream[9 ] = stream_population[idxPopBlock(xm1,  ty, zm1,  9)]; // [idxPopBlock(xp1,  ty, zp1,  9)];
    fStream[10] = stream_population[idxPopBlock(xp1,  ty, zp1, 10)]; // [idxPopBlock(xm1,  ty, zm1, 10)];
    fStream[11] = stream_population[idxPopBlock(tx,  ym1, zm1, 11)]; // [idxPopBlock(tx,  yp1, zp1, 11)];
    fStream[12] = stream_population[idxPopBlock(tx,  yp1, zp1, 12)]; // [idxPopBlock(tx,  ym1, zm1, 12)];
    fStream[13] = stream_population[idxPopBlock(xm1, yp1,  tz, 13)]; // [idxPopBlock(xp1, ym1,  tz, 13)];
    fStream[14] = stream_population[idxPopBlock(xp1, ym1,  tz, 14)]; // [idxPopBlock(xm1, yp1,  tz, 14)];
    fStream[15] = stream_population[idxPopBlock(xm1,  ty, zp1, 15)]; // [idxPopBlock(xp1,  ty, zm1, 15)];
    fStream[16] = stream_population[idxPopBlock(xp1,  ty, zm1, 16)]; // [idxPopBlock(xm1,  ty, zp1, 16)];
    fStream[17] = stream_population[idxPopBlock(tx,  ym1, zp1, 17)]; // [idxPopBlock(tx,  yp1, zm1, 17)];
    fStream[18] = stream_population[idxPopBlock(tx,  yp1, zm1, 18)]; // [idxPopBlock(tx,  ym1, zp1, 18)];
    #ifdef D3Q27    
    fStream[19] = stream_population[idxPopBlock(xm1, ym1, zm1, 19)]];; // [idxPopBlock(xp1, yp1, zp1, 19)]];
    fStream[20] = stream_population[idxPopBlock(xp1, yp1, zp1, 20)]];; // [idxPopBlock(xm1, ym1, zm1, 20)]];
    fStream[21] = stream_population[idxPopBlock(xm1, ym1, zp1, 21)]];; // [idxPopBlock(xp1, yp1, zm1, 21)]];
    fStream[22] = stream_population[idxPopBlock(xp1, yp1, zm1, 22)]];; // [idxPopBlock(xm1, ym1, zp1, 22)]];
    fStream[23] = stream_population[idxPopBlock(xm1, yp1, zm1, 23)]];; // [idxPopBlock(xp1, ym1, zp1, 23)]];
    fStream[24] = stream_population[idxPopBlock(xp1, ym1, zp1, 24)]];; // [idxPopBlock(xm1, yp1, zm1, 24)]];
    fStream[25] = stream_population[idxPopBlock(xp1, ym1, zm1, 25)]];; // [idxPopBlock(xm1, yp1, zp1, 25)]];
    fStream[26] = stream_population[idxPopBlock(xm1, yp1, zp1, 26)]];; // [idxPopBlock(xp1, ym1, zm1, 26)]];
    #endif

    // load populations from interface nodes

    
    if(tx == 0){ //check if is on west face of the block
        
        fStream[ 1] = pop.x[idxPopX(ty,tz,0,bx,by,bz)];
        fStream[ 7] = pop.x[idxPopX(ty,tz,1,bx,by,bz)];
        fStream[ 9] = pop.x[idxPopX(ty,tz,2,bx,by,bz)];
        fStream[13] = pop.x[idxPopX(ty,tz,3,bx,by,bz)];
        fStream[15] = pop.x[idxPopX(ty,tz,4,bx,by,bz)];
        
    }else if (tx == BLOCK_NX-1){ // check if is on east face

        fStream[ 2] = pop.x[idxPopX(ty,tz,5,bx,by,bz)];
        fStream[ 8] = pop.x[idxPopX(ty,tz,6,bx,by,bz)];
        fStream[10] = pop.x[idxPopX(ty,tz,7,bx,by,bz)];
        fStream[14] = pop.x[idxPopX(ty,tz,8,bx,by,bz)];
        fStream[16] = pop.x[idxPopX(ty,tz,9,bx,by,bz)];
    }
    if(ty == 0){ //check if is on south face of the block

        fStream[ 3] = pop.y[idxPopY(tx,tz,0,bx,by,bz)];
        fStream[ 7] = pop.y[idxPopY(tx,tz,1,bx,by,bz)];
        fStream[11] = pop.y[idxPopY(tx,tz,2,bx,by,bz)];
        fStream[14] = pop.y[idxPopY(tx,tz,3,bx,by,bz)];
        fStream[17] = pop.y[idxPopY(tx,tz,4,bx,by,bz)];

    }else if (ty == BLOCK_NY-1){ // check if is on north face
        
        fStream[ 4] = pop.y[idxPopX(tx,tz,5,bx,by,bz)];
        fStream[ 8] = pop.y[idxPopX(tx,tz,6,bx,by,bz)];
        fStream[12] = pop.y[idxPopX(tx,tz,7,bx,by,bz)];
        fStream[13] = pop.y[idxPopX(tx,tz,8,bx,by,bz)];
        fStream[18] = pop.y[idxPopX(tx,tz,9,bx,by,bz)];
        
    }
    if(tz == 0){ //check if is on back face of the block
    
        fStream[ 5] = pop.z[idxPopZ(tx,ty,0,bx,by,bz)];
        fStream[ 9] = pop.z[idxPopZ(tx,ty,1,bx,by,bz)];
        fStream[11] = pop.z[idxPopZ(tx,ty,2,bx,by,bz)];
        fStream[16] = pop.z[idxPopZ(tx,ty,3,bx,by,bz)];
        fStream[18] = pop.z[idxPopZ(tx,ty,4,bx,by,bz)];
    
    }else if (tz == BLOCK_NZ-1){ // check if is on front face
            
        fStream[ 6] = pop.z[idxPopZ(tx,ty,5,bx,by,bz)];
        fStream[10] = pop.z[idxPopZ(tx,ty,6,bx,by,bz)];
        fStream[12] = pop.z[idxPopZ(tx,ty,7,bx,by,bz)];
        fStream[15] = pop.z[idxPopZ(tx,ty,8,bx,by,bz)];
        fStream[17] = pop.z[idxPopZ(tx,ty,9,bx,by,bz)];
        
    }



    //compute new moments

    #ifdef D3Q19
    rhoVar = fStream[0] + fStream[1] + fStream[2] + fStream[3] + fStream[4] 
        + fStream[5] + fStream[6] + fStream[7] + fStream[8] + fStream[9] + fStream[10] 
        + fStream[11] + fStream[12] + fStream[13] + fStream[14] + fStream[15] + fStream[16] 
        + fStream[17] + fStream[18];
    dfloat invRho = 1/rhoVar;
    uxVar = ((fStream[1] + fStream[7] + fStream[9] + fStream[13] + fStream[15])
        - (fStream[2] + fStream[8] + fStream[10] + fStream[14] + fStream[16]) + 0.5*FX) * invRho;
    uyVar = ((fStream[3] + fStream[7] + fStream[11] + fStream[14] + fStream[17])
        - (fStream[4] + fStream[8] + fStream[12] + fStream[13] + fStream[18]) + 0.5*FY) * invRho;
    uzVar = ((fStream[5] + fStream[9] + fStream[11] + fStream[16] + fStream[18])
        - (fStream[6] + fStream[10] + fStream[12] + fStream[15] + fStream[17]) + 0.5*FZ) * invRho;
    #endif
    #ifdef D3Q27
    rhoVar = fStream[0] + fStream[1] + fStream[2] + fStream[3] + fStream[4] 
        + fStream[5] + fStream[6] + fStream[7] + fStream[8] + fStream[9] + fStream[10] 
        + fStream[11] + fStream[12] + fStream[13] + fStream[14] + fStream[15] + fStream[16] 
        + fStream[17] + fStream[18] + fStream[19] + fStream[20] + fStream[21] + fStream[22]
        + fStream[23] + fStream[24] + fStream[25] + fStream[26];
    const dfloat invRho = 1/rhoVar;
    uxVar = ((fStream[1] + fStream[7] + fStream[9] + fStream[13] + fStream[15]
        + fStream[19] + fStream[21] + fStream[23] + fStream[26]) 
        - (fStream[2] + fStream[8] + fStream[10] + fStream[14] + fStream[16] + fStream[20]
        + fStream[22] + fStream[24] + fStream[25]) + 0.5*fxVar) * invRho;
    uyVar = ((fStream[3] + fStream[7] + fStream[11] + fStream[14] + fStream[17]
        + fStream[19] + fStream[21] + fStream[24] + fStream[25])
        - (fStream[4] + fStream[8] + fStream[12] + fStream[13] + fStream[18] + fStream[20]
        + fStream[22] + fStream[23] + fStream[26]) + 0.5*fyVar) * invRho;
    uzVar = ((fStream[5] + fStream[9] + fStream[11] + fStream[16] + fStream[18]
        + fStream[19] + fStream[22] + fStream[23] + fStream[25])
        - (fStream[6] + fStream[10] + fStream[12] + fStream[15] + fStream[17] + fStream[20]
        + fStream[21] + fStream[24] + fStream[26]) + 0.5*fzVar) * invRho;
    #endif


    //Collide Moments
    //Equiblibrium momements
    dfloat mNodeEq[6];
    mNodeEq[0] = rhoVar * (uxVar * uxVar + cs2);
    mNodeEq[1] = rhoVar * (uxVar * uyVar);
    mNodeEq[2] = rhoVar * (uxVar * uzVar);
    mNodeEq[3] = rhoVar * (uyVar * uyVar + cs2);
    mNodeEq[4] = rhoVar * (uyVar * uzVar);
    mNodeEq[5] = rhoVar * (uzVar * uzVar + cs2);

    pxxVar = pxxVar - OMEGA*(pxxVar- mNodeEq[0]) + TT_OMEGA * (FX*uxVar + FX*uxVar);
    pxyVar = pxyVar - OMEGA*(pxyVar- mNodeEq[1]) + TT_OMEGA * (FX*uyVar + FY*uxVar);
    pxzVar = pxzVar - OMEGA*(pxzVar- mNodeEq[2]) + TT_OMEGA * (FX*uzVar + FZ*uxVar);
    pyyVar = pyyVar - OMEGA*(pyyVar- mNodeEq[3]) + TT_OMEGA * (FY*uyVar + FY*uyVar);
    pyzVar = pyzVar - OMEGA*(pyzVar- mNodeEq[4]) + TT_OMEGA * (FY*uzVar + FZ*uyVar);
    pzzVar = pzzVar - OMEGA*(pzzVar- mNodeEq[5]) + TT_OMEGA * (FZ*uzVar + FZ*uzVar);

    //compute new populations


    //compute macroscopics
    #ifdef D3Q19
    rhoVar = fStream[0] + fStream[1] + fStream[2] + fStream[3] + fStream[4] 
        + fStream[5] + fStream[6] + fStream[7] + fStream[8] + fStream[9] + fStream[10] 
        + fStream[11] + fStream[12] + fStream[13] + fStream[14] + fStream[15] + fStream[16] 
        + fStream[17] + fStream[18];
    invRho = 1/rhoVar;
    uxVar = ((fStream[1] + fStream[7] + fStream[9] + fStream[13] + fStream[15])
        - (fStream[2] + fStream[8] + fStream[10] + fStream[14] + fStream[16]) + 0.5*FX) * invRho;
    uyVar = ((fStream[3] + fStream[7] + fStream[11] + fStream[14] + fStream[17])
        - (fStream[4] + fStream[8] + fStream[12] + fStream[13] + fStream[18]) + 0.5*FY) * invRho;
    uzVar = ((fStream[5] + fStream[9] + fStream[11] + fStream[16] + fStream[18])
        - (fStream[6] + fStream[10] + fStream[12] + fStream[15] + fStream[17]) + 0.5*FZ) * invRho;
    #endif
    #ifdef D3Q27
    rhoVar = fStream[0] + fStream[1] + fStream[2] + fStream[3] + fStream[4] 
        + fStream[5] + fStream[6] + fStream[7] + fStream[8] + fStream[9] + fStream[10] 
        + fStream[11] + fStream[12] + fStream[13] + fStream[14] + fStream[15] + fStream[16] 
        + fStream[17] + fStream[18] + fStream[19] + fStream[20] + fStream[21] + fStream[22]
        + fStream[23] + fStream[24] + fStream[25] + fStream[26];
    const dfloat invRho = 1/rhoVar;
    uxVar = ((fStream[1] + fStream[7] + fStream[9] + fStream[13] + fStream[15]
        + fStream[19] + fStream[21] + fStream[23] + fStream[26]) 
        - (fStream[2] + fStream[8] + fStream[10] + fStream[14] + fStream[16] + fStream[20]
        + fStream[22] + fStream[24] + fStream[25]) + 0.5*fxVar) * invRho;
    uyVar = ((fStream[3] + fStream[7] + fStream[11] + fStream[14] + fStream[17]
        + fStream[19] + fStream[21] + fStream[24] + fStream[25])
        - (fStream[4] + fStream[8] + fStream[12] + fStream[13] + fStream[18] + fStream[20]
        + fStream[22] + fStream[23] + fStream[26]) + 0.5*fyVar) * invRho;
    uzVar = ((fStream[5] + fStream[9] + fStream[11] + fStream[16] + fStream[18]
        + fStream[19] + fStream[22] + fStream[23] + fStream[25])
        - (fStream[6] + fStream[10] + fStream[12] + fStream[15] + fStream[17] + fStream[20]
        + fStream[21] + fStream[24] + fStream[26]) + 0.5*fzVar) * invRho;
    #endif
    //write moments in global memory

    
    mom.rho[indexNodeLBM] = rhoVar;

    mom.ux[indexNodeLBM] =  uxVar;
    mom.uy[indexNodeLBM] =  uyVar;
    mom.uz[indexNodeLBM] =  uzVar;

    mom.pxx[indexNodeLBM] = pxxVar;
    mom.pxy[indexNodeLBM] = pxyVar;
    mom.pxz[indexNodeLBM] = pxzVar;
    mom.pyy[indexNodeLBM] = pyyVar;
    mom.pyz[indexNodeLBM] = pyzVar;
    mom.pzz[indexNodeLBM] = pzzVar;
    
    //write populations of the interface

if(tx == 0){ //check if is on west face of the block
        /*
        fPopWest[idxPopX(ty,tz,1,indexBlock)] = fStream[ 1];
        fPopWest[idxPopX(ty,tz,2,indexBlock)] = fStream[ 7];
        fPopWest[idxPopX(ty,tz,3,indexBlock)] = fStream[ 9];
        fPopWest[idxPopX(ty,tz,4,indexBlock)] = fStream[13];
        fPopWest[idxPopX(ty,tz,5,indexBlock)] = fStream[15];
        */
    }else if (tx == BLOCK_NX-1){ // check if is on east face
        /*
        fPopEast[idxPopX(ty,tz,1,indexBlock)] = fStream[ 2];
        fPopEast[idxPopX(ty,tz,2,indexBlock)] = fStream[ 8];
        fPopEast[idxPopX(ty,tz,3,indexBlock)] = fStream[10];
        fPopEast[idxPopX(ty,tz,4,indexBlock)] = fStream[14];
        fPopEast[idxPopX(ty,tz,5,indexBlock)] = fStream[16];
        */
    }
    if(ty == 0){ //check if is on south face of the block
        /*
        fPopSouth[idxPopY(tx,tz,1,indexBlock)] = fStream[ 3];
        fPopSouth[idxPopY(tx,tz,2,indexBlock)] = fStream[ 7];
        fPopSouth[idxPopY(tx,tz,3,indexBlock)] = fStream[11];
        fPopSouth[idxPopY(tx,tz,4,indexBlock)] = fStream[14];
        fPopSouth[idxPopY(tx,tz,5,indexBlock)] = fStream[17];
        */
    }else if (ty == BLOCK_NY-1){ // check if is on north face
        /*
        fPopNorth[idxPopX(tx,tz,1,indexBlock)] = fStream[ 4];
        fPopNorth[idxPopX(tx,tz,2,indexBlock)] = fStream[ 8];
        fPopNorth[idxPopX(tx,tz,3,indexBlock)] = fStream[12];
        fPopNorth[idxPopX(tx,tz,4,indexBlock)] = fStream[13];
        fPopNorth[idxPopX(tx,tz,5,indexBlock)] = fStream[18];
        */
    }
    if(tz == 0){ //check if is on back face of the block
    /*
        fPopBack[idxPopZ(tx,ty,1,indexBlock)] = fStream[ 5];
        fPopBack[idxPopZ(tx,ty,2,indexBlock)] = fStream[ 9];
        fPopBack[idxPopZ(tx,ty,3,indexBlock)] = fStream[11];
        fPopBack[idxPopZ(tx,ty,4,indexBlock)] = fStream[16];
        fPopBack[idxPopZ(tx,ty,5,indexBlock)] = fStream[18];
    */
    }else if (tz == BLOCK_NZ-1){ // check if is on front face
            /*
        fPopFront[idxPopZ(tx,ty,1,indexBlock)] = fStream[ 6];
        fPopFront[idxPopZ(tx,ty,2,indexBlock)] = fStream[10];
        fPopFront[idxPopZ(tx,ty,3,indexBlock)] = fStream[12];
        fPopFront[idxPopZ(tx,ty,4,indexBlock)] = fStream[15];
        fPopFront[idxPopZ(tx,ty,5,indexBlock)] = fStream[17];
        */
    }
    
    
}
