#include <stdio.h>
#include <stdlib.h>

// CUDA INCLUDE
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// FILES INCLUDES
#include "var.h"
#include "errorDef.h"
//#include "structs.h"
//#include "globalFunctions.h"
#include "lbmInitialization.cuh"
#include "mlbm.cuh"

using namespace std;

int main() {

    dfloat* fMom;
    dfloat* fGhostX_0;
    dfloat* fGhostX_1;
    dfloat* fGhostY_0; 
    dfloat* fGhostY_1;
    dfloat* fGhostZ_0; 
    dfloat* fGhostZ_1;

    /* ------------------------- ALLOCATION FOR CPU ------------------------- */

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);    
    hipMalloc((void**)&fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    


    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    gpuInitialization_mom << <gridBlock, threadBlock >> >(fMom);
    checkCudaErrors(hipDeviceSynchronize());
    gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipSetDevice(0));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    
    size_t step;
    for (step=0; step<N_STEPS;step++){
        gpuMomCollisionStream << <gridBlock, threadBlock >> > (fMom,fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1);
    }
    checkCudaErrors(hipDeviceSynchronize());
    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);
    return 0;
}
