#include <stdio.h>
#include <stdlib.h>

// CUDA INCLUDE
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// FILES INCLUDES
#include "var.h"
#include "errorDef.h"
//#include "structs.h"
//#include "globalFunctions.h"
#include "lbmInitialization.cuh"
#include "mlbm.cuh"
#include "saveData.cuh"

using namespace std;

int main() {

    dfloat* fMom;
    dfloat* fGhostX_0;
    dfloat* fGhostX_1;
    dfloat* fGhostY_0; 
    dfloat* fGhostY_1;
    dfloat* fGhostZ_0; 
    dfloat* fGhostZ_1;



    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    checkCudaErrors(hipHostMalloc((void**)&(h_fMom), MEM_SIZE_MOM));
    checkCudaErrors(hipHostMalloc((void**)&(rho), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(ux), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uy), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uz), MEM_SIZE_SCALAR));


    // Setup saving folder
    folderSetup();

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);    
    hipMalloc((void**)&fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    


    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    gpuInitialization_mom << <gridBlock, threadBlock >> >(fMom);
    checkCudaErrors(hipDeviceSynchronize());
    gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipSetDevice(0));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    
    size_t step = 0;
    bool save = false;
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            linearMacr(h_fMom,rho,ux,uy,uz);
            //saveMacr(rho,ux,uy,uz,step);
    for (step=1; step<N_STEPS;step++){
        save =false;

        if(MACR_SAVE)
            save = !(step % MACR_SAVE);

        gpuMomCollisionStream << <gridBlock, threadBlock >> > (fMom,fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1);
        fflush(stdout);

        //save macroscopics
        if(save){
            //printf("step %d \n",step);
            //printf("------------------------------------------------------------------------\n");
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            linearMacr(h_fMom,rho,ux,uy,uz);
            saveMacr(rho,ux,uy,uz,step);
        }
    }
    checkCudaErrors(hipDeviceSynchronize());
    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);



    /* ------------------------------ FREE ------------------------------ */
    hipFree(fMom);
    hipFree(fGhostX_0);
    hipFree(fGhostX_1);
    hipFree(fGhostY_0);
    hipFree(fGhostY_1);
    hipFree(fGhostZ_0);
    hipFree(fGhostZ_1);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    return 0;



}
