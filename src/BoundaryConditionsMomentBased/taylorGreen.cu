#include "hip/hip_runtime.h"
#include "boundaryCondition.cuh"

__device__ void gpuBoundaryConditionMom(
    dim3 threadIdx, dim3 blockIdx,  dfloat* pop, dfloat& rhoVar, char dNodeType,
    dfloat &uxVar , dfloat &uyVar , dfloat& uzVar , 
    dfloat &pixx  , dfloat &pixy  , dfloat& pixz  , 
    dfloat &piyy  , dfloat &piyz  , dfloat& pizz ){
}

__global__ void gpuInitialization_nodeType(
    char *dNodeType)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    dNodeType[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] = BULK;

}