#include "hip/hip_runtime.h"
#include "boundaryCondition.cuh"


__device__ void gpuBoundaryConditionMom(
    dfloat* pop, dfloat& rhoVar, char dNodeType,
    dfloat &uxVar , dfloat &uyVar , dfloat& uzVar , 
    dfloat &pixx  , dfloat &pixy  , dfloat& pixz  , 
    dfloat &piyy  , dfloat &piyz  , dfloat& pizz ){


    //  BOUNDARY CONDITIONS SET TO LID DRIVEN CAVITY
    // Z - DIRECTION : PERIODIC
    // Y = NY-1 : LID MOVING IN THE +X DIRECTION
    // Y = 0 : BOUNCE BACK WALL
    // X - DIRECTION : BOUNCE BACK WALL



    switch (dNodeType){
        case BULK:
            break;
        //corners
        case SOUTH_WEST_BACK:
            gpuBCMomentSW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case SOUTH_WEST_FRONT:
            gpuBCMomentSW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case SOUTH_EAST_BACK:
            gpuBCMomentSE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case SOUTH_EAST_FRONT:
            gpuBCMomentSE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case NORTH_WEST_BACK://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentNW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case NORTH_WEST_FRONT://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentNW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case NORTH_EAST_BACK://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentNE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case NORTH_EAST_FRONT://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentNE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;

            
        //edges
        case NORTH_WEST://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentNW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case NORTH_EAST://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentNE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case NORTH_FRONT://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentN(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case NORTH_BACK://TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentN(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case SOUTH_EAST:
            gpuBCMomentSE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case SOUTH_WEST:
            gpuBCMomentSW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case SOUTH_FRONT:
            gpuBCMomentS(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case SOUTH_BACK:
            gpuBCMomentS(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case WEST_FRONT:
            gpuBCMomentW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case WEST_BACK:
            gpuBCMomentW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case EAST_FRONT:
            gpuBCMomentE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case EAST_BACK:
            gpuBCMomentE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;


        // face
        case SOUTH:
            gpuBCMomentS(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;

        case NORTH: //TODO: this ones have velocity, need make a way to pass the velocity index
            gpuBCMomentN(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        
        case EAST:
            gpuBCMomentE(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        case WEST:
            gpuBCMomentW(pop,rhoVar,dNodeType,uxVar,uyVar,uzVar,pixx,pixy,pixz,piyy,piyz,pizz);
            break;
        //periodic
        case BACK:
            break;
        case FRONT:
            break;
        default:
            break;
    }
}

__global__ void gpuInitialization_nodeType(
    char *dNodeType)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;
    
    char nodeType;

    if (y == 0){ //S
        nodeType = SOUTH;
    }else if (y == (NY - 1)){ // N
        nodeType = NORTH;
    }else{
        nodeType = BULK;
    }
    
    dNodeType[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] = nodeType;

}