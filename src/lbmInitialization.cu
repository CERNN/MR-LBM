#include "hip/hip_runtime.h"
#include "lbmInitialization.cuh"



__host__
void initializationRandomNumbers(
    float* randomNumbers, int seed)
{
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    checkCurandStatus(hiprandCreateGenerator(&gen,
        HIPRAND_RNG_PSEUDO_DEFAULT));
    
    // Set generator seed
    checkCurandStatus(hiprandSetPseudoRandomGeneratorSeed(gen,
        CURAND_SEED));
    
    // Generate NX*NY*NZ floats on device, using normal distribution
    // with mean=0 and std_dev=NORMAL_STD_DEV
    checkCurandStatus(hiprandGenerateNormal(gen, randomNumbers, NUMBER_LBM_NODES,
        0, CURAND_STD_DEV));

    checkCurandStatus(hiprandDestroyGenerator(gen));
}


__global__ void gpuInitialization_mom(
    dfloat *fMom, float* randomNumbers)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalarGlobal(x, y, z);
    //printf("threadIdx.x % d threadIdx.y % d threadIdx.z % d  bix %d biy %d biz %d --  x: %d y: %d z: %d idx %d\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, x, y, z, index);

    //first moments
    dfloat rho, ux, uy, uz;
    #ifdef NON_NEWTONIAN_FLUID
    dfloat omega;
    #endif

    //Taylor Green
	rho = RHO_0;
	ux = U_0_X;
	uy = U_0_Y;
    uz = U_0_Z;

    //if(y == NY-1 && (( x%(NX-1) != 0 ||z%(NZ-1) != 0)))
    //    ux = U_MAX;

    /*dfloat pert = 0.05;
    int l = idxScalarGlobal(x, y, z);
    int Nt = NUMBER_LBM_NODES;
    
    ux += (U_MAX)*pert*randomNumbers[l + x - Nt*((l + x) / Nt)];
    uy += (U_MAX)*pert*randomNumbers[l + y - Nt*((l + y) / Nt)];
    uz += (U_MAX)*pert*randomNumbers[l + z - Nt*((l + z) / Nt)];*/

    #ifdef NON_NEWTONIAN_FLUID
    omega = OMEGA;
    #endif

/*    
	rho = RHO_0 + (1.0/(16.0*cs2))*RHO_0*U_MAX*U_MAX*(cos((dfloat)2.0*(x) / L) + cos((dfloat)2.0*(y) / L))*(cos((dfloat)2.0*(z) / L) + 2.0);
	ux =   U_MAX * sin((dfloat)(x) / L) * cos((dfloat)(y) / L) * cos((dfloat)(z) / L);
	uy = - U_MAX * cos((dfloat)(x) / L) * sin((dfloat)(y) / L) * cos((dfloat)(z) / L);
    uz = 0.0;
*/    

    /*
    // Example of usage of random numbers for turbulence in parallel plates flow in z  
        dfloat y_visc = 6.59, ub_f = 15.6, uc_f = 18.2;
        // logaritimic velocity profile
        dfloat uz_log; 
        dfloat pos = (y < NY/2 ? y + 0.5 : NY - (y + 0.5));
        uz_log = -(uc_f*U_TAU)*(((pos-NY/2)/del)*((pos-NY/2)/del)) + (uc_f*U_TAU);
        
        uz = uz_log;
        ux = 0.0;
        uy = 0.0;
        rho = RHO_0;


        // perturbation
        dfloat pert = 0.1;
        int l = idxScalarGlobal(x, y, z);
        int Nt = NUMBER_LBM_NODES;
        uz += (ub_f*U_TAU)*pert*randomNumbers[l + z - Nt*((l + z) / Nt)];
        ux += (ub_f*U_TAU)*pert*randomNumbers[l + x - Nt*((l + x) / Nt)];
        uy += (ub_f*U_TAU)*pert*randomNumbers[l + y - Nt*((l + y) / Nt)];
    */   

    
    // zeroth moment
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_RHO_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = rho-RHO_0;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_UX_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = ux;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_UY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = uy;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_UZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = uz;

    //second moments
    //define equilibrium populations
    dfloat pop[Q];
    for (int i = 0; i < Q; i++)
    {
        pop[i] = gpu_f_eq(w[i] * RHO_0,
                          3 * (ux * cx[i] + uy * cy[i] + uz * cz[i]),
                          1 - 1.5 * (ux * ux + uy * uy + uz * uz));
    }
    
    dfloat invRho = 1.0/rho;
    dfloat pixx =  (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16]) * invRho - cs2;
    dfloat pixy = ((pop[7] + pop[ 8]) - (pop[13] + pop[14])) * invRho;
    dfloat pixz = ((pop[9] + pop[10]) - (pop[15] + pop[16])) * invRho;
    dfloat piyy =  (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18]) * invRho - cs2;
    dfloat piyz = ((pop[11]+pop[12])-(pop[17]+pop[18])) * invRho;
    dfloat pizz =  (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18]) * invRho - cs2;

    //pixx = pixx + OMEGA * (RHO_0 * ux * ux -  pixx)  + TT_OMEGA * (FX * ux + FX * ux);
    //pixy = pixy + OMEGA * (RHO_0 * ux * uy -  pixy)  + TT_OMEGA * (FX * uy + FY * ux);
    //pixz = pixz + OMEGA * (RHO_0 * ux * uz -  pixz)  + TT_OMEGA * (FX * uz + FZ * ux);
    //piyy = piyy + OMEGA * (RHO_0 * uy * uy -  piyy)  + TT_OMEGA * (FY * uy + FY * uy);
    //piyz = piyz + OMEGA * (RHO_0 * uy * uz -  piyz)  + TT_OMEGA * (FY * uz + FZ * uy);
    //pizz = pizz + OMEGA * (RHO_0 * uz * uz -  pizz)  + TT_OMEGA * (FZ * uz + FZ * uz);

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXX_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = pixx; //= RHO_0*ux*ux+RHO_0*cs2;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = pixy; //= RHO_0*ux*uy;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = pixz; //= RHO_0*ux*uz;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MYY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = piyy; //= RHO_0*uy*uy+RHO_0*cs2;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MYZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = piyz; //= RHO_0*uy*uz;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MZZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = pizz; //= RHO_0*uz*uz+RHO_0*cs2;

    #ifdef NON_NEWTONIAN_FLUID
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_OMEGA_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = omega;
    #endif   
}

__global__ void gpuInitialization_pop(
    dfloat *fMom,
    dfloat *fGhostX_0, dfloat *fGhostX_1,
    dfloat *fGhostY_0, dfloat *fGhostY_1,
    dfloat *fGhostZ_0, dfloat *fGhostZ_1)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalarGlobal(x, y, z);
    // zeroth moment

    dfloat rhoVar = RHO_0 + fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_RHO_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uxVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_UX_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uyVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_UY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uzVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_UZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixx = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXX_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixy = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixz = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat piyy = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MYY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat piyz = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MYZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pizz = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MZZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];

    dfloat pop[Q];
    #pragma unroll //equation 6
    for (int i = 0; i < Q; i++)
    {
        pop[i] = rhoVar * w[i] * (1 
        + as2 * (uxVar * cx[i] + uyVar * cy[i] + uzVar * cz[i]) 
        + 0.5 * as2 * as2 * (
            pixx * (cx[i] * cx[i] - cs2) + 
            2.0*pixy * (cx[i] * cy[i]) + 
            2.0*pixz * (cx[i] * cz[i]) + 
            piyy * (cy[i] * cy[i] - cs2) + 
            2.0*piyz * (cy[i] * cz[i]) + 
            pizz * (cz[i] * cz[i] - cs2))
        );
    }

    
    //thread xyz
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    
    //block xyz
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    if (threadIdx.x == 0) { //w
        fGhostX_0[idxPopX(ty, tz, 0, bx, by, bz)] = pop[ 2]; 
        fGhostX_0[idxPopX(ty, tz, 1, bx, by, bz)] = pop[ 8];
        fGhostX_0[idxPopX(ty, tz, 2, bx, by, bz)] = pop[10];
        fGhostX_0[idxPopX(ty, tz, 3, bx, by, bz)] = pop[14];
        fGhostX_0[idxPopX(ty, tz, 4, bx, by, bz)] = pop[16];
        #ifdef D3Q27                                                                                                           
        fGhostX_0[idxPopX(ty, tz, 5, bx, by, bz)] = pop[20];
        fGhostX_0[idxPopX(ty, tz, 6, bx, by, bz)] = pop[22];
        fGhostX_0[idxPopX(ty, tz, 7, bx, by, bz)] = pop[24];
        fGhostX_0[idxPopX(ty, tz, 8, bx, by, bz)] = pop[25];
        #endif //D3Q27                                                                                                           
    }else if (threadIdx.x == (BLOCK_NX - 1)){                                                                                                                                                                               
        fGhostX_1[idxPopX(ty, tz, 0, bx, by, bz)] = pop[ 1];
        fGhostX_1[idxPopX(ty, tz, 1, bx, by, bz)] = pop[ 7];
        fGhostX_1[idxPopX(ty, tz, 2, bx, by, bz)] = pop[ 9];
        fGhostX_1[idxPopX(ty, tz, 3, bx, by, bz)] = pop[13];
        fGhostX_1[idxPopX(ty, tz, 4, bx, by, bz)] = pop[15];
        #ifdef D3Q27                                                                                                           
        fGhostX_1[idxPopX(ty, tz, 5, bx, by, bz)] = pop[19];
        fGhostX_1[idxPopX(ty, tz, 6, bx, by, bz)] = pop[21];
        fGhostX_1[idxPopX(ty, tz, 7, bx, by, bz)] = pop[23];
        fGhostX_1[idxPopX(ty, tz, 8, bx, by, bz)] = pop[26];
        #endif //D3Q27       
    }

    if (threadIdx.y == 0)  { //s                                                                                                                                                                                        
        fGhostY_0[idxPopY(tx, tz, 0, bx, by, bz)] = pop[ 4];
        fGhostY_0[idxPopY(tx, tz, 1, bx, by, bz)] = pop[ 8];
        fGhostY_0[idxPopY(tx, tz, 2, bx, by, bz)] = pop[12];
        fGhostY_0[idxPopY(tx, tz, 3, bx, by, bz)] = pop[13];
        fGhostY_0[idxPopY(tx, tz, 4, bx, by, bz)] = pop[18];
        #ifdef D3Q27                                                                                                           
        fGhostY_0[idxPopY(tx, tz, 5, bx, by, bz)] = pop[20];
        fGhostY_0[idxPopY(tx, tz, 6, bx, by, bz)] = pop[22];
        fGhostY_0[idxPopY(tx, tz, 7, bx, by, bz)] = pop[23];
        fGhostY_0[idxPopY(tx, tz, 8, bx, by, bz)] = pop[26];
        #endif //D3Q27                                                                                                           
    }else if (threadIdx.y == (BLOCK_NY - 1)){                                                                                                                                                                        
        fGhostY_1[idxPopY(tx, tz, 0, bx, by, bz)] = pop[ 3];
        fGhostY_1[idxPopY(tx, tz, 1, bx, by, bz)] = pop[ 7];
        fGhostY_1[idxPopY(tx, tz, 2, bx, by, bz)] = pop[11];
        fGhostY_1[idxPopY(tx, tz, 3, bx, by, bz)] = pop[14];
        fGhostY_1[idxPopY(tx, tz, 4, bx, by, bz)] = pop[17];
        #ifdef D3Q27                                                                                                           
        fGhostY_1[idxPopY(tx, tz, 5, bx, by, bz)] = pop[19];
        fGhostY_1[idxPopY(tx, tz, 6, bx, by, bz)] = pop[21];
        fGhostY_1[idxPopY(tx, tz, 7, bx, by, bz)] = pop[24];
        fGhostY_1[idxPopY(tx, tz, 8, bx, by, bz)] = pop[25];
        #endif //D3Q27                                                                                                           
    }
    
    if (threadIdx.z == 0){ //b                                                                                                                                                                                     
        fGhostZ_0[idxPopZ(tx, ty, 0, bx, by, bz)] = pop[ 6];
        fGhostZ_0[idxPopZ(tx, ty, 1, bx, by, bz)] = pop[10];
        fGhostZ_0[idxPopZ(tx, ty, 2, bx, by, bz)] = pop[12];
        fGhostZ_0[idxPopZ(tx, ty, 3, bx, by, bz)] = pop[15];
        fGhostZ_0[idxPopZ(tx, ty, 4, bx, by, bz)] = pop[17];
        #ifdef D3Q27                                                                                                           
        fGhostZ_0[idxPopZ(tx, ty, 5, bx, by, bz)] = pop[20];
        fGhostZ_0[idxPopZ(tx, ty, 6, bx, by, bz)] = pop[21];
        fGhostZ_0[idxPopZ(tx, ty, 7, bx, by, bz)] = pop[24];
        fGhostZ_0[idxPopZ(tx, ty, 8, bx, by, bz)] = pop[26];
        #endif //D3Q27                                                                                                           
    }else if (threadIdx.z == (BLOCK_NZ - 1)){                                                                                                               
        fGhostZ_1[idxPopZ(tx, ty, 0, bx, by, bz)] = pop[ 5];
        fGhostZ_1[idxPopZ(tx, ty, 1, bx, by, bz)] = pop[ 9];
        fGhostZ_1[idxPopZ(tx, ty, 2, bx, by, bz)] = pop[11];
        fGhostZ_1[idxPopZ(tx, ty, 3, bx, by, bz)] = pop[16];
        fGhostZ_1[idxPopZ(tx, ty, 4, bx, by, bz)] = pop[18];
        #ifdef D3Q27                                                                                                           
        fGhostZ_1[idxPopZ(tx, ty, 5, bx, by, bz)] = pop[19];
        fGhostZ_1[idxPopZ(tx, ty, 6, bx, by, bz)] = pop[22];
        fGhostZ_1[idxPopZ(tx, ty, 7, bx, by, bz)] = pop[23];
        fGhostZ_1[idxPopZ(tx, ty, 8, bx, by, bz)] = pop[25];
        #endif //D3Q27                                                                                                                                                                                                                    
    }
}


__global__ void gpuInitialization_nodeType(
    unsigned int *dNodeType)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;
    
    unsigned int nodeType;

    #include BC_INIT_PATH

    dNodeType[idxScalarBlock(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] = nodeType;
}


__global__ void gpuInitialization_force(
    dfloat *d_BC_Fx, dfloat* d_BC_Fy, dfloat* d_BC_Fz)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    size_t index = idxScalarGlobal(x, y, z);

    d_BC_Fx[idxScalarBlock(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] = 0.0;
    d_BC_Fy[idxScalarBlock(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] = 0.0;
    d_BC_Fz[idxScalarBlock(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] = 0.0; 
}


__host__ void hostInitialization_nodeType(
    unsigned int *hNodeType)
{
    int x,y,z;
    unsigned int nodeType;

    for (x = 0; x<NX;x++){
        for (y = 0; y<NY;y++){
            for (z = 0; z<NZ_TOTAL;z++){
                #include BC_INIT_PATH
                hNodeType[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = (unsigned int)nodeType;
            }
        }
    }

    printf("bulk boundary condition done\n");
}


void read_xyz_file(
    const std::string& filename,
    unsigned int* dNodeType
) {
    std::ifstream csv_file(filename);
    if (!csv_file)
    {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }

    int x, y, z;
    int value;
    size_t index, index_n;

    int xi, yi, zi;

    std::string line;
    while (std::getline(csv_file, line)) {
        std::stringstream ss(line);
        std::string field;

        std::getline(ss, field, ',');
        x = std::stol(field);

        std::getline(ss, field, ',');
        y = std::stol(field);

        std::getline(ss, field, ',');
        z = std::stol(field);


        index = idxScalarBlock(x % BLOCK_NX, y % BLOCK_NY, z % BLOCK_NZ, x / BLOCK_NX, y / BLOCK_NY, z / BLOCK_NZ);
        //printf("x %d y %d z %d \n",x,y,z); fflush(stdout);
        dNodeType[idxScalarBlock(x % BLOCK_NX, y % BLOCK_NY, z % BLOCK_NZ, x / BLOCK_NX, y / BLOCK_NY, z / BLOCK_NZ)] = SOLID_NODE;


        //set neighborings to be BC
        for (int xn = -1; xn < 2; xn++) {
            for (int yn = -1; yn < 2; yn++) {
                for (int zn = -1; zn < 2; zn++) {

                    xi = (x + xn + NX) % NX;
                    yi = (y + yn + NY) % NY;
                    zi = (z + zn + NZ) % NZ;


                    index_n = idxScalarBlock(xi% BLOCK_NX, yi % BLOCK_NY, zi % BLOCK_NZ, xi / BLOCK_NX, yi / BLOCK_NY, zi / BLOCK_NZ);

                    if ((index_n == index) || dNodeType[index_n] == 255) // check if is the center of the cuboid or if is already a solid node
                        continue;
                    else //set flag to max int 
                        dNodeType[index_n] = MISSING_DEFINITION;
                }
            }
        }
    }
    csv_file.close();
    printf("voxels imported \n");
}

/*
void define_voxel_bc(
    unsigned int *dNodeType
){
    for(int x= 0;x<NX;x++){
        for(int y =0; y<NY;y++){
            for(int z =0; z<NZ_TOTAL;z++){
                unsigned int index = idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ);
                if(dNodeType[index] == MISSING_DEFINITION){
                    dNodeType[index] = bc_id(dNodeType,x,y,z);
                }
            }
        }
    }
}
*/

__global__ 
void define_voxel_bc(
    unsigned int *dNodeType
){
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;

    unsigned int index = idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ);
    if(dNodeType[index] == MISSING_DEFINITION){
        dNodeType[index] = bc_id(dNodeType,x,y,z);
    }
}



/*
Note: Due to the way the BC are set up, it possible when setting a solid node to also set the bit flags of neighboring nodes
However if attempt to perform in device, need to pay attention of two solid nodes setting the same flag at same time 
*/
__host__ __device__
unsigned int bc_id(unsigned int *dNodeType, int x, int y, int z){

    unsigned int bc_d = BULK;

    int xp1 = (x+1+NX)%NX;
    int xm1 = (x-1+NX)%NX;
    int yp1 = (y+1+NY)%NY;
    int ym1 = (y-1+NY)%NY;
    int zp1 = (z+1+NZ)%NZ;
    int zm1 = (z-1+NZ)%NZ;

    // 1
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, xp1/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 1);
        bc_d |= (1 << 3);
        bc_d |= (1 << 5);
        bc_d |= (1 << 7);
    }
     // 2
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, xm1/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 0);
        bc_d |= (1 << 2);
        bc_d |= (1 << 4);
        bc_d |= (1 << 6);
    }
    // 3
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, yp1%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, yp1/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 2);
        bc_d |= (1 << 3);
        bc_d |= (1 << 6);
        bc_d |= (1 << 7);
    }
    // 4
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, ym1%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, ym1/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 0);
        bc_d |= (1 << 1);
        bc_d |= (1 << 4);
        bc_d |= (1 << 5);
    }
    // 5
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, zp1%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 4);
        bc_d |= (1 << 5);
        bc_d |= (1 << 6);
        bc_d |= (1 << 7);
    }
    // 6
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, zm1%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 0);
        bc_d |= (1 << 1);
        bc_d |= (1 << 2);
        bc_d |= (1 << 3);
    }
    // 7
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, yp1%BLOCK_NY, z%BLOCK_NZ, xp1/BLOCK_NX, yp1/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 3);
        bc_d |= (1 << 7);
    }
    // 8
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, ym1%BLOCK_NY, z%BLOCK_NZ, xm1/BLOCK_NX, ym1/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 0);
        bc_d |= (1 << 4);
    }
    // 9
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, y%BLOCK_NY, zp1%BLOCK_NZ, xp1/BLOCK_NX, y/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 5);
        bc_d |= (1 << 7);
    }
    // 10
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, y%BLOCK_NY, zm1%BLOCK_NZ, xm1/BLOCK_NX, y/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 0);
        bc_d |= (1 << 2);
    }
    // 11
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, yp1%BLOCK_NY, zp1%BLOCK_NZ, x/BLOCK_NX, yp1/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 6);
        bc_d |= (1 << 7);
    }
    // 12
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, ym1%BLOCK_NY, zm1%BLOCK_NZ, x/BLOCK_NX, ym1/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 0);
        bc_d |= (1 << 1);
    }
    // 13
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, ym1%BLOCK_NY, z%BLOCK_NZ, xp1/BLOCK_NX, ym1/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 1);
        bc_d |= (1 << 5);
    }
    // 14
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, yp1%BLOCK_NY, z%BLOCK_NZ, xm1/BLOCK_NX, yp1/BLOCK_NY, z/BLOCK_NZ)] == 255){
        bc_d |= (1 << 2);
        bc_d |= (1 << 6);
    }
    // 15
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, y%BLOCK_NY, zm1%BLOCK_NZ, xp1/BLOCK_NX, y/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 1);
        bc_d |= (1 << 3);
    }
    // 16
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, y%BLOCK_NY, zp1%BLOCK_NZ, xm1/BLOCK_NX, y/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 4);
        bc_d |= (1 << 6);
    }
    // 17
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, yp1%BLOCK_NY, zm1%BLOCK_NZ, x/BLOCK_NX, yp1/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 2);
        bc_d |= (1 << 3);
    }
    // 18
    if(dNodeType[idxScalarBlock(x%BLOCK_NX, ym1%BLOCK_NY, zp1%BLOCK_NZ, x/BLOCK_NX, ym1/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 4);
        bc_d |= (1 << 5);
    }
    // 19
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, yp1%BLOCK_NY, zp1%BLOCK_NZ, xp1/BLOCK_NX, yp1/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 7);
    }
    // 20
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, ym1%BLOCK_NY, zm1%BLOCK_NZ, xm1/BLOCK_NX, ym1/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 0);
    }
    // 21
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, yp1%BLOCK_NY, zm1%BLOCK_NZ, xp1/BLOCK_NX, yp1/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 3);
    }
    // 22
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, ym1%BLOCK_NY, zp1%BLOCK_NZ, xm1/BLOCK_NX, ym1/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 4);
    }
    // 23
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, ym1%BLOCK_NY, zp1%BLOCK_NZ, xp1/BLOCK_NX, ym1/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 5);
    }
    // 24
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, ym1%BLOCK_NY, zp1%BLOCK_NZ, xp1/BLOCK_NX, ym1/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 2);
    }
    // 25
    if(dNodeType[idxScalarBlock(xm1%BLOCK_NX, yp1%BLOCK_NY, zp1%BLOCK_NZ, xm1/BLOCK_NX, yp1/BLOCK_NY, zp1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 6);
    }
    // 26
    if(dNodeType[idxScalarBlock(xp1%BLOCK_NX, ym1%BLOCK_NY, zm1%BLOCK_NZ, xp1/BLOCK_NX, ym1/BLOCK_NY, zm1/BLOCK_NZ)] == 255){
        bc_d |= (1 << 1);   
    }

    return bc_d;
}