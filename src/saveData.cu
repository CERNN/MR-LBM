#include "hip/hip_runtime.h"
#include "saveData.cuh"
#ifdef NON_NEWTONIAN_FLUID
#include "nnf.h"
#endif

__host__
void treatData(
    dfloat* h_fMom,
    dfloat* fMom,
    #if MEAN_FLOW
    dfloat* fMom_mean,
    #endif
    unsigned int step
){

    //copy full macroscopic field
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());


    #ifdef THERMAL_MODEL
    std::ostringstream strDataInfo("");
    strDataInfo << std::scientific;
    strDataInfo << std::setprecision(6);
    
    int x0 = 0;
    int x1 = 1;
    int x2 = NX-1;
    int x3 = NX-2;
    dfloat C_x0;
    dfloat C_x1;
    dfloat C_x2;
    dfloat C_x3;
    dfloat Nu_sum = 0.0;


    for (int z = 0; z <NZ_TOTAL; z++){
        for(int y = 0; y< NY-0;y++){
            C_x0 = h_fMom[idxMom(x0%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_C_INDEX, x0/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
            C_x1 = h_fMom[idxMom(x1%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_C_INDEX, x1/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
            C_x2 = h_fMom[idxMom(x2%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_C_INDEX, x2/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
            C_x3 = h_fMom[idxMom(x3%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_C_INDEX, x3/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];

            Nu_sum +=-(C_x1 - C_x0);
            Nu_sum +=(C_x3 - C_x2);
        }
    }

    Nu_sum /= (2*(NY-2)*NZ_TOTAL);
    Nu_sum = Nu_sum/(T_DELTA_T/L);

    strDataInfo <<"step,"<< step<< "," << Nu_sum;// << "," << mean_counter;
    saveTreatData("_Nu_mean",strDataInfo.str(),step);
    #endif

    /*
    dfloat t_ux0, t_ux1;
    dfloat m_ux0_s, m_ux1_s;
    int y0 = NY-1;
    int y1 = NY-2;
    int count = 0;
    m_ux0_s = 0.0;
    m_ux1_s = 0.0;

    //right side of the equation 10
    for (int z = 0 ; z <NZ_TOTAL-1 ; z++){
        for (int x = 0; x< NX-1;x++){
            t_ux0 = h_fMom[idxMom(x%BLOCK_NX, y0%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y0/BLOCK_NY, z/BLOCK_NZ)];
            t_ux1 = h_fMom[idxMom(x%BLOCK_NX, y1%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y1/BLOCK_NY, z/BLOCK_NZ)];

            m_ux0_s += (t_ux0 * t_ux0);
            m_ux1_s += (t_ux1 * t_ux1);
            count++;
        }
    }
    m_ux0_s /= count;
    m_ux1_s /= count;


    dfloat LS = (m_ux0_s-m_ux1_s);
    LS = LS/(4*N);
    */

    //LEFT SIDE
/*
    dfloat t_ux0, t_uy0,t_uz0;
    dfloat t_mxx0,t_mxy0,t_mxz0,t_myy0,t_myz0,t_mzz0;
    dfloat Sxx = 0;
    dfloat Sxy = 0;
    dfloat Sxz = 0;
    dfloat Syy = 0;
    dfloat Syz = 0;
    dfloat Szz = 0;
    dfloat SS = 0;
    int count = 0;


    #if MEAN_FLOW
    dfloat f_ux = 0;
    dfloat f_uy = 0;
    dfloat f_uz = 0;

    dfloat f_Sxx = 0;
    dfloat f_Sxy = 0;
    dfloat f_Sxz = 0;
    dfloat f_Syy = 0;
    dfloat f_Syz = 0;
    dfloat f_Szz = 0;

    dfloat f_SS = 0;

    dfloat m_ux = 0.0;
    dfloat m_uy = 0.0;
    dfloat m_uz = 0.0;

    dfloat m_Sxx = 0;
    dfloat m_Sxy = 0;
    dfloat m_Sxz = 0;
    dfloat m_Syy = 0;
    dfloat m_Syz = 0;
    dfloat m_Szz = 0;
    #endif 

    dfloat mean_counter = 1.0/((dfloat)(step/MACR_SAVE)+1.0);
    count = 0;
    //left side of the equation
    for (int z = 0 ; z <NZ_TOTAL; z++){
        for(int y = 0; y< NY;y++){
            for(int x = 0; x< NX;x++){
                t_ux0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_uy0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_uz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];

                t_mxx0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_mxy0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_mxz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_myy0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MYY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_myz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MYZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_mzz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MZZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];

                Sxx = (as2/(2*TAU))*(t_ux0*t_ux0-t_mxx0);
                Sxy = (as2/(2*TAU))*(t_ux0*t_uy0-t_mxy0);
                Sxz = (as2/(2*TAU))*(t_ux0*t_uz0-t_mxz0);

                Syy = (as2/(2*TAU))*(t_uy0*t_uy0-t_myy0);
                Syz = (as2/(2*TAU))*(t_uy0*t_uz0-t_myz0);

                Szz = (as2/(2*TAU))*(t_uz0*t_uz0-t_mzz0);
                SS += ( Sxx * Sxx + 
                        Syy * Syy + 
                        Szz * Szz + 2*(
                        Sxy * Sxy + 
                        Sxz * Sxz + 
                        Syz * Syz)) ;

                #if MEAN_FLOW
                //STORE AND UPDATE MEANS

                //retrive mean values
                m_ux = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                m_uy = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                m_uz = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                
                m_Sxx = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                m_Sxy = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                m_Sxz = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                m_Syy = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MYY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                m_Syz = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MYZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                m_Szz = fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MZZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];

                //update and store mean values
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_ux + (t_ux0 - m_ux)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_uy + (t_uy0 - m_uy)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_uz + (t_uz0 - m_uz)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_Sxx + (Sxx - m_Sxx)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_Sxy + (Sxy - m_Sxy)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MXZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_Sxz + (Sxz - m_Sxz)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MYY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_Syy + (Syy - m_Syy)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MYZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_Syz + (Syz - m_Syz)*(mean_counter);
                fMom_mean[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_MZZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = m_Szz + (Szz - m_Szz)*(mean_counter);
            
                f_ux = t_ux0 - m_ux;
                f_uy = t_uy0 - m_uy;
                f_uz = t_uz0 - m_uz;
                f_Sxx = Sxx - m_Sxx;
                f_Sxy = Sxy - m_Sxy;
                f_Sxz = Sxz - m_Sxz;
                f_Syy = Syy - m_Syy;
                f_Syz = Syz - m_Syz;
                f_Szz = Szz - m_Szz;

                f_SS += ( f_Sxx * f_Sxx + f_Syy * f_Syy + f_Szz * f_Szz + 2*( f_Sxy * f_Sxy + f_Sxz * f_Sxz + f_Syz * f_Syz));


                #endif
                count++;

            }
        }
    }

    SS = SS/(N*N*N);
    #if MEAN_FLOW
    f_SS = f_SS / (count);
    #endif


    int y0 = NY-1;
    int y1 = NY-2;
    dfloat t_ux1;
    
    dfloat mean_prod = 0.0;
    for (int z = 0 ; z <NZ_TOTAL ; z++){
        for (int x = 0; x< NX;x++){
            t_ux0 = h_fMom[idxMom(x%BLOCK_NX, y0%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y0/BLOCK_NY, z/BLOCK_NZ)];
            t_ux1 = h_fMom[idxMom(x%BLOCK_NX, y1%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y1/BLOCK_NY, z/BLOCK_NZ)];
            mean_prod +=(t_ux0*t_ux0-t_ux1*t_ux1)/4;
        }
    }
    mean_prod = mean_prod/(N*N*N);

    #if MEAN_FLOW
    dfloat epsilon = 2*((TAU-0.5)/3)*f_SS;
    #endif




    //printf("%0.7e\t%0.7e\t%0.7e\n",LS,SS,SS/LS);
    // step << total_energy_dissipated, total_energy_produced, error , epsilon, omega
    strDataInfo <<"step,"<< step<< "," << SS << "," << mean_prod << "," << abs(SS/mean_prod - 1.0);// << "," << mean_counter;
    #if MEAN_FLOW
        strDataInfo <<"," <<  epsilon;
    #endif



    saveTreatData("_treatData",strDataInfo.str(),step);
    */
}

__host__
void velocityProfile(
    dfloat* fMom,
    int dir_index,
    unsigned int step
){

    std::ostringstream strDataInfo("");
    strDataInfo << std::scientific;
    strDataInfo << std::setprecision(6);
    strDataInfo <<"step "<< step;

    int x_loc,y_loc,z_loc;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;
    switch (dir_index)
    {
    case 1: //ux on y-direction
        
        checkCudaErrors(hipHostMalloc((void**)&(ux), sizeof(dfloat)));

        x_loc = NX/2;
        z_loc = NZ/2;
        
        for (y_loc = 0; y_loc < NY ; y_loc++){

            checkCudaErrors(hipMemcpy(ux, fMom + idxMom(x_loc%BLOCK_NX,y_loc%BLOCK_NY, z_loc%BLOCK_NZ, 1, x_loc/BLOCK_NX, y_loc/BLOCK_NY, z_loc/BLOCK_NZ),
            sizeof(dfloat), hipMemcpyDeviceToHost));
            strDataInfo <<"\t"<< *ux;
        }
        saveTreatData("_ux_dy",strDataInfo.str(),step);

        hipFree(ux);
        break;
    case 2: //uy on y-direction
        checkCudaErrors(hipHostMalloc((void**)&(uy), sizeof(dfloat)));

        x_loc = NX/2;
        z_loc = NZ/2;
        
        for (y_loc = 0; y_loc < NY ; y_loc++){

            checkCudaErrors(hipMemcpy(uy, fMom + idxMom(x_loc%BLOCK_NX,y_loc%BLOCK_NY, z_loc%BLOCK_NZ, 2, x_loc/BLOCK_NX, y_loc/BLOCK_NY, z_loc/BLOCK_NZ),
            sizeof(dfloat), hipMemcpyDeviceToHost));
            strDataInfo <<"\t"<< *uy;
        }
        saveTreatData("_uy_dy",strDataInfo.str(),step);

        hipFree(uy);
        break;
    case 3: //uz on y-direction
        checkCudaErrors(hipHostMalloc((void**)&(uz), sizeof(dfloat)));

        x_loc = NX/2;
        z_loc = NZ/2;
        
        for (y_loc = 0; y_loc < NY ; y_loc++){

            checkCudaErrors(hipMemcpy(uz, fMom + idxMom(x_loc%BLOCK_NX,y_loc%BLOCK_NY, z_loc%BLOCK_NZ, 3, x_loc/BLOCK_NX, y_loc/BLOCK_NY, z_loc/BLOCK_NZ),
            sizeof(dfloat), hipMemcpyDeviceToHost));
            strDataInfo <<"\t"<< *uz;
        }
        saveTreatData("_uz_dy",strDataInfo.str(),step);

        hipFree(uz);
        break;
    case 4: //ux on x-direction
        checkCudaErrors(hipHostMalloc((void**)&(ux), sizeof(dfloat)));

        y_loc = NY/2;
        z_loc = NZ/2;
        for (x_loc = 0; x_loc < NX ; x_loc++){

            checkCudaErrors(hipMemcpy(ux, fMom + idxMom(x_loc%BLOCK_NX,y_loc%BLOCK_NY, z_loc%BLOCK_NZ, 1, x_loc/BLOCK_NX, y_loc/BLOCK_NY, z_loc/BLOCK_NZ),
            sizeof(dfloat), hipMemcpyDeviceToHost));
            strDataInfo <<"\t"<< *ux;
        }
        saveTreatData("_ux_dx",strDataInfo.str(),step);

        hipFree(ux);
        break;
    case 5: //uy on x-direction
        checkCudaErrors(hipHostMalloc((void**)&(uy), sizeof(dfloat)));

        y_loc = NY/2;
        z_loc = NZ/2;
        for (x_loc = 0; x_loc < NX ; x_loc++){

            checkCudaErrors(hipMemcpy(uy, fMom + idxMom(x_loc%BLOCK_NX,y_loc%BLOCK_NY, z_loc%BLOCK_NZ, 2, x_loc/BLOCK_NX, y_loc/BLOCK_NY, z_loc/BLOCK_NZ),
            sizeof(dfloat), hipMemcpyDeviceToHost));
            strDataInfo <<"\t"<< *uy;
        }
        saveTreatData("_uy_dx",strDataInfo.str(),step);

        hipFree(uy);
        break;
    case 6: //uz on x-direction
        checkCudaErrors(hipHostMalloc((void**)&(uz), sizeof(dfloat)));

        y_loc = NY/2;
        z_loc = NZ/2;
        for (x_loc = 0; x_loc < NX ; x_loc++){

            checkCudaErrors(hipMemcpy(uz, fMom + idxMom(x_loc%BLOCK_NX,y_loc%BLOCK_NY, z_loc%BLOCK_NZ, 3, x_loc/BLOCK_NX, y_loc/BLOCK_NY, z_loc/BLOCK_NZ),
            sizeof(dfloat), hipMemcpyDeviceToHost));
            strDataInfo <<"\t"<< *ux;
        }
        saveTreatData("_uz_dx",strDataInfo.str(),step);

        hipFree(uz);
        break;
    default:
        break;
    }
}



__host__
void probeExport(
    dfloat* fMom,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    unsigned int step
){
    std::ostringstream strDataInfo("");
    strDataInfo << std::scientific;
    strDataInfo << std::setprecision(6);
    strDataInfo <<"step "<< step;


    int probeNumber = 7;
    
    //probe locations
                //0        1       2       3        4       5   6
    int x[7] = {probe_x,(NX/4),(NX/4),(3*NX/4),(3*NX/4),(NX/4),(NX/4)};
    int y[7] = {probe_y,(NY/4),(3*NY/4),(3*NY/4),(NY/4),(NY/4),(NY/4)};
    int z[7] = {probe_z,probe_z,probe_z,probe_z,probe_z,(NZ_TOTAL/4),(3*NZ_TOTAL/4)};

    dfloat* rho;

    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    /*dfloat* mxx;
    dfloat* mxy;
    dfloat* mxz;
    dfloat* myy;
    dfloat* myz;
    dfloat* mzz;*/
    
    checkCudaErrors(hipHostMalloc((void**)&(rho), sizeof(dfloat)));    
    checkCudaErrors(hipHostMalloc((void**)&(ux), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(uy), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(uz), sizeof(dfloat)));    
    /*checkCudaErrors(hipHostMalloc((void**)&(mxx), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(mxy), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(mxz), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(myy), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(myz), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(mzz), sizeof(dfloat)));*/

    checkCudaErrors(hipDeviceSynchronize());
    for(int i=0; i< probeNumber; i++){
        checkCudaErrors(hipMemcpy(rho, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_RHO_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ux , fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_UX_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(uy , fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_UY_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(uz , fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_UZ_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        /*checkCudaErrors(hipMemcpy(mxx, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_MXX_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(mxy, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_MXY_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(mxz, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_MXZ_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(myy, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_MYY_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(myz, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_MYZ_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(mzz, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, M_MZZ_INDEX, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));*/

        strDataInfo <<"\t"<< *ux << "\t" << *uy << "\t" << *uz;

    }
    saveTreatData("_probeData",strDataInfo.str(),step);




    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);
    /*hipFree(mxx);
    hipFree(mxy);
    hipFree(mxz);
    hipFree(myy);
    hipFree(myz);
    hipFree(mzz);*/

}

__host__
void linearMacr(
    dfloat* h_fMom,
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SECOND_DIST 
    dfloat* C,
    #endif
    #if SAVE_BC
    dfloat* nodeTypeSave,
    unsigned int* hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    dfloat* h_BC_Fx,
    dfloat* h_BC_Fy,
    dfloat* h_BC_Fz,
    #endif
    unsigned int step
){
    size_t indexMacr;
    for(int z = 0; z< NZ;z++){
        ///printf("z %d \n", z);
        for(int y = 0; y< NY;y++){
            for(int x = 0; x< NX;x++){
                indexMacr = idxScalarGlobal(x,y,z);

                rho[indexMacr] = RHO_0+h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_RHO_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                ux[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                uy[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                uz[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];

                #ifdef NON_NEWTONIAN_FLUID
                omega[indexMacr] = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_OMEGA_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)]; 
                #endif

                #ifdef SECOND_DIST 
                C[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_C_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                #endif
                
                #if SAVE_BC
                nodeTypeSave[indexMacr] = (dfloat)hNodeType[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)]; 
                #endif
                //data += rho[indexMacr]*(ux[indexMacr]*ux[indexMacr] + uy[indexMacr]*uy[indexMacr] + uz[indexMacr]*uz[indexMacr]);
                //meanRho += rho[indexMacr];
            }
        }
    }


    #if defined BC_FORCES && defined SAVE_BC_FORCES
        dfloat* temp_x; 
        dfloat* temp_y;
        dfloat* temp_z;
        checkCudaErrors(hipHostMalloc((void**)&(temp_x), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(temp_y), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(temp_z), MEM_SIZE_SCALAR));


        for(int z = 0; z< NZ;z++){
            for(int y = 0; y< NY;y++){
                for(int x = 0; x< NX;x++){
                    indexMacr = idxScalarGlobal(x,y,z);
                    temp_x[indexMacr] = h_BC_Fx[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                    temp_y[indexMacr] = h_BC_Fy[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                    temp_z[indexMacr] = h_BC_Fz[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                }
            }
        }

        checkCudaErrors(hipMemcpy(h_BC_Fx, temp_x, MEM_SIZE_SCALAR, hipMemcpyHostToHost));
        checkCudaErrors(hipMemcpy(h_BC_Fy, temp_y, MEM_SIZE_SCALAR, hipMemcpyHostToHost));
        checkCudaErrors(hipMemcpy(h_BC_Fz, temp_z, MEM_SIZE_SCALAR, hipMemcpyHostToHost));


        hipHostFree(temp_x);
        hipHostFree(temp_y);
        hipHostFree(temp_z);
    #endif

}


__host__
void saveMacr(
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SECOND_DIST 
    dfloat* C,
    #endif
    #if SAVE_BC
    dfloat* nodeTypeSave,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    dfloat* h_BC_Fx,
    dfloat* h_BC_Fy,
    dfloat* h_BC_Fz,
    #endif
    unsigned int nSteps
){
// Names of files
    std::string strFileRho, strFileUx, strFileUy, strFileUz;
    std::string strFileOmega;
    std::string strFileC;
    std::string strFileBc; 
    std::string strFileFx, strFileFy, strFileFz;

    strFileRho = getVarFilename("rho", nSteps, ".bin");
    strFileUx = getVarFilename("ux", nSteps, ".bin");
    strFileUy = getVarFilename("uy", nSteps, ".bin");
    strFileUz = getVarFilename("uz", nSteps, ".bin");
    #ifdef NON_NEWTONIAN_FLUID
    strFileOmega = getVarFilename("omega", nSteps, ".bin");
    #endif
    #ifdef SECOND_DIST 
    strFileC = getVarFilename("C", nSteps, ".bin");
    #endif
    #if SAVE_BC
    strFileBc = getVarFilename("bc", nSteps, ".bin");
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    strFileFx = getVarFilename("fx", nSteps, ".bin");
    strFileFy = getVarFilename("fy", nSteps, ".bin");
    strFileFz = getVarFilename("fz", nSteps, ".bin");
    #endif
    // saving files
    saveVarBin(strFileRho, rho, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUx, ux, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUy, uy, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUz, uz, MEM_SIZE_SCALAR, false);
    #ifdef NON_NEWTONIAN_FLUID
    saveVarBin(strFileOmega, omega, MEM_SIZE_SCALAR, false);
    #endif
    #ifdef SECOND_DIST
    saveVarBin(strFileC, C, MEM_SIZE_SCALAR, false);
    #endif
    #if SAVE_BC
    saveVarBin(strFileBc, nodeTypeSave, MEM_SIZE_SCALAR, false);
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    saveVarBin(strFileFx, h_BC_Fx, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileFy, h_BC_Fy, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileFz, h_BC_Fz, MEM_SIZE_SCALAR, false);
    #endif
}

void saveVarBin(
    std::string strFile, 
    dfloat* var, 
    size_t memSize,
    bool append)
{
    FILE* outFile = nullptr;
    if(append)
        outFile = fopen(strFile.c_str(), "ab");
    else
        outFile = fopen(strFile.c_str(), "wb");
    if(outFile != nullptr)
    {
        fwrite(var, memSize, 1, outFile);
        fclose(outFile);
    }
    else
    {
        printf("Error saving \"%s\" \nProbably wrong path!\n", strFile.c_str());
    }
}



void folderSetup()
{
// Windows
#if defined(_WIN32)
    std::string strPath;
    strPath = PATH_FILES;
    strPath += "\\\\"; // adds "\\"
    strPath += ID_SIM;
    std::string cmd = "md ";
    cmd += strPath;
    system(cmd.c_str());
    return;
#endif // !_WIN32

// Unix
#if defined(__APPLE__) || defined(__MACH__) || defined(__linux__)
    std::string strPath;
    strPath = PATH_FILES;
    strPath += "/";
    strPath += ID_SIM;
    std::string cmd = "mkdir -p ";
    cmd += strPath;
    system(cmd.c_str());
    return;
#endif // !Unix
    printf("I don't know how to setup folders for your operational system :(\n");
    return;
}


std::string getVarFilename(
    const std::string varName, 
    unsigned int step,
    const std::string ext)
{
    unsigned int n_zeros = 0, pot_10 = 10;
    unsigned int aux1 = 1000000;  // 6 numbers on step
    // calculate number of zeros
    if (step != 0)
        for (n_zeros = 0; step * pot_10 < aux1; pot_10 *= 10)
            n_zeros++;
    else
        n_zeros = 6;

    // generates the file name as "PATH_FILES/id/id_varName000000.bin"
    std::string strFile = PATH_FILES;
    strFile += "/";
    strFile += ID_SIM;
    strFile += "/";
    strFile += ID_SIM;
    strFile += "_";
    strFile += varName;
    for (unsigned int i = 0; i < n_zeros; i++)
        strFile += "0";
    strFile += std::to_string(step);
    strFile += ext;

    return strFile;
}

std::string getSimInfoString(int step,dfloat MLUPS)
{
    std::ostringstream strSimInfo("");
    
    strSimInfo << std::scientific;
    strSimInfo << std::setprecision(6);
    
    strSimInfo << "---------------------------- SIMULATION INFORMATION ----------------------------\n";
    strSimInfo << "      Simulation ID: " << ID_SIM << "\n";
    #ifdef D3Q19
    strSimInfo << "       Velocity set: D3Q19\n";
    #endif // !D3Q19
    #ifdef D3Q27
    strSimInfo << "       Velocity set: D3Q27\n";
    #endif // !D3Q27
    #ifdef SINGLE_PRECISION
        strSimInfo << "          Precision: float\n";
    #else
        strSimInfo << "          Precision: double\n";
    #endif
    strSimInfo << "                 NX: " << NX << "\n";
    strSimInfo << "                 NY: " << NY << "\n";
    strSimInfo << "                 NZ: " << NZ << "\n";
    strSimInfo << "           NZ_TOTAL: " << NZ_TOTAL << "\n";
    strSimInfo << std::scientific << std::setprecision(6);
    strSimInfo << "                Tau: " << TAU << "\n";
    strSimInfo << "               Umax: " << U_MAX << "\n";
    strSimInfo << "                 FX: " << FX << "\n";
    strSimInfo << "                 FY: " << FY << "\n";
    strSimInfo << "                 FZ: " << FZ << "\n";
    strSimInfo << "         Save steps: " << MACR_SAVE << "\n";
    strSimInfo << "             Nsteps: " << step << "\n";
    strSimInfo << "              MLUPS: " << MLUPS << "\n";
    strSimInfo << std::scientific << std::setprecision(0);
    strSimInfo << "       Bx x By x Bz: " << BLOCK_NX << "x" << BLOCK_NY << "x"<< BLOCK_NZ << "\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";

    strSimInfo << "\n------------------------------ BOUNDARY CONDITIONS -----------------------------\n";
    #ifdef BC_MOMENT_BASED
    strSimInfo << "            BC mode: Moment Based \n";
    #endif
    strSimInfo << "            BC type: " << STR(BC_PROBLEM) << "\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";


    #ifdef NON_NEWTONIAN_FLUID
    strSimInfo << "\n------------------------------ NON NEWTONIAN FLUID -----------------------------\n";
    strSimInfo << std::scientific << std::setprecision(6);
    
    #ifdef POWERLAW
    strSimInfo << "              Model: Power-Law\n";
    strSimInfo << "        Power index: " << N_INDEX << "\n";
    strSimInfo << " Consistency factor: " << K_CONSISTENCY << "\n";
    strSimInfo << "            Gamma 0: " << GAMMA_0 << "\n";
    #endif // POWERLAW

    #ifdef BINGHAM
    strSimInfo << "              Model: Bingham\n";
    strSimInfo << "  Plastic viscosity: " << VISC << "\n";
    strSimInfo << "       Yield stress: " << S_Y << "\n";
    strSimInfo << "      Plastic omega: " << OMEGA_P << "\n";
    #endif // BINGHAM
    strSimInfo << "--------------------------------------------------------------------------------\n";
    #endif // NON_NEWTONIAN_FLUID
    #ifdef LES_MODEL
    strSimInfo << "\t Smagorisky Constant:" << CONST_SMAGORINSKY <<"\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";
    #endif //LES
    #ifdef THERMAL_MODEL 
    strSimInfo << "\n------------------------------ THERMAL -----------------------------\n";
        strSimInfo << std::scientific << std::setprecision(2);
    strSimInfo << "     Prandtl Number: " << T_PR_NUMBER << "\n";
        strSimInfo << std::scientific << std::setprecision(4);
    strSimInfo << "    Rayleigh Number: " << T_RA_NUMBER << "\n";
    strSimInfo << "     Grashof Number: " << T_GR_NUMBER << "\n";
       strSimInfo << std::scientific << std::setprecision(3);
    strSimInfo << "            Delta T: " << T_DELTA_T << "\n";
    strSimInfo << "        Reference T: " << T_REFERENCE << "\n";
    strSimInfo << "             Cold T: " << T_COLD << "\n";
    strSimInfo << "              Hot T: " << T_HOT << "\n";
    strSimInfo << std::scientific << std::setprecision(6);
    strSimInfo << "       Thermal Diff: " << T_DIFFUSIVITY << "\n";
    strSimInfo << "   Grav_t_Exp.Coeff: " << T_gravity_t_beta << "\n";
       strSimInfo << std::scientific << std::setprecision(2);
    strSimInfo << "          Gravity_x: " << gravity_vector[0] << "\n";
    strSimInfo << "          Gravity_y: " << gravity_vector[1] << "\n";
    strSimInfo << "          Gravity_z: " << gravity_vector[2] << "\n";
       strSimInfo << std::scientific << std::setprecision(6);
    strSimInfo << "              G_TAU: " << G_TAU << "\n";
    strSimInfo << "            G_OMEGA: " << G_OMEGA << "\n";

    strSimInfo << "--------------------------------------------------------------------------------\n";
    #endif// THERMAL_MODEL



    return strSimInfo.str();
}

void saveSimInfo(int step,dfloat MLUPS)
{
    std::string strInf = PATH_FILES;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "_info.txt"; // generate file name (with path)
    FILE* outFile = nullptr;

    outFile = fopen(strInf.c_str(), "w");
    if(outFile != nullptr)
    {
        std::string strSimInfo = getSimInfoString(step,MLUPS);
        fprintf(outFile, strSimInfo.c_str());
        fclose(outFile);
    }
    else
    {
        printf("Error saving \"%s\" \nProbably wrong path!\n", strInf.c_str());
    }
    
}
/**/


void saveTreatData(std::string fileName, std::string dataString, int step)
{
    #if SAVEDATA
    std::string strInf = PATH_FILES;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "/";
    strInf += ID_SIM;
    strInf += fileName;
    strInf += ".txt"; // generate file name (with path)
    std::ifstream file(strInf.c_str());
    std::ofstream outfile;

    if(step == MACR_SAVE){ //check if first time step to save data
        outfile.open(strInf.c_str());
    }else{
        if (file.good()) {
            outfile.open(strInf.c_str(), std::ios::app);
        }else{ 
            outfile.open(strInf.c_str());
        }
    }


    outfile << dataString.c_str() << std::endl; 
    outfile.close(); 
    #endif
    #if CONSOLEPRINT
    printf("%s \n",dataString.c_str());
    #endif
}