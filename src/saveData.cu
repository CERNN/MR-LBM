#include "saveData.cuh"
#ifdef NON_NEWTONIAN_FLUID
#include "nnf.h"
#endif

__host__
void linearMacr(
    dfloat* h_fMom,
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SECOND_DIST 
    dfloat* C,
    #endif
    #if SAVE_BC
    dfloat* nodeTypeSave,
    unsigned int* hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    dfloat* h_BC_Fx,
    dfloat* h_BC_Fy,
    dfloat* h_BC_Fz,
    #endif
    unsigned int step
){
    size_t indexMacr;
    for(int z = 0; z< NZ;z++){
        ///printf("z %d \n", z);
        for(int y = 0; y< NY;y++){
            for(int x = 0; x< NX;x++){
                indexMacr = idxScalarGlobal(x,y,z);

                rho[indexMacr] = RHO_0+h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_RHO_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                ux[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                uy[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                uz[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];

                #ifdef NON_NEWTONIAN_FLUID
                omega[indexMacr] = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_OMEGA_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)]; 
                #endif

                #ifdef SECOND_DIST 
                C[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_C_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                #endif
                
                #if SAVE_BC
                nodeTypeSave[indexMacr] = (dfloat)hNodeType[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)]; 
                #endif
                //data += rho[indexMacr]*(ux[indexMacr]*ux[indexMacr] + uy[indexMacr]*uy[indexMacr] + uz[indexMacr]*uz[indexMacr]);
                //meanRho += rho[indexMacr];
            }
        }
    }


    #if defined BC_FORCES && defined SAVE_BC_FORCES
        dfloat* temp_x; 
        dfloat* temp_y;
        dfloat* temp_z;
        checkCudaErrors(hipHostMalloc((void**)&(temp_x), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(temp_y), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(temp_z), MEM_SIZE_SCALAR));


        for(int z = 0; z< NZ;z++){
            for(int y = 0; y< NY;y++){
                for(int x = 0; x< NX;x++){
                    indexMacr = idxScalarGlobal(x,y,z);
                    temp_x[indexMacr] = h_BC_Fx[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                    temp_y[indexMacr] = h_BC_Fy[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                    temp_z[indexMacr] = h_BC_Fz[idxScalarBlock(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                }
            }
        }

        checkCudaErrors(hipMemcpy(h_BC_Fx, temp_x, MEM_SIZE_SCALAR, hipMemcpyHostToHost));
        checkCudaErrors(hipMemcpy(h_BC_Fy, temp_y, MEM_SIZE_SCALAR, hipMemcpyHostToHost));
        checkCudaErrors(hipMemcpy(h_BC_Fz, temp_z, MEM_SIZE_SCALAR, hipMemcpyHostToHost));


        hipHostFree(temp_x);
        hipHostFree(temp_y);
        hipHostFree(temp_z);
    #endif

}
/*
__host__
void loadMoments(
    dfloat* h_fMom,
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SECOND_DIST
    dfloat* C
    #endif 
){
    size_t indexMacr;


    //first moments
    dfloat rhoVar, uxVar, uyVar, uzVar;
    dfloat pixx, pixy, pixz, piyy, piyz, pizz;
    dfloat invRho;
    dfloat pop[Q];
    #ifdef NON_NEWTONIAN_FLUID
    dfloat omegaVar;
    #endif
    #ifdef SECOND_DIST 
    dfloat cVar, invC, qx_t30, qy_t30, qz_t30;
    dfloat gNode[GQ];
    #endif

    


    for(int z = 0; z< NZ;z++){
        for(int y = 0; y< NY;y++){
            for(int x = 0; x< NX;x++){
                indexMacr = idxScalarGlobal(x,y,z);

                rhoVar = rho[indexMacr];
                h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_RHO_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = rhoVar-RHO_0;
                uxVar = ux[indexMacr];
                h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UX_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = F_M_I_SCALE*uxVar;
                uyVar = uy[indexMacr];
                h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UY_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = F_M_I_SCALE*uyVar;
                uzVar = uz[indexMacr];
                h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_UZ_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = F_M_I_SCALE*uzVar;


                //second moments
                //define equilibrium populations
                for (int i = 0; i < Q; i++)
                {
                    pop[i] = gpu_f_eq(w[i] * RHO_0,
                                    3 * (uxVar * cx[i] + uyVar * cy[i] + uzVar * cz[i]),
                                    1 - 1.5 * (uxVar * uxVar + uyVar * uyVar + uzVar * uzVar));
                }


                invRho = 1.0/rhoVar;
                pixx =  (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16]) * invRho - cs2;
                pixy = ((pop[7] + pop[ 8]) - (pop[13] + pop[14])) * invRho;
                pixz = ((pop[9] + pop[10]) - (pop[15] + pop[16])) * invRho;
                piyy =  (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18]) * invRho - cs2;
                piyz = ((pop[11]+pop[12])-(pop[17]+pop[18])) * invRho;
                pizz =  (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18]) * invRho - cs2;

                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXX_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = F_M_II_SCALE*pixx;
                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = F_M_IJ_SCALE*pixy;
                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MXZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = F_M_IJ_SCALE*pixz;
                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MYY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = F_M_II_SCALE*piyy;
                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MYZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = F_M_IJ_SCALE*piyz;
                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_MZZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = F_M_II_SCALE*pizz;


                #ifdef NON_NEWTONIAN_FLUID
                omegaVar = omega[indexMacr];
                h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_OMEGA_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = omegaVar; 
                #endif

                #ifdef SECOND_DIST 
                cVar = C[indexMacr];
                h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, M_C_INDEX, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)] = cVar;

                dfloat udx_t30 = G_DIFF_FLUC_COEF * (qx_t30*invC - uxVar*F_M_I_SCALE);
                dfloat udy_t30 = G_DIFF_FLUC_COEF * (qy_t30*invC - uyVar*F_M_I_SCALE);
                dfloat udz_t30 = G_DIFF_FLUC_COEF * (qz_t30*invC - uzVar*F_M_I_SCALE);

                dfloat multiplyTerm = cVar * gW0;
                dfloat pics2 = 1.0;

                gNode[ 0] = multiplyTerm * (pics2);
                multiplyTerm = cVar * gW1;
                gNode[ 1] = multiplyTerm * (pics2 + uxVar * F_M_I_SCALE  + udx_t30 );
                gNode[ 2] = multiplyTerm * (pics2 - uxVar * F_M_I_SCALE  - udx_t30 );
                gNode[ 3] = multiplyTerm * (pics2 + uyVar * F_M_I_SCALE  + udy_t30 );
                gNode[ 4] = multiplyTerm * (pics2 - uyVar * F_M_I_SCALE  - udy_t30 );
                gNode[ 5] = multiplyTerm * (pics2 + uzVar * F_M_I_SCALE  + udz_t30 );
                gNode[ 6] = multiplyTerm * (pics2 - uzVar * F_M_I_SCALE  - udz_t30 );
                multiplyTerm = cVar * gW2;
                gNode[ 7] = multiplyTerm * (pics2 + uxVar * F_M_I_SCALE + uyVar * F_M_I_SCALE + udx_t30 + udy_t30 );
                gNode[ 8] = multiplyTerm * (pics2 - uxVar * F_M_I_SCALE - uyVar * F_M_I_SCALE - udx_t30 - udy_t30 );
                gNode[ 9] = multiplyTerm * (pics2 + uxVar * F_M_I_SCALE + uzVar * F_M_I_SCALE + udx_t30 + udz_t30 );
                gNode[10] = multiplyTerm * (pics2 - uxVar * F_M_I_SCALE - uzVar * F_M_I_SCALE - udx_t30 - udz_t30 );
                gNode[11] = multiplyTerm * (pics2 + uyVar * F_M_I_SCALE + uzVar * F_M_I_SCALE + udy_t30 + udz_t30 );
                gNode[12] = multiplyTerm * (pics2 - uyVar * F_M_I_SCALE - uzVar * F_M_I_SCALE - udy_t30 - udz_t30 );
                gNode[13] = multiplyTerm * (pics2 + uxVar * F_M_I_SCALE - uyVar * F_M_I_SCALE + udx_t30 - udy_t30 );
                gNode[14] = multiplyTerm * (pics2 - uxVar * F_M_I_SCALE + uyVar * F_M_I_SCALE - udx_t30 + udy_t30 );
                gNode[15] = multiplyTerm * (pics2 + uxVar * F_M_I_SCALE - uzVar * F_M_I_SCALE + udx_t30 - udz_t30 );
                gNode[16] = multiplyTerm * (pics2 - uxVar * F_M_I_SCALE + uzVar * F_M_I_SCALE - udx_t30 + udz_t30 );
                gNode[17] = multiplyTerm * (pics2 + uyVar * F_M_I_SCALE - uzVar * F_M_I_SCALE + udy_t30 - udz_t30 );
                gNode[18] = multiplyTerm * (pics2 - uyVar * F_M_I_SCALE + uzVar * F_M_I_SCALE - udy_t30 + udz_t30 );

                qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));




                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_CX_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = qx_t30;
                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_CY_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = qy_t30;
                h_fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_CZ_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = qz_t30;
                #endif



            }
        }
    }
}


__host__
void loadSimField(
    dfloat* h_fMom,
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SECOND_DIST
    dfloat* C
    #endif 
){
    std::string strFileRho, strFileUx, strFileUy, strFileUz;
    std::string strFileOmega;
    std::string strFileC;
    std::string strFileBc; 
    std::string strFileFx, strFileFy, strFileFz;

    strFileRho = getVarFilename("rho", LOAD_FIELD_STEP, ".bin");
    strFileUx = getVarFilename("ux", LOAD_FIELD_STEP, ".bin");
    strFileUy = getVarFilename("uy", LOAD_FIELD_STEP, ".bin");
    strFileUz = getVarFilename("uz", LOAD_FIELD_STEP, ".bin");
    #ifdef NON_NEWTONIAN_FLUID
    strFileOmega = getVarFilename("omega", LOAD_FIELD_STEP, ".bin");
    #endif
    #ifdef SECOND_DIST 
    strFileC = getVarFilename("C", LOAD_FIELD_STEP, ".bin");
    #endif
    #if SAVE_BC
    strFileBc = getVarFilename("bc", LOAD_FIELD_STEP, ".bin");
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    strFileFx = getVarFilename("fx", LOAD_FIELD_STEP, ".bin");
    strFileFy = getVarFilename("fy", LOAD_FIELD_STEP, ".bin");
    strFileFz = getVarFilename("fz", LOAD_FIELD_STEP, ".bin");
    #endif

    // load files
    loadVarBin(strFileRho, rho, MEM_SIZE_SCALAR, false);
    loadVarBin(strFileUx, ux, MEM_SIZE_SCALAR, false);
    loadVarBin(strFileUy, uy, MEM_SIZE_SCALAR, false);
    loadVarBin(strFileUz, uz, MEM_SIZE_SCALAR, false);
    #ifdef NON_NEWTONIAN_FLUID
    loadVarBin(strFileOmega, omega, MEM_SIZE_SCALAR, false);
    #endif
    #ifdef SECOND_DIST
    loadVarBin(strFileC, C, MEM_SIZE_SCALAR, false);
    #endif
    #if SAVE_BC
    loadVarBin(strFileBc, nodeTypeSave, MEM_SIZE_SCALAR, false);
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    loadVarBin(strFileFx, h_BC_Fx, MEM_SIZE_SCALAR, false);
    loadVarBin(strFileFy, h_BC_Fy, MEM_SIZE_SCALAR, false);
    loadVarBin(strFileFz, h_BC_Fz, MEM_SIZE_SCALAR, false);
    #endif


    loadMoments(h_fMom,rho,ux,uy,uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST
            C
            #endif 
            );

}


void loadVarBin(
    std::string strFile, 
    dfloat* var, 
    size_t memSize,
    bool append)
{
    FILE* outFile = nullptr;
    if(append)
        outFile = fopen(strFile.c_str(), "ab");
    else
        outFile = fopen(strFile.c_str(), "wb");
    if(outFile != nullptr)
    {
        fread(var, memSize, 1, outFile);
        fclose(outFile);
    }
    else
    {
        printf("Error loading \"%s\" \nProbably wrong path!\n", strFile.c_str());
    }
}
*/

__host__
void saveMacr(
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SECOND_DIST 
    dfloat* C,
    #endif
    #if SAVE_BC
    dfloat* nodeTypeSave,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    dfloat* h_BC_Fx,
    dfloat* h_BC_Fy,
    dfloat* h_BC_Fz,
    #endif
    unsigned int nSteps
){
// Names of files
    std::string strFileRho, strFileUx, strFileUy, strFileUz;
    std::string strFileOmega;
    std::string strFileC;
    std::string strFileBc; 
    std::string strFileFx, strFileFy, strFileFz;

    strFileRho = getVarFilename("rho", nSteps, ".bin");
    strFileUx = getVarFilename("ux", nSteps, ".bin");
    strFileUy = getVarFilename("uy", nSteps, ".bin");
    strFileUz = getVarFilename("uz", nSteps, ".bin");
    #ifdef NON_NEWTONIAN_FLUID
    strFileOmega = getVarFilename("omega", nSteps, ".bin");
    #endif
    #ifdef SECOND_DIST 
    strFileC = getVarFilename("C", nSteps, ".bin");
    #endif
    #if SAVE_BC
    strFileBc = getVarFilename("bc", nSteps, ".bin");
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    strFileFx = getVarFilename("fx", nSteps, ".bin");
    strFileFy = getVarFilename("fy", nSteps, ".bin");
    strFileFz = getVarFilename("fz", nSteps, ".bin");
    #endif
    // saving files
    saveVarBin(strFileRho, rho, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUx, ux, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUy, uy, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUz, uz, MEM_SIZE_SCALAR, false);
    #ifdef NON_NEWTONIAN_FLUID
    saveVarBin(strFileOmega, omega, MEM_SIZE_SCALAR, false);
    #endif
    #ifdef SECOND_DIST
    saveVarBin(strFileC, C, MEM_SIZE_SCALAR, false);
    #endif
    #if SAVE_BC
    saveVarBin(strFileBc, nodeTypeSave, MEM_SIZE_SCALAR, false);
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    saveVarBin(strFileFx, h_BC_Fx, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileFy, h_BC_Fy, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileFz, h_BC_Fz, MEM_SIZE_SCALAR, false);
    #endif
}

void saveVarBin(
    std::string strFile, 
    dfloat* var, 
    size_t memSize,
    bool append)
{
    FILE* outFile = nullptr;
    if(append)
        outFile = fopen(strFile.c_str(), "ab");
    else
        outFile = fopen(strFile.c_str(), "wb");
    if(outFile != nullptr)
    {
        fwrite(var, memSize, 1, outFile);
        fclose(outFile);
    }
    else
    {
        printf("Error saving \"%s\" \nProbably wrong path!\n", strFile.c_str());
    }
}



void folderSetup()
{
// Windows
#if defined(_WIN32)
    std::string strPath;
    strPath = PATH_FILES;
    strPath += "\\\\"; // adds "\\"
    strPath += ID_SIM;
    std::string cmd = "md ";
    cmd += strPath;
    system(cmd.c_str());
    return;
#endif // !_WIN32

// Unix
#if defined(__APPLE__) || defined(__MACH__) || defined(__linux__)
    std::string strPath;
    strPath = PATH_FILES;
    strPath += "/";
    strPath += ID_SIM;
    std::string cmd = "mkdir -p ";
    cmd += strPath;
    system(cmd.c_str());
    return;
#endif // !Unix
    printf("I don't know how to setup folders for your operational system :(\n");
    return;
}


std::string getVarFilename(
    const std::string varName, 
    unsigned int step,
    const std::string ext)
{
    unsigned int n_zeros = 0, pot_10 = 10;
    unsigned int aux1 = 1000000;  // 6 numbers on step
    // calculate number of zeros
    if (step != 0)
        for (n_zeros = 0; step * pot_10 < aux1; pot_10 *= 10)
            n_zeros++;
    else
        n_zeros = 6;

    // generates the file name as "PATH_FILES/id/id_varName000000.bin"
    std::string strFile = PATH_FILES;
    strFile += "/";
    strFile += ID_SIM;
    strFile += "/";
    strFile += ID_SIM;
    strFile += "_";
    strFile += varName;
    for (unsigned int i = 0; i < n_zeros; i++)
        strFile += "0";
    strFile += std::to_string(step);
    strFile += ext;

    return strFile;
}

std::string getSimInfoString(int step,dfloat MLUPS)
{
    std::ostringstream strSimInfo("");
    
    strSimInfo << std::scientific;
    strSimInfo << std::setprecision(6);
    
    strSimInfo << "---------------------------- SIMULATION INFORMATION ----------------------------\n";
    strSimInfo << "      Simulation ID: " << ID_SIM << "\n";
    #ifdef D3Q19
    strSimInfo << "       Velocity set: D3Q19\n";
    #endif // !D3Q19
    #ifdef D3Q27
    strSimInfo << "       Velocity set: D3Q27\n";
    #endif // !D3Q27
    #ifdef SINGLE_PRECISION
        strSimInfo << "          Precision: float\n";
    #else
        strSimInfo << "          Precision: double\n";
    #endif
    strSimInfo << "                 NX: " << NX << "\n";
    strSimInfo << "                 NY: " << NY << "\n";
    strSimInfo << "                 NZ: " << NZ << "\n";
    strSimInfo << "           NZ_TOTAL: " << NZ_TOTAL << "\n";
    strSimInfo << std::scientific << std::setprecision(6);
    strSimInfo << "                Tau: " << TAU << "\n";
    strSimInfo << "               Umax: " << U_MAX << "\n";
    strSimInfo << "                 FX: " << FX << "\n";
    strSimInfo << "                 FY: " << FY << "\n";
    strSimInfo << "                 FZ: " << FZ << "\n";
    strSimInfo << "         Save steps: " << MACR_SAVE << "\n";
    strSimInfo << "             Nsteps: " << step << "\n";
    strSimInfo << "              MLUPS: " << MLUPS << "\n";
    strSimInfo << std::scientific << std::setprecision(0);
    strSimInfo << "       Bx x By x Bz: " << BLOCK_NX << "x" << BLOCK_NY << "x"<< BLOCK_NZ << "\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";

    strSimInfo << "\n------------------------------ BOUNDARY CONDITIONS -----------------------------\n";
    #ifdef BC_MOMENT_BASED
    strSimInfo << "            BC mode: Moment Based \n";
    #endif
    strSimInfo << "            BC type: " << STR(BC_PROBLEM) << "\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";


    #ifdef NON_NEWTONIAN_FLUID
    strSimInfo << "\n------------------------------ NON NEWTONIAN FLUID -----------------------------\n";
    strSimInfo << std::scientific << std::setprecision(6);
    
    #ifdef POWERLAW
    strSimInfo << "              Model: Power-Law\n";
    strSimInfo << "        Power index: " << N_INDEX << "\n";
    strSimInfo << " Consistency factor: " << K_CONSISTENCY << "\n";
    strSimInfo << "            Gamma 0: " << GAMMA_0 << "\n";
    #endif // POWERLAW

    #ifdef BINGHAM
    strSimInfo << "              Model: Bingham\n";
    strSimInfo << "  Plastic viscosity: " << VISC << "\n";
    strSimInfo << "       Yield stress: " << S_Y << "\n";
    strSimInfo << "      Plastic omega: " << OMEGA_P << "\n";
    #endif // BINGHAM
    strSimInfo << "--------------------------------------------------------------------------------\n";
    #endif // NON_NEWTONIAN_FLUID
    #ifdef LES_MODEL
    strSimInfo << "\t Smagorisky Constant:" << CONST_SMAGORINSKY <<"\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";
    #endif //LES
    #ifdef THERMAL_MODEL 
    strSimInfo << "\n------------------------------ THERMAL -----------------------------\n";
        strSimInfo << std::scientific << std::setprecision(2);
    strSimInfo << "     Prandtl Number: " << T_PR_NUMBER << "\n";
        strSimInfo << std::scientific << std::setprecision(4);
    strSimInfo << "    Rayleigh Number: " << T_RA_NUMBER << "\n";
    strSimInfo << "     Grashof Number: " << T_GR_NUMBER << "\n";
       strSimInfo << std::scientific << std::setprecision(3);
    strSimInfo << "            Delta T: " << T_DELTA_T << "\n";
    strSimInfo << "        Reference T: " << T_REFERENCE << "\n";
    strSimInfo << "             Cold T: " << T_COLD << "\n";
    strSimInfo << "              Hot T: " << T_HOT << "\n";
    strSimInfo << std::scientific << std::setprecision(6);
    strSimInfo << "       Thermal Diff: " << T_DIFFUSIVITY << "\n";
    strSimInfo << "   Grav_t_Exp.Coeff: " << T_gravity_t_beta << "\n";
       strSimInfo << std::scientific << std::setprecision(2);
    strSimInfo << "          Gravity_x: " << gravity_vector[0] << "\n";
    strSimInfo << "          Gravity_y: " << gravity_vector[1] << "\n";
    strSimInfo << "          Gravity_z: " << gravity_vector[2] << "\n";
       strSimInfo << std::scientific << std::setprecision(6);
    strSimInfo << "              G_TAU: " << G_TAU << "\n";
    strSimInfo << "            G_OMEGA: " << G_OMEGA << "\n";

    strSimInfo << "--------------------------------------------------------------------------------\n";
    #endif// THERMAL_MODEL



    return strSimInfo.str();
}

void saveSimInfo(int step,dfloat MLUPS)
{
    std::string strInf = PATH_FILES;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "_info.txt"; // generate file name (with path)
    FILE* outFile = nullptr;

    outFile = fopen(strInf.c_str(), "w");
    if(outFile != nullptr)
    {
        std::string strSimInfo = getSimInfoString(step,MLUPS);
        fprintf(outFile, strSimInfo.c_str());
        fclose(outFile);
    }
    else
    {
        printf("Error saving \"%s\" \nProbably wrong path!\n", strInf.c_str());
    }
    
}
/**/


void saveTreatData(std::string fileName, std::string dataString, int step)
{
    #if SAVEDATA
    std::string strInf = PATH_FILES;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "/";
    strInf += ID_SIM;
    strInf += fileName;
    strInf += ".txt"; // generate file name (with path)
    std::ifstream file(strInf.c_str());
    std::ofstream outfile;

    if(step == REPORT_SAVE){ //check if first time step to save data
        outfile.open(strInf.c_str());
    }else{
        if (file.good()) {
            outfile.open(strInf.c_str(), std::ios::app);
        }else{ 
            outfile.open(strInf.c_str());
        }
    }


    outfile << dataString.c_str() << std::endl; 
    outfile.close(); 
    #endif
    #if CONSOLEPRINT
    printf("%s \n",dataString.c_str());
    #endif
}