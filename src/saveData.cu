#include "hip/hip_runtime.h"
#include "saveData.cuh"
#ifdef NON_NEWTONIAN_FLUID
#include "nnf.h"
#endif

__host__
void treatData(
    dfloat* h_fMom,
    dfloat* fMom,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    unsigned int step
){

    //copy full macroscopic field
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());



    std::ostringstream strDataInfo("");
    strDataInfo << std::scientific;
    strDataInfo << std::setprecision(6);


    dfloat t_ux0, t_ux1;
    dfloat m_ux0, m_ux1;
    int y0 = NY-1;
    int y1 = NY-2;
    int count = 0;
    m_ux0 = 0.0;
    m_ux1 = 0.0;

    //right side of the equation 10
    for (int z = 0 ; z <NZ_TOTAL-1 ; z++){
        for (int x = 0; x< NX-1;x++){
            t_ux0 = h_fMom[idxMom(x%BLOCK_NX, y0%BLOCK_NY, z%BLOCK_NZ, 1, x/BLOCK_NX, y0/BLOCK_NY, z/BLOCK_NZ)];
            t_ux1 = h_fMom[idxMom(x%BLOCK_NX, y1%BLOCK_NY, z%BLOCK_NZ, 1, x/BLOCK_NX, y1/BLOCK_NY, z/BLOCK_NZ)];

            m_ux0 += (t_ux0 * t_ux0);
            m_ux1 += (t_ux1 * t_ux1);
            count++;
        }
    }
    m_ux0 /= count;
    m_ux1 /= count;


    dfloat LS = (m_ux0-m_ux1);
    LS = LS/(4*N);

    dfloat t_uy0,t_uz0;
    dfloat t_mxx0,t_mxy0,t_mxz0,t_myy0,t_myz0,t_mzz0;
    dfloat Sxx = 0;
    dfloat Sxy = 0;
    dfloat Sxz = 0;
    dfloat Syy = 0;
    dfloat Syz = 0;
    dfloat Szz = 0;
    dfloat SS = 0;

    count = 0;
    //left side of the equation
    for (int z = 0 ; z <NZ_TOTAL-1; z++){
        for(int y = 0; y< NY-1;y++){
            for(int x = 0; x< NX-1;x++){
                t_ux0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 1, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_uy0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 2, x/BLOCK_NX, y0/BLOCK_NY, z/BLOCK_NZ)];
                t_uz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 3, x/BLOCK_NX, y0/BLOCK_NY, z/BLOCK_NZ)];

                t_mxx0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 4, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_mxy0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 5, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_mxz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 9, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_myy0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 7, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_myz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 8, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                t_mzz0 = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 9, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];

                Sxx = (as2/(2*TAU))*(t_ux0*t_ux0-t_mxx0);
                Sxy = (as2/(2*TAU))*(t_ux0*t_uy0-t_mxy0);
                Sxz = (as2/(2*TAU))*(t_ux0*t_uz0-t_mxz0);

                Syy = (as2/(2*TAU))*(t_uy0*t_uy0-t_myy0);
                Syz = (as2/(2*TAU))*(t_uy0*t_uz0-t_myz0);

                Szz = (as2/(2*TAU))*(t_uz0*t_uz0-t_mzz0);
                SS += ( Sxx * Sxx + 
                        Syy * Syy + 
                        Szz * Szz + 2*(
                        Sxy * Sxy + 
                        Sxz * Sxz + 
                        Syz * Syz)) ;
                count++;

            }
        }
    }

    SS = SS / (count);
    //printf("%0.7e\t%0.7e\t%0.7e\n",LS,SS,SS/LS);
    strDataInfo <<"step,"<< step<< "," << LS << "," << SS << "," <<SS/LS;



    saveTreatData("_treatData",strDataInfo.str(),step);
}

__host__
void probeExport(
    dfloat* fMom,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    unsigned int step
){
    std::ostringstream strDataInfo("");
    strDataInfo << std::scientific;
    strDataInfo << std::setprecision(6);
    strDataInfo <<"step "<< step;


    int probeNumber = 5;
    
    //probe locations
    int x[5] = {probe_x,(NX/4),(NX/4),(3*NX/4),(3*NX/4)};
    int y[5] = {probe_y,(NY/4),(3*NY/4),(3*NY/4),(NY/4)};
    int z[5] = {probe_z,probe_z,probe_z,probe_z,probe_z};

    dfloat* rho;

    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    dfloat* mxx;
    dfloat* mxy;
    dfloat* mxz;
    dfloat* myy;
    dfloat* myz;
    dfloat* mzz;
    
    checkCudaErrors(hipHostMalloc((void**)&(rho), sizeof(dfloat)));    
    checkCudaErrors(hipHostMalloc((void**)&(ux), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(uy), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(uz), sizeof(dfloat)));    
    checkCudaErrors(hipHostMalloc((void**)&(mxx), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(mxy), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(mxz), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(myy), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(myz), sizeof(dfloat)));
    checkCudaErrors(hipHostMalloc((void**)&(mzz), sizeof(dfloat)));

    checkCudaErrors(hipDeviceSynchronize());
    for(int i=0; i< probeNumber; i++){
        checkCudaErrors(hipMemcpy(rho, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 0, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ux , fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 1, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(uy , fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 2, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(uz , fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 3, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(mxx, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 4, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(mxy, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 5, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(mxz, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 6, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(myy, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 7, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(myz, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 8, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(mzz, fMom + idxMom(x[i]%BLOCK_NX, y[i]%BLOCK_NY, z[i]%BLOCK_NZ, 9, x[i]/BLOCK_NX, y[i]/BLOCK_NY, z[i]/BLOCK_NZ),
        sizeof(dfloat), hipMemcpyDeviceToHost));

        strDataInfo <<"\t"<< *ux << "\t" << *uy << "\t" << *uz;

    }
    saveTreatData("_probeData",strDataInfo.str(),step);




    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);
    hipFree(mxx);
    hipFree(mxy);
    hipFree(mxz);
    hipFree(myy);
    hipFree(myz);
    hipFree(mzz);

}

__host__
void linearMacr(
    dfloat* h_fMom,
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SAVE_BC
    dfloat* nodeTypeSave,
    unsigned char* hNodeType,
    #endif
    unsigned int step
){
    size_t indexMacr;

    dfloat meanRho;
    meanRho  =  0;
    dfloat bc;

    for(int z = 0; z< NZ;z++){
        ///printf("z %d \n", z);
        for(int y = 0; y< NY;y++){
            for(int x = 0; x< NX;x++){
                indexMacr = idxScalarGlobal(x,y,z);

                rho[indexMacr] = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 0, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                ux[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 1, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                uy[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 2, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                uz[indexMacr]  = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 3, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)];
                #ifdef NON_NEWTONIAN_FLUID
                omega[indexMacr] = h_fMom[idxMom(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, 10, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)]; 
                #endif
                #ifdef SAVE_BC
                nodeTypeSave[indexMacr] = (dfloat)hNodeType[idxNodeType(x%BLOCK_NX, y%BLOCK_NY, z%BLOCK_NZ, x/BLOCK_NX, y/BLOCK_NY, z/BLOCK_NZ)]; 
                #endif
                //data += rho[indexMacr]*(ux[indexMacr]*ux[indexMacr] + uy[indexMacr]*uy[indexMacr] + uz[indexMacr]*uz[indexMacr]);
                //meanRho += rho[indexMacr];
            }
        }
    }
}


__host__
void saveMacr(
    dfloat* rho,
    dfloat* ux,
    dfloat* uy,
    dfloat* uz,
    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega,
    #endif
    #ifdef SAVE_BC
    dfloat* nodeTypeSave,
    #endif
    unsigned int nSteps
){
// Names of files
    std::string strFileRho, strFileUx, strFileUy, strFileUz, strFileOmega, strFileBc;

    strFileRho = getVarFilename("rho", nSteps, ".bin");
    strFileUx = getVarFilename("ux", nSteps, ".bin");
    strFileUy = getVarFilename("uy", nSteps, ".bin");
    strFileUz = getVarFilename("uz", nSteps, ".bin");
    #ifdef NON_NEWTONIAN_FLUID
    strFileOmega = getVarFilename("omega", nSteps, ".bin");
    #endif
    #ifdef SAVE_BC
    strFileBc = getVarFilename("bc", nSteps, ".bin");
    #endif
    // saving files
    saveVarBin(strFileRho, rho, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUx, ux, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUy, uy, MEM_SIZE_SCALAR, false);
    saveVarBin(strFileUz, uz, MEM_SIZE_SCALAR, false);
    #ifdef NON_NEWTONIAN_FLUID
    saveVarBin(strFileOmega, omega, MEM_SIZE_SCALAR, false);
    #endif
    #ifdef SAVE_BC
    saveVarBin(strFileBc, nodeTypeSave, MEM_SIZE_SCALAR, false);
    #endif
}

void saveVarBin(
    std::string strFile, 
    dfloat* var, 
    size_t memSize,
    bool append)
{
    FILE* outFile = nullptr;
    if(append)
        outFile = fopen(strFile.c_str(), "ab");
    else
        outFile = fopen(strFile.c_str(), "wb");
    if(outFile != nullptr)
    {
        fwrite(var, memSize, 1, outFile);
        fclose(outFile);
    }
    else
    {
        printf("Error saving \"%s\" \nProbably wrong path!\n", strFile.c_str());
    }
}



void folderSetup()
{
// Windows
#if defined(_WIN32)
    std::string strPath;
    strPath = PATH_FILES;
    strPath += "\\\\"; // adds "\\"
    strPath += ID_SIM;
    std::string cmd = "md ";
    cmd += strPath;
    system(cmd.c_str());
    return;
#endif // !_WIN32

// Unix
#if defined(__APPLE__) || defined(__MACH__) || defined(__linux__)
    std::string strPath;
    strPath = PATH_FILES;
    strPath += "/";
    strPath += ID_SIM;
    std::string cmd = "mkdir -p ";
    cmd += strPath;
    system(cmd.c_str());
    return;
#endif // !Unix
    printf("I don't know how to setup folders for your operational system :(\n");
    return;
}


std::string getVarFilename(
    const std::string varName, 
    unsigned int step,
    const std::string ext)
{
    unsigned int n_zeros = 0, pot_10 = 10;
    unsigned int aux1 = 1000000;  // 6 numbers on step
    // calculate number of zeros
    if (step != 0)
        for (n_zeros = 0; step * pot_10 < aux1; pot_10 *= 10)
            n_zeros++;
    else
        n_zeros = 6;

    // generates the file name as "PATH_FILES/id/id_varName000000.bin"
    std::string strFile = PATH_FILES;
    strFile += "/";
    strFile += ID_SIM;
    strFile += "/";
    strFile += ID_SIM;
    strFile += "_";
    strFile += varName;
    for (unsigned int i = 0; i < n_zeros; i++)
        strFile += "0";
    strFile += std::to_string(step);
    strFile += ext;

    return strFile;
}

std::string getSimInfoString(int step)
{
    std::ostringstream strSimInfo("");
    
    strSimInfo << std::scientific;
    strSimInfo << std::setprecision(6);
    
    strSimInfo << "---------------------------- SIMULATION INFORMATION ----------------------------\n";
    strSimInfo << "      Simulation ID: " << ID_SIM << "\n";
    #ifdef D3Q19
    strSimInfo << "       Velocity set: D3Q19\n";
    #endif // !D3Q19
    #ifdef D3Q27
    strSimInfo << "       Velocity set: D3Q27\n";
    #endif // !D3Q27
    #ifdef SINGLE_PRECISION
        strSimInfo << "          Precision: float\n";
    #else
        strSimInfo << "          Precision: double\n";
    #endif
    strSimInfo << "                 NX: " << NX << "\n";
    strSimInfo << "                 NY: " << NY << "\n";
    strSimInfo << "                 NZ: " << NZ << "\n";
    strSimInfo << "           NZ_TOTAL: " << NZ_TOTAL << "\n";
    strSimInfo << std::scientific << std::setprecision(6);
    strSimInfo << "                Tau: " << TAU << "\n";
    strSimInfo << "               Umax: " << U_MAX << "\n";
    strSimInfo << "                 FX: " << FX << "\n";
    strSimInfo << "                 FY: " << FY << "\n";
    strSimInfo << "                 FZ: " << FZ << "\n";
    strSimInfo << "         Save steps: " << MACR_SAVE << "\n";
    strSimInfo << "             Nsteps: " << step << "\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";

    strSimInfo << "\n------------------------------ BOUNDARY CONDITIONS -----------------------------\n";
    #ifdef BC_POPULATION_BASED
    strSimInfo << "            BC mode: Population Based \n";
    #endif
    #ifdef BC_MOMENT_BASED
    strSimInfo << "            BC mode: Moment Based \n";
    #endif
    strSimInfo << "            BC type: " << STR(BC_PROBLEM) << "\n";
    strSimInfo << "--------------------------------------------------------------------------------\n";


    #ifdef NON_NEWTONIAN_FLUID
    strSimInfo << "\n------------------------------ NON NEWTONIAN FLUID -----------------------------\n";
    strSimInfo << std::scientific << std::setprecision(6);
    
    #ifdef POWERLAW
    strSimInfo << "              Model: Power-Law\n";
    strSimInfo << "        Power index: " << N_INDEX << "\n";
    strSimInfo << " Consistency factor: " << K_CONSISTENCY << "\n";
    strSimInfo << "            Gamma 0: " << GAMMA_0 << "\n";
    #endif // POWERLAW

    #ifdef BINGHAM
    strSimInfo << "              Model: Bingham\n";
    strSimInfo << "  Plastic viscosity: " << VISC << "\n";
    strSimInfo << "       Yield stress: " << S_Y << "\n";
    strSimInfo << "      Plastic omega: " << OMEGA_P << "\n";
    #endif // BINGHAM
    strSimInfo << "--------------------------------------------------------------------------------\n";
    #endif // NON_NEWTONIAN_FLUID

    return strSimInfo.str();
}

void saveSimInfo(int step)
{
    std::string strInf = PATH_FILES;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "_info.txt"; // generate file name (with path)
    FILE* outFile = nullptr;

    outFile = fopen(strInf.c_str(), "w");
    if(outFile != nullptr)
    {
        std::string strSimInfo = getSimInfoString(step);
        fprintf(outFile, strSimInfo.c_str());
        fclose(outFile);
    }
    else
    {
        printf("Error saving \"%s\" \nProbably wrong path!\n", strInf.c_str());
    }
    
}
/**/


void saveTreatData(std::string fileName, std::string dataString, int step)
{
    #if SAVEDATA
    std::string strInf = PATH_FILES;
    strInf += "/";
    strInf += ID_SIM;
    strInf += "/";
    strInf += ID_SIM;
    strInf += fileName;
    strInf += ".txt"; // generate file name (with path)
    std::ifstream file(strInf.c_str());
    std::ofstream outfile;

    if(step == MACR_SAVE){ //check if first time step to save data
        outfile.open(strInf.c_str());
    }else{
        if (file.good()) {
            outfile.open(strInf.c_str(), std::ios::app);
        }else{ 
            outfile.open(strInf.c_str());
        }
    }


    outfile << dataString.c_str() << std::endl; 
    outfile.close(); 
    #endif
    #if CONSOLEPRINT
    printf("%s \n",dataString.c_str());
    #endif
}