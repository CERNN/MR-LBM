//#ifdef PARTICLE_MODEL

#include "particle.cuh"
#include <cstdlib>
#include <iostream>


__host__ __device__ Particle::Particle(){
    method = none; // Initialize method
}

__host__ __device__ ParticleMethod Particle::getMethod() const {return this->method;}
__host__ __device__ void Particle::setMethod(ParticleMethod method) { this->method = method;}

__host__ __device__ ParticleCenter* Particle::getPCenter() const {return this->pCenter;}
__host__ __device__ void Particle::setPCenter(ParticleCenter* pCenter) { this->pCenter = pCenter;}

__host__ __device__ const bool& Particle::getCollideParticle() const { return this->collideParticle; }
__host__ __device__ void Particle::setCollideParticle(const bool& value) { this->collideParticle = value; }

__host__ __device__ const bool& Particle::getCollideWall() const { return this->collideWall; }
__host__ __device__ void Particle::setCollideWall(const bool& value) { this->collideWall = value; }

__host__ __device__ ParticleShape* Particle::getShape() const {return this->shape;}
__host__ __device__ void Particle::setShape(ParticleShape* shape) { this->shape = shape;}

// ParticlesSoA class implementation
__host__ 
ParticlesSoA::ParticlesSoA() {
    pCenterArray = nullptr;
    pCenterLastPos = nullptr;
    pCenterLastWPos = nullptr;
    pShape = nullptr;
    pMethod = nullptr;
    pCollideWall = nullptr;
    pCollideParticle = nullptr;
}

__host__ 
ParticlesSoA::~ParticlesSoA() {
    if (pCenterArray) {
        hipFree(pCenterArray);
        pCenterArray = nullptr;
    }
    if (pCenterLastPos) {
        free(pCenterLastPos);
        pCenterLastPos = nullptr;
    }
    if (pCenterLastWPos) {
        free(pCenterLastWPos);
        pCenterLastWPos = nullptr;
    }
    if (pShape) {
        free(pShape);
        pShape = nullptr;
    }
    if (pMethod) {
        free(pMethod);
        pMethod = nullptr;
    }
    if (pCollideWall) {
        free(pCollideWall);
        pCollideWall = nullptr;
    }
    if (pCollideParticle) {
        free(pCollideParticle);
        pCollideParticle = nullptr;
    }
}

__host__ __device__ ParticleCenter* ParticlesSoA::getPCenterArray() const {return this->pCenterArray;}
__host__ __device__ void ParticlesSoA::setPCenterArray(ParticleCenter* pArray) {this->pCenterArray = pArray;}

__host__ __device__ dfloat3* ParticlesSoA::getPCenterLastPos() const {return this->pCenterLastPos;}
__host__ __device__ void ParticlesSoA::setPCenterLastPos(dfloat3* pLastPos) {this->pCenterLastPos = pLastPos;}

__host__ __device__ dfloat3* ParticlesSoA::getPCenterLastWPos() const {return this->pCenterLastWPos;}
__host__ __device__ void ParticlesSoA::setPCenterLastWPos(dfloat3* pLastWPos) {this->pCenterLastWPos = pLastWPos;}

__host__ __device__ ParticleShape* ParticlesSoA::getPShape() const {return this->pShape;}
__host__ __device__ void ParticlesSoA::setPShape(ParticleShape* pShape) {this->pShape = pShape;}

__host__ __device__ ParticleMethod* ParticlesSoA::getPMethod() const {return this->pMethod;}
__host__ __device__ void ParticlesSoA::setPMethod(ParticleMethod* pMethod) {this->pMethod = pMethod;}

__host__ __device__ bool* ParticlesSoA::getPCollideWall() const {return this->pCollideWall;}
__host__ __device__ void ParticlesSoA::setPCollideWall(bool* pMethod) {this->pCollideWall = pCollideWall;}

__host__ __device__ bool* ParticlesSoA::getPCollideParticle() const {return this->pCollideParticle;}
__host__ __device__ void ParticlesSoA::setPCollideParticle(bool* pMethod) {this->pCollideParticle = pCollideParticle;}

__device__ __host__
const MethodRange& ParticlesSoA::getMethodRange(ParticleMethod method) const {
    static const MethodRange empty{-1, -1};
    auto it = methodRanges.find(method);
    return (it != methodRanges.end()) ? it->second : empty;
}

__device__ __host__
void ParticlesSoA::setMethodRange(ParticleMethod method, int first, int last) {
    methodRanges[method] = {first, last};
}

__device__ __host__
int ParticlesSoA::getMethodCount(ParticleMethod method) const {
    MethodRange range = getMethodRange(method);
    if (range.first == -1 || range.last == -1 || range.last < range.first)
        return 0;  // No particles of this method
    return range.last - range.first + 1;
}

__host__ void ParticlesSoA::createParticles(Particle *particles){
   
    #include CASE_PARTICLE_CREATE
    if (pShape == nullptr) {
        pShape = new ParticleShape[NUM_PARTICLES]; 
        for (int i = 0; i < NUM_PARTICLES; i++) {
            pShape[i] = SPHERE;
            particles[i].setShape(&pShape[i]);
        }
    }

    for(int i = 0; i <NUM_PARTICLES ; i++){

        switch (particles[i].getMethod())
        {
        case PIBM:
            /* code */
            break;
        case IBM:
            //particles[i].makeSpherePolar(PARTICLE_DIAMETER, center[i], MESH_COULOMB, true, PARTICLE_DENSITY, vel, w);
          //  particles[i].makeEllipsoid(dfloat3(40.0,20.0,10.0), center[i], dfloat3(0.5,1.0,0.6), 0.3*M_PI/4,true, PARTICLE_DENSITY, vel, w);
            //particles[i].makeCapsule(PARTICLE_DIAMETER, center1, center2, true,PARTICLE_DENSITY, vel, w);
            break;
        case TRACER:
            break;
        default:
            break;
        }
        
    }
}

__host__ __device__ void ParticlesSoA::updateParticlesAsSoA(Particle* particles){
    if (particles == nullptr) {
        printf("ERROR: particles is nullptr!\n\n"); fflush(stdout);
        return;
    }
    if (NUM_PARTICLES <= 0) {
        printf("ERROR: Invalid NUM_PARTICLES!\n\n"); fflush(stdout);
        return;
    }

    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipMallocManaged((void**)&this->pCenterArray,       sizeof(ParticleCenter) * NUM_PARTICLES));
    checkCudaErrors(hipMallocManaged((void**)&this->pCenterLastPos,     sizeof(dfloat3)        * NUM_PARTICLES));
    checkCudaErrors(hipMallocManaged((void**)&this->pCenterLastWPos,    sizeof(dfloat3)        * NUM_PARTICLES));
    checkCudaErrors(hipMallocManaged((void**)&this->pShape,             sizeof(ParticleShape)  * NUM_PARTICLES));
    checkCudaErrors(hipMallocManaged((void**)&this->pMethod,            sizeof(ParticleMethod) * NUM_PARTICLES));
    checkCudaErrors(hipMallocManaged((void**)&this->pCollideWall,       sizeof(bool)           * NUM_PARTICLES));
    checkCudaErrors(hipMallocManaged((void**)&this->pCollideParticle,   sizeof(bool)           * NUM_PARTICLES));

    if (!this->pCenterArray || !pCenterLastPos || !pCenterLastWPos ||
        !this->pShape || !this->pMethod || !this->pCollideWall || !this->pCollideParticle) {
        printf("ERRO: Memory allocation failed!!\n"); fflush(stdout);
        return;
    }

    auto insertByMethod = [&](ParticleMethod method) {
        int firstIndex = -1;
        int lastIndex = -1;

        for (int p = 0; p < NUM_PARTICLES; ++p) {     
            if (particles[p].getMethod() != method)
                continue; // <- Adicionado: só copia se for do tipo correto
            ParticleCenter* pc = particles[p].getPCenter();

            if (!pc) {
                printf("NOTICE: Particle %d com pc == nullptr\n", p); fflush(stdout);
                continue;
            }

            this->pCenterArray[p]       = *pc;
            this->pCenterLastPos[p]     = pc->getPos_old();
            this->pCenterLastWPos[p]    = pc->getW_old();
            this->pShape[p]             = *(particles[p].getShape());
            this->pMethod[p]            = particles[p].getMethod();
            this->pCollideWall[p]       = particles[p].getCollideWall();
            this->pCollideParticle[p]   = particles[p].getCollideParticle();

            if (firstIndex == -1) firstIndex = p;
            lastIndex = p;
        }
       if (firstIndex != -1) {
              this->setMethodRange(method, firstIndex, lastIndex);
        }
     };

    insertByMethod(IBM);
    insertByMethod(PIBM);
    insertByMethod(TRACER);
    checkCudaErrors(hipSetDevice(0));

}


#ifdef IBM
#endif // !IBM


//#endif //PARTICLE_MODEL