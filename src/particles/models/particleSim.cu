

#include "particleSim.cuh"

#ifdef PARTICLE_MODEL

void particleSimulation(
    ParticlesSoA *particles,
    dfloat *fMom,
    hipStream_t *streamParticles,
    unsigned int step
){

    int numIBM    = particles->getMethodCount(IBM);
    int numPIBM   = particles->getMethodCount(PIBM);
    int numTRACER = particles->getMethodCount(TRACER);

    //printf("Number of particles IBM %d PIBM %d Tracer %d \n",numIBM,numPIBM,numTRACER);

    if(numIBM>0){
       ibmSimulation(particles,fMom,streamParticles[0],step);
    }
    if(numPIBM>0){
        pibmSimulation(particles,fMom,streamParticles[0],step);
    }
    if(numTRACER>0){
        tracerSimulation(particles,fMom,streamParticles[0],step);
    }

}

#endif //PARTICLE_MODEL