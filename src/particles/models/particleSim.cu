

#include "particleSim.cuh"

#ifdef PARTICLE_MODEL

void particleSimulation(
    ParticlesSoA *particles,
    dfloat *fMom,
    hipStream_t *streamParticles,
    unsigned int step
){

    int numIBM    = particles->getMethodCount(IBM);
    int numPIBM   = particles->getMethodCount(PIBM);
    int numTRACER = particles->getMethodCount(TRACER);

    if(numIBM>0){
       ibmSimulation(particles,fMom,streamParticles[0],step);
    }
    if(numPIBM>0){
        pibmSimulation(particles,fMom,streamParticles[0],step);
    }
    if(numTRACER>0){
        tracerSimulation(particles,fMom,streamParticles[0],step);
    }

}

#endif //PARTICLE_MODEL