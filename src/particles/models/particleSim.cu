//#ifdef PARTICLE_MODEL

#include "particleSim.cuh"

void particleSimulation(
    ParticlesSoA *particles,
    dfloat *fMom,
    hipStream_t *streamParticles,
    unsigned int step
){

    int numIBM    = particles->getMethodCount(IBM);
    int numPIBM   = particles->getMethodCount(PIBM);
    int numTRACER = particles->getMethodCount(TRACER);

    if(numIBM>0){
        /*code*/
    }
    if(numPIBM>0){
        /*code*/
    }
    if(numTRACER>0){
        tracerSimulation(particles,fMom,streamParticles[0],step);
    }

}

//#endif //PARTICLE_MODEL