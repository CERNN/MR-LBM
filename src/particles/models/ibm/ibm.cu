#include "hip/hip_runtime.h"
// #ifdef PARTICLE_MODEL

#include "ibm.cuh"

// Functions for the immersed boundary method
//void gpuForceInterpolationSpread();
//gpuResetNodesForces
//gpuParticleNodeMovement

__global__
void gpuUpdateParticleOldValues(
    ParticleCenter particleCenters[NUM_PARTICLES])
{
    unsigned int p = threadIdx.x + blockDim.x * blockIdx.x;

    if(p >= NUM_PARTICLES)
        return;

    ParticleCenter *pc = &(particleCenters[p]);

    // Internal linear momentum delta = rho*volume*delta(v)/delta(t)
    // https://doi.org/10.1016/j.compfluid.2011.05.011
    pc->setDPInternalX(0.0); //RHO_0 * pc->getVolume() * (pc->getVelX() - pc->getVelOldX());
    pc->setDPInternalY(0.0); //RHO_0 * pc->getVolume() * (pc->getVelY() - pc->getVelOldY());
    pc->setDPInternalY(0.0); //RHO_0 * pc->getVolume() * (pc->getVelZ() - pc->getVelOldZ());

    // Internal angular momentum delta = (rho_f/rho_p)*I*delta(omega)/delta(t)
    // https://doi.org/10.1016/j.compfluid.2011.05.011
    
    pc->setDLInternalX(0.0); //(RHO_0 / pc->getDensity()) * pc->getIXX() * (pc->getWX() - pc->getWOldX());
    pc->setDLInternalX(0.0); //(RHO_0 / pc->getDensity()) * pc->getIYY() * (pc->getWY() - pc->getWOldY());
    pc->setDLInternalX(0.0); //(RHO_0 / pc->getDensity()) * pc->getIZZ() * (pc->getWZ() - pc->getWOldZ());

    pc->setPosOldX(pc->getPosX());
    pc->setPosOldY(pc->getPosY());
    pc->setPosOldZ(pc->getPosZ());

    pc->setVelOldX(pc->getVelX());
    pc->setVelOldY(pc->getVelY());
    pc->setVelOldZ(pc->getVelZ());

    pc->setWOldX(pc->getWX());
    pc->setWOldY(pc->getWY());
    pc->setWOldZ(pc->getWZ());

    pc->setFOldX(pc->getFX());
    pc->setFOldY(pc->getFY());
    pc->setFOldZ(pc->getFZ());

    // Reset force, because kernel is always added
    pc->setFX(0);
    pc->setFY(0);
    pc->setFZ(0);

    pc->setMX(0);
    pc->setMY(0);
    pc->setMZ(0);
}

__global__ 
void gpuResetNodesForces(IbmNodesSoA* particlesNodes)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= particlesNodes->getNumNodes())
        return;

    const dfloat3SoA force = particlesNodes->getF();
    const dfloat3SoA delta_force = particlesNodes->getDeltaF();

    force.x[idx] = 0;
    force.y[idx] = 0;
    force.z[idx] = 0;
    delta_force.x[idx] = 0;
    delta_force.y[idx] = 0;
    delta_force.z[idx] = 0;
}

__global__ 
void gpuUpdateParticleCenterVelocityAndRotation(
    ParticleCenter particleCenters[NUM_PARTICLES])
{
    unsigned int p = threadIdx.x + blockDim.x * blockIdx.x;

    if(p >= NUM_PARTICLES)
        return;

    ParticleCenter *pc = &(particleCenters[p]);

    #ifdef IBM_DEBUG
    printf("gpuUpdateParticleCenterVelocityAndRotation 1 pos  x: %f y: %f z: %f\n",pc->pos.x,pc->pos.y,pc->pos.z);
    printf("gpuUpdateParticleCenterVelocityAndRotation 1 vel  x: %f y: %f z: %f\n",pc->vel.x,pc->vel.y,pc->vel.z);
    printf("gpuUpdateParticleCenterVelocityAndRotation 1 w  x: %f y: %f z: %f\n",pc->w.x,pc->w.y,pc->w.z);
    printf("gpuUpdateParticleCenterVelocityAndRotation 1 f  x: %f y: %f z: %f\n",pc->f.x,pc->f.y,pc->f.z);
    printf("gpuUpdateParticleCenterVelocityAndRotation 1 f_old  x: %f y: %f z: %f\n",pc->f_old.x,pc->f_old.y,pc->f_old.z);
    printf("gpuUpdateParticleCenterVelocityAndRotation 1 dP_internal  x: %f y: %f z: %f\n",pc->dP_internal.x,pc->dP_internal.y,pc->dP_internal.z);
    #endif

    if(!pc->getMovable())
        return;

    const dfloat inv_volume = 1 / pc->getVolume();

    // Update particle center velocity using its surface forces and the body forces
    
    pc->setVelX(pc->getVelOldX() + (( (pc->getFOldX() * (1.0 - IBM_MOVEMENT_DISCRETIZATION ) 
                + pc->getFX() * IBM_MOVEMENT_DISCRETIZATION) + pc->getDPInternalX()) * inv_volume 
                + (pc->getDensity() - FLUID_DENSITY) * GX) / (pc->getDensity()));
    pc->setVelY(pc->getVelOldY() + (( (pc->getFOldY() * (1.0 - IBM_MOVEMENT_DISCRETIZATION )  
                + pc->getFY() * IBM_MOVEMENT_DISCRETIZATION) + pc->getDPInternalY()) * inv_volume 
                + (pc->getDensity() - FLUID_DENSITY) * GY) / (pc->getDensity()));
    pc->setVelZ(pc->getVelOldZ() + (( (pc->getFOldZ() * (1.0 - IBM_MOVEMENT_DISCRETIZATION )  
                + pc->getFZ() * IBM_MOVEMENT_DISCRETIZATION) + pc->getDPInternalZ()) * inv_volume 
                + (pc->getDensity() - FLUID_DENSITY) * GZ) / (pc->getDensity()));

    // Auxiliary variables for angular velocity update
    dfloat error = 1;
    dfloat3 wNew = dfloat3(), wAux;
    const dfloat3 M = pc->getM();
    const dfloat3 M_old = pc->getM_old();
    const dfloat3 w_old = pc->getW_old();
    dfloat6 I = pc->getI();

    dfloat6 Iaux6;
    dfloat4 q_rot;

    wAux.x = w_old.x;
    wAux.y = w_old.y;
    wAux.z = w_old.z;

    //dfloat I_det_neg = (I.zz*I.xy*I.xy + I.yy*I.xz*I.xz + I.xx*I.yz*I.yz - I.xx*I.yy*I.zz - 2*I.xy*I.xz*I.yz);
    dfloat inv_I_det_neg = 1.0/(I.zz*I.xy*I.xy + I.yy*I.xz*I.xz + I.xx*I.yz*I.yz - I.xx*I.yy*I.zz - 2*I.xy*I.xz*I.yz);
    dfloat3 wAvg, LM_avg, M_avg;

    wAvg.x = (w_old.x * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.x * IBM_MOVEMENT_DISCRETIZATION);
    wAvg.y = (w_old.y * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.y * IBM_MOVEMENT_DISCRETIZATION);
    wAvg.z = (w_old.z * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.z * IBM_MOVEMENT_DISCRETIZATION);

    LM_avg.x = pc->getDLInternalX() + (M.x * IBM_MOVEMENT_DISCRETIZATION + M_old.x * (1.0 - IBM_MOVEMENT_DISCRETIZATION));
    LM_avg.y = pc->getDLInternalY() + (M.y * IBM_MOVEMENT_DISCRETIZATION + M_old.y * (1.0 - IBM_MOVEMENT_DISCRETIZATION));
    LM_avg.z = pc->getDLInternalZ() + (M.z * IBM_MOVEMENT_DISCRETIZATION + M_old.z * (1.0 - IBM_MOVEMENT_DISCRETIZATION));

    //OLD CODE
    // Iteration process to upadate angular velocity 
    // (Crank-Nicolson implicit scheme)
    //for (int i = 0; error > 1e-4; i++)
    {
        //TODO the last term should be present in dL equation, but since it does not affect spheres, it will stay for now.
        /*
        wNew.x = pc->getWOldX() + (((M.x * IBM_MOVEMENT_DISCRETIZATION + M_old.x * (1.0 - IBM_MOVEMENT_DISCRETIZATION)) + pc->getDLInternalX()) 
                - (I.zz - I.yy)*(w_old.y * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.y * IBM_MOVEMENT_DISCRETIZATION ) 
                               *(w_old.z * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.z * IBM_MOVEMENT_DISCRETIZATION))/I.xx;
        wNew.y = pc->getWOldY() + (((M.y * IBM_MOVEMENT_DISCRETIZATION + M_old.y * (1.0 - IBM_MOVEMENT_DISCRETIZATION)) + pc->getDLInternalY()) 
                - (I.xx - I.zz)*(w_old.x * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.x * IBM_MOVEMENT_DISCRETIZATION ) 
                               *(w_old.z * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.z * IBM_MOVEMENT_DISCRETIZATION))/I.yy;
        wNew.z = pc->getWOldZ() + (((M.z * IBM_MOVEMENT_DISCRETIZATION + M_old.z * (1.0 - IBM_MOVEMENT_DISCRETIZATION)) + pc->getDLInternalZ()) 
                - (I.yy - I.xx)*(w_old.x * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.x * IBM_MOVEMENT_DISCRETIZATION ) 
                               *(w_old.y * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.y * IBM_MOVEMENT_DISCRETIZATION))/I.zz;
        */


        wNew.x = pc->getWOldX() + ((I.yz*I.yz - I.yy*I.zz)*(LM_avg.x + (wAvg.z)*(I.xy*wAvg.x + I.yy*wAvg.y + I.yz*wAvg.z) - (wAvg.y)*(I.xz*wAvg.x + I.yz*wAvg.y + I.zz*wAvg.z))
                              - (I.xy*I.yz - I.xz*I.yy)*(LM_avg.z + (wAvg.y)*(I.xx*wAvg.x + I.xy*wAvg.y + I.xz*wAvg.z) - (wAvg.x)*(I.xy*wAvg.x + I.yy*wAvg.y + I.yz*wAvg.z))
                              - (I.xz*I.yz - I.xy*I.zz)*(LM_avg.y + (wAvg.x)*(I.xz*wAvg.x + I.yz*wAvg.y + I.zz*wAvg.z) - (wAvg.z)*(I.xx*wAvg.x + I.xy*wAvg.y + I.xz*wAvg.z)))*inv_I_det_neg;
        wNew.y = pc->getWOldY() + ((I.xz*I.xz - I.xx*I.zz)*(LM_avg.y + (wAvg.x)*(I.xz*wAvg.x + I.yz*wAvg.y + I.zz*wAvg.z) - (wAvg.z)*(I.xx*wAvg.x + I.xy*wAvg.y + I.xz*wAvg.z))
                              - (I.xy*I.xz - I.xx*I.yz)*(LM_avg.z + (wAvg.y)*(I.xx*wAvg.x + I.xy*wAvg.y + I.xz*wAvg.z) - (wAvg.x)*(I.xy*wAvg.x + I.yy*wAvg.y + I.yz*wAvg.z))
                              - (I.xz*I.yz - I.xy*I.zz)*(LM_avg.x + (wAvg.z)*(I.xy*wAvg.x + I.yy*wAvg.y + I.yz*wAvg.z) - (wAvg.y)*(I.xz*wAvg.x + I.yz*wAvg.y + I.zz*wAvg.z)))*inv_I_det_neg;
        wNew.z = pc->getWOldZ() + ((I.xy*I.xy - I.xx*I.yy)*(LM_avg.z + (wAvg.y)*(I.xx*wAvg.x + I.xy*wAvg.y + I.xz*wAvg.z) - (wAvg.x)*(I.xy*wAvg.x + I.yy*wAvg.y + I.yz*wAvg.z))
                              - (I.xy*I.xz - I.xx*I.yz)*(LM_avg.y + (wAvg.x)*(I.xz*wAvg.x + I.yz*wAvg.y + I.zz*wAvg.z) - (wAvg.z)*(I.xx*wAvg.x + I.xy*wAvg.y + I.xz*wAvg.z))
                              - (I.xy*I.yz - I.xz*I.yy)*(LM_avg.x + (wAvg.z)*(I.xy*wAvg.x + I.yy*wAvg.y + I.yz*wAvg.z) - (wAvg.y)*(I.xz*wAvg.x + I.yz*wAvg.y + I.zz*wAvg.z)))*inv_I_det_neg;
                              
        //inertia update
        wAvg.x = (w_old.x * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.x * IBM_MOVEMENT_DISCRETIZATION);
        wAvg.y = (w_old.y * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.y * IBM_MOVEMENT_DISCRETIZATION);
        wAvg.z = (w_old.z * (1.0 - IBM_MOVEMENT_DISCRETIZATION) + wAux.z * IBM_MOVEMENT_DISCRETIZATION);

        //calculate rotation quartention
        q_rot = axis_angle_to_quart(wAvg,vector_length(wAvg));

        //compute new moment of inertia       
        Iaux6 = rotate_inertia_by_quart(q_rot,I);

        error =  (Iaux6.xx-I.xx)*(Iaux6.xx-I.xx)/(Iaux6.xx*Iaux6.xx);
        error += (Iaux6.yy-I.yy)*(Iaux6.yy-I.yy)/(Iaux6.yy*Iaux6.yy);
        error += (Iaux6.zz-I.zz)*(Iaux6.zz-I.zz)/(Iaux6.zz*Iaux6.zz);
        error += (Iaux6.xy-I.xy)*(Iaux6.xy-I.xy)/(Iaux6.xy*Iaux6.xy);
        error += (Iaux6.xz-I.xz)*(Iaux6.xz-I.xz)/(Iaux6.xz*Iaux6.xz);
        error += (Iaux6.yz-I.yz)*(Iaux6.yz-I.yz)/(Iaux6.yz*Iaux6.yz);

        //printf("error: %e \n",error);

        wAux.x = wNew.x;
        wAux.y = wNew.y;
        wAux.z = wNew.z;

        I.xx = Iaux6.xx;
        I.yy = Iaux6.yy;
        I.zz = Iaux6.zz;
        I.xy = Iaux6.xy;
        I.xz = Iaux6.xz;
        I.yz = Iaux6.yz;
    
       }

    // Store new velocities in particle center
    pc->setWX(wNew.x);
    pc->setWY(wNew.y);
    pc->setWZ(wNew.z);

    pc->setIXX(Iaux6.xx);
    pc->setIYY(Iaux6.yy);
    pc->setIZZ(Iaux6.zz);
    pc->setIXY(Iaux6.xy);
    pc->setIXZ(Iaux6.xz);
    pc->setIYZ(Iaux6.yz);

    #ifdef IBM_DEBUG
    printf("gpuUpdateParticleCenterVelocityAndRotation 2 pos  x: %f y: %f z: %f\n",pc->pos.x,pc->pos.y,pc->pos.z);
    printf("gpuUpdateParticleCenterVelocityAndRotation 2 vel  x: %f y: %f z: %f\n",pc->vel.x,pc->vel.y,pc->vel.z);
    printf("gpuUpdateParticleCenterVelocityAndRotation 2 w  x: %f y: %f z: %f\n",pc->w.x,pc->w.y,pc->w.z);
    #endif
}

__global__
void gpuParticleMovement(
    ParticleCenter particleCenters[NUM_PARTICLES])
{
    unsigned int p = threadIdx.x + blockDim.x * blockIdx.x;

    if(p >= NUM_PARTICLES)
        return;

    ParticleCenter *pc = &(particleCenters[p]);

    #ifdef IBM_DEBUG
    printf("gpuParticleMovement 1 pos  x: %f y: %f z: %f\n",pc->pos.x,pc->pos.y,pc->pos.z);
    printf("gpuParticleMovement 1 vel  x: %f y: %f z: %f\n",pc->vel.x,pc->vel.y,pc->vel.z);
    printf("gpuParticleMovement 1 w  x: %f y: %f z: %f\n",pc->w.x,pc->w.y,pc->w.z);
    #endif
    
    
    if(!pc->getMovable())
        return;

    #ifdef IBM_BC_X_WALL
        pc->setPosX((pc->getPosX() + (pc->getVelX() * IBM_MOVEMENT_DISCRETIZATION + pc->getVelOldX() * (1.0 - IBM_MOVEMENT_DISCRETIZATION))));
    #endif //IBM_BC_X_WALL
    #ifdef IBM_BC_X_PERIODIC
        dfloat dx =  (pc->getVelX() * IBM_MOVEMENT_DISCRETIZATION + pc->getVelOldX() * (1.0 - IBM_MOVEMENT_DISCRETIZATION));
        pc->setPosX(IBM_BC_X_0 + std::fmod((dfloat)(pc->getPosX() + dx + IBM_BC_X_E - IBM_BC_X_0 - IBM_BC_X_0) , (dfloat)(IBM_BC_X_E - IBM_BC_X_0))); 
    #endif //IBM_BC_X_PERIODIC

    #ifdef IBM_BC_Y_WALL
        pc->setPosY((pc->getPosY() + (pc->getVelY() * IBM_MOVEMENT_DISCRETIZATION + pc->getVelOldY() * (1.0 - IBM_MOVEMENT_DISCRETIZATION))));
    #endif //IBM_BC_Y_WALL
    #ifdef IBM_BC_Y_PERIODIC
        dfloat dy =  (pc->getVelY() * IBM_MOVEMENT_DISCRETIZATION + pc->getVelOldY() * (1.0 - IBM_MOVEMENT_DISCRETIZATION));
        pc->setPosY(IBM_BC_Y_0 + std::fmod((dfloat)(pc->getPosY() + dy + IBM_BC_Y_E - IBM_BC_Y_0 - IBM_BC_Y_0) , (dfloat)(IBM_BC_Y_E - IBM_BC_Y_0)));
    #endif //IBM_BC_Y_PERIODIC

    // #ifdef IBM_BC_Z_WALL
        pc->setPosZ((pc->getPosZ() + (pc->getVelZ() * IBM_MOVEMENT_DISCRETIZATION + pc->getVelOldZ() * (1.0 - IBM_MOVEMENT_DISCRETIZATION))));
    // #endif //IBM_BC_Z_WALL
    #ifdef IBM_BC_Z_PERIODIC
        dfloat dz =  (pc->getVelZ() * IBM_MOVEMENT_DISCRETIZATION + pc->getVelOldZ() * (1.0 - IBM_MOVEMENT_DISCRETIZATION));
        pc->setPosZ(IBM_BC_Z_0 + std::fmod((dfloat)(pc->getPosZ() + dz + IBM_BC_Z_E - IBM_BC_Z_0 - IBM_BC_Z_0) , (dfloat)(IBM_BC_Z_E - IBM_BC_Z_0))); 
    #endif //IBM_BC_Z_PERIODIC

    pc->setWAvgX((pc->getWX()   * IBM_MOVEMENT_DISCRETIZATION + pc->getWOldX()   * (1.0 - IBM_MOVEMENT_DISCRETIZATION)));
    pc->setWAvgY((pc->getWY()   * IBM_MOVEMENT_DISCRETIZATION + pc->getWOldY()   * (1.0 - IBM_MOVEMENT_DISCRETIZATION)));
    pc->setWAvgZ((pc->getWZ()   * IBM_MOVEMENT_DISCRETIZATION + pc->getWOldZ()   * (1.0 - IBM_MOVEMENT_DISCRETIZATION)));
    pc->setWPosX(pc->getWAvgX());
    pc->setWPosY(pc->getWAvgY());
    pc->setWPosZ(pc->getWAvgZ());

    #ifdef IBM_DEBUG
    printf("gpuParticleMovement 2 pos  x: %f y: %f z: %f\n",pc->pos.x,pc->pos.y,pc->pos.z);
    printf("gpuParticleMovement 2 vel  x: %f y: %f z: %f\n",pc->vel.x,pc->vel.y,pc->vel.z);
    printf("gpuParticleMovement 2 w  x: %f y: %f z: %f\n",pc->w.x,pc->w.y,pc->w.z);
    #endif


    //update orientation vector
    const dfloat w_norm = sqrt((pc->getWAvgX() * pc->getWAvgX()) 
                             + (pc->getWAvgY() * pc->getWAvgY()) 
                             + (pc->getWAvgZ() * pc->getWAvgZ()));
    const dfloat q0 = cos(0.5*w_norm);
    const dfloat qi = (pc->getWAvgX()/w_norm) * sin (0.5*w_norm);
    const dfloat qj = (pc->getWAvgY()/w_norm) * sin (0.5*w_norm);
    const dfloat qk = (pc->getWAvgZ()/w_norm) * sin (0.5*w_norm);
    const dfloat tq0m1 = (q0*q0) - 0.5;

    dfloat x_vec = pc->getSemiAxis1X() - pc->getPosOldX();
    dfloat y_vec = pc->getSemiAxis1Y() - pc->getPosOldY();
    dfloat z_vec = pc->getSemiAxis1Z() - pc->getPosOldZ();

    
    pc->setSemiAxis1X(pc->getPosX() + 2 * (   (tq0m1 + (qi*qi))*x_vec + ((qi*qj) - (q0*qk))*y_vec + ((qi*qk) + (q0*qj))*z_vec));
    pc->setSemiAxis1Y(pc->getPosY() + 2 * ( ((qi*qj) + (q0*qk))*x_vec +   (tq0m1 + (qj*qj))*y_vec + ((qj*qk) - (q0*qi))*z_vec));
    pc->setSemiAxis1Z(pc->getPosZ() + 2 * ( ((qi*qj) - (q0*qj))*x_vec + ((qj*qk) + (q0*qi))*y_vec +   (tq0m1 + (qk*qk))*z_vec));

    x_vec = pc->getSemiAxis2X() - pc->getPosOldX();
    y_vec = pc->getSemiAxis2Y() - pc->getPosOldY();
    z_vec = pc->getSemiAxis2Z() - pc->getPosOldZ();

    pc->setSemiAxis2X(pc->getPosX() +  2 * (   (tq0m1 + (qi*qi))*x_vec + ((qi*qj) - (q0*qk))*y_vec + ((qi*qk) + (q0*qj))*z_vec));
    pc->setSemiAxis2Y(pc->getPosY() +  2 * ( ((qi*qj) + (q0*qk))*x_vec +   (tq0m1  + (qj*qj))*y_vec + ((qj*qk) - (q0*qi))*z_vec));
    pc->setSemiAxis2Z(pc->getPosZ() +  2 * ( ((qi*qj) - (q0*qj))*x_vec + ((qj*qk) + (q0*qi))*y_vec +   (tq0m1  + (qk*qk))*z_vec));

    x_vec = pc->getSemiAxis3X() - pc->getPosOldX();
    y_vec = pc->getSemiAxis3Y() - pc->getPosOldY();
    z_vec = pc->getSemiAxis3Z() - pc->getPosOldZ();

    pc->setSemiAxis3X(pc->getPosX() +  2 * (   (tq0m1 + (qi*qi))*x_vec + ((qi*qj) - (q0*qk))*y_vec + ((qi*qk) + (q0*qj))*z_vec));
    pc->setSemiAxis3X(pc->getPosY() +  2 * ( ((qi*qj) + (q0*qk))*x_vec +   (tq0m1  + (qj*qj))*y_vec + ((qj*qk) - (q0*qi))*z_vec));
    pc->setSemiAxis3X(pc->getPosZ() +  2 * ( ((qi*qj) - (q0*qj))*x_vec + ((qj*qk) + (q0*qi))*y_vec +   (tq0m1  + (qk*qk))*z_vec));

}


void ibmSimulation(
    ParticlesSoA particles,
    dfloat *fMom,
    hipStream_t streamParticles,
    unsigned int step
){
    checkCudaErrors(hipSetDevice(GPUS_TO_USE[0]));
    gpuUpdateParticleOldValues<<<GRID_PARTICLES_IBM, THREADS_PARTICLES_IBM, 0, streamParticles>>>(
        particles.getPCenterArray());
    checkCudaErrors(hipStreamSynchronize(streamParticles));

     // Grid for only  z-borders
    //  dim3 copyMacrGrid = gridLBM;
     // Grid for full domain, including z-borders
    //  dim3 borderMacrGrid = gridLBM; 
     // Only 1 in z
    //  copyMacrGrid.z = MACR_BORDER_NODES;
    //  borderMacrGrid.z += MACR_BORDER_NODES*2;
 
    unsigned int gridNodesIBM[N_GPUS];
    unsigned int threadsNodesIBM[N_GPUS];
    for(int i = 0; i < N_GPUS; i++){
        threadsNodesIBM[i] = 64;
        checkCudaErrors(hipSetDevice(GPUS_TO_USE[i]));
        unsigned int pNumNodes = particles.getNodesSoA()->getNumNodes();
        gridNodesIBM[i] = pNumNodes % threadsNodesIBM[i] ? pNumNodes / threadsNodesIBM[i] + 1 : pNumNodes / threadsNodesIBM[i];
    }
    checkCudaErrors(hipSetDevice(GPUS_TO_USE[0]));

    for(int i = 0; i < N_GPUS; i++){
        checkCudaErrors(hipSetDevice(GPUS_TO_USE[i]));
        //int nxt = (i+1) % N_GPUS;
        //Copy macroscopics
        //gpuCopyBorderMacr<<<copyMacrGrid, threadsLBM, 0, streamLBM[i]>>>(macr[i], macr[nxt]); Verificar se é necessário
        checkCudaErrors(hipStreamSynchronize(streamParticles));
        getLastCudaError("Copy macroscopics border error\n");
        // If GPU has nodes in it
        if(particles.getNodesSoA()->getNumNodes() > 0){
            // Reset forces in all IBM nodes;
            gpuResetNodesForces<<<gridNodesIBM[i], threadsNodesIBM[i], 0, streamParticles>>>(particles.getNodesSoA());
            checkCudaErrors(hipStreamSynchronize(streamParticles));
            getLastCudaError("Reset IBM nodes forces error\n");
        }
    }

     // Calculate collision force between particles
     checkCudaErrors(hipSetDevice(GPUS_TO_USE[0]));
     checkCudaErrors(hipStreamSynchronize(streamParticles)); 
 
     // First update particle velocity using body center force and constant forces
     checkCudaErrors(hipSetDevice(GPUS_TO_USE[0]));
     gpuUpdateParticleCenterVelocityAndRotation <<<GRID_PARTICLES_IBM, THREADS_PARTICLES_IBM, 0, streamParticles >>>(
         particles.getPCenterArray());
     getLastCudaError("IBM update particle center velocity error\n");
     checkCudaErrors(hipStreamSynchronize(streamParticles));
 
    //  for (int i = 0; i < IBM_MAX_ITERATION; i++)
    //  {
    //      for(int j = 0; j < N_GPUS; j++){
    //          // If GPU has nodes in it
    //          if(particles.nodesSoA[j].numNodes > 0){
    //              checkCudaErrors(hipSetDevice(GPUS_TO_USE[j]));
    //              // Make the interpolation of LBM and spreading of IBM forces
    //              gpuForceInterpolationSpread<<<gridNodesIBM[j], threadsNodesIBM[j], 
    //                  0, streamIBM[j]>>>(
    //                  particles.nodesSoA[j], particles.pCenterArray, macr[j], ibmMacrsAux, j);
    //              checkCudaErrors(hipStreamSynchronize(streamIBM[j]));
    //              getLastCudaError("IBM interpolation spread error\n");
    //          }
    //      }
 
    //      checkCudaErrors(hipSetDevice(GPUS_TO_USE[0]));
    //      // Update particle velocity using body center force and constant forces
    //      // Migrar
    //      gpuUpdateParticleCenterVelocityAndRotation<<<GRID_PARTICLES_IBM, THREADS_PARTICLES_IBM, 0, streamIBM[0]>>>(
    //          particles.pCenterArray);
    //      checkCudaErrors(hipStreamSynchronize(streamIBM[0]));
    //      getLastCudaError("IBM update particle center velocity error\n");
 
    //      // Sum border macroscopics
    //      // for(int j = 0; j < N_GPUS; j++){
    //      //     checkCudaErrors(hipSetDevice(GPUS_TO_USE[j]));
    //      //     int nxt = (j+1) % N_GPUS;
    //      //     int prv = (j-1+N_GPUS) % N_GPUS;
    //      //     bool run_nxt = nxt != 0;
    //      //     bool run_prv = prv != (N_GPUS-1);
    //      //     #ifdef IBM_BC_Z_PERIODIC
    //      //     run_nxt = true;
    //      //     run_prv = true;
    //      //     #endif
             
    //      //     if(run_nxt){
    //      //         gpuSumBorderMacr<<<copyMacrGrid, threadsLBM, 0, streamLBM[j]>>>(macr[nxt], ibmMacrsAux, j, 1);
    //      //         checkCudaErrors(hipStreamSynchronize(streamLBM[j]));
    //      //     }
    //      //     if(run_prv){
    //      //         gpuSumBorderMacr<<<copyMacrGrid, threadsLBM, 0, streamLBM[j]>>>(macr[prv], ibmMacrsAux, j, -1);
    //      //         checkCudaErrors(hipStreamSynchronize(streamLBM[j]));
    //      //     }
    //      //     getLastCudaError("Sum border macroscopics error\n");
    //      // }
 
    //      // #if IBM_EULER_OPTIMIZATION
 
    //      // for(int j = 0; j < N_GPUS; j++){
    //      //     if(pEulerNodes->currEulerNodes[j] > 0){
    //      //         checkCudaErrors(hipSetDevice(GPUS_TO_USE[j]));
    //      //         dim3 currGrid(pEulerNodes->currEulerNodes[j]/64+(pEulerNodes->currEulerNodes[j]%64? 1 : 0), 1, 1);
    //      //         gpuEulerSumIBMAuxsReset<<<currGrid, 64, 0, streamLBM[j]>>>(macr[j], ibmMacrsAux,
    //      //             pEulerNodes->eulerIndexesUpdate[j], pEulerNodes->currEulerNodes[j], j);
    //      //         checkCudaErrors(hipStreamSynchronize(streamLBM[j]));
    //      //         getLastCudaError("IBM sum auxiliary values error\n");
    //      //     }
    //      // }
    //      // #else
    //      // for(int j = 0; j < N_GPUS; j++){
    //      //     checkCudaErrors(hipSetDevice(GPUS_TO_USE[j]));
    //      //     gpuEulerSumIBMAuxsReset<<<borderMacrGrid, threadsLBM, 0, streamLBM[j]>>>(macr[j], ibmMacrsAux, j);
    //      //     checkCudaErrors(hipStreamSynchronize(streamLBM[j]));
    //      // }
    //      // #endif
 
    //  }

    checkCudaErrors(hipSetDevice(GPUS_TO_USE[0]));
    // Update particle center position and its old values
    // Migrar
    gpuParticleMovement<<<GRID_PARTICLES_IBM, THREADS_PARTICLES_IBM, 0, streamParticles>>>(
        particles.getPCenterArray());
    checkCudaErrors(hipStreamSynchronize(streamParticles));
    getLastCudaError("IBM particle movement error\n");

    // for(int i = 0; i < N_GPUS; i++){
    //     // If GPU has nodes in it
    //     if(particles.getNodesSoA()[i].getNumNodes() > 0){ // particles.nodesSoA[i].numNodes
    //         checkCudaErrors(hipSetDevice(GPUS_TO_USE[i]));
    //         // Update particle nodes positions
    //         gpuParticleNodeMovement<<<gridNodesIBM[i], threadsNodesIBM[i], 0, streamParticles>>>(
    //             particles.getNodesSoA()[i], particles.getPCenterArray());
    //         checkCudaErrors(hipStreamSynchronize(streamParticles));
    //         getLastCudaError("IBM particle movement error\n");
    //     }
    // }

    checkCudaErrors(hipDeviceSynchronize());
 
}

// #endif //PARTICLE_MODEL