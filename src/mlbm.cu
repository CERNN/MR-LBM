#include "hip/hip_runtime.h"
#include "mlbm.cuh"

__global__ void gpuMomCollisionStream(
    dfloat *fMom, unsigned int *dNodeType,ghostInterfaceData ghostInterface,
    DENSITY_CORRECTION_PARAMS_DECLARATION(d_)
    BC_FORCES_PARAMS_DECLARATION(d_)
    unsigned int step,
    bool save)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;
    dfloat pop[Q];
    #ifdef CONVECTION_DIFFUSION_TRANSPORT
    dfloat gNode[GQ];
    #endif
    dfloat pics2;
    dfloat multiplyTerm;

    #ifdef DYNAMIC_SHARED_MEMORY
    extern __shared__ dfloat s_pop[]; 
    #else
    __shared__ dfloat s_pop[MAX_SHARED_MEMORY_SIZE];
    #endif
    
    const int baseIdx = idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 0, blockIdx.x, blockIdx.y, blockIdx.z);
    const int baseIdxPop = idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  0);

    // Load moments from global memory

    //rho'
    unsigned int nodeType = dNodeType[idxScalarBlock(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)];
    if (nodeType == 0b11111111)  return;

    dfloat rhoVar = RHO_0 + fMom[baseIdx + M_RHO_INDEX];
    dfloat ux_t30     = fMom[baseIdx + BLOCK_LBM_SIZE * M_UX_INDEX];
    dfloat uy_t30     = fMom[baseIdx + BLOCK_LBM_SIZE * M_UY_INDEX];
    dfloat uz_t30     = fMom[baseIdx + BLOCK_LBM_SIZE * M_UZ_INDEX];
    dfloat m_xx_t45   = fMom[baseIdx + BLOCK_LBM_SIZE * M_MXX_INDEX];
    dfloat m_xy_t90   = fMom[baseIdx + BLOCK_LBM_SIZE * M_MXY_INDEX];
    dfloat m_xz_t90   = fMom[baseIdx + BLOCK_LBM_SIZE * M_MXZ_INDEX];
    dfloat m_yy_t45   = fMom[baseIdx + BLOCK_LBM_SIZE * M_MYY_INDEX];
    dfloat m_yz_t90   = fMom[baseIdx + BLOCK_LBM_SIZE * M_MYZ_INDEX];
    dfloat m_zz_t45   = fMom[baseIdx + BLOCK_LBM_SIZE * M_MZZ_INDEX];

    #ifdef OMEGA_FIELD
        //dfloat omegaVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_OMEGA_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)];
        dfloat omegaVar = fMom[baseIdx + BLOCK_LBM_SIZE * M_OMEGA_INDEX];
        dfloat t_omegaVar = 1 - omegaVar;
        dfloat tt_omegaVar = 1 - omegaVar/2;
        dfloat omegaVar_d2 = omegaVar / 2;
        dfloat tt_omega_t3 = tt_omegaVar * 3;
    #else
        const dfloat omegaVar = OMEGA;
        const dfloat t_omegaVar = 1 - omegaVar;
        const dfloat tt_omegaVar = 1 - omegaVar/2;
        const dfloat omegaVar_d2 = omegaVar / 2;
        const dfloat tt_omega_t3 = tt_omegaVar * 3;
    #endif
    
    /*
    if(z > (NZ_TOTAL-50)){
        dfloat dist = (z - (NZ_TOTAL-50))/((NZ_TOTAL)- (NZ_TOTAL-50));
        dfloat ttau = 0.5+ 3*VISC*(1000.0*dist*dist*dist+1.0);
        omegaVar = 1/ttau;
    }*/

    //Local forces
    //dfloat K_const = 2.0*M_PI/(dfloat)N;
   // dfloat xx = 2.0 * M_PI * x / L;
   // dfloat yy = 2.0 * M_PI * y / L;
   // dfloat zz = 2.0 * M_PI * z / L;

   dfloat L_Fx = 0.00; // F_0 * sin(K_const*x) * cos(K_const*y) ;
   dfloat L_Fy = 0.00; //-F_0 * sin(K_const*y) * cos(K_const*x) ;
   dfloat L_Fz = 0.00;

    #ifdef BC_FORCES
    dfloat L_BC_Fx = 0.0;
    dfloat L_BC_Fy = 0.0;
    dfloat L_BC_Fz = 0.0;
    #endif


    #include COLREC_RECONSTRUCTION

    const unsigned short int xp1 = (threadIdx.x + 1 + BLOCK_NX) % BLOCK_NX;
    const unsigned short int xm1 = (threadIdx.x - 1 + BLOCK_NX) % BLOCK_NX;

    const unsigned short int yp1 = (threadIdx.y + 1 + BLOCK_NY) % BLOCK_NY;
    const unsigned short int ym1 = (threadIdx.y - 1 + BLOCK_NY) % BLOCK_NY;

    const unsigned short int zp1 = (threadIdx.z + 1 + BLOCK_NZ) % BLOCK_NZ;
    const unsigned short int zm1 = (threadIdx.z - 1 + BLOCK_NZ) % BLOCK_NZ;
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;

    const int txm1 = (tx-1+BLOCK_NX)%BLOCK_NX;
    const int txp1 = (tx+1+BLOCK_NX)%BLOCK_NX;

    const int tym1 = (ty-1+BLOCK_NY)%BLOCK_NY;
    const int typ1 = (ty+1+BLOCK_NY)%BLOCK_NY;

    const int tzm1 = (tz-1+BLOCK_NZ)%BLOCK_NZ;
    const int tzp1 = (tz+1+BLOCK_NZ)%BLOCK_NZ;

    const int bxm1 = (bx-1+NUM_BLOCK_X)%NUM_BLOCK_X;
    const int bxp1 = (bx+1+NUM_BLOCK_X)%NUM_BLOCK_X;

    const int bym1 = (by-1+NUM_BLOCK_Y)%NUM_BLOCK_Y;
    const int byp1 = (by+1+NUM_BLOCK_Y)%NUM_BLOCK_Y;

    const int bzm1 = (bz-1+NUM_BLOCK_Z)%NUM_BLOCK_Z;
    const int bzp1 = (bz+1+NUM_BLOCK_Z)%NUM_BLOCK_Z;

    //need to compute the gradient before the moments are recalculated
    #ifdef COMPUTE_VEL_GRADIENT_FINITE_DIFFERENCE
        #include "includeFiles/velocity_gradient.inc"
    #endif //COMPUTE_VEL_GRADIENT_FINITE_DIFFERENCE

    
    #ifdef CONFORMATION_TENSOR
        #ifdef A_XX_DIST
            dfloat AxxVar = fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_C_INDEX];
        #endif //A_XX_DIST
        #ifdef A_XY_DIST
            dfloat AxyVar = fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_C_INDEX];
        #endif //A_XY_DIST
        #ifdef A_XZ_DIST
            dfloat AxzVar = fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_C_INDEX];
        #endif //A_XZ_DIST
        #ifdef A_YY_DIST
            dfloat AyyVar = fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_C_INDEX];
        #endif //A_YY_DIST
        #ifdef A_YZ_DIST
            dfloat AyzVar = fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_C_INDEX];
        #endif //A_YZ_DIST
        #ifdef A_ZZ_DIST
            dfloat AzzVar = fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_C_INDEX];
        #endif //A_ZZ_DIST

        #ifdef COMPUTE_CONF_GRADIENT_FINITE_DIFFERENCE
            #include "includeFiles/conformationTransport/conformation_gradient.inc"   
        #endif

        #include "includeFiles/conformationTransport/conformation_evolution.inc"
    #endif

    #ifdef CONVECTION_DIFFUSION_TRANSPORT
        #ifdef SECOND_DIST 
            dfloat cVar = fMom[baseIdx + BLOCK_LBM_SIZE * M2_C_INDEX];
            dfloat invC = 1/cVar;
            dfloat qx_t30   = fMom[baseIdx + BLOCK_LBM_SIZE * M2_CX_INDEX];
            dfloat qy_t30   = fMom[baseIdx + BLOCK_LBM_SIZE * M2_CY_INDEX];
            dfloat qz_t30   = fMom[baseIdx + BLOCK_LBM_SIZE * M2_CZ_INDEX];

            dfloat udx_t30 = G_DIFF_FLUC_COEF * (qx_t30*invC - ux_t30);
            dfloat udy_t30 = G_DIFF_FLUC_COEF * (qy_t30*invC - uy_t30);
            dfloat udz_t30 = G_DIFF_FLUC_COEF * (qz_t30*invC - uz_t30);

            #include  COLREC_G_RECONSTRUCTION

            __syncthreads();

            #include "includeFiles/convection_diffusion_streaming.inc"
            /* load pop from global in cover nodes */        
            #include "includeFiles/g_popLoad.inc"


            if(nodeType != BULK){
                #include CASE_G_BC_DEF
            }else{
                cVar = gNode[0] + gNode[1] + gNode[2] + gNode[3] + gNode[4] + gNode[5] + gNode[6] + gNode[7] + gNode[8] + gNode[9] + gNode[10] + gNode[11] + gNode[12] + gNode[13] + gNode[14] + gNode[15] + gNode[16] + gNode[17] + gNode[18];
                cVar = cVar + T_Q_INTERNAL_D_Cp;
                invC= 1.0/cVar;

                qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));
            }
        #endif
        #ifdef A_XX_DIST
            dfloat GxxVar = fMom[baseIdx + BLOCK_LBM_SIZE * G_XX_C_INDEX];
            dfloat invAxx = 1/AxxVar;
            dfloat Axx_qx_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_CX_INDEX];
            dfloat Axx_qy_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_CY_INDEX];
            dfloat Axx_qz_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_CZ_INDEX];


            dfloat Axx_udx_t30 = CONF_DIFF_FLUC_COEF * (Axx_qx_t30*invAxx - ux_t30);
            dfloat Axx_udy_t30 = CONF_DIFF_FLUC_COEF * (Axx_qy_t30*invAxx - uy_t30);
            dfloat Axx_udz_t30 = CONF_DIFF_FLUC_COEF * (Axx_qz_t30*invAxx - uz_t30);

            #include COLREC_AXX_RECONSTRUCTION

            __syncthreads();

            #include "includeFiles/convection_diffusion_streaming.inc"
            /* load pop from global in cover nodes */
            #include "includeFiles/conformationTransport/popLoad_Axx.inc"

            if(nodeType != BULK){
                 #include CASE_AXX_BC_DEF
            }else{
                AxxVar = gNode[0] + gNode[1] + gNode[2] + gNode[3] + gNode[4] + gNode[5] + gNode[6] + gNode[7] + gNode[8] + gNode[9] + gNode[10] + gNode[11] + gNode[12] + gNode[13] + gNode[14] + gNode[15] + gNode[16] + gNode[17] + gNode[18];
                AxxVar = AxxVar + GxxVar;
                invAxx= 1.0/AxxVar;

                Axx_qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                Axx_qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                Axx_qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));
            }
        #endif //A_XX_DIST
        #ifdef A_XY_DIST
            dfloat GxyVar = fMom[baseIdx + BLOCK_LBM_SIZE * G_XY_C_INDEX];
            dfloat invAxy = 1/AxyVar;
            dfloat Axy_qx_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_CX_INDEX];
            dfloat Axy_qy_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_CY_INDEX];
            dfloat Axy_qz_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_CZ_INDEX];


            dfloat Axy_udx_t30 = CONF_DIFF_FLUC_COEF * (Axy_qx_t30*invAxy - ux_t30);
            dfloat Axy_udy_t30 = CONF_DIFF_FLUC_COEF * (Axy_qy_t30*invAxy - uy_t30);
            dfloat Axy_udz_t30 = CONF_DIFF_FLUC_COEF * (Axy_qz_t30*invAxy - uz_t30);

            #include COLREC_AXY_RECONSTRUCTION

            __syncthreads();

            #include "includeFiles/convection_diffusion_streaming.inc"
            /* load pop from global in cover nodes */
            #include "includeFiles/conformationTransport/popLoad_Axy.inc"

            if(nodeType != BULK){
                    #include CASE_AXY_BC_DEF
            }else{
                AxyVar = gNode[0] + gNode[1] + gNode[2] + gNode[3] + gNode[4] + gNode[5] + gNode[6] + gNode[7] + gNode[8] + gNode[9] + gNode[10] + gNode[11] + gNode[12] + gNode[13] + gNode[14] + gNode[15] + gNode[16] + gNode[17] + gNode[18];
                AxyVar = AxyVar + GxyVar;
                invAxy= 1.0/AxyVar;

                Axy_qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                Axy_qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                Axy_qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));
            }
        #endif //A_XY_DIST
        #ifdef A_XZ_DIST
            dfloat GxzVar = fMom[baseIdx + BLOCK_LBM_SIZE * G_XZ_C_INDEX];
            dfloat invAxz = 1/AxzVar;
            dfloat Axz_qx_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_CX_INDEX];
            dfloat Axz_qy_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_CY_INDEX];
            dfloat Axz_qz_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_CZ_INDEX];


            dfloat Axz_udx_t30 = CONF_DIFF_FLUC_COEF * (Axz_qx_t30*invAxz - ux_t30);
            dfloat Axz_udy_t30 = CONF_DIFF_FLUC_COEF * (Axz_qy_t30*invAxz - uy_t30);
            dfloat Axz_udz_t30 = CONF_DIFF_FLUC_COEF * (Axz_qz_t30*invAxz - uz_t30);

            #include COLREC_AXZ_RECONSTRUCTION

            __syncthreads();

            #include "includeFiles/convection_diffusion_streaming.inc"
            /* load pop from global in cover nodes */
            #include "includeFiles/conformationTransport/popLoad_Axz.inc"

            if(nodeType != BULK){
                    #include CASE_AXZ_BC_DEF
            }else{
                AxzVar = gNode[0] + gNode[1] + gNode[2] + gNode[3] + gNode[4] + gNode[5] + gNode[6] + gNode[7] + gNode[8] + gNode[9] + gNode[10] + gNode[11] + gNode[12] + gNode[13] + gNode[14] + gNode[15] + gNode[16] + gNode[17] + gNode[18];
                AxzVar = AxzVar + GxzVar;
                invAxz= 1.0/AxzVar;

                Axz_qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                Axz_qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                Axz_qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));
            }
        #endif //A_XZ_DIST
        #ifdef A_YY_DIST
            dfloat GyyVar = fMom[baseIdx + BLOCK_LBM_SIZE * G_YY_C_INDEX];
            dfloat invAyy = 1/AyyVar;
            dfloat Ayy_qx_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_CX_INDEX];
            dfloat Ayy_qy_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_CY_INDEX];
            dfloat Ayy_qz_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_CZ_INDEX];


            dfloat Ayy_udx_t30 = CONF_DIFF_FLUC_COEF * (Ayy_qx_t30*invAyy - ux_t30);
            dfloat Ayy_udy_t30 = CONF_DIFF_FLUC_COEF * (Ayy_qy_t30*invAyy - uy_t30);
            dfloat Ayy_udz_t30 = CONF_DIFF_FLUC_COEF * (Ayy_qz_t30*invAyy - uz_t30);

            #include COLREC_AYY_RECONSTRUCTION

            __syncthreads();

            #include "includeFiles/convection_diffusion_streaming.inc"
            /* load pop from global in cover nodes */
            #include "includeFiles/conformationTransport/popLoad_Ayy.inc"

            if(nodeType != BULK){
                    #include CASE_AYY_BC_DEF
            }else{
                AyyVar = gNode[0] + gNode[1] + gNode[2] + gNode[3] + gNode[4] + gNode[5] + gNode[6] + gNode[7] + gNode[8] + gNode[9] + gNode[10] + gNode[11] + gNode[12] + gNode[13] + gNode[14] + gNode[15] + gNode[16] + gNode[17] + gNode[18];
                AyyVar = AyyVar + GyyVar;
                invAyy= 1.0/AyyVar;

                Ayy_qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                Ayy_qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                Ayy_qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));
            }
        #endif //A_YY_DIST
        #ifdef A_YZ_DIST
            dfloat GyzVar = fMom[baseIdx + BLOCK_LBM_SIZE * G_YZ_C_INDEX];
            dfloat invAyz = 1/AyzVar;
            dfloat Ayz_qx_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_CX_INDEX];
            dfloat Ayz_qy_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_CY_INDEX];
            dfloat Ayz_qz_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_CZ_INDEX];


            dfloat Ayz_udx_t30 = CONF_DIFF_FLUC_COEF * (Ayz_qx_t30*invAyz - ux_t30);
            dfloat Ayz_udy_t30 = CONF_DIFF_FLUC_COEF * (Ayz_qy_t30*invAyz - uy_t30);
            dfloat Ayz_udz_t30 = CONF_DIFF_FLUC_COEF * (Ayz_qz_t30*invAyz - uz_t30);

            #include COLREC_AYZ_RECONSTRUCTION

            __syncthreads();

            #include "includeFiles/convection_diffusion_streaming.inc"
            /* load pop from global in cover nodes */
            #include "includeFiles/conformationTransport/popLoad_Ayz.inc"

            if(nodeType != BULK){
                    #include CASE_AYZ_BC_DEF
            }else{
                AyzVar = gNode[0] + gNode[1] + gNode[2] + gNode[3] + gNode[4] + gNode[5] + gNode[6] + gNode[7] + gNode[8] + gNode[9] + gNode[10] + gNode[11] + gNode[12] + gNode[13] + gNode[14] + gNode[15] + gNode[16] + gNode[17] + gNode[18];
                AyzVar = AyzVar + GyzVar;
                invAyz= 1.0/AyzVar;

                Ayz_qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                Ayz_qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                Ayz_qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));
            }
        #endif //A_YZ_DIST
        #ifdef A_ZZ_DIST
            dfloat GzzVar = fMom[baseIdx + BLOCK_LBM_SIZE * G_ZZ_C_INDEX];
            dfloat invAzz = 1/AzzVar;
            dfloat Azz_qx_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_CX_INDEX];
            dfloat Azz_qy_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_CY_INDEX];
            dfloat Azz_qz_t30 = fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_CZ_INDEX];


            dfloat Azz_udx_t30 = CONF_DIFF_FLUC_COEF * (Azz_qx_t30*invAzz - ux_t30);
            dfloat Azz_udy_t30 = CONF_DIFF_FLUC_COEF * (Azz_qy_t30*invAzz - uy_t30);
            dfloat Azz_udz_t30 = CONF_DIFF_FLUC_COEF * (Azz_qz_t30*invAzz - uz_t30);

            #include COLREC_AZZ_RECONSTRUCTION

            __syncthreads();

            #include "includeFiles/convection_diffusion_streaming.inc"
            /* load pop from global in cover nodes */
            #include "includeFiles/conformationTransport/popLoad_Azz.inc"

            if(nodeType != BULK){
                    #include CASE_AZZ_BC_DEF
            }else{
                AzzVar = gNode[0] + gNode[1] + gNode[2] + gNode[3] + gNode[4] + gNode[5] + gNode[6] + gNode[7] + gNode[8] + gNode[9] + gNode[10] + gNode[11] + gNode[12] + gNode[13] + gNode[14] + gNode[15] + gNode[16] + gNode[17] + gNode[18];
                AzzVar = AzzVar + GzzVar;
                invAzz= 1.0/AzzVar;

                Azz_qx_t30 = F_M_I_SCALE*((gNode[1] - gNode[2] + gNode[7] - gNode[ 8] + gNode[ 9] - gNode[10] + gNode[13] - gNode[14] + gNode[15] - gNode[16]));
                Azz_qy_t30 = F_M_I_SCALE*((gNode[3] - gNode[4] + gNode[7] - gNode[ 8] + gNode[11] - gNode[12] + gNode[14] - gNode[13] + gNode[17] - gNode[18]));
                Azz_qz_t30 = F_M_I_SCALE*((gNode[5] - gNode[6] + gNode[9] - gNode[10] + gNode[11] - gNode[12] + gNode[16] - gNode[15] + gNode[18] - gNode[17]));
            }
        #endif //A_ZZ_DIST
        

    #endif //CONVECTION_DIFFUSION_TRANSPORT

    //save populations in shared memory

    s_pop[baseIdxPop +  0*BLOCK_LBM_SIZE] = pop[ 1];
    s_pop[baseIdxPop +  1*BLOCK_LBM_SIZE] = pop[ 2];
    s_pop[baseIdxPop +  2*BLOCK_LBM_SIZE] = pop[ 3];
    s_pop[baseIdxPop +  3*BLOCK_LBM_SIZE] = pop[ 4];
    s_pop[baseIdxPop +  4*BLOCK_LBM_SIZE] = pop[ 5];
    s_pop[baseIdxPop +  5*BLOCK_LBM_SIZE] = pop[ 6];
    s_pop[baseIdxPop +  6*BLOCK_LBM_SIZE] = pop[ 7];
    s_pop[baseIdxPop +  7*BLOCK_LBM_SIZE] = pop[ 8];
    s_pop[baseIdxPop +  8*BLOCK_LBM_SIZE] = pop[ 9];
    s_pop[baseIdxPop +  9*BLOCK_LBM_SIZE] = pop[10];
    s_pop[baseIdxPop + 10*BLOCK_LBM_SIZE] = pop[11];
    s_pop[baseIdxPop + 11*BLOCK_LBM_SIZE] = pop[12];
    s_pop[baseIdxPop + 12*BLOCK_LBM_SIZE] = pop[13];
    s_pop[baseIdxPop + 13*BLOCK_LBM_SIZE] = pop[14];
    s_pop[baseIdxPop + 14*BLOCK_LBM_SIZE] = pop[15];
    s_pop[baseIdxPop + 15*BLOCK_LBM_SIZE] = pop[16];
    s_pop[baseIdxPop + 16*BLOCK_LBM_SIZE] = pop[17];
    s_pop[baseIdxPop + 17*BLOCK_LBM_SIZE] = pop[18];
    #ifdef D3Q27
    s_pop[baseIdxPop + 18*BLOCK_LBM_SIZE] = pop[19];
    s_pop[baseIdxPop + 19*BLOCK_LBM_SIZE] = pop[20];
    s_pop[baseIdxPop + 20*BLOCK_LBM_SIZE] = pop[21];
    s_pop[baseIdxPop + 21*BLOCK_LBM_SIZE] = pop[22];
    s_pop[baseIdxPop + 22*BLOCK_LBM_SIZE] = pop[23];
    s_pop[baseIdxPop + 23*BLOCK_LBM_SIZE] = pop[24];
    s_pop[baseIdxPop + 24*BLOCK_LBM_SIZE] = pop[25];
    s_pop[baseIdxPop + 25*BLOCK_LBM_SIZE] = pop[26];
    #endif //D3Q27


    //sync threads of the block so all populations are saved
    __syncthreads();

    /* pull */

    pop[ 1] = s_pop[idxPopBlock(xm1, threadIdx.y, threadIdx.z, 0)];
    pop[ 2] = s_pop[idxPopBlock(xp1, threadIdx.y, threadIdx.z, 1)];
    pop[ 3] = s_pop[idxPopBlock(threadIdx.x, ym1, threadIdx.z, 2)];
    pop[ 4] = s_pop[idxPopBlock(threadIdx.x, yp1, threadIdx.z, 3)];
    pop[ 5] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zm1, 4)];
    pop[ 6] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zp1, 5)];
    pop[ 7] = s_pop[idxPopBlock(xm1, ym1, threadIdx.z, 6)];
    pop[ 8] = s_pop[idxPopBlock(xp1, yp1, threadIdx.z, 7)];
    pop[ 9] = s_pop[idxPopBlock(xm1, threadIdx.y, zm1, 8)];
    pop[10] = s_pop[idxPopBlock(xp1, threadIdx.y, zp1, 9)];
    pop[11] = s_pop[idxPopBlock(threadIdx.x, ym1, zm1, 10)];
    pop[12] = s_pop[idxPopBlock(threadIdx.x, yp1, zp1, 11)];
    pop[13] = s_pop[idxPopBlock(xm1, yp1, threadIdx.z, 12)];
    pop[14] = s_pop[idxPopBlock(xp1, ym1, threadIdx.z, 13)];
    pop[15] = s_pop[idxPopBlock(xm1, threadIdx.y, zp1, 14)];
    pop[16] = s_pop[idxPopBlock(xp1, threadIdx.y, zm1, 15)];
    pop[17] = s_pop[idxPopBlock(threadIdx.x, ym1, zp1, 16)];
    pop[18] = s_pop[idxPopBlock(threadIdx.x, yp1, zm1, 17)];
    #ifdef D3Q27
    pop[19] = s_pop[idxPopBlock(xm1, ym1, zm1, 18)];
    pop[20] = s_pop[idxPopBlock(xp1, yp1, zp1, 19)];
    pop[21] = s_pop[idxPopBlock(xm1, ym1, zp1, 20)];
    pop[22] = s_pop[idxPopBlock(xp1, yp1, zm1, 21)];
    pop[23] = s_pop[idxPopBlock(xm1, yp1, zm1, 22)];
    pop[24] = s_pop[idxPopBlock(xp1, ym1, zp1, 23)];
    pop[25] = s_pop[idxPopBlock(xp1, ym1, zm1, 24)];
    pop[26] = s_pop[idxPopBlock(xm1, yp1, zp1, 25)];
    #endif

    /* load pop from global in cover nodes */

    #include "includeFiles/popLoad.inc"

    dfloat invRho;
    if(nodeType != BULK){
        #include CASE_BC_DEF

        invRho = 1.0 / rhoVar;               
    }else{

        //calculate streaming moments
        #ifdef D3Q19
            //equation3
            rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18];
            invRho = 1 / rhoVar;
            //equation4 + force correction
            ux_t30 = ((pop[1] - pop[2] + pop[7] - pop[ 8] + pop[ 9] - pop[10] + pop[13] - pop[14] + pop[15] - pop[16])) * invRho;
            uy_t30 = ((pop[3] - pop[4] + pop[7] - pop[ 8] + pop[11] - pop[12] + pop[14] - pop[13] + pop[17] - pop[18])) * invRho;
            uz_t30 = ((pop[5] - pop[6] + pop[9] - pop[10] + pop[11] - pop[12] + pop[16] - pop[15] + pop[18] - pop[17])) * invRho;

            //equation5
            m_xx_t45 = (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16])* invRho - cs2;
            m_xy_t90 = (pop[7] - pop[13] + pop[8] - pop[14])* invRho;
            m_xz_t90 = (pop[9] - pop[15] + pop[10] - pop[16])* invRho;
            m_yy_t45 = (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18])* invRho - cs2;
            m_yz_t90 = (pop[11] - pop[17] + pop[12] - pop[18])* invRho;
            m_zz_t45 = (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18])* invRho - cs2;


        #endif
        #ifdef D3Q27
            rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26];
            invRho = 1 / rhoVar;
            ux_t30 = ((pop[1] + pop[7] + pop[9] + pop[13] + pop[15] + pop[19] + pop[21] + pop[23] + pop[26])  - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16] + pop[20] + pop[22] + pop[24] + pop[25])) * invRho;
            uy_t30 = ((pop[3] + pop[7] + pop[11] + pop[14] + pop[17] + pop[19] + pop[21] + pop[24] + pop[25]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18] + pop[20] + pop[22] + pop[23] + pop[26])) * invRho;
            uz_t30 = ((pop[5] + pop[9] + pop[11] + pop[16] + pop[18] + pop[19] + pop[22] + pop[23] + pop[25]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17] + pop[20] + pop[21] + pop[24] + pop[26])) * invRho;

            m_xx_t45 = ( (pop[ 1] + pop[ 2] + pop[ 7] + pop[ 8] + pop[ 9] + pop[10]  +  pop[13] + pop[14] + pop[15] + pop[16] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
            m_xy_t90 = (((pop[ 7] + pop[ 8] + pop[19] + pop[20] + pop[21] + pop[22]) - (pop[13] + pop[14] + pop[23] + pop[24] + pop[25] + pop[26])) )* invRho;
            m_xz_t90 = (((pop[ 9] + pop[10] + pop[19] + pop[20] + pop[23] + pop[24]) - (pop[15] + pop[16] + pop[21] + pop[22] + pop[25] + pop[26])) )* invRho;
            m_yy_t45 = ( (pop[ 3] + pop[ 4] + pop[ 7] + pop[ 8] + pop[11] + pop[12]  +  pop[13] + pop[14] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
            m_yz_t90 = (((pop[11] + pop[12] + pop[19] + pop[20] + pop[25] + pop[26]) - (pop[17] + pop[18] + pop[21] + pop[22] + pop[23] + pop[24])))* invRho;
            m_zz_t45 = ( (pop[ 5] + pop[ 6] + pop[ 9] + pop[10] + pop[11] + pop[12]  +  pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
        #endif
    }

    // multiply moments by as2 -- as4*0.5 -- as4 - add correction to m_alpha_beta
    ux_t30 = F_M_I_SCALE * ux_t30;
    uy_t30 = F_M_I_SCALE * uy_t30;
    uz_t30 = F_M_I_SCALE * uz_t30;

    m_xx_t45 = F_M_II_SCALE * (m_xx_t45);
    m_xy_t90 = F_M_IJ_SCALE * (m_xy_t90);
    m_xz_t90 = F_M_IJ_SCALE * (m_xz_t90);
    m_yy_t45 = F_M_II_SCALE * (m_yy_t45);
    m_yz_t90 = F_M_IJ_SCALE * (m_yz_t90);
    m_zz_t45 = F_M_II_SCALE * (m_zz_t45);


    #ifdef DENSITY_CORRECTION
        //printf("%f ",d_mean_rho[0]-1.0) ;
        rhoVar -= (d_mean_rho[0]) ;
        invRho = 1/rhoVar;
    #endif // DENSITY_CORRECTION
    #ifdef THERMAL_MODEL //Boussinesq Approximation
        if(nodeType == BULK && T_BOUYANCY){
                L_Fx += gravity_vector[0] * T_gravity_t_beta * RHO_0*((cVar-T_REFERENCE));
                L_Fy += gravity_vector[1] * T_gravity_t_beta * RHO_0*((cVar-T_REFERENCE));
                L_Fz += gravity_vector[2] * T_gravity_t_beta * RHO_0*((cVar-T_REFERENCE));
        }
            
    #endif
    
    #ifdef COMPUTE_SHEAR
            //TODO change to fix perfomance
        const dfloat S_XX = rhoVar * (m_xx_t45/F_M_II_SCALE - ux_t30*ux_t30/(F_M_I_SCALE*F_M_I_SCALE));
        const dfloat S_YY = rhoVar * (m_yy_t45/F_M_II_SCALE - uy_t30*uy_t30/(F_M_I_SCALE*F_M_I_SCALE));
        const dfloat S_ZZ = rhoVar * (m_zz_t45/F_M_II_SCALE - uz_t30*uz_t30/(F_M_I_SCALE*F_M_I_SCALE));
        const dfloat S_XY = rhoVar * (m_xy_t90/F_M_IJ_SCALE - ux_t30*uy_t30/(F_M_I_SCALE*F_M_I_SCALE));
        const dfloat S_XZ = rhoVar * (m_xz_t90/F_M_IJ_SCALE - ux_t30*uz_t30/(F_M_I_SCALE*F_M_I_SCALE));
        const dfloat S_YZ = rhoVar * (m_yz_t90/F_M_IJ_SCALE - uy_t30*uz_t30/(F_M_I_SCALE*F_M_I_SCALE));

        const dfloat uFxxd2 = ux_t30*L_Fx/F_M_I_SCALE; // d2 = uFxx Divided by two
        const dfloat uFyyd2 = uy_t30*L_Fy/F_M_I_SCALE;
        const dfloat uFzzd2 = uz_t30*L_Fz/F_M_I_SCALE;
        const dfloat uFxyd2 = (ux_t30*L_Fy + uy_t30*L_Fx) / (2.0*F_M_I_SCALE);
        const dfloat uFxzd2 = (ux_t30*L_Fz + uz_t30*L_Fx) / (2.0*F_M_I_SCALE);
        const dfloat uFyzd2 = (uy_t30*L_Fz + uz_t30*L_Fy) / (2.0*F_M_I_SCALE);

        const dfloat auxStressMag = sqrt(0.5 * (
            (S_XX + uFxxd2) * (S_XX + uFxxd2) +(S_YY + uFyyd2) * (S_YY + uFyyd2) + (S_ZZ + uFzzd2) * (S_ZZ + uFzzd2) +
            2 * ((S_XY + uFxyd2) * (S_XY + uFxyd2) + (S_XZ + uFxzd2) * (S_XZ + uFxzd2) + (S_YZ + uFyzd2) * (S_YZ + uFyzd2))));

    #endif
    // MOMENTS DETERMINED, COMPUTE OMEGA IF NON-NEWTONIAN FLUID
    #if defined(OMEGA_FIELD)
            #ifdef NON_NEWTONIAN_FLUID 
                omegaVar = calcOmega_nnf(omegaVar, auxStressMag,step);
            #endif//  OMEGA_FIELD

            #ifdef LES_MODEL
                dfloat tau_t = 0.5*sqrt(TAU*TAU+Implicit_const*auxStressMag)-0.5*TAU;
                dfloat visc_turb_var = tau_t/3.0;

                omegaVar = 1.0/(TAU + tau_t);
            #endif

            //Compute new auxiliary variables
            t_omegaVar = 1 - omegaVar;
            tt_omegaVar = 1 - 0.5*omegaVar;
            omegaVar_d2 = omegaVar / 2.0;
            tt_omega_t3 = tt_omegaVar * 3.0;
    #endif 
    
        // COLLIDE
    #include COLREC_COLLISION
    

    //calculate post collision populations
    #include COLREC_RECONSTRUCTION
    
    
    /* write to global mom */

    fMom[baseIdx + M_RHO_INDEX] = rhoVar - RHO_0;

    fMom[baseIdx + BLOCK_LBM_SIZE * M_UX_INDEX] = ux_t30;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_UY_INDEX] = uy_t30;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_UZ_INDEX] = uz_t30;

    fMom[baseIdx + BLOCK_LBM_SIZE * M_MXX_INDEX] = m_xx_t45;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_MXY_INDEX] = m_xy_t90;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_MXZ_INDEX] = m_xz_t90;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_MYY_INDEX] = m_yy_t45;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_MYZ_INDEX] = m_yz_t90;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_MZZ_INDEX] = m_zz_t45;
    
    #ifdef OMEGA_FIELD
    //fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, M_OMEGA_INDEX, blockIdx.x, blockIdx.y, blockIdx.z)] = omegaVar;
    fMom[baseIdx + BLOCK_LBM_SIZE * M_OMEGA_INDEX] = omegaVar;
    #endif


    if(save){
        #ifdef BC_FORCES
        //update local forces
        const int baseIdxScalarBlock = idxScalarBlock(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z);
        d_BC_Fx[baseIdxScalarBlock] = (L_BC_Fx);
        d_BC_Fy[baseIdxScalarBlock] = (L_BC_Fy);
        d_BC_Fz[baseIdxScalarBlock] = (L_BC_Fz);
        #endif 
    }
    #ifdef CONVECTION_DIFFUSION_TRANSPORT
        #ifdef SECOND_DIST 
            udx_t30 = G_DIFF_FLUC_COEF * (qx_t30*invC - ux_t30);
            udy_t30 = G_DIFF_FLUC_COEF * (qy_t30*invC - uy_t30);
            udz_t30 = G_DIFF_FLUC_COEF * (qz_t30*invC - uz_t30);

            #include COLREC_G_RECONSTRUCTION

            #include "includeFiles/g_popSave.inc"
            
            fMom[baseIdx + BLOCK_LBM_SIZE * M2_C_INDEX] = cVar;
            fMom[baseIdx + BLOCK_LBM_SIZE * M2_CX_INDEX] = qx_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * M2_CY_INDEX] = qy_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * M2_CZ_INDEX] = qz_t30;

        #endif
        #ifdef A_XX_DIST
            Axx_udx_t30 = CONF_DIFF_FLUC_COEF * (Axx_qx_t30*invAxx - ux_t30);
            Axx_udy_t30 = CONF_DIFF_FLUC_COEF * (Axx_qy_t30*invAxx - uy_t30);
            Axx_udz_t30 = CONF_DIFF_FLUC_COEF * (Axx_qz_t30*invAxx - uz_t30);

            #include COLREC_AXX_RECONSTRUCTION

            #include "includeFiles/conformationTransport\popSave_Axx.inc"
           
           fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_C_INDEX] = AxxVar;
           fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_CX_INDEX] = Axx_qx_t30;
           fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_CY_INDEX] = Axx_qy_t30;
           fMom[baseIdx + BLOCK_LBM_SIZE * A_XX_CZ_INDEX] = Axx_qz_t30;
        #endif //A_XX_DIST
        #ifdef A_XY_DIST
            Axy_udx_t30 = CONF_DIFF_FLUC_COEF * (Axy_qx_t30*invAxy - ux_t30);
            Axy_udy_t30 = CONF_DIFF_FLUC_COEF * (Axy_qy_t30*invAxy - uy_t30);
            Axy_udz_t30 = CONF_DIFF_FLUC_COEF * (Axy_qz_t30*invAxy - uz_t30);

            #include COLREC_AXY_RECONSTRUCTION

            #include "includeFiles/conformationTransport\popSave_Axy.inc"
           
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_C_INDEX] = AxyVar;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_CX_INDEX] = Axy_qx_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_CY_INDEX] = Axy_qy_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XY_CZ_INDEX] = Axy_qz_t30;
        #endif //A_XY_DIST
        #ifdef A_XZ_DIST
            Axz_udx_t30 = CONF_DIFF_FLUC_COEF * (Axz_qx_t30*invAxz - ux_t30);
            Axz_udy_t30 = CONF_DIFF_FLUC_COEF * (Axz_qy_t30*invAxz - uy_t30);
            Axz_udz_t30 = CONF_DIFF_FLUC_COEF * (Axz_qz_t30*invAxz - uz_t30);

            #include COLREC_AXZ_RECONSTRUCTION

            #include "includeFiles/conformationTransport\popSave_Axz.inc"
           
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_C_INDEX] = AxzVar;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_CX_INDEX] = Axz_qx_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_CY_INDEX] = Axz_qy_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_XZ_CZ_INDEX] = Axz_qz_t30;
        #endif //A_XZ_DIST
        #ifdef A_YY_DIST
            Ayy_udx_t30 = CONF_DIFF_FLUC_COEF * (Ayy_qx_t30*invAyy - ux_t30);
            Ayy_udy_t30 = CONF_DIFF_FLUC_COEF * (Ayy_qy_t30*invAyy - uy_t30);
            Ayy_udz_t30 = CONF_DIFF_FLUC_COEF * (Ayy_qz_t30*invAyy - uz_t30);

            #include COLREC_AYY_RECONSTRUCTION

            #include "includeFiles/conformationTransport\popSave_Ayy.inc"
           
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_C_INDEX] = AyyVar;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_CX_INDEX] = Ayy_qx_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_CY_INDEX] = Ayy_qy_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YY_CZ_INDEX] = Ayy_qz_t30;
        #endif //A_YY_DIST
        #ifdef A_YZ_DIST
            Ayz_udx_t30 = CONF_DIFF_FLUC_COEF * (Ayz_qx_t30*invAyz - ux_t30);
            Ayz_udy_t30 = CONF_DIFF_FLUC_COEF * (Ayz_qy_t30*invAyz - uy_t30);
            Ayz_udz_t30 = CONF_DIFF_FLUC_COEF * (Ayz_qz_t30*invAyz - uz_t30);

            #include COLREC_AYZ_RECONSTRUCTION

            #include "includeFiles/conformationTransport\popSave_Ayz.inc"
           
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_C_INDEX] = AyzVar;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_CX_INDEX] = Ayz_qx_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_CY_INDEX] = Ayz_qy_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_YZ_CZ_INDEX] = Ayz_qz_t30;
        #endif //A_YZ_DIST
        #ifdef A_ZZ_DIST
            Azz_udx_t30 = CONF_DIFF_FLUC_COEF * (Azz_qx_t30*invAzz - ux_t30);
            Azz_udy_t30 = CONF_DIFF_FLUC_COEF * (Azz_qy_t30*invAzz - uy_t30);
            Azz_udz_t30 = CONF_DIFF_FLUC_COEF * (Azz_qz_t30*invAzz - uz_t30);

            #include COLREC_AZZ_RECONSTRUCTION

            #include "includeFiles/conformationTransport\popSave_Azz.inc"

            fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_C_INDEX] = AzzVar;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_CX_INDEX] = Azz_qx_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_CY_INDEX] = Azz_qy_t30;
            fMom[baseIdx + BLOCK_LBM_SIZE * A_ZZ_CZ_INDEX] = Azz_qz_t30;
        #endif //A_ZZ_DIST
    #endif //CONVECTION_DIFFUSION_TRANSPORT

    #include "includeFiles/popSave.inc"

    //save velocities in the end in order to load next step to compute the gradient
    #ifdef COMPUTE_VEL_GRADIENT_FINITE_DIFFERENCE
    //#include "includeFiles/velSave.inc"
    //save conformation tensor components in the halo
    //#include "includeFIles/conformationTransport/confSave.inc"
    #endif //COMPUTE_VEL_GRADIENT_FINITE_DIFFERENCE

}
