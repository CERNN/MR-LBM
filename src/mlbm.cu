#include "hip/hip_runtime.h"
#include "mlbm.cuh"

__global__ void gpuMomCollisionStream(
    dfloat *fMom, unsigned char *dNodeType,
    dfloat *fGhostX_0, dfloat *fGhostX_1,
    dfloat *fGhostY_0, dfloat *fGhostY_1,
    dfloat *fGhostZ_0, dfloat *fGhostZ_1,
    dfloat *gGhostX_0, dfloat *gGhostX_1,
    dfloat *gGhostY_0, dfloat *gGhostY_1,
    dfloat *gGhostZ_0, dfloat *gGhostZ_1,
    #ifdef DENSITY_CORRECTION
    dfloat *d_mean_rho,
    #endif
    #ifdef LOCAL_FORCES
    dfloat *d_L_Fx, dfloat *d_L_Fy, dfloat *d_L_Fz,
    #endif 
    unsigned int step)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;
    dfloat pop[Q];

    // Load moments from global memory

    //rho'
    unsigned char nodeType = dNodeType[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)];
    if (nodeType == 0b11111111)  return;
    dfloat rhoVar = RHO_0 + fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 0, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat ux_t30     = 3*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 1, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uy_t30     = 3*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 2, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uz_t30     = 3*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 3, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat m_xx_t45   = 9*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 4, blockIdx.x, blockIdx.y, blockIdx.z)]/2;
    dfloat m_xy_t90   = 9*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 5, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat m_xz_t90   = 9*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 6, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat m_yy_t45   = 9*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 7, blockIdx.x, blockIdx.y, blockIdx.z)]/2;
    dfloat m_yz_t90   = 9*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 8, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat m_zz_t45   = 9*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 9, blockIdx.x, blockIdx.y, blockIdx.z)]/2;

    #ifdef NON_NEWTONIAN_FLUID
    dfloat omegaVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 10, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat t_omegaVar = 1 - omegaVar;
    dfloat tt_omegaVar = 1 - omegaVar/2;
    dfloat omegaVar_d2 = omegaVar / 2;
    dfloat tt_omega_t3 = tt_omegaVar * 3;
    #else
    dfloat omegaVar = OMEGA;
    #endif

    #ifdef LOCAL_FORCES
    dfloat L_Fx = d_L_Fx[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat L_Fy = d_L_Fy[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat L_Fz = d_L_Fz[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)];
    #else
    dfloat L_Fx = FX;
    dfloat L_Fy = FY;
    dfloat L_Fz = FZ;
    #endif 

    dfloat pics2;
    #ifndef HIGH_ORDER_COLLISION
    //calculate post collision populations
    dfloat multiplyTerm;
    multiplyTerm = rhoVar * W0;
    pics2 = 1.0 - cs2 * (m_xx_t45 + m_yy_t45 + m_zz_t45);

    pop[ 0] = multiplyTerm * (pics2);
    multiplyTerm = rhoVar * W1;
    pop[ 1] = multiplyTerm * (pics2 + ux_t30 + m_xx_t45);
    pop[ 2] = multiplyTerm * (pics2 - ux_t30 + m_xx_t45);
    pop[ 3] = multiplyTerm * (pics2 + uy_t30 + m_yy_t45);
    pop[ 4] = multiplyTerm * (pics2 - uy_t30 + m_yy_t45);
    pop[ 5] = multiplyTerm * (pics2 + uz_t30 + m_zz_t45);
    pop[ 6] = multiplyTerm * (pics2 - uz_t30 + m_zz_t45);
    multiplyTerm = rhoVar * W2;
    pop[ 7] = multiplyTerm * (pics2 +ux_t30 + uy_t30 + m_xx_t45 + m_yy_t45 + m_xy_t90);
    pop[ 8] = multiplyTerm * (pics2 -ux_t30 + m_xx_t45 - uy_t30 + m_yy_t45 + m_xy_t90);
    pop[ 9] = multiplyTerm * (pics2 +ux_t30 + uz_t30 + m_xx_t45 + m_zz_t45 + m_xz_t90);
    pop[10] = multiplyTerm * (pics2 -ux_t30 + m_xx_t45 - uz_t30 + m_zz_t45 + m_xz_t90);
    pop[11] = multiplyTerm * (pics2 +uy_t30 + uz_t30 + m_yy_t45 + m_zz_t45 + m_yz_t90);
    pop[12] = multiplyTerm * (pics2 -uy_t30 + m_yy_t45 - uz_t30 + m_zz_t45 + m_yz_t90);
    pop[13] = multiplyTerm * (pics2 +ux_t30 - uy_t30 + m_xx_t45 + m_yy_t45 - m_xy_t90);
    pop[14] = multiplyTerm * (pics2 -ux_t30 + uy_t30 + m_xx_t45 + m_yy_t45 - m_xy_t90);
    pop[15] = multiplyTerm * (pics2 +ux_t30 - uz_t30 + m_xx_t45 + m_zz_t45 - m_xz_t90);
    pop[16] = multiplyTerm * (pics2 -ux_t30 + uz_t30 + m_xx_t45 + m_zz_t45 - m_xz_t90);
    pop[17] = multiplyTerm * (pics2 +uy_t30 - uz_t30 + m_yy_t45 + m_zz_t45 - m_yz_t90);
    pop[18] = multiplyTerm * (pics2 -uy_t30 + uz_t30 + m_yy_t45 + m_zz_t45 - m_yz_t90);
    #ifdef D3Q27
    multiplyTerm = rhoVar * W3;
    pop[19] = multiplyTerm * (pics2 + ux_t30 + uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 + m_xz_t90 + m_yz_t90));
    pop[20] = multiplyTerm * (pics2 - ux_t30 - uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 + m_xz_t90 + m_yz_t90));
    pop[21] = multiplyTerm * (pics2 + ux_t30 + uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 - m_xz_t90 - m_yz_t90));
    pop[22] = multiplyTerm * (pics2 - ux_t30 - uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 - m_xz_t90 - m_yz_t90));
    pop[23] = multiplyTerm * (pics2 + ux_t30 - uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 - m_xz_t90 + m_yz_t90));
    pop[24] = multiplyTerm * (pics2 - ux_t30 + uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 - m_xz_t90 + m_yz_t90));
    pop[25] = multiplyTerm * (pics2 - ux_t30 + uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 + m_xz_t90 - m_yz_t90));
    pop[26] = multiplyTerm * (pics2 + ux_t30 - uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 + m_xz_t90 - m_yz_t90));
    #endif //D3Q27
    #endif //!HIGH_ORDER_COLLISION
    #ifdef HIGH_ORDER_COLLISION
            #ifdef HOME_LBM
            dfloat multiplyTerm;
            multiplyTerm = rhoVar * W0;
            pics2 = 1.0 - cs2 * (m_xx_t45 + m_yy_t45 + m_zz_t45);

            pop[ 0] = multiplyTerm * (pics2);
            multiplyTerm = rhoVar * W1;
            pop[ 1] = multiplyTerm * (pics2 + ux_t30 + m_xx_t45 + (ux_t30*uy_t30*uy_t30)/3 - (m_zz_t45*ux_t30)/3 - (m_xy_t90*uy_t30)/3 - (m_xz_t90*uz_t30)/3 - (m_yy_t45*ux_t30)/3 + (ux_t30*uz_t30*uz_t30)/3);
            pop[ 2] = multiplyTerm * (pics2 - ux_t30 + m_xx_t45 + (m_yy_t45*ux_t30)/3 + (m_zz_t45*ux_t30)/3 + (m_xy_t90*uy_t30)/3 + (m_xz_t90*uz_t30)/3 - (ux_t30*uy_t30*uy_t30)/3 - (ux_t30*uz_t30*uz_t30)/3);
            pop[ 3] = multiplyTerm * (pics2 + uy_t30 + m_yy_t45 + (ux_t30*ux_t30*uy_t30)/3 - (m_xx_t45*uy_t30)/3 - (m_zz_t45*uy_t30)/3 - (m_yz_t90*uz_t30)/3 - (m_xy_t90*ux_t30)/3 + (uy_t30*uz_t30*uz_t30)/3);
            pop[ 4] = multiplyTerm * (pics2 - uy_t30 + m_yy_t45 + (m_xy_t90*ux_t30)/3 + (m_xx_t45*uy_t30)/3 + (m_zz_t45*uy_t30)/3 + (m_yz_t90*uz_t30)/3 - (ux_t30*ux_t30*uy_t30)/3 - (uy_t30*uz_t30*uz_t30)/3);
            pop[ 5] = multiplyTerm * (pics2 + uz_t30 + m_zz_t45 + (ux_t30*ux_t30*uz_t30)/3 - (m_yz_t90*uy_t30)/3 - (m_xx_t45*uz_t30)/3 - (m_yy_t45*uz_t30)/3 - (m_xz_t90*ux_t30)/3 + (uy_t30*uy_t30*uz_t30)/3);
            pop[ 6] = multiplyTerm * (pics2 - uz_t30 + m_zz_t45 + (m_xz_t90*ux_t30)/3 + (m_yz_t90*uy_t30)/3 + (m_xx_t45*uz_t30)/3 + (m_yy_t45*uz_t30)/3 - (ux_t30*ux_t30*uz_t30)/3 - (uy_t30*uy_t30*uz_t30)/3);
            multiplyTerm = rhoVar * W2;
            pop[ 7] = multiplyTerm * (pics2 +ux_t30 + uy_t30 + m_xx_t45 + m_yy_t45 + m_xy_t90 + (2*m_xy_t90*ux_t30)/3 + (2*m_yy_t45*ux_t30)/3 - (m_zz_t45*ux_t30)/3 + (2*m_xx_t45*uy_t30)/3 + (2*m_xy_t90*uy_t30)/3 - (m_zz_t45*uy_t30)/3 - (m_xz_t90*uz_t30)/3 - (m_yz_t90*uz_t30)/3 - (2*ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*ux_t30*uy_t30)/3 + (ux_t30*uz_t30*uz_t30)/3 + (uy_t30*uz_t30*uz_t30)/3);
            pop[ 8] = multiplyTerm * (pics2 -ux_t30 + m_xx_t45 - uy_t30 + m_yy_t45 + m_xy_t90 + (m_zz_t45*ux_t30)/3 - (2*m_yy_t45*ux_t30)/3 - (2*m_xy_t90*ux_t30)/3 - (2*m_xx_t45*uy_t30)/3 - (2*m_xy_t90*uy_t30)/3 + (m_zz_t45*uy_t30)/3 + (m_xz_t90*uz_t30)/3 + (m_yz_t90*uz_t30)/3 + (2*ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*ux_t30*uy_t30)/3 - (ux_t30*uz_t30*uz_t30)/3 - (uy_t30*uz_t30*uz_t30)/3);
            pop[ 9] = multiplyTerm * (pics2 +ux_t30 + uz_t30 + m_xx_t45 + m_zz_t45 + m_xz_t90 + (2*m_xz_t90*ux_t30)/3 - (m_yy_t45*ux_t30)/3 + (2*m_zz_t45*ux_t30)/3 - (m_xy_t90*uy_t30)/3 - (m_yz_t90*uy_t30)/3 + (2*m_xx_t45*uz_t30)/3 + (2*m_xz_t90*uz_t30)/3 - (m_yy_t45*uz_t30)/3 + (ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*uz_t30*uz_t30)/3 - (2*ux_t30*ux_t30*uz_t30)/3 + (uy_t30*uy_t30*uz_t30)/3);
            pop[10] = multiplyTerm * (pics2 -ux_t30 + m_xx_t45 - uz_t30 + m_zz_t45 + m_xz_t90 + (m_yy_t45*ux_t30)/3 - (2*m_xz_t90*ux_t30)/3 - (2*m_zz_t45*ux_t30)/3 + (m_xy_t90*uy_t30)/3 + (m_yz_t90*uy_t30)/3 - (2*m_xx_t45*uz_t30)/3 - (2*m_xz_t90*uz_t30)/3 + (m_yy_t45*uz_t30)/3 - (ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*uz_t30*uz_t30)/3 + (2*ux_t30*ux_t30*uz_t30)/3 - (uy_t30*uy_t30*uz_t30)/3);
            pop[11] = multiplyTerm * (pics2 +uy_t30 + uz_t30 + m_yy_t45 + m_zz_t45 + m_yz_t90 + (2*m_yz_t90*uy_t30)/3 - (m_xz_t90*ux_t30)/3 - (m_xx_t45*uy_t30)/3 - (m_xy_t90*ux_t30)/3 + (2*m_zz_t45*uy_t30)/3 - (m_xx_t45*uz_t30)/3 + (2*m_yy_t45*uz_t30)/3 + (2*m_yz_t90*uz_t30)/3 + (ux_t30*ux_t30*uy_t30)/3 + (ux_t30*ux_t30*uz_t30)/3 - (2*uy_t30*uz_t30*uz_t30)/3 - (2*uy_t30*uy_t30*uz_t30)/3);
            pop[12] = multiplyTerm * (pics2 -uy_t30 + m_yy_t45 - uz_t30 + m_zz_t45 + m_yz_t90 + (m_xy_t90*ux_t30)/3 + (m_xz_t90*ux_t30)/3 + (m_xx_t45*uy_t30)/3 - (2*m_yz_t90*uy_t30)/3 - (2*m_zz_t45*uy_t30)/3 + (m_xx_t45*uz_t30)/3 - (2*m_yy_t45*uz_t30)/3 - (2*m_yz_t90*uz_t30)/3 - (ux_t30*ux_t30*uy_t30)/3 - (ux_t30*ux_t30*uz_t30)/3 + (2*uy_t30*uz_t30*uz_t30)/3 + (2*uy_t30*uy_t30*uz_t30)/3);
            pop[13] = multiplyTerm * (pics2 +ux_t30 - uy_t30 + m_xx_t45 + m_yy_t45 - m_xy_t90 + (2*m_yy_t45*ux_t30)/3 - (2*m_xy_t90*ux_t30)/3 - (m_zz_t45*ux_t30)/3 - (2*m_xx_t45*uy_t30)/3 + (2*m_xy_t90*uy_t30)/3 + (m_zz_t45*uy_t30)/3 - (m_xz_t90*uz_t30)/3 + (m_yz_t90*uz_t30)/3 - (2*ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*ux_t30*uy_t30)/3 + (ux_t30*uz_t30*uz_t30)/3 - (uy_t30*uz_t30*uz_t30)/3);
            pop[14] = multiplyTerm * (pics2 -ux_t30 + uy_t30 + m_xx_t45 + m_yy_t45 - m_xy_t90 + (2*m_xy_t90*ux_t30)/3 - (2*m_yy_t45*ux_t30)/3 + (m_zz_t45*ux_t30)/3 + (2*m_xx_t45*uy_t30)/3 - (2*m_xy_t90*uy_t30)/3 - (m_zz_t45*uy_t30)/3 + (m_xz_t90*uz_t30)/3 - (m_yz_t90*uz_t30)/3 + (2*ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*ux_t30*uy_t30)/3 - (ux_t30*uz_t30*uz_t30)/3 + (uy_t30*uz_t30*uz_t30)/3);
            pop[15] = multiplyTerm * (pics2 +ux_t30 - uz_t30 + m_xx_t45 + m_zz_t45 - m_xz_t90 + (2*m_zz_t45*ux_t30)/3 - (m_yy_t45*ux_t30)/3 - (2*m_xz_t90*ux_t30)/3 - (m_xy_t90*uy_t30)/3 + (m_yz_t90*uy_t30)/3 - (2*m_xx_t45*uz_t30)/3 + (2*m_xz_t90*uz_t30)/3 + (m_yy_t45*uz_t30)/3 + (ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*uz_t30*uz_t30)/3 + (2*ux_t30*ux_t30*uz_t30)/3 - (uy_t30*uy_t30*uz_t30)/3);
            pop[16] = multiplyTerm * (pics2 -ux_t30 + uz_t30 + m_xx_t45 + m_zz_t45 - m_xz_t90 + (2*m_xz_t90*ux_t30)/3 + (m_yy_t45*ux_t30)/3 - (2*m_zz_t45*ux_t30)/3 + (m_xy_t90*uy_t30)/3 - (m_yz_t90*uy_t30)/3 + (2*m_xx_t45*uz_t30)/3 - (2*m_xz_t90*uz_t30)/3 - (m_yy_t45*uz_t30)/3 - (ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*uz_t30*uz_t30)/3 - (2*ux_t30*ux_t30*uz_t30)/3 + (uy_t30*uy_t30*uz_t30)/3);
            pop[17] = multiplyTerm * (pics2 +uy_t30 - uz_t30 + m_yy_t45 + m_zz_t45 - m_yz_t90 + (m_xz_t90*ux_t30)/3 - (m_xy_t90*ux_t30)/3 - (m_xx_t45*uy_t30)/3 - (2*m_yz_t90*uy_t30)/3 + (2*m_zz_t45*uy_t30)/3 + (m_xx_t45*uz_t30)/3 - (2*m_yy_t45*uz_t30)/3 + (2*m_yz_t90*uz_t30)/3 + (ux_t30*ux_t30*uy_t30)/3 - (ux_t30*ux_t30*uz_t30)/3 - (2*uy_t30*uz_t30*uz_t30)/3 + (2*uy_t30*uy_t30*uz_t30)/3);
            pop[18] = multiplyTerm * (pics2 -uy_t30 + uz_t30 + m_yy_t45 + m_zz_t45 - m_yz_t90 + (m_xy_t90*ux_t30)/3 - (m_xz_t90*ux_t30)/3 + (m_xx_t45*uy_t30)/3 + (2*m_yz_t90*uy_t30)/3 - (2*m_zz_t45*uy_t30)/3 - (m_xx_t45*uz_t30)/3 + (2*m_yy_t45*uz_t30)/3 - (2*m_yz_t90*uz_t30)/3 - (ux_t30*ux_t30*uy_t30)/3 + (ux_t30*ux_t30*uz_t30)/3 + (2*uy_t30*uz_t30*uz_t30)/3 - (2*uy_t30*uy_t30*uz_t30)/3);
           #endif
    #endif //HIGH_ORDER_COLLISION

    #ifdef DYNAMIC_SHARED_MEMORY
    extern __shared__ dfloat s_pop[]; 
    #else
    __shared__ dfloat s_pop[BLOCK_LBM_SIZE * (Q - 1)];
    #endif
    //save populations in shared memory
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  0)] = pop[ 1];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  1)] = pop[ 2];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  2)] = pop[ 3];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  3)] = pop[ 4];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  4)] = pop[ 5];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  5)] = pop[ 6];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  6)] = pop[ 7];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  7)] = pop[ 8];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  8)] = pop[ 9];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  9)] = pop[10];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 10)] = pop[11];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 11)] = pop[12];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 12)] = pop[13];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 13)] = pop[14];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 14)] = pop[15];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 15)] = pop[16];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 16)] = pop[17];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 17)] = pop[18];
    #ifdef D3Q27
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 18)] = pop[19];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 19)] = pop[20];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 20)] = pop[21];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 21)] = pop[22];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 22)] = pop[23];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 23)] = pop[24];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 24)] = pop[25];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 25)] = pop[26];
    #endif //D3Q27

    //sync threads of the block so all populations are saved
    __syncthreads();

    /* pull */

    const unsigned short int xp1 = (threadIdx.x + 1 + BLOCK_NX) % BLOCK_NX;
    const unsigned short int xm1 = (threadIdx.x - 1 + BLOCK_NX) % BLOCK_NX;

    const unsigned short int yp1 = (threadIdx.y + 1 + BLOCK_NY) % BLOCK_NY;
    const unsigned short int ym1 = (threadIdx.y - 1 + BLOCK_NY) % BLOCK_NY;

    const unsigned short int zp1 = (threadIdx.z + 1 + BLOCK_NZ) % BLOCK_NZ;
    const unsigned short int zm1 = (threadIdx.z - 1 + BLOCK_NZ) % BLOCK_NZ;

    pop[ 1] = s_pop[idxPopBlock(xm1, threadIdx.y, threadIdx.z, 0)];
    pop[ 2] = s_pop[idxPopBlock(xp1, threadIdx.y, threadIdx.z, 1)];
    pop[ 3] = s_pop[idxPopBlock(threadIdx.x, ym1, threadIdx.z, 2)];
    pop[ 4] = s_pop[idxPopBlock(threadIdx.x, yp1, threadIdx.z, 3)];
    pop[ 5] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zm1, 4)];
    pop[ 6] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zp1, 5)];
    pop[ 7] = s_pop[idxPopBlock(xm1, ym1, threadIdx.z, 6)];
    pop[ 8] = s_pop[idxPopBlock(xp1, yp1, threadIdx.z, 7)];
    pop[ 9] = s_pop[idxPopBlock(xm1, threadIdx.y, zm1, 8)];
    pop[10] = s_pop[idxPopBlock(xp1, threadIdx.y, zp1, 9)];
    pop[11] = s_pop[idxPopBlock(threadIdx.x, ym1, zm1, 10)];
    pop[12] = s_pop[idxPopBlock(threadIdx.x, yp1, zp1, 11)];
    pop[13] = s_pop[idxPopBlock(xm1, yp1, threadIdx.z, 12)];
    pop[14] = s_pop[idxPopBlock(xp1, ym1, threadIdx.z, 13)];
    pop[15] = s_pop[idxPopBlock(xm1, threadIdx.y, zp1, 14)];
    pop[16] = s_pop[idxPopBlock(xp1, threadIdx.y, zm1, 15)];
    pop[17] = s_pop[idxPopBlock(threadIdx.x, ym1, zp1, 16)];
    pop[18] = s_pop[idxPopBlock(threadIdx.x, yp1, zm1, 17)];
    #ifdef D3Q27
    pop[19] = s_pop[idxPopBlock(xm1, ym1, zm1, 18)];
    pop[20] = s_pop[idxPopBlock(xp1, yp1, zp1, 19)];
    pop[21] = s_pop[idxPopBlock(xm1, ym1, zp1, 20)];
    pop[22] = s_pop[idxPopBlock(xp1, yp1, zm1, 21)];
    pop[23] = s_pop[idxPopBlock(xm1, yp1, zm1, 22)];
    pop[24] = s_pop[idxPopBlock(xp1, ym1, zp1, 23)];
    pop[25] = s_pop[idxPopBlock(xp1, ym1, zm1, 24)];
    pop[26] = s_pop[idxPopBlock(xm1, yp1, zp1, 25)];
    #endif

    /* load pop from global in cover nodes */


   
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;

    const int txm1 = (tx-1+BLOCK_NX)%BLOCK_NX;
    const int txp1 = (tx+1+BLOCK_NX)%BLOCK_NX;

    const int tym1 = (ty-1+BLOCK_NY)%BLOCK_NY;
    const int typ1 = (ty+1+BLOCK_NY)%BLOCK_NY;

    const int tzm1 = (tz-1+BLOCK_NZ)%BLOCK_NZ;
    const int tzp1 = (tz+1+BLOCK_NZ)%BLOCK_NZ;

    const int bxm1 = (bx-1+NUM_BLOCK_X)%NUM_BLOCK_X;
    const int bxp1 = (bx+1+NUM_BLOCK_X)%NUM_BLOCK_X;

    const int bym1 = (by-1+NUM_BLOCK_Y)%NUM_BLOCK_Y;
    const int byp1 = (by+1+NUM_BLOCK_Y)%NUM_BLOCK_Y;

    const int bzm1 = (bz-1+NUM_BLOCK_Z)%NUM_BLOCK_Z;
    const int bzp1 = (bz+1+NUM_BLOCK_Z)%NUM_BLOCK_Z;


    #include "interfaceInclude/popLoad"

    //NOTE : STREAMING DONE, APPLY BOUNDARY CONDITION AND COMPUTE POST STREAMING MOMENTS
    #ifdef BC_POPULATION_BASED

        if (nodeType){
            #include BC_PATH
        }
            //calculate streaming moments
        #ifdef D3Q19
            //equation3
            rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18];
            dfloat invRho = 1 / rhoVar;
            //equation4 + force correction
            ux_t30 = ((pop[1] - pop[2] + pop[7] - pop[8] + pop[9] - pop[10] + pop[13] - pop[14] + pop[15] - pop[16]) + 0.5 * L_Fx) * invRho;
            uy_t30 = ((pop[3] - pop[4] + pop[7] - pop[8] + pop[11] - pop[12] + pop[14] - pop[13] + pop[17] - pop[18]) + 0.5 * L_Fy) * invRho;
            uz_t30 = ((pop[5] - pop[6] + pop[9] - pop[10] + pop[11] - pop[12] + pop[16] - pop[15] + pop[18] - pop[17]) + 0.5 * L_Fz) * invRho;


            //equation5
            m_xx_t45 = (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16])* invRho - cs2;
            m_xy_t90 = (pop[7] - pop[13] + pop[8] - pop[14])* invRho;
            m_xz_t90 = (pop[9] - pop[15] + pop[10] - pop[16])* invRho;
            m_yy_t45 = (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18])* invRho - cs2;
            m_yz_t90 = (pop[11] - pop[17] + pop[12] - pop[18])* invRho;
            m_zz_t45 = (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18])* invRho - cs2;


        #endif
        #ifdef D3Q27
            rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26];
            dfloat invRho = 1 / rhoVar;
            ux_t30 = ((pop[1] + pop[7] + pop[9] + pop[13] + pop[15] + pop[19] + pop[21] + pop[23] + pop[26])  - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16] + pop[20] + pop[22] + pop[24] + pop[25]) + 0.5 * L_Fx) * invRho;
            uy_t30 = ((pop[3] + pop[7] + pop[11] + pop[14] + pop[17] + pop[19] + pop[21] + pop[24] + pop[25]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18] + pop[20] + pop[22] + pop[23] + pop[26]) + 0.5 * L_Fy) * invRho;
            uz_t30 = ((pop[5] + pop[9] + pop[11] + pop[16] + pop[18] + pop[19] + pop[22] + pop[23] + pop[25]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17] + pop[20] + pop[21] + pop[24] + pop[26]) + 0.5 * L_Fz) * invRho;

            m_xx_t45 = ( (pop[ 1] + pop[ 2] + pop[ 7] + pop[ 8] + pop[ 9] + pop[10]  +  pop[13] + pop[14] + pop[15] + pop[16] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
            m_xy_t90 = (((pop[ 7] + pop[ 8] + pop[19] + pop[20] + pop[21] + pop[22]) - (pop[13] + pop[14] + pop[23] + pop[24] + pop[25] + pop[26])) )* invRho;
            m_xz_t90 = (((pop[ 9] + pop[10] + pop[19] + pop[20] + pop[23] + pop[24]) - (pop[15] + pop[16] + pop[21] + pop[22] + pop[25] + pop[26])) )* invRho;
            m_yy_t45 = ( (pop[ 3] + pop[ 4] + pop[ 7] + pop[ 8] + pop[11] + pop[12]  +  pop[13] + pop[14] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
            m_yz_t90 = (((pop[11] + pop[12] + pop[19] + pop[20] + pop[25] + pop[26]) - (pop[17] + pop[18] + pop[21] + pop[22] + pop[23] + pop[24])))* invRho;
            m_zz_t45 = ( (pop[ 5] + pop[ 6] + pop[ 9] + pop[10] + pop[11] + pop[12]  +  pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
        #endif
    #endif

    #ifdef BC_MOMENT_BASED
        dfloat invRho;
        if(nodeType != BULK){
            #include BC_PATH

            invRho = 1.0 / rho;
        }else{

            //calculate streaming moments
            #ifdef D3Q19
                //equation3
                rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18];
                invRho = 1 / rhoVar;
                //equation4 + force correction
                ux_t30 = ((pop[1] - pop[2] + pop[7] - pop[ 8] + pop[ 9] - pop[10] + pop[13] - pop[14] + pop[15] - pop[16]) + L_Fx/2) * invRho;
                uy_t30 = ((pop[3] - pop[4] + pop[7] - pop[ 8] + pop[11] - pop[12] + pop[14] - pop[13] + pop[17] - pop[18]) + L_Fy/2) * invRho;
                uz_t30 = ((pop[5] - pop[6] + pop[9] - pop[10] + pop[11] - pop[12] + pop[16] - pop[15] + pop[18] - pop[17]) + L_Fz/2) * invRho;

                //equation5
                m_xx_t45 = (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16])* invRho - cs2;
                m_xy_t90 = (pop[7] - pop[13] + pop[8] - pop[14])* invRho;
                m_xz_t90 = (pop[9] - pop[15] + pop[10] - pop[16])* invRho;
                m_yy_t45 = (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18])* invRho - cs2;
                m_yz_t90 = (pop[11] - pop[17] + pop[12] - pop[18])* invRho;
                m_zz_t45 = (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18])* invRho - cs2;


            #endif
            #ifdef D3Q27
                rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26];
                invRho = 1 / rhoVar;
                ux_t30 = ((pop[1] + pop[7] + pop[9] + pop[13] + pop[15] + pop[19] + pop[21] + pop[23] + pop[26])  - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16] + pop[20] + pop[22] + pop[24] + pop[25]) + 0.5 * L_Fx) * invRho;
                uy_t30 = ((pop[3] + pop[7] + pop[11] + pop[14] + pop[17] + pop[19] + pop[21] + pop[24] + pop[25]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18] + pop[20] + pop[22] + pop[23] + pop[26]) + 0.5 * L_Fy) * invRho;
                uz_t30 = ((pop[5] + pop[9] + pop[11] + pop[16] + pop[18] + pop[19] + pop[22] + pop[23] + pop[25]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17] + pop[20] + pop[21] + pop[24] + pop[26]) + 0.5 * L_Fz) * invRho;

                m_xx_t45 = ( (pop[ 1] + pop[ 2] + pop[ 7] + pop[ 8] + pop[ 9] + pop[10]  +  pop[13] + pop[14] + pop[15] + pop[16] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
                m_xy_t90 = (((pop[ 7] + pop[ 8] + pop[19] + pop[20] + pop[21] + pop[22]) - (pop[13] + pop[14] + pop[23] + pop[24] + pop[25] + pop[26])) )* invRho;
                m_xz_t90 = (((pop[ 9] + pop[10] + pop[19] + pop[20] + pop[23] + pop[24]) - (pop[15] + pop[16] + pop[21] + pop[22] + pop[25] + pop[26])) )* invRho;
                m_yy_t45 = ( (pop[ 3] + pop[ 4] + pop[ 7] + pop[ 8] + pop[11] + pop[12]  +  pop[13] + pop[14] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
                m_yz_t90 = (((pop[11] + pop[12] + pop[19] + pop[20] + pop[25] + pop[26]) - (pop[17] + pop[18] + pop[21] + pop[22] + pop[23] + pop[24])))* invRho;
                m_zz_t45 = ( (pop[ 5] + pop[ 6] + pop[ 9] + pop[10] + pop[11] + pop[12]  +  pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]))* invRho - cs2;
            #endif
        }
    #endif // BC_MOMENT_BASED


    // MOMENTS DETERMINED, COMPUTE OMEGA IF NON-NEWTONIAN FLUID
    #ifdef NON_NEWTONIAN_FLUID

    const dfloat S_XX = rhoVar * (m_xx_t45 - ux_t30*ux_t30);
    const dfloat S_YY = rhoVar * (m_yy_t45 - uy_t30*uy_t30);
    const dfloat S_ZZ = rhoVar * (m_zz_t45 - uz_t30*uz_t30);
    const dfloat S_XY = rhoVar * (m_xy_t90 - ux_t30*uy_t30);
    const dfloat S_XZ = rhoVar * (m_xz_t90 - ux_t30*uz_t30);
    const dfloat S_YZ = rhoVar * (m_yz_t90 - uy_t30*uz_t30);

    const dfloat uFxxd2 = ux_t30*L_Fx; // d2 = uFxx Divided by two
    const dfloat uFyyd2 = uy_t30*L_Fy;
    const dfloat uFzzd2 = uz_t30*L_Fz;
    const dfloat uFxyd2 = (ux_t30*L_Fy + uy_t30*L_Fx) / 2;
    const dfloat uFxzd2 = (ux_t30*L_Fz + uz_t30*L_Fx) / 2;
    const dfloat uFyzd2 = (uy_t30*L_Fz + uz_t30*L_Fy) / 2;

    const dfloat auxStressMag = sqrt(0.5 * (
        (S_XX + uFxxd2) * (S_XX + uFxxd2) +(S_YY + uFyyd2) * (S_YY + uFyyd2) + (S_ZZ + uFzzd2) * (S_ZZ + uFzzd2) +
        2 * ((S_XY + uFxyd2) * (S_XY + uFxyd2) + (S_XZ + uFxzd2) * (S_XZ + uFxzd2) + (S_YZ + uFyzd2) * (S_YZ + uFyzd2))));

    /*
    dfloat eta = (1.0/omegaVar - 0.5) / 3.0;
    dfloat gamma_dot = (1 - 0.5 * (omegaVar)) * auxStressMag / eta;
    eta = VISC + S_Y/gamma_dot;
    omegaVar = omegaVar;// 1.0 / (0.5 + 3.0 * eta);
    */

    omegaVar = calcOmega(omegaVar, auxStressMag);

    t_omegaVar = 1 - omegaVar;
    tt_omegaVar = 1 - 0.5*omegaVar;
    omegaVar_d2 = omegaVar / 2.0;
    tt_omega_t3 = tt_omegaVar * 3.0;
    #endif//  NON_NEWTONIAN_FLUID


    // COLLIDE

    //Collide Moments
    // multiply moments by as2 -- as4*0.5 -- as4 - add correction to m_alpha_beta
    #ifndef HIGH_ORDER_COLLISION
        ux_t30 = 3 * ux_t30;
        uy_t30 = 3 * uy_t30;
        uz_t30 = 3 * uz_t30;

        m_xx_t45 = 9 * (m_xx_t45)/2;
        m_xy_t90 = 9 * (m_xy_t90);
        m_xz_t90 = 9 * (m_xz_t90);
        m_yy_t45 = 9 * (m_yy_t45)/2;
        m_yz_t90 = 9 * (m_yz_t90);
        m_zz_t45 = 9 * (m_zz_t45)/2;

        #ifdef DENSITY_CORRECTION
            //printf("%f ",d_mean_rho[0]-1.0) ;
            rhoVar -= (d_mean_rho[0]-1e-7) ;
            invRho = 1/rhoVar;
        #endif // DENSITY_CORRECTION
        #ifdef NON_NEWTONIAN_FLUID
            dfloat invRho_mt15 = -3*invRho/2;
            ux_t30 = (t_omegaVar * (ux_t30 + invRho_mt15 * L_Fx ) + omegaVar * ux_t30 + tt_omega_t3 * L_Fx);
            uy_t30 = (t_omegaVar * (uy_t30 + invRho_mt15 * L_Fy ) + omegaVar * uy_t30 + tt_omega_t3 * L_Fy);
            uz_t30 = (t_omegaVar * (uz_t30 + invRho_mt15 * L_Fz ) + omegaVar * uz_t30 + tt_omega_t3 * L_Fz);
            
            //equation 90
            m_xx_t45 = (t_omegaVar * m_xx_t45  +   omegaVar_d2 * ux_t30 * ux_t30  - invRho_mt15 * tt_omegaVar * (L_Fx * ux_t30 + L_Fx * ux_t30));
            m_yy_t45 = (t_omegaVar * m_yy_t45  +   omegaVar_d2 * uy_t30 * uy_t30  - invRho_mt15 * tt_omegaVar * (L_Fy * uy_t30 + L_Fy * uy_t30));
            m_zz_t45 = (t_omegaVar * m_zz_t45  +   omegaVar_d2 * uz_t30 * uz_t30  - invRho_mt15 * tt_omegaVar * (L_Fz * uz_t30 + L_Fz * uz_t30));

            m_xy_t90 = (t_omegaVar * m_xy_t90  +   omegaVar * ux_t30 * uy_t30    +    tt_omega_t3 *invRho* (L_Fx * uy_t30 + L_Fy * ux_t30));
            m_xz_t90 = (t_omegaVar * m_xz_t90  +   omegaVar * ux_t30 * uz_t30    +    tt_omega_t3 *invRho* (L_Fx * uz_t30 + L_Fz * ux_t30));
            m_yz_t90 = (t_omegaVar * m_yz_t90  +   omegaVar * uy_t30 * uz_t30    +    tt_omega_t3 *invRho* (L_Fy * uz_t30 + L_Fz * uy_t30));
        #endif // NON_NEWTONIAN_FLUID
        #ifndef NON_NEWTONIAN_FLUID 
            dfloat invRho_mt15 = -3*invRho/2;
            ux_t30 = (T_OMEGA * (ux_t30 + invRho_mt15 * L_Fx ) + OMEGA * ux_t30 + TT_OMEGA_T3 * L_Fx);
            uy_t30 = (T_OMEGA * (uy_t30 + invRho_mt15 * L_Fy ) + OMEGA * uy_t30 + TT_OMEGA_T3 * L_Fy);
            uz_t30 = (T_OMEGA * (uz_t30 + invRho_mt15 * L_Fz ) + OMEGA * uz_t30 + TT_OMEGA_T3 * L_Fz);
            
            //equation 90
            m_xx_t45 = (T_OMEGA * m_xx_t45  +   OMEGAd2 * ux_t30 * ux_t30    - invRho_mt15 * TT_OMEGA * (L_Fx * ux_t30 + L_Fx * ux_t30));
            m_yy_t45 = (T_OMEGA * m_yy_t45  +   OMEGAd2 * uy_t30 * uy_t30    - invRho_mt15 * TT_OMEGA * (L_Fy * uy_t30 + L_Fy * uy_t30));
            m_zz_t45 = (T_OMEGA * m_zz_t45  +   OMEGAd2 * uz_t30 * uz_t30    - invRho_mt15 * TT_OMEGA * (L_Fz * uz_t30 + L_Fz * uz_t30));

            m_xy_t90 = (T_OMEGA * m_xy_t90  +     OMEGA * ux_t30 * uy_t30    +    TT_OMEGA_T3 *invRho* (L_Fx * uy_t30 + L_Fy * ux_t30));
            m_xz_t90 = (T_OMEGA * m_xz_t90  +     OMEGA * ux_t30 * uz_t30    +    TT_OMEGA_T3 *invRho* (L_Fx * uz_t30 + L_Fz * ux_t30));
            m_yz_t90 = (T_OMEGA * m_yz_t90  +     OMEGA * uy_t30 * uz_t30    +    TT_OMEGA_T3 *invRho* (L_Fy * uz_t30 + L_Fz * uy_t30));
        #endif //!_NON_NEWTONIAN_FLUID
    #endif //!_HIGH_ORDER_COLLISION

    //USING HIGH
    #ifdef HIGH_ORDER_COLLISION

    #ifdef HO_RR

        dfloat ux = ux_t30 + L_Fx*invRho/2;
        dfloat uy = uy_t30 + L_Fy*invRho/2;
        dfloat uz = uz_t30 + L_Fz*invRho/2;

        //matlab original
        dfloat m_xx = (ux_t30*ux_t30 - (9*uy_t30*uy_t30*uz_t30*uz_t30)/2 + (3*m_zz_t45*uy_t30*uy_t30)/4 + 3*m_yz_t90*uy_t30*uz_t30 + (3*m_yy_t45*uz_t30*uz_t30)/4 - m_xx_t45)*omegaVar + ((15*uy_t30*uy_t30*uz_t30*uz_t30)/4 - (3*m_zz_t45*uy_t30*uy_t30)/4 - 3*m_yz_t90*uy_t30*uz_t30 - (3*m_yy_t45*uz_t30*uz_t30)/4 + m_xx_t45);
        dfloat m_yy = ((3*m_zz_t45*ux_t30*ux_t30)/4 - (9*ux_t30*ux_t30*uz_t30*uz_t30)/2 + 3*m_xz_t90*ux_t30*uz_t30 + uy_t30*uy_t30 + (3*m_xx_t45*uz_t30*uz_t30)/4 - m_yy_t45)*omegaVar + ((15*ux_t30*ux_t30*uz_t30*uz_t30)/4 - (3*m_zz_t45*ux_t30*ux_t30)/4 - 3*m_xz_t90*ux_t30*uz_t30 - (3*m_xx_t45*uz_t30*uz_t30)/4 + m_yy_t45);
        dfloat m_zz = ((3*m_yy_t45*ux_t30*ux_t30)/4 - (9*ux_t30*ux_t30*uy_t30*uy_t30)/2 + 3*m_xy_t90*ux_t30*uy_t30 + (3*m_xx_t45*uy_t30*uy_t30)/4 + uz_t30*uz_t30 - m_zz_t45)*omegaVar + ((15*ux_t30*ux_t30*uy_t30*uy_t30)/4 - (3*m_yy_t45*ux_t30*ux_t30)/4 - 3*m_xy_t90*ux_t30*uy_t30 - (3*m_xx_t45*uy_t30*uy_t30)/4 + m_zz_t45);
        dfloat m_xy = (ux_t30*uy_t30 - m_xy_t90)*omegaVar + m_xy_t90;
        dfloat m_xz = (ux_t30*uz_t30 - m_xz_t90)*omegaVar + m_xz_t90;
        dfloat m_yz = (uy_t30*uz_t30 - m_yz_t90)*omegaVar + m_yz_t90;


        //dfloat m_xx = ((((m_zz_t45*uy_t30*uy_t30 + m_yy_t45*uz_t30*uz_t30) - 6*uy_t30*uy_t30*uz_t30*uz_t30)/4 + m_yz_t90*uy_t30*uz_t30)*3 + ux_t30*ux_t30 - m_xx_t45)*omegaVar + (((5*uy_t30*uy_t30*uz_t30*uz_t30 - m_zz_t45*uy_t30*uy_t30 - m_yy_t45*uz_t30*uz_t30)/4 - m_yz_t90*uy_t30*uz_t30)*3 + m_xx_t45);
        //dfloat m_yy = ((((m_zz_t45*ux_t30*ux_t30 + m_xx_t45*uz_t30*uz_t30) - 6*ux_t30*ux_t30*uz_t30*uz_t30)/4 + m_xz_t90*ux_t30*uz_t30)*3 + uy_t30*uy_t30 - m_yy_t45)*omegaVar + (((5*ux_t30*ux_t30*uz_t30*uz_t30 - m_zz_t45*ux_t30*ux_t30 - m_xx_t45*uz_t30*uz_t30)/4 - m_xz_t90*ux_t30*uz_t30)*3 + m_yy_t45);
        //dfloat m_zz = ((((m_yy_t45*ux_t30*ux_t30 + m_xx_t45*uy_t30*uy_t30) - 6*ux_t30*ux_t30*uy_t30*uy_t30)/4 + m_xy_t90*ux_t30*uy_t30)*3 + uz_t30*uz_t30 - m_zz_t45)*omegaVar + (((5*ux_t30*ux_t30*uy_t30*uy_t30 - m_yy_t45*ux_t30*ux_t30 - m_xx_t45*uy_t30*uy_t30)/4 - m_xy_t90*ux_t30*uy_t30)*3 + m_zz_t45);
        //dfloat m_xy = (ux_t30*uy_t30 - m_xy_t90)*omegaVar + m_xy_t90;
        //dfloat m_xz = (ux_t30*uz_t30 - m_xz_t90)*omegaVar + m_xz_t90;
        //dfloat m_yz = (uy_t30*uz_t30 - m_yz_t90)*omegaVar + m_yz_t90;
    #endif //HO_RR
    #ifdef HOME_LBM
        dfloat ux = ux_t30 + L_Fx*invRho/2;
        dfloat uy = uy_t30 + L_Fy*invRho/2;
        dfloat uz = uz_t30 + L_Fz*invRho/2;


        dfloat m_xy = T_OMEGA * m_xy_t90 + OMEGA*ux_t30*uy_t30 + TT_OMEGA * invRho * (L_Fx * uy_t30 + L_Fy * ux_t30);
        dfloat m_xz = T_OMEGA * m_xz_t90 + OMEGA*ux_t30*uz_t30 + TT_OMEGA * invRho * (L_Fx * uz_t30 + L_Fz * ux_t30);
        dfloat m_yz = T_OMEGA * m_yz_t90 + OMEGA*uy_t30*uz_t30 + TT_OMEGA * invRho * (L_Fy * uz_t30 + L_Fz * uy_t30);

        dfloat m_xx = ONETHIRD* (T_OMEGA * (2*m_xx_t45 - m_yy_t45 - m_zz_t45) +  (ux_t30*ux_t30 + uy_t30*uy_t30 + uz_t30*uz_t30) + OMEGA*(2*ux_t30*ux_t30 - uy_t30*uy_t30 - uz_t30*uz_t30) + invRho*T_OMEGA*(L_Fx*ux_t30*2 - L_Fy*uy_t30 - L_Fz*uz_t30)) + invRho*L_Fx*ux_t30;
        dfloat m_yy = ONETHIRD* (T_OMEGA * (2*m_yy_t45 - m_xx_t45 - m_zz_t45) +  (ux_t30*ux_t30 + uy_t30*uy_t30 + uz_t30*uz_t30) + OMEGA*(2*uy_t30*uy_t30 - ux_t30*ux_t30 - uz_t30*uz_t30) + invRho*T_OMEGA*(L_Fy*uy_t30*2 - L_Fx*ux_t30 - L_Fz*uz_t30)) + invRho*L_Fy*uy_t30;
        dfloat m_zz = ONETHIRD* (T_OMEGA * (2*m_zz_t45 - m_xx_t45 - m_yy_t45) +  (ux_t30*ux_t30 + uy_t30*uy_t30 + uz_t30*uz_t30) + OMEGA*(2*uz_t30*uz_t30 - ux_t30*ux_t30 - uy_t30*uy_t30) + invRho*T_OMEGA*(L_Fz*uz_t30*2 - L_Fx*ux_t30 - L_Fy*uy_t30)) + invRho*L_Fz*uz_t30;
    #endif
    ux_t30 = 3 * ux;
    uy_t30 = 3 * uy;
    uz_t30 = 3 * uz;

    m_xx_t45 = 9*(m_xx)/2;
    m_xy_t90 = 9*(m_xy);
    m_xz_t90 = 9*(m_xz);
    m_yy_t45 = 9*(m_yy)/2;
    m_yz_t90 = 9*(m_yz);
    m_zz_t45 = 9*(m_zz)/2;



    #endif //HIGH_ORDER_COLLISION



    //calculate post collision populations
    #ifndef HIGH_ORDER_COLLISION
    multiplyTerm = rhoVar * W0;
    pics2 = 1.0 - cs2 * (m_xx_t45 + m_yy_t45 + m_zz_t45);

    pop[ 0] = multiplyTerm * (pics2);
    multiplyTerm = rhoVar * W1;
    pop[ 1] = multiplyTerm * (pics2 + ux_t30 + m_xx_t45);
    pop[ 2] = multiplyTerm * (pics2 - ux_t30 + m_xx_t45);
    pop[ 3] = multiplyTerm * (pics2 + uy_t30 + m_yy_t45);
    pop[ 4] = multiplyTerm * (pics2 - uy_t30 + m_yy_t45);
    pop[ 5] = multiplyTerm * (pics2 + uz_t30 + m_zz_t45);
    pop[ 6] = multiplyTerm * (pics2 - uz_t30 + m_zz_t45);
    multiplyTerm = rhoVar * W2;
    pop[ 7] = multiplyTerm * (pics2 + ( ux_t30 + uy_t30) + (m_xx_t45 + m_yy_t45) + m_xy_t90);
    pop[ 8] = multiplyTerm * (pics2 + (-ux_t30 - uy_t30) + (m_xx_t45 + m_yy_t45) + m_xy_t90);
    pop[ 9] = multiplyTerm * (pics2 + ( ux_t30 + uz_t30) + (m_xx_t45 + m_zz_t45) + m_xz_t90);
    pop[10] = multiplyTerm * (pics2 + (-ux_t30 - uz_t30) + (m_xx_t45 + m_zz_t45) + m_xz_t90);
    pop[11] = multiplyTerm * (pics2 + ( uy_t30 + uz_t30) + (m_yy_t45 + m_zz_t45) + m_yz_t90);
    pop[12] = multiplyTerm * (pics2 + (-uy_t30 - uz_t30) + (m_yy_t45 + m_zz_t45) + m_yz_t90);
    pop[13] = multiplyTerm * (pics2 + ( ux_t30 - uy_t30) + (m_xx_t45 + m_yy_t45) - m_xy_t90);
    pop[14] = multiplyTerm * (pics2 + (-ux_t30 + uy_t30) + (m_xx_t45 + m_yy_t45) - m_xy_t90);
    pop[15] = multiplyTerm * (pics2 + ( ux_t30 - uz_t30) + (m_xx_t45 + m_zz_t45) - m_xz_t90);
    pop[16] = multiplyTerm * (pics2 + (-ux_t30 + uz_t30) + (m_xx_t45 + m_zz_t45) - m_xz_t90);
    pop[17] = multiplyTerm * (pics2 + ( uy_t30 - uz_t30) + (m_yy_t45 + m_zz_t45) - m_yz_t90);
    pop[18] = multiplyTerm * (pics2 + (-uy_t30 + uz_t30) + (m_yy_t45 + m_zz_t45) - m_yz_t90);   
    #ifdef D3Q27
    multiplyTerm = rhoVar * W3;
    pop[19] = multiplyTerm * (pics2 + ux_t30 + uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 + m_xz_t90 + m_yz_t90));
    pop[20] = multiplyTerm * (pics2 - ux_t30 - uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 + m_xz_t90 + m_yz_t90));
    pop[21] = multiplyTerm * (pics2 + ux_t30 + uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 - m_xz_t90 - m_yz_t90));
    pop[22] = multiplyTerm * (pics2 - ux_t30 - uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 + (m_xy_t90 - m_xz_t90 - m_yz_t90));
    pop[23] = multiplyTerm * (pics2 + ux_t30 - uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 - m_xz_t90 + m_yz_t90));
    pop[24] = multiplyTerm * (pics2 - ux_t30 + uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 - m_xz_t90 + m_yz_t90));
    pop[25] = multiplyTerm * (pics2 - ux_t30 + uy_t30 + uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 + m_xz_t90 - m_yz_t90));
    pop[26] = multiplyTerm * (pics2 + ux_t30 - uy_t30 - uz_t30 + m_xx_t45 + m_yy_t45 + m_zz_t45 - (m_xy_t90 + m_xz_t90 - m_yz_t90));
    #endif //D3Q27
    #endif //!HIGH_ORDER_COLLISION
    #ifdef HIGH_ORDER_COLLISION
    
        #ifdef HOME_LBM
    multiplyTerm = rhoVar * W0;
    pics2 = 1.0 - cs2 * (m_xx_t45 + m_yy_t45 + m_zz_t45);

    pop[ 0] = multiplyTerm * (pics2);
    multiplyTerm = rhoVar * W1;
    pop[ 1] = multiplyTerm * (pics2 + ux_t30 + m_xx_t45 + (ux_t30*uy_t30*uy_t30)/3 - (m_zz_t45*ux_t30)/3 - (m_xy_t90*uy_t30)/3 - (m_xz_t90*uz_t30)/3 - (m_yy_t45*ux_t30)/3 + (ux_t30*uz_t30*uz_t30)/3);
    pop[ 2] = multiplyTerm * (pics2 - ux_t30 + m_xx_t45 + (m_yy_t45*ux_t30)/3 + (m_zz_t45*ux_t30)/3 + (m_xy_t90*uy_t30)/3 + (m_xz_t90*uz_t30)/3 - (ux_t30*uy_t30*uy_t30)/3 - (ux_t30*uz_t30*uz_t30)/3);
    pop[ 3] = multiplyTerm * (pics2 + uy_t30 + m_yy_t45 + (ux_t30*ux_t30*uy_t30)/3 - (m_xx_t45*uy_t30)/3 - (m_zz_t45*uy_t30)/3 - (m_yz_t90*uz_t30)/3 - (m_xy_t90*ux_t30)/3 + (uy_t30*uz_t30*uz_t30)/3);
    pop[ 4] = multiplyTerm * (pics2 - uy_t30 + m_yy_t45 + (m_xy_t90*ux_t30)/3 + (m_xx_t45*uy_t30)/3 + (m_zz_t45*uy_t30)/3 + (m_yz_t90*uz_t30)/3 - (ux_t30*ux_t30*uy_t30)/3 - (uy_t30*uz_t30*uz_t30)/3);
    pop[ 5] = multiplyTerm * (pics2 + uz_t30 + m_zz_t45 + (ux_t30*ux_t30*uz_t30)/3 - (m_yz_t90*uy_t30)/3 - (m_xx_t45*uz_t30)/3 - (m_yy_t45*uz_t30)/3 - (m_xz_t90*ux_t30)/3 + (uy_t30*uy_t30*uz_t30)/3);
    pop[ 6] = multiplyTerm * (pics2 - uz_t30 + m_zz_t45 + (m_xz_t90*ux_t30)/3 + (m_yz_t90*uy_t30)/3 + (m_xx_t45*uz_t30)/3 + (m_yy_t45*uz_t30)/3 - (ux_t30*ux_t30*uz_t30)/3 - (uy_t30*uy_t30*uz_t30)/3);
    
    multiplyTerm = rhoVar * W2;
    pop[ 7] = multiplyTerm * (pics2 +ux_t30 + uy_t30 + m_xx_t45 + m_yy_t45 + m_xy_t90 + (2*m_xy_t90*ux_t30)/3 + (2*m_yy_t45*ux_t30)/3 - (m_zz_t45*ux_t30)/3 + (2*m_xx_t45*uy_t30)/3 + (2*m_xy_t90*uy_t30)/3 - (m_zz_t45*uy_t30)/3 - (m_xz_t90*uz_t30)/3 - (m_yz_t90*uz_t30)/3 - (2*ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*ux_t30*uy_t30)/3 + (ux_t30*uz_t30*uz_t30)/3 + (uy_t30*uz_t30*uz_t30)/3);
    pop[ 8] = multiplyTerm * (pics2 -ux_t30 + m_xx_t45 - uy_t30 + m_yy_t45 + m_xy_t90 + (m_zz_t45*ux_t30)/3 - (2*m_yy_t45*ux_t30)/3 - (2*m_xy_t90*ux_t30)/3 - (2*m_xx_t45*uy_t30)/3 - (2*m_xy_t90*uy_t30)/3 + (m_zz_t45*uy_t30)/3 + (m_xz_t90*uz_t30)/3 + (m_yz_t90*uz_t30)/3 + (2*ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*ux_t30*uy_t30)/3 - (ux_t30*uz_t30*uz_t30)/3 - (uy_t30*uz_t30*uz_t30)/3);
    pop[ 9] = multiplyTerm * (pics2 +ux_t30 + uz_t30 + m_xx_t45 + m_zz_t45 + m_xz_t90 + (2*m_xz_t90*ux_t30)/3 - (m_yy_t45*ux_t30)/3 + (2*m_zz_t45*ux_t30)/3 - (m_xy_t90*uy_t30)/3 - (m_yz_t90*uy_t30)/3 + (2*m_xx_t45*uz_t30)/3 + (2*m_xz_t90*uz_t30)/3 - (m_yy_t45*uz_t30)/3 + (ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*uz_t30*uz_t30)/3 - (2*ux_t30*ux_t30*uz_t30)/3 + (uy_t30*uy_t30*uz_t30)/3);
    pop[10] = multiplyTerm * (pics2 -ux_t30 + m_xx_t45 - uz_t30 + m_zz_t45 + m_xz_t90 + (m_yy_t45*ux_t30)/3 - (2*m_xz_t90*ux_t30)/3 - (2*m_zz_t45*ux_t30)/3 + (m_xy_t90*uy_t30)/3 + (m_yz_t90*uy_t30)/3 - (2*m_xx_t45*uz_t30)/3 - (2*m_xz_t90*uz_t30)/3 + (m_yy_t45*uz_t30)/3 - (ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*uz_t30*uz_t30)/3 + (2*ux_t30*ux_t30*uz_t30)/3 - (uy_t30*uy_t30*uz_t30)/3);
    pop[11] = multiplyTerm * (pics2 +uy_t30 + uz_t30 + m_yy_t45 + m_zz_t45 + m_yz_t90 + (2*m_yz_t90*uy_t30)/3 - (m_xz_t90*ux_t30)/3 - (m_xx_t45*uy_t30)/3 - (m_xy_t90*ux_t30)/3 + (2*m_zz_t45*uy_t30)/3 - (m_xx_t45*uz_t30)/3 + (2*m_yy_t45*uz_t30)/3 + (2*m_yz_t90*uz_t30)/3 + (ux_t30*ux_t30*uy_t30)/3 + (ux_t30*ux_t30*uz_t30)/3 - (2*uy_t30*uz_t30*uz_t30)/3 - (2*uy_t30*uy_t30*uz_t30)/3);
    pop[12] = multiplyTerm * (pics2 -uy_t30 + m_yy_t45 - uz_t30 + m_zz_t45 + m_yz_t90 + (m_xy_t90*ux_t30)/3 + (m_xz_t90*ux_t30)/3 + (m_xx_t45*uy_t30)/3 - (2*m_yz_t90*uy_t30)/3 - (2*m_zz_t45*uy_t30)/3 + (m_xx_t45*uz_t30)/3 - (2*m_yy_t45*uz_t30)/3 - (2*m_yz_t90*uz_t30)/3 - (ux_t30*ux_t30*uy_t30)/3 - (ux_t30*ux_t30*uz_t30)/3 + (2*uy_t30*uz_t30*uz_t30)/3 + (2*uy_t30*uy_t30*uz_t30)/3);
    pop[13] = multiplyTerm * (pics2 +ux_t30 - uy_t30 + m_xx_t45 + m_yy_t45 - m_xy_t90 + (2*m_yy_t45*ux_t30)/3 - (2*m_xy_t90*ux_t30)/3 - (m_zz_t45*ux_t30)/3 - (2*m_xx_t45*uy_t30)/3 + (2*m_xy_t90*uy_t30)/3 + (m_zz_t45*uy_t30)/3 - (m_xz_t90*uz_t30)/3 + (m_yz_t90*uz_t30)/3 - (2*ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*ux_t30*uy_t30)/3 + (ux_t30*uz_t30*uz_t30)/3 - (uy_t30*uz_t30*uz_t30)/3);
    pop[14] = multiplyTerm * (pics2 -ux_t30 + uy_t30 + m_xx_t45 + m_yy_t45 - m_xy_t90 + (2*m_xy_t90*ux_t30)/3 - (2*m_yy_t45*ux_t30)/3 + (m_zz_t45*ux_t30)/3 + (2*m_xx_t45*uy_t30)/3 - (2*m_xy_t90*uy_t30)/3 - (m_zz_t45*uy_t30)/3 + (m_xz_t90*uz_t30)/3 - (m_yz_t90*uz_t30)/3 + (2*ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*ux_t30*uy_t30)/3 - (ux_t30*uz_t30*uz_t30)/3 + (uy_t30*uz_t30*uz_t30)/3);
    pop[15] = multiplyTerm * (pics2 +ux_t30 - uz_t30 + m_xx_t45 + m_zz_t45 - m_xz_t90 + (2*m_zz_t45*ux_t30)/3 - (m_yy_t45*ux_t30)/3 - (2*m_xz_t90*ux_t30)/3 - (m_xy_t90*uy_t30)/3 + (m_yz_t90*uy_t30)/3 - (2*m_xx_t45*uz_t30)/3 + (2*m_xz_t90*uz_t30)/3 + (m_yy_t45*uz_t30)/3 + (ux_t30*uy_t30*uy_t30)/3 - (2*ux_t30*uz_t30*uz_t30)/3 + (2*ux_t30*ux_t30*uz_t30)/3 - (uy_t30*uy_t30*uz_t30)/3);
    pop[16] = multiplyTerm * (pics2 -ux_t30 + uz_t30 + m_xx_t45 + m_zz_t45 - m_xz_t90 + (2*m_xz_t90*ux_t30)/3 + (m_yy_t45*ux_t30)/3 - (2*m_zz_t45*ux_t30)/3 + (m_xy_t90*uy_t30)/3 - (m_yz_t90*uy_t30)/3 + (2*m_xx_t45*uz_t30)/3 - (2*m_xz_t90*uz_t30)/3 - (m_yy_t45*uz_t30)/3 - (ux_t30*uy_t30*uy_t30)/3 + (2*ux_t30*uz_t30*uz_t30)/3 - (2*ux_t30*ux_t30*uz_t30)/3 + (uy_t30*uy_t30*uz_t30)/3);
    pop[17] = multiplyTerm * (pics2 +uy_t30 - uz_t30 + m_yy_t45 + m_zz_t45 - m_yz_t90 + (m_xz_t90*ux_t30)/3 - (m_xy_t90*ux_t30)/3 - (m_xx_t45*uy_t30)/3 - (2*m_yz_t90*uy_t30)/3 + (2*m_zz_t45*uy_t30)/3 + (m_xx_t45*uz_t30)/3 - (2*m_yy_t45*uz_t30)/3 + (2*m_yz_t90*uz_t30)/3 + (ux_t30*ux_t30*uy_t30)/3 - (ux_t30*ux_t30*uz_t30)/3 - (2*uy_t30*uz_t30*uz_t30)/3 + (2*uy_t30*uy_t30*uz_t30)/3);
    pop[18] = multiplyTerm * (pics2 -uy_t30 + uz_t30 + m_yy_t45 + m_zz_t45 - m_yz_t90 + (m_xy_t90*ux_t30)/3 - (m_xz_t90*ux_t30)/3 + (m_xx_t45*uy_t30)/3 + (2*m_yz_t90*uy_t30)/3 - (2*m_zz_t45*uy_t30)/3 - (m_xx_t45*uz_t30)/3 + (2*m_yy_t45*uz_t30)/3 - (2*m_yz_t90*uz_t30)/3 - (ux_t30*ux_t30*uy_t30)/3 + (ux_t30*ux_t30*uz_t30)/3 + (2*uy_t30*uz_t30*uz_t30)/3 - (2*uy_t30*uy_t30*uz_t30)/3);

        #endif
    #endif //HIGH_ORDER_COLLISION
    
    
    /* write to global mom */

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 0, blockIdx.x, blockIdx.y, blockIdx.z)] = rhoVar - RHO_0;

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 1, blockIdx.x, blockIdx.y, blockIdx.z)] = ux_t30/3;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 2, blockIdx.x, blockIdx.y, blockIdx.z)] = uy_t30/3;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 3, blockIdx.x, blockIdx.y, blockIdx.z)] = uz_t30/3;

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 4, blockIdx.x, blockIdx.y, blockIdx.z)] = 2*m_xx_t45/9;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 5, blockIdx.x, blockIdx.y, blockIdx.z)] = m_xy_t90/9;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 6, blockIdx.x, blockIdx.y, blockIdx.z)] = m_xz_t90/9;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 7, blockIdx.x, blockIdx.y, blockIdx.z)] = 2*m_yy_t45/9;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 8, blockIdx.x, blockIdx.y, blockIdx.z)] = m_yz_t90/9;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 9, blockIdx.x, blockIdx.y, blockIdx.z)] = 2*m_zz_t45/9;
    
    #ifdef NON_NEWTONIAN_FLUID
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 10, blockIdx.x, blockIdx.y, blockIdx.z)] = omegaVar;
    #endif


    #ifdef LOCAL_FORCES
    //update local forces
    d_L_Fx[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] =  L_Fx;
    d_L_Fy[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] =  L_Fy;
    d_L_Fz[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)] =  L_Fz;
    #endif 


    #include "interfaceInclude/popSave"
}
