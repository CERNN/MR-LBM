#include "hip/hip_runtime.h"
#include "mlbm.cuh"

__global__ void gpuMomCollisionStream(
    dfloat *fMom, char *dNodeType,
    dfloat *fGhostX_0, dfloat *fGhostX_1,
    dfloat *fGhostY_0, dfloat *fGhostY_1,
    dfloat *fGhostZ_0, dfloat *fGhostZ_1,
    dfloat *gGhostX_0, dfloat *gGhostX_1,
    dfloat *gGhostY_0, dfloat *gGhostY_1,
    dfloat *gGhostZ_0, dfloat *gGhostZ_1)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    const int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= NX || y >= NY || z >= NZ)
        return;
    dfloat pop[Q];

    // Load moments from global memory

    //rho'
    char nodeType = dNodeType[idxNodeType(threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat rhoVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 0, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat ux_t30  = 3.0*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 1, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uy_t30  = 3.0*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 2, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat uz_t30  = 3.0*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 3, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixx_t45   = 4.5*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 4, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixy_t90   = 9.0*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 5, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pixz_t90   = 9.0*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 6, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat piyy_t45   = 4.5*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 7, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat piyz_t90   = 9.0*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 8, blockIdx.x, blockIdx.y, blockIdx.z)];
    dfloat pizz_t45   = 4.5*fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 9, blockIdx.x, blockIdx.y, blockIdx.z)];

    #ifdef NON_NEWTONIAN_FLUID
    dfloat omegaVar = fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 10, blockIdx.x, blockIdx.y, blockIdx.z)];
    #endif
    #ifndef NON_NEWTONIAN_FLUID
    dfloat omegaVar = OMEGA;
    #endif
    dfloat t_omegaVar = 1 - omegaVar;
    dfloat tt_omegaVar = 1 - 0.5*omegaVar;
    dfloat omegaVar_d2 = omegaVar / 2.0;
    dfloat omegaVar_d9 = omegaVar / 9.0;
    dfloat omegaVar_p1 = 1.0 + omegaVar;
    dfloat tt_omega_t3 = tt_omegaVar * 3.0;

    
    //calculate post collision populations
    dfloat multiplyTerm;
    multiplyTerm = rhoVar * W0;
    dfloat pics2 = 1.0 - cs2 * (pixx_t45 + piyy_t45 + pizz_t45);

    pop[ 0] = multiplyTerm * (pics2);
    multiplyTerm = rhoVar * W1;
    pop[ 1] = multiplyTerm * (pics2 + ux_t30 + pixx_t45);
    pop[ 2] = multiplyTerm * (pics2 - ux_t30 + pixx_t45);
    pop[ 3] = multiplyTerm * (pics2 + uy_t30 + piyy_t45);
    pop[ 4] = multiplyTerm * (pics2 - uy_t30 + piyy_t45);
    pop[ 5] = multiplyTerm * (pics2 + uz_t30 + pizz_t45);
    pop[ 6] = multiplyTerm * (pics2 - uz_t30 + pizz_t45);
    multiplyTerm = rhoVar * W2;
    pop[ 7] = multiplyTerm * (pics2 +ux_t30 + uy_t30 + pixx_t45 + piyy_t45 + pixy_t90);
    pop[ 8] = multiplyTerm * (pics2 -ux_t30 - uy_t30 + pixx_t45 + piyy_t45 + pixy_t90);
    pop[ 9] = multiplyTerm * (pics2 +ux_t30 + uz_t30 + pixx_t45 + pizz_t45 + pixz_t90);
    pop[10] = multiplyTerm * (pics2 -ux_t30 - uz_t30 + pixx_t45 + pizz_t45 + pixz_t90);
    pop[11] = multiplyTerm * (pics2 +uy_t30 + uz_t30 + piyy_t45 + pizz_t45 + piyz_t90);
    pop[12] = multiplyTerm * (pics2 -uy_t30 - uz_t30 + piyy_t45 + pizz_t45 + piyz_t90);
    pop[13] = multiplyTerm * (pics2 +ux_t30 - uy_t30 + pixx_t45 + piyy_t45 - pixy_t90);
    pop[14] = multiplyTerm * (pics2 -ux_t30 + uy_t30 + pixx_t45 + piyy_t45 - pixy_t90);
    pop[15] = multiplyTerm * (pics2 +ux_t30 - uz_t30 + pixx_t45 + pizz_t45 - pixz_t90);
    pop[16] = multiplyTerm * (pics2 -ux_t30 + uz_t30 + pixx_t45 + pizz_t45 - pixz_t90);
    pop[17] = multiplyTerm * (pics2 +uy_t30 - uz_t30 + piyy_t45 + pizz_t45 - piyz_t90);
    pop[18] = multiplyTerm * (pics2 -uy_t30 + uz_t30 + piyy_t45 + pizz_t45 - piyz_t90);   
    #ifdef D3Q27
    multiplyTerm = rhoVar * W3;
    pop[19] = multiplyTerm * (pics2 + ux_t30 + uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 + pixz_t90 + piyz_t90));
    pop[20] = multiplyTerm * (pics2 - ux_t30 - uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 + pixz_t90 + piyz_t90));
    pop[21] = multiplyTerm * (pics2 + ux_t30 + uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 - pixz_t90 - piyz_t90));
    pop[22] = multiplyTerm * (pics2 - ux_t30 - uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 - pixz_t90 - piyz_t90));
    pop[23] = multiplyTerm * (pics2 + ux_t30 - uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 - pixz_t90 + piyz_t90));
    pop[24] = multiplyTerm * (pics2 - ux_t30 + uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 - pixz_t90 + piyz_t90));
    pop[25] = multiplyTerm * (pics2 - ux_t30 + uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 + pixz_t90 - piyz_t90));
    pop[26] = multiplyTerm * (pics2 + ux_t30 - uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 + pixz_t90 - piyz_t90));
    #endif //D3Q27

    __shared__ dfloat s_pop[BLOCK_LBM_SIZE * (Q - 1)];

    //save populations in shared memory
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  0)] = pop[ 1];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  1)] = pop[ 2];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  2)] = pop[ 3];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  3)] = pop[ 4];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  4)] = pop[ 5];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  5)] = pop[ 6];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  6)] = pop[ 7];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  7)] = pop[ 8];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  8)] = pop[ 9];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z,  9)] = pop[10];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 10)] = pop[11];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 11)] = pop[12];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 12)] = pop[13];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 13)] = pop[14];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 14)] = pop[15];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 15)] = pop[16];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 16)] = pop[17];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 17)] = pop[18];
    #ifdef D3Q27
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 18)] = pop[19];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 19)] = pop[20];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 20)] = pop[21];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 21)] = pop[22];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 22)] = pop[23];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 23)] = pop[24];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 24)] = pop[25];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, threadIdx.z, 25)] = pop[26];
    #endif //D3Q27

    //sync threads of the block so all populations are saved
    __syncthreads();

    /* pull */

    const unsigned short int xp1 = (threadIdx.x + 1 + BLOCK_NX) % BLOCK_NX;
    const unsigned short int xm1 = (threadIdx.x - 1 + BLOCK_NX) % BLOCK_NX;

    const unsigned short int yp1 = (threadIdx.y + 1 + BLOCK_NY) % BLOCK_NY;
    const unsigned short int ym1 = (threadIdx.y - 1 + BLOCK_NY) % BLOCK_NY;

    const unsigned short int zp1 = (threadIdx.z + 1 + BLOCK_NZ) % BLOCK_NZ;
    const unsigned short int zm1 = (threadIdx.z - 1 + BLOCK_NZ) % BLOCK_NZ;

    pop[ 1] = s_pop[idxPopBlock(xm1, threadIdx.y, threadIdx.z, 0)];
    pop[ 2] = s_pop[idxPopBlock(xp1, threadIdx.y, threadIdx.z, 1)];
    pop[ 3] = s_pop[idxPopBlock(threadIdx.x, ym1, threadIdx.z, 2)];
    pop[ 4] = s_pop[idxPopBlock(threadIdx.x, yp1, threadIdx.z, 3)];
    pop[ 5] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zm1, 4)];
    pop[ 6] = s_pop[idxPopBlock(threadIdx.x, threadIdx.y, zp1, 5)];
    pop[ 7] = s_pop[idxPopBlock(xm1, ym1, threadIdx.z, 6)];
    pop[ 8] = s_pop[idxPopBlock(xp1, yp1, threadIdx.z, 7)];
    pop[ 9] = s_pop[idxPopBlock(xm1, threadIdx.y, zm1, 8)];
    pop[10] = s_pop[idxPopBlock(xp1, threadIdx.y, zp1, 9)];
    pop[11] = s_pop[idxPopBlock(threadIdx.x, ym1, zm1, 10)];
    pop[12] = s_pop[idxPopBlock(threadIdx.x, yp1, zp1, 11)];
    pop[13] = s_pop[idxPopBlock(xm1, yp1, threadIdx.z, 12)];
    pop[14] = s_pop[idxPopBlock(xp1, ym1, threadIdx.z, 13)];
    pop[15] = s_pop[idxPopBlock(xm1, threadIdx.y, zp1, 14)];
    pop[16] = s_pop[idxPopBlock(xp1, threadIdx.y, zm1, 15)];
    pop[17] = s_pop[idxPopBlock(threadIdx.x, ym1, zp1, 16)];
    pop[18] = s_pop[idxPopBlock(threadIdx.x, yp1, zm1, 17)];
    #ifdef D3Q27
    pop[19] = s_pop[idxPopBlock(xm1, ym1, zm1, 18)];
    pop[20] = s_pop[idxPopBlock(xp1, yp1, zp1, 19)];
    pop[21] = s_pop[idxPopBlock(xm1, ym1, zp1, 20)];
    pop[22] = s_pop[idxPopBlock(xp1, yp1, zm1, 21)];
    pop[23] = s_pop[idxPopBlock(xm1, yp1, zm1, 22)];
    pop[24] = s_pop[idxPopBlock(xp1, ym1, zp1, 23)];
    pop[25] = s_pop[idxPopBlock(xp1, ym1, zm1, 24)];
    pop[26] = s_pop[idxPopBlock(xm1, yp1, zp1, 25)];
    #endif

    /* load pop from global in cover nodes */


   
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;

    const int txm1 = (tx-1+BLOCK_NX)%BLOCK_NX;
    const int txp1 = (tx+1)%BLOCK_NX ;
    const int tym1 = (ty-1+BLOCK_NY)%BLOCK_NY;
    const int typ1 = (ty+1)%BLOCK_NY;
    const int tzm1 = (tz-1+BLOCK_NZ)%BLOCK_NZ;
    const int tzp1 = (tz+1)%BLOCK_NZ ;

    const int bxm1 = (bx-1+NUM_BLOCK_X)%NUM_BLOCK_X;
    const int bxp1 = (bx+1)%NUM_BLOCK_X;
    const int bym1 = (by-1+NUM_BLOCK_Y)%NUM_BLOCK_Y;
    const int byp1 = (by+1)%NUM_BLOCK_Y;
    const int bzm1 = (bz-1+NUM_BLOCK_Z)%NUM_BLOCK_Z;
    const int bzp1 = (bz+1)%NUM_BLOCK_Z;


    if (tx == 0) { //w
        pop[ 1] = fGhostX_1[idxPopX(ty  , tz, 0, bxm1, by                                       , bz)];
        pop[ 7] = fGhostX_1[idxPopX(tym1, tz, 1, bxm1, ((ty == 0) ? bym1 : by)                  , bz)];
        pop[ 9] = fGhostX_1[idxPopX(ty, tzm1, 2, bxm1, by                                       , ((tz == 0) ? bzm1 : bz))];
        pop[13] = fGhostX_1[idxPopX(typ1, tz, 3, bxm1, ((ty == (BLOCK_NY - 1)) ? byp1 : by)     , bz)];
        pop[15] = fGhostX_1[idxPopX(ty, tzp1, 4, bxm1, by                                       , ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        #ifdef D3Q27
        pop[19] = fGhostX_1[idxPopX(tym1, tzm1, 5, bxm1, ((ty == 0) ? bym1 : by)                , ((tz == 0) ? bzm1 : bz))];
        pop[21] = fGhostX_1[idxPopX(tym1, tzp1, 6, bxm1, ((ty == 0) ? bym1 : by)                , ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[23] = fGhostX_1[idxPopX(typ1, tzm1, 7, bxm1, ((ty == (BLOCK_NY - 1)) ? byp1 : by)   , ((tz == 0) ? bzm1 : bz))];
        pop[26] = fGhostX_1[idxPopX(typ1, tzp1, 8, bxm1, ((ty == (BLOCK_NY - 1)) ? byp1 : by)   , ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        #endif //D3Q27
    }
    else if (tx == (BLOCK_NX - 1))
    { //e
        pop[ 2] = fGhostX_0[idxPopX(ty  , tz, 0, bxp1, by                                       , bz)];
        pop[ 8] = fGhostX_0[idxPopX(typ1, tz, 1, bxp1, ((ty == (BLOCK_NY - 1)) ? byp1 : by)     , bz)];
        pop[10] = fGhostX_0[idxPopX(ty, tzp1, 2, bxp1, by                                       , ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[14] = fGhostX_0[idxPopX(tym1, tz, 3, bxp1, ((ty == 0) ? bym1 : by)                  , bz)];
        pop[16] = fGhostX_0[idxPopX(ty, tzm1, 4, bxp1, by                                       , ((tz == 0) ? bzm1 : bz))];
        #ifdef D3Q27
        pop[20] = fGhostX_0[idxPopX(typ1, tzp1, 5, bxp1, ((ty == (BLOCK_NY - 1)) ? byp1 : by)   , ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[22] = fGhostX_0[idxPopX(typ1, tzm1, 6, bxp1, ((ty == (BLOCK_NY - 1)) ? byp1 : by)   , ((tz == 0) ? bzm1 : bz))];
        pop[24] = fGhostX_0[idxPopX(tym1, tzp1, 7, bxp1, ((ty == 0) ? bym1 : by)                , ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[25] = fGhostX_0[idxPopX(tym1, tzm1, 8, bxp1, ((ty == 0) ? bym1 : by)                , ((tz == 0) ? bzm1 : bz))];
        #endif //D3Q27
    }

    if (ty == 0)
    { //s
        pop[ 3] = fGhostY_1[idxPopY(tx  , tz, 0, bx                                     , bym1, bz)];
        pop[ 7] = fGhostY_1[idxPopY(txm1, tz, 1, ((tx == 0) ? bxm1 : bx)                , bym1, bz)];
        pop[11] = fGhostY_1[idxPopY(tx, tzm1, 2, bx                                     , bym1, ((tz == 0) ? bzm1 : bz))];
        pop[14] = fGhostY_1[idxPopY(txp1, tz, 3, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx)   , bym1, bz)];
        pop[17] = fGhostY_1[idxPopY(tx, tzp1, 4, bx                                     , bym1, ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        #ifdef D3Q27
        pop[19] = fGhostY_1[idxPopY(txm1, tzm1, 5, ((tx == 0) ? bxm1 : bx)              , bym1, ((tz == 0) ? bzm1 : bz))];
        pop[21] = fGhostY_1[idxPopY(txm1, tzp1, 6, ((tx == 0) ? bxm1 : bx)              , bym1, ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[24] = fGhostY_1[idxPopY(txp1, tzp1, 7, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , bym1, ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[25] = fGhostY_1[idxPopY(txp1, tzm1, 8, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , bym1, ((tz == 0) ? bzm1 : bz))];
        #endif //D3Q27
    }
    else if (ty == (BLOCK_NY - 1))
    { //n
        pop[ 4] = fGhostY_0[idxPopY(tx  , tz, 0, bx                                     , byp1, bz)];
        pop[ 8] = fGhostY_0[idxPopY(txp1, tz, 1, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx)   , byp1, bz)];
        pop[12] = fGhostY_0[idxPopY(tx, tzp1, 2, bx                                     , byp1, ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[13] = fGhostY_0[idxPopY(txm1, tz, 3, ((tx == 0) ? bxm1 : bx)                , byp1, bz)];
        pop[18] = fGhostY_0[idxPopY(tx, tzm1, 4, bx                                     , byp1, ((tz == 0) ? bzm1 : bz))];
        #ifdef D3Q27
        pop[20] = fGhostY_0[idxPopY(txp1, tzp1, 5, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , byp1, ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        pop[22] = fGhostY_0[idxPopY(txp1, tzm1, 6, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , byp1, ((tz == 0) ? bzm1 : bz))];
        pop[23] = fGhostY_0[idxPopY(txm1, tzm1, 7, ((tx == 0) ? bxm1 : bx)              , byp1, ((tz == 0) ? bzm1 : bz))];
        pop[26] = fGhostY_0[idxPopY(txm1, tzp1, 8, ((tx == 0) ? bxm1 : bx)              , byp1, ((tz == (BLOCK_NZ - 1)) ? bzp1 : bz))];
        #endif //D3Q27
    }

    if (tz == 0)
    { //b
        pop[ 5] = fGhostZ_1[idxPopZ(tx  , ty, 0, bx                                     , by                                    , bzm1)];
        pop[ 9] = fGhostZ_1[idxPopZ(txm1, ty, 1, ((tx == 0) ? bxm1 : bx)                , by                                    , bzm1)];
        pop[11] = fGhostZ_1[idxPopZ(tx, tym1, 2, bx                                     , ((ty == 0) ? bym1 : by)               , bzm1)];
        pop[16] = fGhostZ_1[idxPopZ(txp1, ty, 3, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx)   , by                                    , bzm1)];
        pop[18] = fGhostZ_1[idxPopZ(tx, typ1, 4, bx                                     , ((ty == (BLOCK_NY - 1)) ? byp1 : by)  , bzm1)];
        #ifdef D3Q27
        pop[19] = fGhostZ_1[idxPopZ(txm1, tym1, 5, ((tx == 0) ? bxm1 : bx)              , ((ty == 0) ? bym1 : by)               , bzm1)];
        pop[22] = fGhostZ_1[idxPopZ(txp1, typ1, 6, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , ((ty == (BLOCK_NY - 1)) ? byp1 : by)  , bzm1)];
        pop[23] = fGhostZ_1[idxPopZ(txm1, typ1, 7, ((tx == 0) ? bxm1 : bx)              , ((ty == (BLOCK_NY - 1)) ? byp1 : by)  , bzm1)];
        pop[25] = fGhostZ_1[idxPopZ(txp1, tym1, 8, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , ((ty == 0) ? bym1 : by)               , bzm1)];
        #endif //D3Q27
    }
    else if (tz == (BLOCK_NZ - 1))
    { //f
        pop[ 6] = fGhostZ_0[idxPopZ(tx  , ty, 0, bx                                     , by                                    , bzp1)];
        pop[10] = fGhostZ_0[idxPopZ(txp1, ty, 1, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx)   , by                                    , bzp1)];
        pop[12] = fGhostZ_0[idxPopZ(tx, typ1, 2, bx                                     , ((ty == (BLOCK_NY - 1)) ? byp1 : by)  , bzp1)];
        pop[15] = fGhostZ_0[idxPopZ(txm1, ty, 3, ((tx == 0) ? bxm1 : bx)                , by                                    , bzp1)];
        pop[17] = fGhostZ_0[idxPopZ(tx, tym1, 4, bx                                     , ((ty == 0) ? bym1 : by)               , bzp1)];
        #ifdef D3Q27
        pop[20] = fGhostZ_0[idxPopZ(txp1, typ1, 5, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , ((ty == (BLOCK_NY - 1)) ? byp1 : by)  , bzp1)];
        pop[21] = fGhostZ_0[idxPopZ(txm1, tym1, 6, ((tx == 0) ? bxm1 : bx)              , ((ty == 0) ? bym1 : by)               , bzp1)];
        pop[24] = fGhostZ_0[idxPopZ(txp1, tym1, 7, ((tx == (BLOCK_NX - 1)) ? bxp1 : bx) , ((ty == 0) ? bym1 : by)               , bzp1)];
        pop[26] = fGhostZ_0[idxPopZ(txm1, typ1, 8, ((tx == 0) ? bxm1 : bx)              , ((ty == (BLOCK_NY - 1)) ? byp1 : by)  , bzp1)];
        #endif //D3Q27
    }

    #ifdef BC_POPULATION_BASED

        if (nodeType){
            #include BC_PATH

            ux_t30 = ux_t30 /3.0;
            uy_t30 = uy_t30 /3.0;
            uz_t30 = uz_t30 /3.0;

            pixx_t45 = (pixx_t45 / 4.5 + cs2 ) * rhoVar;
            pixy_t90 = (pixy_t90 / 9.0 ) * rhoVar;
            pixz_t90 = (pixz_t90 / 9.0 ) * rhoVar;
            piyy_t45 = (piyy_t45 / 4.5 + cs2) * rhoVar;
            piyz_t90 = (piyz_t90 / 9.0 ) * rhoVar;
            pizz_t45 = (pizz_t45 / 4.5 + cs2) * rhoVar;

        }
            

            //calculate streaming moments
        #ifdef D3Q19
            //equation3
            rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18];
            dfloat invRho = 1 / rhoVar;
            //equation4 + force correction
            ux_t30 = ((pop[ 1] + pop[7] + pop[ 9] + pop[13] + pop[15]) - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16]) + 0.5 * FX) * invRho;
            uy_t30 = ((pop[ 3] + pop[7] + pop[11] + pop[14] + pop[17]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18]) + 0.5 * FY) * invRho;
            uz_t30 = ((pop[ 5] + pop[9] + pop[11] + pop[16] + pop[18]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17]) + 0.5 * FZ) * invRho;

            //equation5
            pixx_t45 = ( (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16]) );
            pixy_t90 = (((pop[7] + pop[ 8]) - (pop[13] + pop[14])));
            pixz_t90 = (((pop[9] + pop[10]) - (pop[15] + pop[16])));
            piyy_t45 = ( (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18]));
            piyz_t90 = (((pop[11]+pop[12])-(pop[17]+pop[18])));
            pizz_t45 = ( (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18]));


        #endif
        #ifdef D3Q27
            rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26];
            dfloat invRho = 1 / rhoVar;
            ux_t30 = ((pop[1] + pop[7] + pop[9] + pop[13] + pop[15] + pop[19] + pop[21] + pop[23] + pop[26])  - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16] + pop[20] + pop[22] + pop[24] + pop[25]) + 0.5 * FX) * invRho;
            uy_t30 = ((pop[3] + pop[7] + pop[11] + pop[14] + pop[17] + pop[19] + pop[21] + pop[24] + pop[25]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18] + pop[20] + pop[22] + pop[23] + pop[26]) + 0.5 * FY) * invRho;
            uz_t30 = ((pop[5] + pop[9] + pop[11] + pop[16] + pop[18] + pop[19] + pop[22] + pop[23] + pop[25]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17] + pop[20] + pop[21] + pop[24] + pop[26]) + 0.5 * FZ) * invRho;

            pixx_t45 = ( (pop[ 1] + pop[ 2] + pop[ 7] + pop[ 8] + pop[ 9] + pop[10]  +  pop[13] + pop[14] + pop[15] + pop[16] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
            pixy_t90 = (((pop[ 7] + pop[ 8] + pop[19] + pop[20] + pop[21] + pop[22]) - (pop[13] + pop[14] + pop[23] + pop[24] + pop[25] + pop[26])) );
            pixz_t90 = (((pop[ 9] + pop[10] + pop[19] + pop[20] + pop[23] + pop[24]) - (pop[15] + pop[16] + pop[21] + pop[22] + pop[25] + pop[26])) );
            piyy_t45 = ( (pop[ 3] + pop[ 4] + pop[ 7] + pop[ 8] + pop[11] + pop[12]  +  pop[13] + pop[14] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
            piyz_t90 = (((pop[11] + pop[12] + pop[19] + pop[20] + pop[25] + pop[26]) - (pop[17] + pop[18] + pop[21] + pop[22] + pop[23] + pop[24])));
            pizz_t45 = ( (pop[ 5] + pop[ 6] + pop[ 9] + pop[10] + pop[11] + pop[12]  +  pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
        #endif
    #endif

    #ifdef BC_MOMENT_BASED
        dfloat invRho;
        if(nodeType){
            #include BC_PATH
            //gpuBoundaryConditionMom(pop,rhoVar,nodeType,ux_t30,uy_t30,uz_t30,pixx_t45,pixy_t90,pixz_t90,piyy_t45,piyz_t90,pizz_t45);
            invRho = 1.0 / rhoVar;
        }else{

            //calculate streaming moments
            #ifdef D3Q19
                //equation3
                rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18];
                invRho = 1 / rhoVar;
                //equation4 + force correction
                ux_t30 = ((pop[ 1] + pop[7] + pop[ 9] + pop[13] + pop[15]) - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16]) + 0.5 * FX) * invRho;
                uy_t30 = ((pop[ 3] + pop[7] + pop[11] + pop[14] + pop[17]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18]) + 0.5 * FY) * invRho;
                uz_t30 = ((pop[ 5] + pop[9] + pop[11] + pop[16] + pop[18]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17]) + 0.5 * FZ) * invRho;

            //equation5
                pixx_t45 = ( (pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16]) );
                pixy_t90 = (((pop[7] + pop[ 8]) - (pop[13] + pop[14])));
                pixz_t90 = (((pop[9] + pop[10]) - (pop[15] + pop[16])));
                piyy_t45 = ( (pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18]));
                piyz_t90 = (((pop[11]+pop[12])-(pop[17]+pop[18])));
                pizz_t45 = ( (pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18]));


            #endif
            #ifdef D3Q27
                rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26];
                dfloat invRho = 1 / rhoVar;
                ux_t30 = ((pop[1] + pop[7] + pop[9] + pop[13] + pop[15] + pop[19] + pop[21] + pop[23] + pop[26])  - (pop[ 2] + pop[ 8] + pop[10] + pop[14] + pop[16] + pop[20] + pop[22] + pop[24] + pop[25]) + 0.5 * FX) * invRho;
                uy_t30 = ((pop[3] + pop[7] + pop[11] + pop[14] + pop[17] + pop[19] + pop[21] + pop[24] + pop[25]) - (pop[ 4] + pop[ 8] + pop[12] + pop[13] + pop[18] + pop[20] + pop[22] + pop[23] + pop[26]) + 0.5 * FY) * invRho;
                uz_t30 = ((pop[5] + pop[9] + pop[11] + pop[16] + pop[18] + pop[19] + pop[22] + pop[23] + pop[25]) - (pop[ 6] + pop[10] + pop[12] + pop[15] + pop[17] + pop[20] + pop[21] + pop[24] + pop[26]) + 0.5 * FZ) * invRho;

                pixx_t45 = ( (pop[ 1] + pop[ 2] + pop[ 7] + pop[ 8] + pop[ 9] + pop[10]  +  pop[13] + pop[14] + pop[15] + pop[16] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
                pixy_t90 = (((pop[ 7] + pop[ 8] + pop[19] + pop[20] + pop[21] + pop[22]) - (pop[13] + pop[14] + pop[23] + pop[24] + pop[25] + pop[26])) );
                pixz_t90 = (((pop[ 9] + pop[10] + pop[19] + pop[20] + pop[23] + pop[24]) - (pop[15] + pop[16] + pop[21] + pop[22] + pop[25] + pop[26])) );
                piyy_t45 = ( (pop[ 3] + pop[ 4] + pop[ 7] + pop[ 8] + pop[11] + pop[12]  +  pop[13] + pop[14] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
                piyz_t90 = (((pop[11] + pop[12] + pop[19] + pop[20] + pop[25] + pop[26]) - (pop[17] + pop[18] + pop[21] + pop[22] + pop[23] + pop[24])));
                pizz_t45 = ( (pop[ 5] + pop[ 6] + pop[ 9] + pop[10] + pop[11] + pop[12]  +  pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
            #endif
        }
    #endif // moment based

#ifdef NON_NEWTONIAN_FLUID
    const dfloat momNeqXX = pixx_t45 - rhoVar*(ux_t30*ux_t30 + cs2);
    const dfloat momNeqYY = piyy_t45 - rhoVar*(uy_t30*uy_t30 + cs2); 
    const dfloat momNeqZZ = pizz_t45 - rhoVar*(uz_t30*uz_t30 + cs2);
    const dfloat momNeqXYt2 = (pixy_t90 - rhoVar*ux_t30*uy_t30) * 2;
    const dfloat momNeqXZt2 = (pixz_t90 - rhoVar*ux_t30*uz_t30) * 2;
    const dfloat momNeqYZt2 = (piyz_t90 - rhoVar*uy_t30*uz_t30) * 2;

    const dfloat uFxxd2 = ux_t30*FX; // d2 = uFxx Divided by two
    const dfloat uFyyd2 = uy_t30*FY;
    const dfloat uFzzd2 = uz_t30*FZ;
    const dfloat uFxyd2 = (ux_t30*FY + uy_t30*FX) / 2;
    const dfloat uFxzd2 = (ux_t30*FZ + uz_t30*FX) / 2;
    const dfloat uFyzd2 = (uy_t30*FZ + uz_t30*FY) / 2;

    const dfloat auxStressMag = sqrt(0.5 * (
        (momNeqXX + uFxxd2) * (momNeqXX + uFxxd2) +
        (momNeqYY + uFyyd2) * (momNeqYY + uFyyd2) +
        (momNeqZZ + uFzzd2) * (momNeqZZ + uFzzd2) +
        2 * ((momNeqXYt2/2 + uFxyd2) * (momNeqXYt2/2 + uFxyd2) +
        (momNeqXZt2/2 + uFxzd2) * (momNeqXZt2/2 + uFxzd2) + 
        (momNeqYZt2/2 + uFyzd2) * (momNeqYZt2/2 + uFyzd2))));

    dfloat eta = (1.0/omegaVar - 0.5) / 3.0;
    dfloat gamma_dot = (1 - 0.5 * (omegaVar)) * auxStressMag / eta;
    eta = VISC + S_Y/gamma_dot;
    omegaVar = omegaVar;// 1.0 / (0.5 + 3.0 * eta);

    omegaVar = calcOmega(omegaVar, auxStressMag);

    t_omegaVar = 1 - omegaVar;
    tt_omegaVar = 1 - 0.5*omegaVar;
    omegaVar_d2 = omegaVar / 2.0;
    omegaVar_d9 = omegaVar / 9.0;
    omegaVar_p1 = 1.0 + omegaVar;
    tt_omega_t3 = tt_omegaVar * 3.0;
#endif


    ux_t30 = 3.0 * ux_t30;
    uy_t30 = 3.0 * uy_t30;
    uz_t30 = 3.0 * uz_t30;

    pixx_t45 = 4.5 * (pixx_t45 * invRho - cs2);
    pixy_t90 = 9.0 * (pixy_t90 * invRho);
    pixz_t90 = 9.0 * (pixz_t90 * invRho);
    piyy_t45 = 4.5 * (piyy_t45 * invRho - cs2);
    piyz_t90 = 9.0 * (piyz_t90 * invRho);
    pizz_t45 = 4.5 * (pizz_t45 * invRho - cs2);

   //NOTE : STREAMING DONE, NOW COLLIDE

    //Collide Moments
    //Equiblibrium momements
    
    dfloat invRho_mt15 = -1.5*invRho;
    ux_t30 = (t_omegaVar * (ux_t30 + invRho_mt15 * FX ) + omegaVar * ux_t30 + tt_omega_t3 * FX);
    uy_t30 = (t_omegaVar * (uy_t30 + invRho_mt15 * FY ) + omegaVar * uy_t30 + tt_omega_t3 * FY);
    uz_t30 = (t_omegaVar * (uz_t30 + invRho_mt15 * FZ ) + omegaVar * uz_t30 + tt_omega_t3 * FZ);
    
    //equation 90
    pixx_t45 = (t_omegaVar * pixx_t45  +   omegaVar_d2 * ux_t30 * ux_t30    - invRho_mt15 * tt_omegaVar * (FX * ux_t30 + FX * ux_t30));
    piyy_t45 = (t_omegaVar * piyy_t45  +   omegaVar_d2 * uy_t30 * uy_t30    - invRho_mt15 * tt_omegaVar * (FY * uy_t30 + FY * uy_t30));
    pizz_t45 = (t_omegaVar * pizz_t45  +   omegaVar_d2 * uz_t30 * uz_t30    - invRho_mt15 * tt_omegaVar * (FZ * uz_t30 + FZ * uz_t30));

    pixy_t90 = (t_omegaVar * pixy_t90  +     omegaVar * ux_t30 * uy_t30    +    tt_omega_t3 *invRho* (FX * uy_t30 + FY * ux_t30));
    pixz_t90 = (t_omegaVar * pixz_t90  +     omegaVar * ux_t30 * uz_t30    +    tt_omega_t3 *invRho* (FX * uz_t30 + FZ * ux_t30));
    piyz_t90 = (t_omegaVar * piyz_t90  +     omegaVar * uy_t30 * uz_t30    +    tt_omega_t3 *invRho* (FY * uz_t30 + FZ * uy_t30));
    

    //calculate post collision populations
    
    multiplyTerm = rhoVar * W0;
    pics2 = 1.0 - cs2 * (pixx_t45 + piyy_t45 + pizz_t45);

    pop[ 0] = multiplyTerm * (pics2);
    multiplyTerm = rhoVar * W1;
    pop[ 1] = multiplyTerm * (pics2 + ux_t30 + pixx_t45);
    pop[ 2] = multiplyTerm * (pics2 - ux_t30 + pixx_t45);
    pop[ 3] = multiplyTerm * (pics2 + uy_t30 + piyy_t45);
    pop[ 4] = multiplyTerm * (pics2 - uy_t30 + piyy_t45);
    pop[ 5] = multiplyTerm * (pics2 + uz_t30 + pizz_t45);
    pop[ 6] = multiplyTerm * (pics2 - uz_t30 + pizz_t45);
    multiplyTerm = rhoVar * W2;
    pop[ 7] = multiplyTerm * (pics2 + ( ux_t30 + uy_t30) + (pixx_t45 + piyy_t45) + pixy_t90);
    pop[ 8] = multiplyTerm * (pics2 + (-ux_t30 - uy_t30) + (pixx_t45 + piyy_t45) + pixy_t90);
    pop[ 9] = multiplyTerm * (pics2 + ( ux_t30 + uz_t30) + (pixx_t45 + pizz_t45) + pixz_t90);
    pop[10] = multiplyTerm * (pics2 + (-ux_t30 - uz_t30) + (pixx_t45 + pizz_t45) + pixz_t90);
    pop[11] = multiplyTerm * (pics2 + ( uy_t30 + uz_t30) + (piyy_t45 + pizz_t45) + piyz_t90);
    pop[12] = multiplyTerm * (pics2 + (-uy_t30 - uz_t30) + (piyy_t45 + pizz_t45) + piyz_t90);
    pop[13] = multiplyTerm * (pics2 + ( ux_t30 - uy_t30) + (pixx_t45 + piyy_t45) - pixy_t90);
    pop[14] = multiplyTerm * (pics2 + (-ux_t30 + uy_t30) + (pixx_t45 + piyy_t45) - pixy_t90);
    pop[15] = multiplyTerm * (pics2 + ( ux_t30 - uz_t30) + (pixx_t45 + pizz_t45) - pixz_t90);
    pop[16] = multiplyTerm * (pics2 + (-ux_t30 + uz_t30) + (pixx_t45 + pizz_t45) - pixz_t90);
    pop[17] = multiplyTerm * (pics2 + ( uy_t30 - uz_t30) + (piyy_t45 + pizz_t45) - piyz_t90);
    pop[18] = multiplyTerm * (pics2 + (-uy_t30 + uz_t30) + (piyy_t45 + pizz_t45) - piyz_t90);   
    #ifdef D3Q27
    multiplyTerm = rhoVar * W3;
    pop[19] = multiplyTerm * (pics2 + ux_t30 + uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 + pixz_t90 + piyz_t90));
    pop[20] = multiplyTerm * (pics2 - ux_t30 - uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 + pixz_t90 + piyz_t90));
    pop[21] = multiplyTerm * (pics2 + ux_t30 + uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 - pixz_t90 - piyz_t90));
    pop[22] = multiplyTerm * (pics2 - ux_t30 - uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 + (pixy_t90 - pixz_t90 - piyz_t90));
    pop[23] = multiplyTerm * (pics2 + ux_t30 - uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 - pixz_t90 + piyz_t90));
    pop[24] = multiplyTerm * (pics2 - ux_t30 + uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 - pixz_t90 + piyz_t90));
    pop[25] = multiplyTerm * (pics2 - ux_t30 + uy_t30 + uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 + pixz_t90 - piyz_t90));
    pop[26] = multiplyTerm * (pics2 + ux_t30 - uy_t30 - uz_t30 + pixx_t45 + piyy_t45 + pizz_t45 - (pixy_t90 + pixz_t90 - piyz_t90));
    #endif //D3Q27
    
    /* write to global mom */

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 0, blockIdx.x, blockIdx.y, blockIdx.z)] = rhoVar;

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 1, blockIdx.x, blockIdx.y, blockIdx.z)] = ux_t30/3.0;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 2, blockIdx.x, blockIdx.y, blockIdx.z)] = uy_t30/3.0;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 3, blockIdx.x, blockIdx.y, blockIdx.z)] = uz_t30/3.0;

    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 4, blockIdx.x, blockIdx.y, blockIdx.z)] = pixx_t45/4.5;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 5, blockIdx.x, blockIdx.y, blockIdx.z)] = pixy_t90/9.0;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 6, blockIdx.x, blockIdx.y, blockIdx.z)] = pixz_t90/9.0;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 7, blockIdx.x, blockIdx.y, blockIdx.z)] = piyy_t45/4.5;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 8, blockIdx.x, blockIdx.y, blockIdx.z)] = piyz_t90/9.0;
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 9, blockIdx.x, blockIdx.y, blockIdx.z)] = pizz_t45/4.5;
    
    #ifdef NON_NEWTONIAN_FLUID
    fMom[idxMom(threadIdx.x, threadIdx.y, threadIdx.z, 10, blockIdx.x, blockIdx.y, blockIdx.z)] = omegaVar;
    #endif

    /* write to global pop */
    if (threadIdx.x == 0) { //w
        gGhostX_0[idxPopX(ty, tz, 0, bx, by, bz)] = pop[ 2]; 
        gGhostX_0[idxPopX(ty, tz, 1, bx, by, bz)] = pop[ 8];
        gGhostX_0[idxPopX(ty, tz, 2, bx, by, bz)] = pop[10];
        gGhostX_0[idxPopX(ty, tz, 3, bx, by, bz)] = pop[14];
        gGhostX_0[idxPopX(ty, tz, 4, bx, by, bz)] = pop[16];
        #ifdef D3Q27                                                                                                           
        gGhostX_0[idxPopX(ty, tz, 5, bx, by, bz)] = pop[20];
        gGhostX_0[idxPopX(ty, tz, 6, bx, by, bz)] = pop[22];
        gGhostX_0[idxPopX(ty, tz, 7, bx, by, bz)] = pop[24];
        gGhostX_0[idxPopX(ty, tz, 8, bx, by, bz)] = pop[25];
        #endif //D3Q27                                                                                                           
    }else if (threadIdx.x == (BLOCK_NX - 1)){                                                                                                                                                                               
        gGhostX_1[idxPopX(ty, tz, 0, bx, by, bz)] = pop[ 1];
        gGhostX_1[idxPopX(ty, tz, 1, bx, by, bz)] = pop[ 7];
        gGhostX_1[idxPopX(ty, tz, 2, bx, by, bz)] = pop[ 9];
        gGhostX_1[idxPopX(ty, tz, 3, bx, by, bz)] = pop[13];
        gGhostX_1[idxPopX(ty, tz, 4, bx, by, bz)] = pop[15];
        #ifdef D3Q27                                                                                                           
        gGhostX_1[idxPopX(ty, tz, 5, bx, by, bz)] = pop[19];
        gGhostX_1[idxPopX(ty, tz, 6, bx, by, bz)] = pop[21];
        gGhostX_1[idxPopX(ty, tz, 7, bx, by, bz)] = pop[23];
        gGhostX_1[idxPopX(ty, tz, 8, bx, by, bz)] = pop[26];
        #endif //D3Q27       
    }

    if (threadIdx.y == 0)  { //s                                                                                                                                                                                        
        gGhostY_0[idxPopY(tx, tz, 0, bx, by, bz)] = pop[ 4];
        gGhostY_0[idxPopY(tx, tz, 1, bx, by, bz)] = pop[ 8];
        gGhostY_0[idxPopY(tx, tz, 2, bx, by, bz)] = pop[12];
        gGhostY_0[idxPopY(tx, tz, 3, bx, by, bz)] = pop[13];
        gGhostY_0[idxPopY(tx, tz, 4, bx, by, bz)] = pop[18];
        #ifdef D3Q27                                                                                                           
        gGhostY_0[idxPopY(tx, tz, 5, bx, by, bz)] = pop[20];
        gGhostY_0[idxPopY(tx, tz, 6, bx, by, bz)] = pop[22];
        gGhostY_0[idxPopY(tx, tz, 7, bx, by, bz)] = pop[23];
        gGhostY_0[idxPopY(tx, tz, 8, bx, by, bz)] = pop[26];
        #endif //D3Q27                                                                                                           
    }else if (threadIdx.y == (BLOCK_NY - 1)){                                                                                                                                                                        
        gGhostY_1[idxPopY(tx, tz, 0, bx, by, bz)] = pop[ 3];
        gGhostY_1[idxPopY(tx, tz, 1, bx, by, bz)] = pop[ 7];
        gGhostY_1[idxPopY(tx, tz, 2, bx, by, bz)] = pop[11];
        gGhostY_1[idxPopY(tx, tz, 3, bx, by, bz)] = pop[14];
        gGhostY_1[idxPopY(tx, tz, 4, bx, by, bz)] = pop[17];
        #ifdef D3Q27                                                                                                           
        gGhostY_1[idxPopY(tx, tz, 5, bx, by, bz)] = pop[19];
        gGhostY_1[idxPopY(tx, tz, 6, bx, by, bz)] = pop[21];
        gGhostY_1[idxPopY(tx, tz, 7, bx, by, bz)] = pop[24];
        gGhostY_1[idxPopY(tx, tz, 8, bx, by, bz)] = pop[25];
        #endif //D3Q27                                                                                                           
    }
    
    if (threadIdx.z == 0){ //b                                                                                                                                                                                     
        gGhostZ_0[idxPopZ(tx, ty, 0, bx, by, bz)] = pop[ 6];
        gGhostZ_0[idxPopZ(tx, ty, 1, bx, by, bz)] = pop[10];
        gGhostZ_0[idxPopZ(tx, ty, 2, bx, by, bz)] = pop[12];
        gGhostZ_0[idxPopZ(tx, ty, 3, bx, by, bz)] = pop[15];
        gGhostZ_0[idxPopZ(tx, ty, 4, bx, by, bz)] = pop[17];
        #ifdef D3Q27                                                                                                           
        gGhostZ_0[idxPopZ(tx, ty, 5, bx, by, bz)] = pop[20];
        gGhostZ_0[idxPopZ(tx, ty, 6, bx, by, bz)] = pop[21];
        gGhostZ_0[idxPopZ(tx, ty, 7, bx, by, bz)] = pop[24];
        gGhostZ_0[idxPopZ(tx, ty, 8, bx, by, bz)] = pop[26];
        #endif //D3Q27                                                                                                           
    }else if (threadIdx.z == (BLOCK_NZ - 1)){                                                                                                               
        gGhostZ_1[idxPopZ(tx, ty, 0, bx, by, bz)] = pop[ 5];
        gGhostZ_1[idxPopZ(tx, ty, 1, bx, by, bz)] = pop[ 9];
        gGhostZ_1[idxPopZ(tx, ty, 2, bx, by, bz)] = pop[11];
        gGhostZ_1[idxPopZ(tx, ty, 3, bx, by, bz)] = pop[16];
        gGhostZ_1[idxPopZ(tx, ty, 4, bx, by, bz)] = pop[18];
        #ifdef D3Q27                                                                                                           
        gGhostZ_1[idxPopZ(tx, ty, 5, bx, by, bz)] = pop[19];
        gGhostZ_1[idxPopZ(tx, ty, 6, bx, by, bz)] = pop[22];
        gGhostZ_1[idxPopZ(tx, ty, 7, bx, by, bz)] = pop[23];
        gGhostZ_1[idxPopZ(tx, ty, 8, bx, by, bz)] = pop[25];
        #endif //D3Q27                                                                                                                                                                                                                    
    }
}
