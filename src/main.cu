#include <stdio.h>
#include <stdlib.h>

// CUDA INCLUDE
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// FILES INCLUDES
#include "var.h"
#include "errorDef.h"
//#include "structs.h"
//#include "globalFunctions.h"
#include "lbmInitialization.cuh"
#include "mlbm.cuh"
#include "saveData.cuh"

using namespace std;

__host__ __device__
void interfaceSwap(dfloat* &pt1, dfloat* &pt2){
  dfloat *temp = pt1;
  pt1 = pt2;
  pt2 = temp;
} 

int main() {

    dfloat* fMom;
    dfloat* fGhostX_0;
    dfloat* fGhostX_1;
    dfloat* fGhostY_0; 
    dfloat* fGhostY_1;
    dfloat* fGhostZ_0; 
    dfloat* fGhostZ_1;

    dfloat* gGhostX_0;
    dfloat* gGhostX_1;
    dfloat* gGhostY_0; 
    dfloat* gGhostY_1;
    dfloat* gGhostZ_0; 
    dfloat* gGhostZ_1;

    char* dNodeType;


    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    checkCudaErrors(hipHostMalloc((void**)&(h_fMom), MEM_SIZE_MOM));
    checkCudaErrors(hipHostMalloc((void**)&(rho), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(ux), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uy), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uz), MEM_SIZE_SCALAR));


    // Setup saving folder
    folderSetup();

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);  
    hipMalloc((void**)&dNodeType, sizeof(char) * NUMBER_LBM_NODES);  

    hipMalloc((void**)&fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);

    hipMalloc((void**)&gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);        
    printf("Allocated memory \n");fflush(stdout);



    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    printf("Moments initialized \n");fflush(stdout);
    gpuInitialization_nodeType << <gridBlock, threadBlock >> >(dNodeType);
    checkCudaErrors(hipDeviceSynchronize());
    gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1);
    printf("Interface Populations initialized \n");fflush(stdout);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(gGhostX_0, fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostX_1, fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostY_0, fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostY_1, fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostZ_0, fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostZ_1, fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    size_t step = 0;
    printf("%d,",step); fflush(stdout);


    bool save = false;
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    linearMacr(h_fMom,rho,ux,uy,uz,step);

    printf("Initializing loop \n");fflush(stdout);
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    

    for (step=1; step<N_STEPS;step++){
        save =false;

        if(MACR_SAVE)
            save = !(step % MACR_SAVE);

        gpuMomCollisionStream << <gridBlock, threadBlock >> > (fMom,dNodeType,
        fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1,
        gGhostX_0,gGhostX_1,gGhostY_0,gGhostY_1,gGhostZ_0,gGhostZ_1);


        
        //swap interface pointers
        checkCudaErrors(hipDeviceSynchronize());
        interfaceSwap(fGhostX_0,gGhostX_0);
        interfaceSwap(fGhostX_1,gGhostX_1);
        interfaceSwap(fGhostY_0,gGhostY_0);
        interfaceSwap(fGhostY_1,gGhostY_1);
        interfaceSwap(fGhostZ_0,gGhostZ_0);
        interfaceSwap(fGhostZ_1,gGhostZ_1);
        

        //save macroscopics

        if(save){
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            checkCudaErrors(hipDeviceSynchronize());
            
            printf("step %d \n",step);
            linearMacr(h_fMom,rho,ux,uy,uz,step); 
            fflush(stdout);
            printf("------------------------------------------------------------------------\n");
            saveMacr(rho,ux,uy,uz,step);
        }

    }
    checkCudaErrors(hipDeviceSynchronize());
    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);

    //save info file
    saveSimInfo(step);


    /* ------------------------------ FREE ------------------------------ */
    hipFree(fMom);
    hipFree(dNodeType);
    hipFree(fGhostX_0);
    hipFree(fGhostX_1);
    hipFree(fGhostY_0);
    hipFree(fGhostY_1);
    hipFree(fGhostZ_0);
    hipFree(fGhostZ_1);

    hipFree(gGhostX_0);
    hipFree(gGhostX_1);
    hipFree(gGhostY_0);
    hipFree(gGhostY_1);
    hipFree(gGhostZ_0);
    hipFree(gGhostZ_1);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    return 0;



}
