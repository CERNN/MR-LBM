#include <stdio.h>
#include <stdlib.h>

// CUDA INCLUDE
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// FILES INCLUDES
#include "var.h"
#include "globalStructs.h"
#include "auxFunctions.cuh"
#ifdef NON_NEWTONIAN_FLUID
    #include "nnf.h"
#endif
#ifdef PARTICLE_TRACER
    #include "particleTracer.cuh"
#endif
#include "errorDef.h"
//#include "structs.h"
//#include "globalFunctions.h"
#include "lbmInitialization.cuh"
#include "mlbm.cuh"
#include "saveData.cuh"
#include "checkpoint.cuh"

using namespace std;

__host__ __device__
void interfaceSwap(dfloat* &pt1, dfloat* &pt2){
  dfloat *temp = pt1;
  pt1 = pt2;
  pt2 = temp;
} 

int main() {
    checkCudaErrors(hipSetDevice(GPU_INDEX));

    dfloat* fMom;
    ghostData fGhost;
    ghostData gGhost;
    ghostData h_fGhost;

    unsigned int* dNodeType;
    unsigned int* hNodeType;
    #if SAVE_BC
    dfloat* nodeTypeSave;
    #endif

    #ifdef DENSITY_CORRECTION
    dfloat* h_mean_rho;
    dfloat* d_mean_rho;
    #endif

    #ifdef PARTICLE_TRACER
    dfloat3* h_particlePos;
    dfloat3* d_particlePos;
    #endif

    #if MEAN_FLOW
        dfloat* m_fMom;
        dfloat* m_rho;
        dfloat* m_ux;
        dfloat* m_uy;
        dfloat* m_uz;
        #ifdef SECOND_DIST
        dfloat* m_c;
        #endif
    #endif //MEAN_FLOW

    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        dfloat* h_BC_Fx;
        dfloat* h_BC_Fy;
        dfloat* h_BC_Fz;
        #endif

        dfloat* d_BC_Fx;
        dfloat* d_BC_Fy;
        dfloat* d_BC_Fz;
    #endif //_BC_FORCES

    #ifdef SECOND_DIST
        ghostData g_fGhost;
        ghostData g_gGhost;
        ghostData g_h_fGhost;
    #endif




    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    int step = 0;

    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega;
    #endif

    #ifdef SECOND_DIST
    dfloat* C;
    #endif 


    float** randomNumbers = nullptr; // useful for turbulence

    checkCudaErrors(hipHostMalloc((void**)&(h_fMom), MEM_SIZE_MOM));
    checkCudaErrors(hipHostMalloc((void**)&(rho), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(ux), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uy), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uz), MEM_SIZE_SCALAR));
    #ifdef NON_NEWTONIAN_FLUID
    checkCudaErrors(hipHostMalloc((void**)&(omega), MEM_SIZE_SCALAR));
    #endif
    #ifdef SECOND_DIST
    checkCudaErrors(hipHostMalloc((void**)&(C), MEM_SIZE_SCALAR));
    #endif 
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipHostMalloc((void**)&(h_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif
    #if MEAN_FLOW
        checkCudaErrors(hipHostMalloc((void**)&(m_fMom), MEM_SIZE_MOM));
        checkCudaErrors(hipHostMalloc((void**)&(m_rho), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_ux), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uz), MEM_SIZE_SCALAR));
        #ifdef SECOND_DIST
        checkCudaErrors(hipHostMalloc((void**)&(m_c), MEM_SIZE_SCALAR));
        #endif
    #endif //MEAN_FLOW
    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fx), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fz), MEM_SIZE_SCALAR));
        #endif
    #endif //_BC_FORCES
    randomNumbers = (float**)malloc(sizeof(float*));


    // Setup saving folder
    folderSetup();

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);  
    hipMalloc((void**)&dNodeType, sizeof(int) * NUMBER_LBM_NODES);  

    hipMalloc((void**)&(fGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(fGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(fGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(fGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(fGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&(fGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);

    hipMalloc((void**)&(gGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(gGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(gGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(gGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(gGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&(gGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    

    #ifdef SECOND_DIST
    hipMalloc((void**)&g_fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    hipMalloc((void**)&g_fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);

    hipMalloc((void**)&g_gGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_gGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_gGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_gGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_gGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    hipMalloc((void**)&g_gGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    #endif 
     

    #ifdef DENSITY_CORRECTION
        checkCudaErrors(hipHostMalloc((void**)&(h_mean_rho), sizeof(dfloat)));
        hipMalloc((void**)&d_mean_rho, sizeof(dfloat));  
    #endif
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipMalloc((void**)&(d_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif

    #ifdef BC_FORCES
        hipMalloc((void**)&d_BC_Fx, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fy, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fz, MEM_SIZE_SCALAR);            
    #endif //_BC_FORCES
    //printf("Allocated memory \n"); if(console_flush){fflush(stdout);}
    

    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef PARTICLE_TRACER
    hipStream_t streamsPart[1];
    checkCudaErrors(hipStreamCreate(&streamsPart[0]));
    #endif

    if(RANDOM_NUMBERS)
    {   
        //printf("Initializing random numbers\n");if(console_flush){fflush(stdout);}
        checkCudaErrors(hipMallocManaged((void**)&randomNumbers[0], 
            sizeof(float)*NUMBER_LBM_NODES));
        initializationRandomNumbers(randomNumbers[0], CURAND_SEED);
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("random numbers transfer error");
        //printf("random numbers initialized \n");if(console_flush){fflush(stdout);}
    }

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    if(LOAD_CHECKPOINT || CHECKPOINT_SAVE){
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));

        #ifdef SECOND_DIST
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF));  
        #endif 
    }
    if(LOAD_CHECKPOINT){
        printf("Loading checkpoint");
        step = INI_STEP;
        loadSimCheckpoint(h_fMom, h_fGhost,
        #ifdef SECOND_DIST 
        g_fGhost,
        #endif 
        &step);

        checkCudaErrors(hipMemcpy(fMom, h_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyHostToDevice));

        checkCudaErrors(hipMemcpy(fGhost.X_0, h_fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhost.X_1, h_fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhost.Y_0, h_fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhost.Y_1, h_fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhost.Z_0, h_fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhost.Z_1, h_fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
       
        #ifdef SECOND_DIST 
        checkCudaErrors(hipMemcpy(g_fGhost.X_0, g_h_fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhost.X_1, g_h_fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhost.Y_0, g_h_fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhost.Y_1, g_h_fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhost.Z_0, g_h_fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhost.Z_1, g_h_fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyHostToDevice));
        #endif 
       

    }else{
        if(LOAD_FIELD){
        }else{
            gpuInitialization_mom << <gridBlock, threadBlock >> >(fMom, randomNumbers[0]);
        }
        //printf("Moments initialized \n");if(console_flush){fflush(stdout);}
        gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,fGhost
        #ifdef SECOND_DIST 
        ,g_fGhost
        #endif 
        );
    }

    #if MEAN_FLOW
        //initialize mean moments
        checkCudaErrors(hipMemcpy(m_fMom,fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToDevice));
    #endif //MEAN_FLOW
    checkCudaErrors(hipHostMalloc((void**)&(hNodeType), sizeof(unsigned int) * NUMBER_LBM_NODES));
    #if SAVE_BC
    checkCudaErrors(hipHostMalloc((void**)&(nodeTypeSave), sizeof(dfloat) * NUMBER_LBM_NODES));
    #endif 

    #ifndef VOXEL_FILENAME
    //gpuInitialization_nodeType << <gridBlock, threadBlock >> >(dNodeType);
    //checkCudaErrors(hipDeviceSynchronize());
        hostInitialization_nodeType(hNodeType);
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  
        checkCudaErrors(hipDeviceSynchronize());
    #endif
    #ifdef VOXEL_FILENAME
        hostInitialization_nodeType_bulk(hNodeType); //initialize the domain with  BULK
        read_xyz_file(VOXEL_FILENAME,hNodeType); //overwrite the domain with the voxels information + add missing defintion 
        hostInitialization_nodeType(hNodeType); //initialize the domain with BC
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  // copy inform\ation to device
        checkCudaErrors(hipDeviceSynchronize());
        define_voxel_bc << <gridBlock, threadBlock >> >(dNodeType); //update information of BC condition nearby the voxels
        checkCudaErrors(hipMemcpy(hNodeType, dNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyDeviceToHost)); 
    #endif

    #ifdef BC_FORCES
    gpuInitialization_force << <gridBlock, threadBlock >> >(d_BC_Fx,d_BC_Fy,d_BC_Fz);
    #endif //_BC_FORCES

    //printf("Interface Populations initialized \n");if(console_flush){fflush(stdout);}
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(gGhost.X_0, fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhost.X_1, fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhost.Y_0, fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhost.Y_1, fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhost.Z_0, fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhost.Z_1, fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));

    #ifdef SECOND_DIST 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(g_gGhost.X_0, g_fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhost.X_1, g_fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhost.Y_0, g_fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhost.Y_1, g_fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhost.Z_0, g_fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhost.Z_1, g_fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyDeviceToDevice));
    #endif 
    #ifdef DENSITY_CORRECTION
        h_mean_rho[0] = RHO_0;
        checkCudaErrors(hipMemcpy(d_mean_rho, h_mean_rho, sizeof(dfloat), hipMemcpyHostToDevice)); 
    #endif
    checkCudaErrors(hipDeviceSynchronize());

    #ifdef PARTICLE_TRACER
        initializeParticles(h_particlePos,d_particlePos);
    #endif

    
    //printf("step %zu\t",step); if(console_flush){fflush(stdout);}


    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    // Free random numbers
    if (RANDOM_NUMBERS) {
        checkCudaErrors(hipSetDevice(GPU_INDEX));
        hipFree(randomNumbers[0]);
        free(randomNumbers);
    }

   
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    #ifdef DYNAMIC_SHARED_MEMORY
    hipFuncSetAttribute(reinterpret_cast<const void*>(gpuMomCollisionStream), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEMORY_SIZE); // DOESNT WORK: DYNAMICALLY SHARED MEMORY HAS WORSE PERFORMANCE
    #endif
    for (step=INI_STEP; step<N_STEPS;step++){

        int aux = step-INI_STEP;
        bool checkpoint = false;
        #ifdef DENSITY_CORRECTION
        mean_rho(fMom,step,d_mean_rho);
        #endif 
        bool save =false;
        bool reportSave = false;
        bool macrSave = false;
        if(aux != 0){
            if(REPORT_SAVE){
                reportSave = !(step % REPORT_SAVE);
                //reportSave = true;
            }                
            if(MACR_SAVE){
                macrSave = !(step % MACR_SAVE);
                //macrSave = true;
            }
            if(MACR_SAVE || REPORT_SAVE)
                save = (reportSave || macrSave);
            if(CHECKPOINT_SAVE)
                checkpoint = !(aux % CHECKPOINT_SAVE);
        }
       



        gpuMomCollisionStream << <gridBlock, threadBlock 
        #ifdef DYNAMIC_SHARED_MEMORY
        , SHARED_MEMORY_SIZE
        #endif
        >> > (fMom,dNodeType,
        fGhost,
        gGhost,
        #ifdef SECOND_DIST 
        g_fGhost,
        g_gGhost,
        #endif 
        #ifdef DENSITY_CORRECTION
        d_mean_rho,
        #endif
        #ifdef BC_FORCES
        d_BC_Fx,d_BC_Fy,d_BC_Fz,
        #endif 
        step,
        save); 

        #ifdef PARTICLE_TRACER
            checkCudaErrors(hipDeviceSynchronize());
            updateParticlePos(d_particlePos, h_particlePos, fMom, streamsPart[0],step);
        #endif

        if(checkpoint){
            printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            // throwing a warning for being used without being initialized. But does not matter since we are overwriting it;
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhost.X_0,gGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhost.X_1,gGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhost.Y_0,gGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhost.Y_1,gGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhost.Z_0,gGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhost.Z_1,gGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipDeviceSynchronize());
           
            saveSimCheckpoint(fMom,gGhost,
            #ifdef SECOND_DIST 
            g_fGhost,
            #endif 
            &step);
        }


        
        //swap interface pointers
        checkCudaErrors(hipDeviceSynchronize());
        interfaceSwap(fGhost.X_0,gGhost.X_0);
        interfaceSwap(fGhost.X_1,gGhost.X_1);
        interfaceSwap(fGhost.Y_0,gGhost.Y_0);
        interfaceSwap(fGhost.Y_1,gGhost.Y_1);
        interfaceSwap(fGhost.Z_0,gGhost.Z_0);
        interfaceSwap(fGhost.Z_1,gGhost.Z_1);

        #ifdef SECOND_DIST
        interfaceSwap(g_fGhost.X_0,g_gGhost.X_0);
        interfaceSwap(g_fGhost.X_1,g_gGhost.X_1);
        interfaceSwap(g_fGhost.Y_0,g_gGhost.Y_0);
        interfaceSwap(g_fGhost.Y_1,g_gGhost.Y_1);
        interfaceSwap(g_fGhost.Z_0,g_gGhost.Z_0);
        interfaceSwap(g_fGhost.Z_1,g_gGhost.Z_1);
        #endif
        

        //save macroscopics

        //if(save){
            //if (N_STEPS - step < 4*((int)turn_over_time)){
            if(reportSave){
                printf("\n--------------------------- Saving report %06d ---------------------------\n", step);
                #if TREATFIELD
                treatData(h_fMom,fMom,
                #if MEAN_FLOW
                m_fMom,
                #endif //MEAN_FLOW
                step);
                //totalKineticEnergy(fMom,step);
                #endif //TREATFIELD
            
                #if TREATPOINT
                    probeExport(fMom,
                    #ifdef NON_NEWTONIAN_FLUID
                    omega,
                    #endif
                    step);
                #endif
                #if TREATLINE
                velocityProfile(fMom,1,step);
                velocityProfile(fMom,2,step);
                velocityProfile(fMom,3,step);
                velocityProfile(fMom,4,step);
                velocityProfile(fMom,5,step);
                velocityProfile(fMom,6,step);
                #endif
            }
            if(macrSave){
                #if defined BC_FORCES && defined SAVE_BC_FORCES
                checkCudaErrors(hipDeviceSynchronize()); 
                checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                #endif
                //if (!(step%((int)turn_over_time/10))){
                //if((step>N_STEPS-80*(int)(MACR_SAVE))){ 
                //    if((step%((int)(turn_over_time/2))) == 0){
                        checkCudaErrors(hipDeviceSynchronize()); 
                        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
                        linearMacr(h_fMom,rho,ux,uy,uz,
                        #ifdef NON_NEWTONIAN_FLUID
                        omega,
                        #endif
                        #ifdef SECOND_DIST 
                        C,
                        #endif 
                        #if SAVE_BC
                        nodeTypeSave,
                        hNodeType,
                        #endif
                        #if defined BC_FORCES && defined SAVE_BC_FORCES
                        h_BC_Fx,
                        h_BC_Fy,
                        h_BC_Fz,
                        #endif
                        step); 

                        printf("\n--------------------------- Saving macro %06d ---------------------------\n", step);
                        if(console_flush){fflush(stdout);}
                        //if(step > N_STEPS - 14000){
                        if(!ONLY_FINAL_MACRO){
                        saveMacr(rho,ux,uy,uz,
                        #ifdef NON_NEWTONIAN_FLUID
                        omega,
                        #endif
                        #ifdef SECOND_DIST 
                        C,
                        #endif 
                        #if SAVE_BC
                        nodeTypeSave,
                        #endif
                        #if defined BC_FORCES && defined SAVE_BC_FORCES
                        h_BC_Fx,
                        h_BC_Fy,
                        h_BC_Fz,
                        #endif
                        step);
                    // }
                      //  }
                    //}
                }

                #ifdef BC_FORCES
                    totalBcDrag(d_BC_Fx, d_BC_Fy, d_BC_Fz, step);
                #endif
            }

        //}

    } // end of the loop
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));

    #if defined BC_FORCES && defined SAVE_BC_FORCES
    checkCudaErrors(hipDeviceSynchronize()); 
    checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    #endif

    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    if(console_flush){fflush(stdout);}
    
    saveMacr(rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step);

    #ifdef PARTICLE_TRACER
        checkCudaErrors(hipMemcpy(h_particlePos, d_particlePos, sizeof(dfloat3)*NUM_PARTICLES, hipMemcpyDeviceToHost)); 
        saveParticleInfo(h_particlePos,step);
    #endif
    if(CHECKPOINT_SAVE){
        printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            
        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhost.X_0,gGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhost.X_1,gGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhost.Y_0,gGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhost.Y_1,gGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhost.Z_0,gGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhost.Z_1,gGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipDeviceSynchronize());
        
        saveSimCheckpoint(fMom,gGhost,
        #ifdef SECOND_DIST 
        g_fGhost,
        #endif 
        &step);

    }
    checkCudaErrors(hipDeviceSynchronize());
    #if MEAN_FLOW
            linearMacr(m_fMom,m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            hNodeType,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX); 

            saveMacr(m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX);
    #endif //MEAN_FLOW

    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);

    //save info file
    saveSimInfo(step,MLUPS);


    /* ------------------------------ FREE ------------------------------ */
    hipFree(fMom);
    hipFree(dNodeType);
    hipFree(hNodeType);

    hipFree(fGhost.X_0);
    hipFree(fGhost.X_1);
    hipFree(fGhost.Y_0);
    hipFree(fGhost.Y_1);
    hipFree(fGhost.Z_0);
    hipFree(fGhost.Z_1);

    hipFree(gGhost.X_0);
    hipFree(gGhost.X_1);
    hipFree(gGhost.Y_0);
    hipFree(gGhost.Y_1);
    hipFree(gGhost.Z_0);
    hipFree(gGhost.Z_1);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    #ifdef SECOND_DIST 
    hipFree(g_fGhost.X_0);
    hipFree(g_fGhost.X_1);
    hipFree(g_fGhost.Y_0);
    hipFree(g_fGhost.Y_1);
    hipFree(g_fGhost.Z_0);
    hipFree(g_fGhost.Z_1);

    hipFree(g_gGhost.X_0);
    hipFree(g_gGhost.X_1);
    hipFree(g_gGhost.Y_0);
    hipFree(g_gGhost.Y_1);
    hipFree(g_gGhost.Z_0);
    hipFree(g_gGhost.Z_1);

    hipFree(C);
    #endif 

    #if MEAN_FLOW
        hipFree(m_fMom);
        hipFree(m_rho);
        hipFree(m_ux);
        hipFree(m_uy);
        hipFree(m_uz);
        #ifdef SECOND_DIST
        hipFree(m_c);
        #endif
    #endif //MEAN_FLOW

    if(LOAD_CHECKPOINT){
        hipFree(h_fGhost.X_0);
        hipFree(h_fGhost.X_1);
        hipFree(h_fGhost.Y_0);
        hipFree(h_fGhost.Y_1);
        hipFree(h_fGhost.Z_0);
        hipFree(h_fGhost.Z_1);
        #ifdef SECOND_DIST 
        hipFree(g_h_fGhost.X_0);
        hipFree(g_h_fGhost.X_1);
        hipFree(g_h_fGhost.Y_0);
        hipFree(g_h_fGhost.Y_1);
        hipFree(g_h_fGhost.Z_0);
        hipFree(g_h_fGhost.Z_1);
        #endif 
    }

    #ifdef DENSITY_CORRECTION
        hipFree(d_mean_rho);
        free(h_mean_rho);
    #endif
    #ifdef PARTICLE_TRACER
    hipFree(h_particlePos);
    hipFree(d_particlePos);
    #endif

    #if SAVE_BC
    hipFree(nodeTypeSave);
    #endif


    return 0;



}
