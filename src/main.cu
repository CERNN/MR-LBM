#include <stdio.h>
#include <stdlib.h>

// CUDA INCLUDE
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// FILES INCLUDES
#include "var.h"
#include "globalStructs.h"
#include "auxFunctions.cuh"
#ifdef NON_NEWTONIAN_FLUID
    #include "nnf.h"
#endif
#ifdef PARTICLE_TRACER
    #include "particleTracer.cuh"
#endif
#include "errorDef.h"
//#include "structs.h"
//#include "globalFunctions.h"
#include "lbmInitialization.cuh"
#include "mlbm.cuh"
#include "saveData.cuh"
#include "checkpoint.cuh"

using namespace std;

__host__ __device__
void interfaceSwap(dfloat* &pt1, dfloat* &pt2){
  dfloat *temp = pt1;
  pt1 = pt2;
  pt2 = temp;
} 

int main() {
    checkCudaErrors(hipSetDevice(GPU_INDEX));

    dfloat* fMom;
    dfloat* fGhostX_0;
    dfloat* fGhostX_1;
    dfloat* fGhostY_0; 
    dfloat* fGhostY_1;
    dfloat* fGhostZ_0; 
    dfloat* fGhostZ_1;

    dfloat* gGhostX_0;
    dfloat* gGhostX_1;
    dfloat* gGhostY_0; 
    dfloat* gGhostY_1;
    dfloat* gGhostZ_0; 
    dfloat* gGhostZ_1;

    dfloat* h_fGhostX_0;
    dfloat* h_fGhostX_1;
    dfloat* h_fGhostY_0; 
    dfloat* h_fGhostY_1;
    dfloat* h_fGhostZ_0; 
    dfloat* h_fGhostZ_1;

    unsigned int* dNodeType;
    unsigned int* hNodeType;
    #if SAVE_BC
    dfloat* nodeTypeSave;
    #endif

    #ifdef DENSITY_CORRECTION
    dfloat* h_mean_rho;
    dfloat* d_mean_rho;
    #endif

    #ifdef PARTICLE_TRACER
    dfloat3* h_particlePos;
    dfloat3* d_particlePos;
    #endif

    #if MEAN_FLOW
        dfloat* m_fMom;
        dfloat* m_rho;
        dfloat* m_ux;
        dfloat* m_uy;
        dfloat* m_uz;
        #ifdef SECOND_DIST
        dfloat* m_c;
        #endif
    #endif //MEAN_FLOW

    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        dfloat* h_BC_Fx;
        dfloat* h_BC_Fy;
        dfloat* h_BC_Fz;
        #endif

        dfloat* d_BC_Fx;
        dfloat* d_BC_Fy;
        dfloat* d_BC_Fz;
    #endif //_BC_FORCES

    #ifdef SECOND_DIST
        dfloat* g_fGhostX_0;
        dfloat* g_fGhostX_1;
        dfloat* g_fGhostY_0; 
        dfloat* g_fGhostY_1;
        dfloat* g_fGhostZ_0; 
        dfloat* g_fGhostZ_1;

        dfloat* g_gGhostX_0;
        dfloat* g_gGhostX_1;
        dfloat* g_gGhostY_0; 
        dfloat* g_gGhostY_1;
        dfloat* g_gGhostZ_0; 
        dfloat* g_gGhostZ_1;

        dfloat* g_h_fGhostX_0;
        dfloat* g_h_fGhostX_1;
        dfloat* g_h_fGhostY_0; 
        dfloat* g_h_fGhostY_1;
        dfloat* g_h_fGhostZ_0; 
        dfloat* g_h_fGhostZ_1;
    #endif




    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    int step = 0;

    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega;
    #endif

    #ifdef SECOND_DIST
    dfloat* C;
    #endif 


    float** randomNumbers = nullptr; // useful for turbulence

    checkCudaErrors(hipHostMalloc((void**)&(h_fMom), MEM_SIZE_MOM));
    checkCudaErrors(hipHostMalloc((void**)&(rho), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(ux), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uy), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uz), MEM_SIZE_SCALAR));
    #ifdef NON_NEWTONIAN_FLUID
    checkCudaErrors(hipHostMalloc((void**)&(omega), MEM_SIZE_SCALAR));
    #endif
    #ifdef SECOND_DIST
    checkCudaErrors(hipHostMalloc((void**)&(C), MEM_SIZE_SCALAR));
    #endif 
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipHostMalloc((void**)&(h_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif
    #if MEAN_FLOW
        checkCudaErrors(hipHostMalloc((void**)&(m_fMom), MEM_SIZE_MOM));
        checkCudaErrors(hipHostMalloc((void**)&(m_rho), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_ux), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uz), MEM_SIZE_SCALAR));
        #ifdef SECOND_DIST
        checkCudaErrors(hipHostMalloc((void**)&(m_c), MEM_SIZE_SCALAR));
        #endif
    #endif //MEAN_FLOW
    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fx), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fz), MEM_SIZE_SCALAR));
        #endif
    #endif //_BC_FORCES
    randomNumbers = (float**)malloc(sizeof(float*));


    // Setup saving folder
    folderSetup();

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);  
    hipMalloc((void**)&dNodeType, sizeof(int) * NUMBER_LBM_NODES);  

    hipMalloc((void**)&fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);

    hipMalloc((void**)&gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    

    #ifdef SECOND_DIST
    hipMalloc((void**)&g_fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    hipMalloc((void**)&g_fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);

    hipMalloc((void**)&g_gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&g_gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&g_gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    hipMalloc((void**)&g_gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    #endif 
     

    #ifdef DENSITY_CORRECTION
        checkCudaErrors(hipHostMalloc((void**)&(h_mean_rho), sizeof(dfloat)));
        hipMalloc((void**)&d_mean_rho, sizeof(dfloat));  
    #endif
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipMalloc((void**)&(d_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif

    #ifdef BC_FORCES
        hipMalloc((void**)&d_BC_Fx, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fy, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fz, MEM_SIZE_SCALAR);            
    #endif //_BC_FORCES
    //printf("Allocated memory \n"); if(console_flush){fflush(stdout);}
    

    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef PARTICLE_TRACER
    hipStream_t streamsPart[1];
    checkCudaErrors(hipStreamCreate(&streamsPart[0]));
    #endif

    if(RANDOM_NUMBERS)
    {   
        //printf("Initializing random numbers\n");if(console_flush){fflush(stdout);}
        checkCudaErrors(hipMallocManaged((void**)&randomNumbers[0], 
            sizeof(float)*NUMBER_LBM_NODES));
        initializationRandomNumbers(randomNumbers[0], CURAND_SEED);
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("random numbers transfer error");
        //printf("random numbers initialized \n");if(console_flush){fflush(stdout);}
    }

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    if(LOAD_CHECKPOINT || CHECKPOINT_SAVE){
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostX_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostX_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostY_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostY_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostZ_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostZ_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));

        #ifdef SECOND_DIST
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhostX_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhostX_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhostY_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhostY_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhostZ_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF));
        checkCudaErrors(hipHostMalloc((void**)&(g_h_fGhostZ_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF));  
        #endif 
    }
    if(LOAD_CHECKPOINT){
        step = INI_STEP;
        loadSimCheckpoint(h_fMom, h_fGhostX_0,h_fGhostX_1,h_fGhostY_0,h_fGhostY_1,h_fGhostZ_0,h_fGhostZ_1,
        #ifdef SECOND_DIST 
        g_fGhostX_0, g_fGhostX_1,g_fGhostY_0, g_fGhostY_1, g_fGhostZ_0, g_fGhostZ_1,
        #endif 
        &step);

        checkCudaErrors(hipMemcpy(fMom, h_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyHostToDevice));

        checkCudaErrors(hipMemcpy(fGhostX_0, h_fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostX_1, h_fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostY_0, h_fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostY_1, h_fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostZ_0, h_fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostZ_1, h_fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
       
        #ifdef SECOND_DIST 
        checkCudaErrors(hipMemcpy(g_fGhostX_0, g_h_fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhostX_1, g_h_fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhostY_0, g_h_fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhostY_1, g_h_fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhostZ_0, g_h_fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(g_fGhostZ_1, g_h_fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyHostToDevice));
        #endif 
       

    }else{
        gpuInitialization_mom << <gridBlock, threadBlock >> >(fMom, randomNumbers[0]);
        //printf("Moments initialized \n");if(console_flush){fflush(stdout);}
        gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1
        #ifdef SECOND_DIST 
        ,g_fGhostX_0, g_fGhostX_1, g_fGhostY_0, g_fGhostY_1,g_fGhostZ_0, g_fGhostZ_1
        #endif 
        );
    }

    #if MEAN_FLOW
        //initialize mean moments
        checkCudaErrors(hipMemcpy(m_fMom,fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToDevice));
    #endif //MEAN_FLOW
    checkCudaErrors(hipHostMalloc((void**)&(hNodeType), sizeof(unsigned int) * NUMBER_LBM_NODES));
    #if SAVE_BC
    checkCudaErrors(hipHostMalloc((void**)&(nodeTypeSave), sizeof(dfloat) * NUMBER_LBM_NODES));
    #endif 

    #ifndef VOXEL_FILENAME
    //gpuInitialization_nodeType << <gridBlock, threadBlock >> >(dNodeType);
    //checkCudaErrors(hipDeviceSynchronize());
        hostInitialization_nodeType(hNodeType);
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  
        checkCudaErrors(hipDeviceSynchronize());
    #endif
    #ifdef VOXEL_FILENAME
        hostInitialization_nodeType_bulk(hNodeType); //initialize the domain with  BULK
        read_xyz_file(VOXEL_FILENAME,hNodeType); //overwrite the domain with the voxels information + add missing defintion 
        hostInitialization_nodeType(hNodeType); //initialize the domain with BC
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  // copy inform\ation to device
        checkCudaErrors(hipDeviceSynchronize());
        define_voxel_bc << <gridBlock, threadBlock >> >(dNodeType); //update information of BC condition nearby the voxels
        checkCudaErrors(hipMemcpy(hNodeType, dNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyDeviceToHost)); 
    #endif

    #ifdef BC_FORCES
    gpuInitialization_force << <gridBlock, threadBlock >> >(d_BC_Fx,d_BC_Fy,d_BC_Fz);
    #endif //_BC_FORCES

    //printf("Interface Populations initialized \n");if(console_flush){fflush(stdout);}
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(gGhostX_0, fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostX_1, fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostY_0, fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostY_1, fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostZ_0, fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostZ_1, fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));

    #ifdef SECOND_DIST 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(g_gGhostX_0, g_fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhostX_1, g_fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhostY_0, g_fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhostY_1, g_fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhostZ_0, g_fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(g_gGhostZ_1, g_fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyDeviceToDevice));
    #endif 
    #ifdef DENSITY_CORRECTION
        h_mean_rho[0] = RHO_0;
        checkCudaErrors(hipMemcpy(d_mean_rho, h_mean_rho, sizeof(dfloat), hipMemcpyHostToDevice)); 
    #endif
    checkCudaErrors(hipDeviceSynchronize());

    #ifdef PARTICLE_TRACER
        initializeParticles(h_particlePos,d_particlePos);
    #endif

    
    //printf("step %zu\t",step); if(console_flush){fflush(stdout);}


    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    // Free random numbers
    if (RANDOM_NUMBERS) {
        checkCudaErrors(hipSetDevice(GPU_INDEX));
        hipFree(randomNumbers[0]);
        free(randomNumbers);
    }

   
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    #ifdef DYNAMIC_SHARED_MEMORY
    hipFuncSetAttribute(reinterpret_cast<const void*>(gpuMomCollisionStream), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEMORY_SIZE); // DOESNT WORK: DYNAMICALLY SHARED MEMORY HAS WORSE PERFORMANCE
    #endif
    for (step=1; step<N_STEPS;step++){

        int aux = step-INI_STEP;
        bool checkpoint = false;
        #ifdef DENSITY_CORRECTION
        mean_rho(fMom,step,d_mean_rho);
        #endif 
        bool save =false;
        if(aux != 0){
            if(MACR_SAVE)
                save = !(step % MACR_SAVE);
            if(CHECKPOINT_SAVE)
                checkpoint = !(aux % CHECKPOINT_SAVE);
        }
       



        gpuMomCollisionStream << <gridBlock, threadBlock 
        #ifdef DYNAMIC_SHARED_MEMORY
        , SHARED_MEMORY_SIZE
        #endif
        >> > (fMom,dNodeType,
        fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1,
        gGhostX_0,gGhostX_1,gGhostY_0,gGhostY_1,gGhostZ_0,gGhostZ_1,
        #ifdef SECOND_DIST 
        g_fGhostX_0,g_fGhostX_1,g_fGhostY_0,g_fGhostY_1,g_fGhostZ_0,g_fGhostZ_1,
        g_gGhostX_0,g_gGhostX_1,g_gGhostY_0,g_gGhostY_1,g_gGhostZ_0,g_gGhostZ_1,
        #endif 
        #ifdef DENSITY_CORRECTION
        d_mean_rho,
        #endif
        #ifdef BC_FORCES
        d_BC_Fx,d_BC_Fy,d_BC_Fz,
        #endif 
        step,
        save); 

        #ifdef PARTICLE_TRACER
            checkCudaErrors(hipDeviceSynchronize());
            updateParticlePos(d_particlePos, h_particlePos, fMom, streamsPart[0],step);
        #endif

        if(checkpoint){
            printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            // throwing a warning for being used without being initialized. But does not matter since we are overwriting it;
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostX_0,gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostX_1,gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostY_0,gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostY_1,gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostZ_0,gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostZ_1,gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipDeviceSynchronize());
           
            saveSimCheckpoint(fMom,gGhostX_0,gGhostX_1,gGhostY_0,gGhostY_1,gGhostZ_0,gGhostZ_1,
            #ifdef SECOND_DIST 
            g_fGhostX_0, g_fGhostX_1, g_fGhostY_0, g_fGhostY_1,g_fGhostZ_0, g_fGhostZ_1,
            #endif 
            &step);
        }


        
        //swap interface pointers
        checkCudaErrors(hipDeviceSynchronize());
        interfaceSwap(fGhostX_0,gGhostX_0);
        interfaceSwap(fGhostX_1,gGhostX_1);
        interfaceSwap(fGhostY_0,gGhostY_0);
        interfaceSwap(fGhostY_1,gGhostY_1);
        interfaceSwap(fGhostZ_0,gGhostZ_0);
        interfaceSwap(fGhostZ_1,gGhostZ_1);

        #ifdef SECOND_DIST
        interfaceSwap(g_fGhostX_0,g_gGhostX_0);
        interfaceSwap(g_fGhostX_1,g_gGhostX_1);
        interfaceSwap(g_fGhostY_0,g_gGhostY_0);
        interfaceSwap(g_fGhostY_1,g_gGhostY_1);
        interfaceSwap(g_fGhostZ_0,g_gGhostZ_0);
        interfaceSwap(g_fGhostZ_1,g_gGhostZ_1);
        #endif
        

        //save macroscopics

        if(save){
            //if (N_STEPS - step < 4*((int)turn_over_time)){

            #if TREATFIELD
            treatData(h_fMom,fMom,
            #if MEAN_FLOW
            m_fMom,
            #endif //MEAN_FLOW
            step);
            //totalKineticEnergy(fMom,step);
            #endif //TREATFIELD
         
            #if TREATPOINT
                probeExport(fMom,
                #ifdef NON_NEWTONIAN_FLUID
                omega,
                #endif
                step);
            #endif
            #if TREATLINE
            velocityProfile(fMom,1,step);
            velocityProfile(fMom,2,step);
            velocityProfile(fMom,3,step);
            velocityProfile(fMom,4,step);
            velocityProfile(fMom,5,step);
            velocityProfile(fMom,6,step);
            #endif
            
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            checkCudaErrors(hipDeviceSynchronize()); 
            checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
            #endif
            //if (!(step%((int)turn_over_time/10))){
            //if((step>N_STEPS-25*(int)(turn_over_time))){ 
            //    if((step%((int)(turn_over_time/2))) == 0){
                    checkCudaErrors(hipDeviceSynchronize()); 
                    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
                    linearMacr(h_fMom,rho,ux,uy,uz,
                    #ifdef NON_NEWTONIAN_FLUID
                    omega,
                    #endif
                    #ifdef SECOND_DIST 
                    C,
                    #endif 
                    #if SAVE_BC
                    nodeTypeSave,
                    hNodeType,
                    #endif
                    #if defined BC_FORCES && defined SAVE_BC_FORCES
                    h_BC_Fx,
                    h_BC_Fy,
                    h_BC_Fz,
                    #endif
                    step); 

                    printf("\n--------------------------- Saving macro %06d ---------------------------\n", step);
                    if(console_flush){fflush(stdout);}
                    if(!ONLY_FINAL_MACRO){
                    saveMacr(rho,ux,uy,uz,
                    #ifdef NON_NEWTONIAN_FLUID
                    omega,
                    #endif
                    #ifdef SECOND_DIST 
                    C,
                    #endif 
                    #if SAVE_BC
                    nodeTypeSave,
                    #endif
                    #if defined BC_FORCES && defined SAVE_BC_FORCES
                    h_BC_Fx,
                    h_BC_Fy,
                    h_BC_Fz,
                    #endif
                    step);
                    }
                //}
            //}

            #ifdef BC_FORCES
                totalBcDrag(d_BC_Fx, d_BC_Fy, d_BC_Fz, step);
            #endif

        }

    } // end of the loop
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));

    #if defined BC_FORCES && defined SAVE_BC_FORCES
    checkCudaErrors(hipDeviceSynchronize()); 
    checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    #endif

    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    if(console_flush){fflush(stdout);}
    
    saveMacr(rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step);

    #ifdef PARTICLE_TRACER
        checkCudaErrors(hipMemcpy(h_particlePos, d_particlePos, sizeof(dfloat3)*NUM_PARTICLES, hipMemcpyDeviceToHost)); 
        saveParticleInfo(h_particlePos,step);
    #endif
    if(CHECKPOINT_SAVE){
        printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            
        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostX_0,gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostX_1,gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostY_0,gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostY_1,gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostZ_0,gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostZ_1,gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipDeviceSynchronize());
        
        saveSimCheckpoint(fMom,gGhostX_0,gGhostX_1,gGhostY_0,gGhostY_1,gGhostZ_0,gGhostZ_1,
        #ifdef SECOND_DIST 
        g_fGhostX_0, g_fGhostX_1, g_fGhostY_0, g_fGhostY_1,g_fGhostZ_0, g_fGhostZ_1,
        #endif 
        &step);

    }
    checkCudaErrors(hipDeviceSynchronize());
    #if MEAN_FLOW
            linearMacr(m_fMom,m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            hNodeType,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX); 

            saveMacr(m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX);
    #endif //MEAN_FLOW

    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);

    //save info file
    saveSimInfo(step,MLUPS);


    /* ------------------------------ FREE ------------------------------ */
    hipFree(fMom);
    hipFree(dNodeType);
    hipFree(hNodeType);

    hipFree(fGhostX_0);
    hipFree(fGhostX_1);
    hipFree(fGhostY_0);
    hipFree(fGhostY_1);
    hipFree(fGhostZ_0);
    hipFree(fGhostZ_1);

    hipFree(gGhostX_0);
    hipFree(gGhostX_1);
    hipFree(gGhostY_0);
    hipFree(gGhostY_1);
    hipFree(gGhostZ_0);
    hipFree(gGhostZ_1);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    #ifdef SECOND_DIST 
    hipFree(g_fGhostX_0);
    hipFree(g_fGhostX_1);
    hipFree(g_fGhostY_0);
    hipFree(g_fGhostY_1);
    hipFree(g_fGhostZ_0);
    hipFree(g_fGhostZ_1);

    hipFree(g_gGhostX_0);
    hipFree(g_gGhostX_1);
    hipFree(g_gGhostY_0);
    hipFree(g_gGhostY_1);
    hipFree(g_gGhostZ_0);
    hipFree(g_gGhostZ_1);

    hipFree(C);
    #endif 

    #if MEAN_FLOW
        hipFree(m_fMom);
        hipFree(m_rho);
        hipFree(m_ux);
        hipFree(m_uy);
        hipFree(m_uz);
        #ifdef SECOND_DIST
        hipFree(m_c);
        #endif
    #endif //MEAN_FLOW

    if(LOAD_CHECKPOINT){
        hipFree(h_fGhostX_0);
        hipFree(h_fGhostX_1);
        hipFree(h_fGhostY_0);
        hipFree(h_fGhostY_1);
        hipFree(h_fGhostZ_0);
        hipFree(h_fGhostZ_1);
        #ifdef SECOND_DIST 
        hipFree(g_h_fGhostX_0);
        hipFree(g_h_fGhostX_1);
        hipFree(g_h_fGhostY_0);
        hipFree(g_h_fGhostY_1);
        hipFree(g_h_fGhostZ_0);
        hipFree(g_h_fGhostZ_1);
        #endif 
    }

    #ifdef DENSITY_CORRECTION
        hipFree(d_mean_rho);
        free(h_mean_rho);
    #endif
    #ifdef PARTICLE_TRACER
    hipFree(h_particlePos);
    hipFree(d_particlePos);
    #endif

    #if SAVE_BC
    hipFree(nodeTypeSave);
    #endif


    return 0;



}
