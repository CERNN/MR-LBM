#include "main.cuh"

using namespace std;

int main() {
    // Setup saving folder
    folderSetup();

    //set cuda device
    checkCudaErrors(hipSetDevice(GPU_INDEX));

    //variable declaration
    dfloat* d_fMom;
    ghostInterfaceData ghostInterface;

    unsigned int* dNodeType;
    unsigned int* hNodeType;

    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;
    
    
    #ifdef OMEGA_FIELD
    dfloat* omega;
    #endif

    #ifdef SECOND_DIST
    dfloat* C;
    #endif 

    #ifdef A_XX_DIST
    dfloat* Axx;
    #endif
    #ifdef A_XY_DIST
    dfloat* Axy;
    #endif
    #ifdef A_XZ_DIST
    dfloat* Axz;
    #endif
    #ifdef A_YY_DIST
    dfloat* Ayy;
    #endif
    #ifdef A_YZ_DIST
    dfloat* Ayz;
    #endif
    #ifdef A_ZZ_DIST
    dfloat* Azz;
    #endif

    #if NODE_TYPE_SAVE
    dfloat* nodeTypeSave;
    #endif

    #ifdef DENSITY_CORRECTION
    dfloat* h_mean_rho;
    dfloat* d_mean_rho;
    #endif

    #ifdef PARTICLE_TRACER
    dfloat3* h_particlePos;
    dfloat3* d_particlePos;
    #endif

    #if MEAN_FLOW
        dfloat* m_fMom;
        dfloat* m_rho;
        dfloat* m_ux;
        dfloat* m_uy;
        dfloat* m_uz;
        #ifdef SECOND_DIST
        dfloat* m_c;
        #endif
    #endif //MEAN_FLOW

    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        dfloat* h_BC_Fx;
        dfloat* h_BC_Fy;
        dfloat* h_BC_Fz;
        #endif

        dfloat* d_BC_Fx;
        dfloat* d_BC_Fy;
        dfloat* d_BC_Fz;
    #endif //_BC_FORCES


    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    int step = 0;

    float** randomNumbers = nullptr; // useful for turbulence
    randomNumbers = (float**)malloc(sizeof(float*));

    allocateHostMemory(
        &h_fMom, &rho, &ux, &uy, &uz
        OMEGA_FIELD_PARAMS_PTR
        SECOND_DIST_PARAMS_PTR
        A_XX_DIST_PARAMS_PTR
        A_XY_DIST_PARAMS_PTR
        A_XZ_DIST_PARAMS_PTR
        A_YY_DIST_PARAMS_PTR
        A_YZ_DIST_PARAMS_PTR
        A_ZZ_DIST_PARAMS_PTR
        PARTICLE_TRACER_PARAMS_PTR(h_)
        MEAN_FLOW_PARAMS_PTR
        MEAN_FLOW_SECOND_DIST_PARAMS_PTR
        BC_FORCES_PARAMS_PTR(h_)
    );
    printf("Host Memory Allocated \n"); if(console_flush) fflush(stdout);
    /* -------------- ALLOCATION FOR GPU ------------- */
    allocateDeviceMemory(
        &d_fMom, &dNodeType, &ghostInterface
        PARTICLE_TRACER_PARAMS_PTR(d_)
        BC_FORCES_PARAMS_PTR(d_)
    );
    printf("Device Memory Allocated \n"); if(console_flush) fflush(stdout);
    #ifdef DENSITY_CORRECTION
        checkCudaErrors(hipHostMalloc((void**)&(h_mean_rho), sizeof(dfloat)));
        hipMalloc((void**)&d_mean_rho, sizeof(dfloat));  
        printf("Density Correction Memory Allocated \n"); if(console_flush) fflush(stdout);
    #endif
    //printf("Allocated memory \n"); if(console_flush){fflush(stdout);}

    // Setup Streams
    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef PARTICLE_TRACER
    hipStream_t streamsPart[1];
    checkCudaErrors(hipStreamCreate(&streamsPart[0]));
    #endif

    initializeDomain(ghostInterface,     
                     d_fMom, h_fMom, 
                     #if MEAN_FLOW
                     m_fMom,
                     #endif
                     hNodeType, dNodeType, randomNumbers, 
                     BC_FORCES_PARAMS(d_)
                     DENSITY_CORRECTION_PARAMS(h_)
                     DENSITY_CORRECTION_PARAMS(d_)
                     PARTICLE_TRACER_PARAMS_PTR(h_)
                     PARTICLE_TRACER_PARAMS_PTR(d_)
                     &step, gridBlock, threadBlock);

    printf("Domain Initialized\n"); if(console_flush) fflush(stdout);
    
    /* ------------------------------ TIMER EVENTS  ------------------------------ */
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    hipEvent_t start, stop, start_step, stop_step;
    initializeCudaEvents(start, stop, start_step, stop_step);
    /* ------------------------------ LBM LOOP ------------------------------ */

    #ifdef DYNAMIC_SHARED_MEMORY
    hipFuncSetAttribute(reinterpret_cast<const void*>(gpuMomCollisionStream), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEMORY_SIZE); // DOESNT WORK: DYNAMICALLY SHARED MEMORY HAS WORSE PERFORMANCE
    #endif

    /* --------------------------------------------------------------------- */
    /* ---------------------------- BEGIN LOOP ----------------------------- */
    /* --------------------------------------------------------------------- */
    for (step=INI_STEP; step<N_STEPS;step++){

        int aux = step-INI_STEP;
        bool checkpoint = false;

        #ifdef DENSITY_CORRECTION
        mean_rho(d_fMom,step,d_mean_rho);
        #endif 

        bool save =false;
        bool reportSave = false;
        bool macrSave = false;

        if(aux != 0){
            if(REPORT_SAVE){ reportSave = !(step % REPORT_SAVE);}                
            if(MACR_SAVE){ macrSave   = !(step % MACR_SAVE);}
            if(MACR_SAVE || REPORT_SAVE){ save = (reportSave || macrSave);}
            if(CHECKPOINT_SAVE){ checkpoint = !(aux % CHECKPOINT_SAVE);}
        }
       
        gpuMomCollisionStream << <gridBlock, threadBlock DYNAMIC_SHARED_MEMORY_PARAMS>> >(d_fMom, dNodeType,ghostInterface, DENSITY_CORRECTION_PARAMS(d_) BC_FORCES_PARAMS(d_) step, save); 

        #ifdef PARTICLE_TRACER
            checkCudaErrors(hipDeviceSynchronize());
            updateParticlePos(d_particlePos, h_particlePos, d_fMom, streamsPart[0],step);
        #endif

        if(checkpoint){
            printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            // throwing a warning for being used without being initialized. But does not matter since we are overwriting it;
            checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            interfaceCudaMemcpy(ghostInterface,ghostInterface.h_fGhost,ghostInterface.gGhost,hipMemcpyDeviceToHost,QF);       
            #ifdef SECOND_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.g_h_fGhost,ghostInterface.g_fGhost,hipMemcpyDeviceToHost,GF);
            #endif    
            #ifdef A_XX_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.Axx_h_fGhost,ghostInterface.Axx_fGhost,hipMemcpyDeviceToHost,GF);
            #endif       
            #ifdef A_XY_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.Axy_h_fGhost,ghostInterface.Axy_fGhost,hipMemcpyDeviceToHost,GF);
            #endif           
            #ifdef A_XZ_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.Axz_h_fGhost,ghostInterface.Axz_fGhost,hipMemcpyDeviceToHost,GF);
            #endif           
            #ifdef A_YY_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.Ayy_h_fGhost,ghostInterface.Ayy_fGhost,hipMemcpyDeviceToHost,GF);
            #endif           
            #ifdef A_YZ_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.Ayz_h_fGhost,ghostInterface.Ayz_fGhost,hipMemcpyDeviceToHost,GF);
            #endif           
            #ifdef A_ZZ_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.Azz_h_fGhost,ghostInterface.Azz_fGhost,hipMemcpyDeviceToHost,GF);
            #endif                 
            saveSimCheckpoint(d_fMom, ghostInterface, &step);
        }
       
        //swap interface pointers
        swapGhostInterfaces(ghostInterface);
        
        //save macroscopics

        //if (N_STEPS - step < 4*((int)turn_over_time)){
        if(reportSave){
            printf("\n--------------------------- Saving report %06d ---------------------------\n", step);
            treatData(h_fMom,d_fMom,
            #if MEAN_FLOW
            m_fMom,
            #endif //MEAN_FLOW
            step); 
        }
        if(macrSave){
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            checkCudaErrors(hipDeviceSynchronize()); 
            checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
            #endif
            //if (!(step%((int)turn_over_time/10))){
            //if((step>N_STEPS-80*(int)(MACR_SAVE))){ 
            //if((step%((int)(turn_over_time/2))) == 0){
                checkCudaErrors(hipDeviceSynchronize()); 
                checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));

                printf("\n--------------------------- Saving macro %06d ---------------------------\n", step);
                if(console_flush){fflush(stdout);}
                //if(step > N_STEPS - 14000){
                if(!ONLY_FINAL_MACRO){
                    saveMacr(h_fMom,rho,ux,uy,uz, OMEGA_FIELD_PARAMS
                    #ifdef SECOND_DIST 
                    C,
                    #endif 
                    #ifdef A_XX_DIST 
                    Axx,
                    #endif 
                    #ifdef A_XY_DIST 
                    Axy,
                    #endif
                    #ifdef A_XZ_DIST 
                    Axz,
                    #endif
                    #ifdef A_YY_DIST 
                    Ayy,
                    #endif
                    #ifdef A_YZ_DIST 
                    Ayz,
                    #endif
                    #ifdef A_ZZ_DIST 
                    Azz,
                    #endif
                    NODE_TYPE_SAVE_PARAMS BC_FORCES_PARAMS(h_) step);
                }
            //}

            #ifdef BC_FORCES
                totalBcDrag(d_BC_Fx, d_BC_Fy, d_BC_Fz, step);
            #endif
        }

    } 
    /* --------------------------------------------------------------------- */
    /* ------------------------------ END LOO ------------------------------ */
    /* --------------------------------------------------------------------- */

    checkCudaErrors(hipDeviceSynchronize());

    //Calculate MLUPS

    dfloat MLUPS = recordElapsedTime(start_step, stop_step, step);
    printf("MLUPS: %f\n",MLUPS);
    
    /* ------------------------------ POST ------------------------------ */
    checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));

    #if defined BC_FORCES && defined SAVE_BC_FORCES
    checkCudaErrors(hipDeviceSynchronize()); 
    checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    #endif



    if(console_flush){fflush(stdout);}
    
    saveMacr(h_fMom,rho,ux,uy,uz, OMEGA_FIELD_PARAMS 
    #ifdef SECOND_DIST 
    C,
    #endif 
    #ifdef A_XX_DIST 
    Axx,
    #endif 
    #ifdef A_XY_DIST 
    Axy,
    #endif
    #ifdef A_XZ_DIST 
    Axz,
    #endif
    #ifdef A_YY_DIST 
    Ayy,
    #endif
    #ifdef A_YZ_DIST 
    Ayz,
    #endif
    #ifdef A_ZZ_DIST 
    Azz,
    #endif
    NODE_TYPE_SAVE_PARAMS BC_FORCES_PARAMS(PREFIX) step);

    #ifdef PARTICLE_TRACER
        checkCudaErrors(hipMemcpy(h_particlePos, d_particlePos, sizeof(dfloat3)*NUM_PARTICLES, hipMemcpyDeviceToHost)); 
        saveParticleInfo(h_particlePos,step);
    #endif
    if(CHECKPOINT_SAVE){
        printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            
        checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
        interfaceCudaMemcpy(ghostInterface,ghostInterface.h_fGhost,ghostInterface.gGhost,hipMemcpyDeviceToHost,QF);    
        #ifdef SECOND_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.g_h_fGhost,ghostInterface.g_fGhost,hipMemcpyDeviceToHost,GF);
        #endif  
        #ifdef A_XX_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.Axx_h_fGhost,ghostInterface.Axx_fGhost,hipMemcpyDeviceToHost,GF);
        #endif 
        #ifdef A_XY_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.Axy_h_fGhost,ghostInterface.Axy_fGhost,hipMemcpyDeviceToHost,GF);
        #endif
        #ifdef A_XZ_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.Axz_h_fGhost,ghostInterface.Axz_fGhost,hipMemcpyDeviceToHost,GF);
        #endif
        #ifdef A_YY_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.Ayy_h_fGhost,ghostInterface.Ayy_fGhost,hipMemcpyDeviceToHost,GF);
        #endif
        #ifdef A_YZ_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.Ayz_h_fGhost,ghostInterface.Ayz_fGhost,hipMemcpyDeviceToHost,GF);
        #endif
        #ifdef A_ZZ_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.Azz_h_fGhost,ghostInterface.Azz_fGhost,hipMemcpyDeviceToHost,GF);
        #endif    
        saveSimCheckpoint(d_fMom,ghostInterface,&step);
    }
    checkCudaErrors(hipDeviceSynchronize());
    #if MEAN_FLOW
            saveMacr(m_fMom,m_rho,m_ux,m_uy,m_uz, OMEGA_FIELD_PARAMS
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            NODE_TYPE_SAVE_PARAMS BC_FORCES_PARAMS(h_) INT_MAX);
    #endif //MEAN_FLOW



    //save info file
    saveSimInfo(step,MLUPS);


    /* ------------------------------ FREE ------------------------------ */
    hipFree(d_fMom);
    hipFree(dNodeType);
    hipFree(hNodeType);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    #ifdef SECOND_DIST 
    hipFree(C);
    #endif 
    #ifdef A_XX_DIST 
    hipFree(Axx);
    #endif 
    #ifdef A_XY_DIST 
    hipFree(Axy);
    #endif
    #ifdef A_XZ_DIST 
    hipFree(Axz);
    #endif
    #ifdef A_YY_DIST 
    hipFree(Ayy);
    #endif
    #ifdef A_YZ_DIST 
    hipFree(Ayz);
    #endif
    #ifdef A_ZZ_DIST 
    hipFree(Azz);
    #endif

    interfaceFree(ghostInterface);

    #if MEAN_FLOW
        hipFree(m_fMom);
        hipFree(m_rho);
        hipFree(m_ux);
        hipFree(m_uy);
        hipFree(m_uz);
        #ifdef SECOND_DIST
        hipFree(m_c);
        #endif
    #endif //MEAN_FLOW



    #ifdef DENSITY_CORRECTION
        hipFree(d_mean_rho);
        free(h_mean_rho);
    #endif
    #ifdef PARTICLE_TRACER
    hipFree(h_particlePos);
    hipFree(d_particlePos);
    #endif

    #if NODE_TYPE_SAVE
    hipFree(nodeTypeSave);
    #endif
    return 0;
}