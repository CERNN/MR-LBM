#include <stdio.h>
#include <stdlib.h>

// CUDA INCLUDE
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// FILES INCLUDES
#include "var.h"
#include "globalStructs.h"
#include "auxFunctions.cuh"
#ifdef NON_NEWTONIAN_FLUID
    #include "nnf.h"
#endif
#ifdef PARTICLE_TRACER
    #include "particleTracer.cuh"
#endif
#include "errorDef.h"
//#include "structs.h"
//#include "globalFunctions.h"
#include "lbmInitialization.cuh"
#include "mlbm.cuh"
#include "saveData.cuh"
#include "checkpoint.cuh"

using namespace std;

__host__ __device__
void interfaceSwap(dfloat* &pt1, dfloat* &pt2){
  dfloat *temp = pt1;
  pt1 = pt2;
  pt2 = temp;
} 

int main() {
    checkCudaErrors(hipSetDevice(GPU_INDEX));

    dfloat* fMom;
    dfloat* fGhostX_0;
    dfloat* fGhostX_1;
    dfloat* fGhostY_0; 
    dfloat* fGhostY_1;
    dfloat* fGhostZ_0; 
    dfloat* fGhostZ_1;

    dfloat* gGhostX_0;
    dfloat* gGhostX_1;
    dfloat* gGhostY_0; 
    dfloat* gGhostY_1;
    dfloat* gGhostZ_0; 
    dfloat* gGhostZ_1;

    dfloat* h_fGhostX_0;
    dfloat* h_fGhostX_1;
    dfloat* h_fGhostY_0; 
    dfloat* h_fGhostY_1;
    dfloat* h_fGhostZ_0; 
    dfloat* h_fGhostZ_1;

    unsigned char* dNodeType;
    unsigned char* hNodeType;
    #if SAVE_BC
    dfloat* nodeTypeSave;
    #endif

    #ifdef DENSITY_CORRECTION
    dfloat* h_mean_rho;
    dfloat* d_mean_rho;
    #endif

    #ifdef PARTICLE_TRACER
    dfloat3* h_particlePos;
    dfloat3* d_particlePos;
    #endif

    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    int step = 0;

    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega;
    #endif

    float** randomNumbers = nullptr; // useful for turbulence

    checkCudaErrors(hipHostMalloc((void**)&(h_fMom), MEM_SIZE_MOM));
    checkCudaErrors(hipHostMalloc((void**)&(rho), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(ux), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uy), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uz), MEM_SIZE_SCALAR));
    #ifdef NON_NEWTONIAN_FLUID
    checkCudaErrors(hipHostMalloc((void**)&(omega), MEM_SIZE_SCALAR));
    #endif
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipHostMalloc((void**)&(h_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif
    randomNumbers = (float**)malloc(sizeof(float*));


    // Setup saving folder
    folderSetup();

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);  
    hipMalloc((void**)&dNodeType, sizeof(char) * NUMBER_LBM_NODES);  

    hipMalloc((void**)&fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);

    hipMalloc((void**)&gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    

    #ifdef DENSITY_CORRECTION
    checkCudaErrors(hipHostMalloc((void**)&(h_mean_rho), sizeof(dfloat)));
    hipMalloc((void**)&d_mean_rho, sizeof(dfloat));  
    #endif
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipMalloc((void**)&(d_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif

    //printf("Allocated memory \n");fflush(stdout);
    


    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef PARTICLE_TRACER
    hipStream_t streamsPart[1];
    checkCudaErrors(hipStreamCreate(&streamsPart[0]));
    #endif

    if(RANDOM_NUMBERS)
    {   
        //printf("Initializing random numbers\n");fflush(stdout);
        checkCudaErrors(hipMallocManaged((void**)&randomNumbers[0], 
            sizeof(float)*NUMBER_LBM_NODES));
        initializationRandomNumbers(randomNumbers[0], CURAND_SEED);
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("random numbers transfer error");
        //printf("random numbers initialized \n");fflush(stdout);
    }

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    if(LOAD_CHECKPOINT || CHECKPOINT_SAVE){
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostX_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostX_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostY_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostY_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostZ_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));
        checkCudaErrors(hipHostMalloc((void**)&(h_fGhostZ_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));
    }
    if(LOAD_CHECKPOINT){
        step = INI_STEP;
        loadSimCheckpoint(h_fMom, h_fGhostX_0,h_fGhostX_1,h_fGhostY_0,h_fGhostY_1,h_fGhostZ_0,h_fGhostZ_1,&step);

        checkCudaErrors(hipMemcpy(fMom, h_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyHostToDevice));

        checkCudaErrors(hipMemcpy(fGhostX_0, h_fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostX_1, h_fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostY_0, h_fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostY_1, h_fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostZ_0, h_fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(fGhostZ_1, h_fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
       

    }else{
        gpuInitialization_mom << <gridBlock, threadBlock >> >(fMom, randomNumbers[0]);
        //printf("Moments initialized \n");fflush(stdout);
        gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1);
    }

    checkCudaErrors(hipHostMalloc((void**)&(hNodeType), sizeof(unsigned char) * NUMBER_LBM_NODES));
    #if SAVE_BC
    checkCudaErrors(hipHostMalloc((void**)&(nodeTypeSave), sizeof(dfloat) * NUMBER_LBM_NODES));
    #endif 

    #ifndef voxel_
    gpuInitialization_nodeType << <gridBlock, threadBlock >> >(dNodeType);
    checkCudaErrors(hipDeviceSynchronize());
    #endif
    #ifdef voxel_
    read_voxel_csv(VOXEL_FILENAME,hNodeType);
    checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned char) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  
    checkCudaErrors(hipDeviceSynchronize());  
    #endif

    //printf("Interface Populations initialized \n");fflush(stdout);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(gGhostX_0, fGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostX_1, fGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostY_0, fGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostY_1, fGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostZ_0, fGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(gGhostZ_1, fGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));
    #ifdef DENSITY_CORRECTION
    h_mean_rho[0] = RHO_0;
    checkCudaErrors(hipMemcpy(d_mean_rho, h_mean_rho, sizeof(dfloat), hipMemcpyHostToDevice)); 
    #endif
    checkCudaErrors(hipDeviceSynchronize());

    #ifdef PARTICLE_TRACER
        initializeParticles(h_particlePos,d_particlePos);
    #endif

    
    //printf("step %zu\t",step); fflush(stdout);


    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    step);

    // Free random numbers
    if (RANDOM_NUMBERS) {
        checkCudaErrors(hipSetDevice(GPU_INDEX));
        hipFree(randomNumbers[0]);
        free(randomNumbers);
    }

   
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    for (step=1; step<N_STEPS;step++){

        int aux = step-INI_STEP;
        bool checkpoint = false;
        #if DENSITY_CORRECTION
        bool densityCorrection = false;
        #endif 
        bool save =false;
        if(aux != 0){
            if(MACR_SAVE)
                save = !(step % MACR_SAVE);
            if(CHECKPOINT_SAVE)
                checkpoint = !(aux % CHECKPOINT_SAVE);
            #ifdef DENSITY_CORRECTION
                densityCorrection = true;
            #endif
        }
       



        gpuMomCollisionStream << <gridBlock, threadBlock >> > (fMom,dNodeType,
        fGhostX_0,fGhostX_1,fGhostY_0,fGhostY_1,fGhostZ_0,fGhostZ_1,
        gGhostX_0,gGhostX_1,gGhostY_0,gGhostY_1,gGhostZ_0,gGhostZ_1,
        #ifdef DENSITY_CORRECTION
        d_mean_rho,
        #endif
        step); 

        #ifdef DENSITY_CORRECTION
            mean_moment(fMom,d_mean_rho,0,step);
        #endif
        #ifdef PARTICLE_TRACER
            checkCudaErrors(hipDeviceSynchronize());
            updateParticlePos(d_particlePos, h_particlePos, fMom, streamsPart[0],step);
        #endif

        if(checkpoint){
            printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            // throwing a warning for being used without being initialized. But does not matter since we are overwriting it;
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostX_0,gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostX_1,gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostY_0,gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostY_1,gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostZ_0,gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_fGhostZ_1,gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipDeviceSynchronize());
           
            saveSimCheckpoint(fMom,gGhostX_0,gGhostX_1,gGhostY_0,gGhostY_1,gGhostZ_0,gGhostZ_1,&step);
        }


        
        //swap interface pointers
        checkCudaErrors(hipDeviceSynchronize());
        interfaceSwap(fGhostX_0,gGhostX_0);
        interfaceSwap(fGhostX_1,gGhostX_1);
        interfaceSwap(fGhostY_0,gGhostY_0);
        interfaceSwap(fGhostY_1,gGhostY_1);
        interfaceSwap(fGhostZ_0,gGhostZ_0);
        interfaceSwap(fGhostZ_1,gGhostZ_1);
        

        //save macroscopics

        if(save){
            //if (N_STEPS - step < 4*((int)turn_over_time)){

            #if TREATFIELD
            treatData(h_fMom,fMom,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            step);
            #endif

            #if TREATPOINT
                probeExport(fMom,
                #ifdef NON_NEWTONIAN_FLUID
                omega,
                #endif
                step);
            #endif
            
            //if (!(step%((int)turn_over_time/10))){
            //if((step>N_STEPS-500*(int)(turn_over_time))){ 
                if((step%((int)(turn_over_time))) == 0){
                    checkCudaErrors(hipDeviceSynchronize()); 
                    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
                    linearMacr(h_fMom,rho,ux,uy,uz,
                    #ifdef NON_NEWTONIAN_FLUID
                    omega,
                    #endif
                    #if SAVE_BC
                    nodeTypeSave,
                    hNodeType,
                    #endif
                    step); 

                    printf("\n--------------------------- Saving macro %06d ---------------------------\n", step);
                    fflush(stdout);

                    saveMacr(rho,ux,uy,uz,
                    #ifdef NON_NEWTONIAN_FLUID
                    omega,
                    #endif
                    #if SAVE_BC
                    nodeTypeSave,
                    #endif
                    step);
                }
            //}
        }

    } // end of the loop
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            linearMacr(h_fMom,rho,ux,uy,uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #if SAVE_BC
            nodeTypeSave,
            hNodeType,
            #endif
            step); 
            fflush(stdout);
            saveMacr(rho,ux,uy,uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #if SAVE_BC
            nodeTypeSave,
            #endif
            step);

    #ifdef PARTICLE_TRACER
        checkCudaErrors(hipMemcpy(h_particlePos, d_particlePos, sizeof(dfloat3)*NUM_PARTICLES, hipMemcpyDeviceToHost)); 
        saveParticleInfo(h_particlePos,step);
    #endif
    if(CHECKPOINT_SAVE){
        printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            
        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostX_0,gGhostX_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostX_1,gGhostX_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostY_0,gGhostY_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostY_1,gGhostY_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostZ_0,gGhostZ_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_fGhostZ_1,gGhostZ_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipDeviceSynchronize());
        
        saveSimCheckpoint(fMom,gGhostX_0,gGhostX_1,gGhostY_0,gGhostY_1,gGhostZ_0,gGhostZ_1,&step);

    }
    checkCudaErrors(hipDeviceSynchronize());
    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);

    //save info file
    saveSimInfo(step,MLUPS);


    /* ------------------------------ FREE ------------------------------ */
    hipFree(fMom);
    hipFree(dNodeType);
    hipFree(hNodeType);

    hipFree(fGhostX_0);
    hipFree(fGhostX_1);
    hipFree(fGhostY_0);
    hipFree(fGhostY_1);
    hipFree(fGhostZ_0);
    hipFree(fGhostZ_1);

    hipFree(gGhostX_0);
    hipFree(gGhostX_1);
    hipFree(gGhostY_0);
    hipFree(gGhostY_1);
    hipFree(gGhostZ_0);
    hipFree(gGhostZ_1);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    if(LOAD_CHECKPOINT){
        hipFree(h_fGhostX_0);
        hipFree(h_fGhostX_1);
        hipFree(h_fGhostY_0);
        hipFree(h_fGhostY_1);
        hipFree(h_fGhostZ_0);
        hipFree(h_fGhostZ_1);
    }

    #ifdef DENSITY_CORRECTION
    hipFree(d_mean_rho);
    free(h_mean_rho);
    #endif
    #ifdef PARTICLE_TRACER
    hipFree(h_particlePos);
    hipFree(d_particlePos);
    #endif

    #if SAVE_BC
    hipFree(nodeTypeSave);
    #endif


    return 0;



}
