#include <stdio.h>
#include <stdlib.h>

// CUDA INCLUDE
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


// FILES INCLUDES
#include "var.h"
#include "globalStructs.h"
#include "auxFunctions.cuh"
#ifdef NON_NEWTONIAN_FLUID
    #include "nnf.h"
#endif
#ifdef PARTICLE_TRACER
    #include "particleTracer.cuh"
#endif
#include "errorDef.h"
//#include "structs.h"
//#include "globalFunctions.h"
#include "lbmInitialization.cuh"
#include "mlbm.cuh"
#include "saveData.cuh"
#include "checkpoint.cuh"

using namespace std;

__host__ __device__
void interfaceSwap(dfloat* &pt1, dfloat* &pt2){
  dfloat *temp = pt1;
  pt1 = pt2;
  pt2 = temp;
} 

int main() {
    checkCudaErrors(hipSetDevice(GPU_INDEX));

    dfloat* fMom;
    ghostInterfaceData ghostInterface;

    unsigned int* dNodeType;
    unsigned int* hNodeType;
    #if SAVE_BC
    dfloat* nodeTypeSave;
    #endif

    #ifdef DENSITY_CORRECTION
    dfloat* h_mean_rho;
    dfloat* d_mean_rho;
    #endif

    #ifdef PARTICLE_TRACER
    dfloat3* h_particlePos;
    dfloat3* d_particlePos;
    #endif

    #if MEAN_FLOW
        dfloat* m_fMom;
        dfloat* m_rho;
        dfloat* m_ux;
        dfloat* m_uy;
        dfloat* m_uz;
        #ifdef SECOND_DIST
        dfloat* m_c;
        #endif
    #endif //MEAN_FLOW

    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        dfloat* h_BC_Fx;
        dfloat* h_BC_Fy;
        dfloat* h_BC_Fz;
        #endif

        dfloat* d_BC_Fx;
        dfloat* d_BC_Fy;
        dfloat* d_BC_Fz;
    #endif //_BC_FORCES





    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    int step = 0;

    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega;
    #endif

    #ifdef SECOND_DIST
    dfloat* C;
    #endif 


    float** randomNumbers = nullptr; // useful for turbulence

    checkCudaErrors(hipHostMalloc((void**)&(h_fMom), MEM_SIZE_MOM));
    checkCudaErrors(hipHostMalloc((void**)&(rho), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(ux), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uy), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uz), MEM_SIZE_SCALAR));
    #ifdef NON_NEWTONIAN_FLUID
    checkCudaErrors(hipHostMalloc((void**)&(omega), MEM_SIZE_SCALAR));
    #endif
    #ifdef SECOND_DIST
    checkCudaErrors(hipHostMalloc((void**)&(C), MEM_SIZE_SCALAR));
    #endif 
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipHostMalloc((void**)&(h_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif
    #if MEAN_FLOW
        checkCudaErrors(hipHostMalloc((void**)&(m_fMom), MEM_SIZE_MOM));
        checkCudaErrors(hipHostMalloc((void**)&(m_rho), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_ux), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uz), MEM_SIZE_SCALAR));
        #ifdef SECOND_DIST
        checkCudaErrors(hipHostMalloc((void**)&(m_c), MEM_SIZE_SCALAR));
        #endif
    #endif //MEAN_FLOW
    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fx), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fz), MEM_SIZE_SCALAR));
        #endif
    #endif //_BC_FORCES
    randomNumbers = (float**)malloc(sizeof(float*));


    // Setup saving folder
    folderSetup();

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);  
    hipMalloc((void**)&dNodeType, sizeof(int) * NUMBER_LBM_NODES);  

    hipMalloc((void**)&(ghostInterface.fGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(ghostInterface.fGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(ghostInterface.fGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(ghostInterface.fGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(ghostInterface.fGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&(ghostInterface.fGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);

    hipMalloc((void**)&(ghostInterface.gGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(ghostInterface.gGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF);    
    hipMalloc((void**)&(ghostInterface.gGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(ghostInterface.gGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF);    
    hipMalloc((void**)&(ghostInterface.gGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    
    hipMalloc((void**)&(ghostInterface.gGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF);    

    #ifdef SECOND_DIST
    hipMalloc((void**)&(ghostInterface.g_fGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_fGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_fGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_fGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_fGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    hipMalloc((void**)&(ghostInterface.g_fGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);

    hipMalloc((void**)&(ghostInterface.g_gGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_gGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_gGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_gGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF);    
    hipMalloc((void**)&(ghostInterface.g_gGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    hipMalloc((void**)&(ghostInterface.g_gGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF);    
    #endif 
     

    #ifdef DENSITY_CORRECTION
        checkCudaErrors(hipHostMalloc((void**)&(h_mean_rho), sizeof(dfloat)));
        hipMalloc((void**)&d_mean_rho, sizeof(dfloat));  
    #endif
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipMalloc((void**)&(d_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif

    #ifdef BC_FORCES
        hipMalloc((void**)&d_BC_Fx, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fy, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fz, MEM_SIZE_SCALAR);            
    #endif //_BC_FORCES
    //printf("Allocated memory \n"); if(console_flush){fflush(stdout);}
    

    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef PARTICLE_TRACER
    hipStream_t streamsPart[1];
    checkCudaErrors(hipStreamCreate(&streamsPart[0]));
    #endif

    if(RANDOM_NUMBERS)
    {   
        //printf("Initializing random numbers\n");if(console_flush){fflush(stdout);}
        checkCudaErrors(hipMallocManaged((void**)&randomNumbers[0], 
            sizeof(float)*NUMBER_LBM_NODES));
        initializationRandomNumbers(randomNumbers[0], CURAND_SEED);
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("random numbers transfer error");
        //printf("random numbers initialized \n");if(console_flush){fflush(stdout);}
    }

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    if(LOAD_CHECKPOINT || CHECKPOINT_SAVE){
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.h_fGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.h_fGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.h_fGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.h_fGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.h_fGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.h_fGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF));

        #ifdef SECOND_DIST
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.g_h_fGhost.X_0), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.g_h_fGhost.X_1), sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.g_h_fGhost.Y_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.g_h_fGhost.Y_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.g_h_fGhost.Z_0), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF));
        checkCudaErrors(hipHostMalloc((void**)&(ghostInterface.g_h_fGhost.Z_1), sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF));  
        #endif 
    }
    if(LOAD_CHECKPOINT){
        printf("Loading checkpoint");
        step = INI_STEP;
        loadSimCheckpoint(h_fMom, ghostInterface, &step);

        checkCudaErrors(hipMemcpy(fMom, h_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyHostToDevice));

        checkCudaErrors(hipMemcpy(ghostInterface.fGhost.X_0, ghostInterface.h_fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.fGhost.X_1, ghostInterface.h_fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.fGhost.Y_0, ghostInterface.h_fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.fGhost.Y_1, ghostInterface.h_fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.fGhost.Z_0, ghostInterface.h_fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.fGhost.Z_1, ghostInterface.h_fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyHostToDevice));
       
        #ifdef SECOND_DIST 
        checkCudaErrors(hipMemcpy(ghostInterface.g_fGhost.X_0, ghostInterface.g_h_fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.g_fGhost.X_1, ghostInterface.g_h_fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.g_fGhost.Y_0, ghostInterface.g_h_fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.g_fGhost.Y_1, ghostInterface.g_h_fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.g_fGhost.Z_0, ghostInterface.g_h_fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(ghostInterface.g_fGhost.Z_1, ghostInterface.g_h_fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyHostToDevice));
        #endif 
       

    }else{
        if(LOAD_FIELD){
        }else{
            gpuInitialization_mom << <gridBlock, threadBlock >> >(fMom, randomNumbers[0]);
        }
        //printf("Moments initialized \n");if(console_flush){fflush(stdout);}
        gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,ghostInterface);
    }

    #if MEAN_FLOW
        //initialize mean moments
        checkCudaErrors(hipMemcpy(m_fMom,fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToDevice));
    #endif //MEAN_FLOW
    checkCudaErrors(hipHostMalloc((void**)&(hNodeType), sizeof(unsigned int) * NUMBER_LBM_NODES));
    #if SAVE_BC
    checkCudaErrors(hipHostMalloc((void**)&(nodeTypeSave), sizeof(dfloat) * NUMBER_LBM_NODES));
    #endif 

    #ifndef VOXEL_FILENAME
    //gpuInitialization_nodeType << <gridBlock, threadBlock >> >(dNodeType);
    //checkCudaErrors(hipDeviceSynchronize());
        hostInitialization_nodeType(hNodeType);
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  
        checkCudaErrors(hipDeviceSynchronize());
    #endif
    #ifdef VOXEL_FILENAME
        hostInitialization_nodeType_bulk(hNodeType); //initialize the domain with  BULK
        read_xyz_file(VOXEL_FILENAME,hNodeType); //overwrite the domain with the voxels information + add missing defintion 
        hostInitialization_nodeType(hNodeType); //initialize the domain with BC
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  // copy inform\ation to device
        checkCudaErrors(hipDeviceSynchronize());
        define_voxel_bc << <gridBlock, threadBlock >> >(dNodeType); //update information of BC condition nearby the voxels
        checkCudaErrors(hipMemcpy(hNodeType, dNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyDeviceToHost)); 
    #endif

    #ifdef BC_FORCES
    gpuInitialization_force << <gridBlock, threadBlock >> >(d_BC_Fx,d_BC_Fy,d_BC_Fz);
    #endif //_BC_FORCES

    //printf("Interface Populations initialized \n");if(console_flush){fflush(stdout);}
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(ghostInterface.gGhost.X_0, ghostInterface.fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.gGhost.X_1, ghostInterface.fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.gGhost.Y_0, ghostInterface.fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.gGhost.Y_1, ghostInterface.fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.gGhost.Z_0, ghostInterface.fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.gGhost.Z_1, ghostInterface.fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToDevice));

    #ifdef SECOND_DIST 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(ghostInterface.g_gGhost.X_0, ghostInterface.g_fGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.g_gGhost.X_1, ghostInterface.g_fGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.g_gGhost.Y_0, ghostInterface.g_fGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.g_gGhost.Y_1, ghostInterface.g_fGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.g_gGhost.Z_0, ghostInterface.g_fGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(ghostInterface.g_gGhost.Z_1, ghostInterface.g_fGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * GF, hipMemcpyDeviceToDevice));
    #endif 
    #ifdef DENSITY_CORRECTION
        h_mean_rho[0] = RHO_0;
        checkCudaErrors(hipMemcpy(d_mean_rho, h_mean_rho, sizeof(dfloat), hipMemcpyHostToDevice)); 
    #endif
    checkCudaErrors(hipDeviceSynchronize());

    #ifdef PARTICLE_TRACER
        initializeParticles(h_particlePos,d_particlePos);
    #endif

    
    //printf("step %zu\t",step); if(console_flush){fflush(stdout);}


    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    // Free random numbers
    if (RANDOM_NUMBERS) {
        checkCudaErrors(hipSetDevice(GPU_INDEX));
        hipFree(randomNumbers[0]);
        free(randomNumbers);
    }

   
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    #ifdef DYNAMIC_SHARED_MEMORY
    hipFuncSetAttribute(reinterpret_cast<const void*>(gpuMomCollisionStream), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEMORY_SIZE); // DOESNT WORK: DYNAMICALLY SHARED MEMORY HAS WORSE PERFORMANCE
    #endif
    for (step=INI_STEP; step<N_STEPS;step++){

        int aux = step-INI_STEP;
        bool checkpoint = false;
        #ifdef DENSITY_CORRECTION
        mean_rho(fMom,step,d_mean_rho);
        #endif 
        bool save =false;
        bool reportSave = false;
        bool macrSave = false;
        if(aux != 0){
            if(REPORT_SAVE){
                reportSave = !(step % REPORT_SAVE);
                //reportSave = true;
            }                
            if(MACR_SAVE){
                macrSave = !(step % MACR_SAVE);
                //macrSave = true;
            }
            if(MACR_SAVE || REPORT_SAVE)
                save = (reportSave || macrSave);
            if(CHECKPOINT_SAVE)
                checkpoint = !(aux % CHECKPOINT_SAVE);
        }
       



        gpuMomCollisionStream << <gridBlock, threadBlock 
        #ifdef DYNAMIC_SHARED_MEMORY
        , SHARED_MEMORY_SIZE
        #endif
        >> > (fMom,dNodeType,ghostInterface,
        #ifdef DENSITY_CORRECTION
        d_mean_rho,
        #endif
        #ifdef BC_FORCES
        d_BC_Fx,d_BC_Fy,d_BC_Fz,
        #endif 
        step,
        save); 

        #ifdef PARTICLE_TRACER
            checkCudaErrors(hipDeviceSynchronize());
            updateParticlePos(d_particlePos, h_particlePos, fMom, streamsPart[0],step);
        #endif

        if(checkpoint){
            printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            // throwing a warning for being used without being initialized. But does not matter since we are overwriting it;
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.X_0,ghostInterface.gGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.X_1,ghostInterface.gGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Y_0,ghostInterface.gGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Y_1,ghostInterface.gGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Z_0,ghostInterface.gGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Z_1,ghostInterface.gGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
            checkCudaErrors(hipDeviceSynchronize());
           
            saveSimCheckpoint(fMom, ghostInterface, &step);
        }


        
        //swap interface pointers
        checkCudaErrors(hipDeviceSynchronize());
        interfaceSwap(ghostInterface.fGhost.X_0,ghostInterface.gGhost.X_0);
        interfaceSwap(ghostInterface.fGhost.X_1,ghostInterface.gGhost.X_1);
        interfaceSwap(ghostInterface.fGhost.Y_0,ghostInterface.gGhost.Y_0);
        interfaceSwap(ghostInterface.fGhost.Y_1,ghostInterface.gGhost.Y_1);
        interfaceSwap(ghostInterface.fGhost.Z_0,ghostInterface.gGhost.Z_0);
        interfaceSwap(ghostInterface.fGhost.Z_1,ghostInterface.gGhost.Z_1);

        #ifdef SECOND_DIST
        interfaceSwap(ghostInterface.g_fGhost.X_0,ghostInterface.g_gGhost.X_0);
        interfaceSwap(ghostInterface.g_fGhost.X_1,ghostInterface.g_gGhost.X_1);
        interfaceSwap(ghostInterface.g_fGhost.Y_0,ghostInterface.g_gGhost.Y_0);
        interfaceSwap(ghostInterface.g_fGhost.Y_1,ghostInterface.g_gGhost.Y_1);
        interfaceSwap(ghostInterface.g_fGhost.Z_0,ghostInterface.g_gGhost.Z_0);
        interfaceSwap(ghostInterface.g_fGhost.Z_1,ghostInterface.g_gGhost.Z_1);
        #endif
        

        //save macroscopics

        //if(save){
            //if (N_STEPS - step < 4*((int)turn_over_time)){
            if(reportSave){
                printf("\n--------------------------- Saving report %06d ---------------------------\n", step);
                #if TREATFIELD
                treatData(h_fMom,fMom,
                #if MEAN_FLOW
                m_fMom,
                #endif //MEAN_FLOW
                step);
                //totalKineticEnergy(fMom,step);
                #endif //TREATFIELD
            
                #if TREATPOINT
                    probeExport(fMom,
                    #ifdef NON_NEWTONIAN_FLUID
                    omega,
                    #endif
                    step);
                #endif
                #if TREATLINE
                velocityProfile(fMom,1,step);
                velocityProfile(fMom,2,step);
                velocityProfile(fMom,3,step);
                velocityProfile(fMom,4,step);
                velocityProfile(fMom,5,step);
                velocityProfile(fMom,6,step);
                #endif
            }
            if(macrSave){
                #if defined BC_FORCES && defined SAVE_BC_FORCES
                checkCudaErrors(hipDeviceSynchronize()); 
                checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                #endif
                //if (!(step%((int)turn_over_time/10))){
                //if((step>N_STEPS-80*(int)(MACR_SAVE))){ 
                //    if((step%((int)(turn_over_time/2))) == 0){
                        checkCudaErrors(hipDeviceSynchronize()); 
                        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
                        linearMacr(h_fMom,rho,ux,uy,uz,
                        #ifdef NON_NEWTONIAN_FLUID
                        omega,
                        #endif
                        #ifdef SECOND_DIST 
                        C,
                        #endif 
                        #if SAVE_BC
                        nodeTypeSave,
                        hNodeType,
                        #endif
                        #if defined BC_FORCES && defined SAVE_BC_FORCES
                        h_BC_Fx,
                        h_BC_Fy,
                        h_BC_Fz,
                        #endif
                        step); 

                        printf("\n--------------------------- Saving macro %06d ---------------------------\n", step);
                        if(console_flush){fflush(stdout);}
                        //if(step > N_STEPS - 14000){
                        if(!ONLY_FINAL_MACRO){
                        saveMacr(rho,ux,uy,uz,
                        #ifdef NON_NEWTONIAN_FLUID
                        omega,
                        #endif
                        #ifdef SECOND_DIST 
                        C,
                        #endif 
                        #if SAVE_BC
                        nodeTypeSave,
                        #endif
                        #if defined BC_FORCES && defined SAVE_BC_FORCES
                        h_BC_Fx,
                        h_BC_Fy,
                        h_BC_Fz,
                        #endif
                        step);
                    // }
                      //  }
                    //}
                }

                #ifdef BC_FORCES
                    totalBcDrag(d_BC_Fx, d_BC_Fy, d_BC_Fz, step);
                #endif
            }

        //}

    } // end of the loop
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));

    #if defined BC_FORCES && defined SAVE_BC_FORCES
    checkCudaErrors(hipDeviceSynchronize()); 
    checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    #endif

    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    if(console_flush){fflush(stdout);}
    
    saveMacr(rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step);

    #ifdef PARTICLE_TRACER
        checkCudaErrors(hipMemcpy(h_particlePos, d_particlePos, sizeof(dfloat3)*NUM_PARTICLES, hipMemcpyDeviceToHost)); 
        saveParticleInfo(h_particlePos,step);
    #endif
    if(CHECKPOINT_SAVE){
        printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            
        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.X_0,ghostInterface.gGhost.X_0, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.X_1,ghostInterface.gGhost.X_1, sizeof(dfloat) * NUMBER_GHOST_FACE_YZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Y_0,ghostInterface.gGhost.Y_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Y_1,ghostInterface.gGhost.Y_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XZ * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Z_0,ghostInterface.gGhost.Z_0, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(ghostInterface.h_fGhost.Z_1,ghostInterface.gGhost.Z_1, sizeof(dfloat) * NUMBER_GHOST_FACE_XY * QF, hipMemcpyDeviceToHost));
        checkCudaErrors(hipDeviceSynchronize());
        
        saveSimCheckpoint(fMom,ghostInterface,&step);

    }
    checkCudaErrors(hipDeviceSynchronize());
    #if MEAN_FLOW
            linearMacr(m_fMom,m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            hNodeType,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX); 

            saveMacr(m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX);
    #endif //MEAN_FLOW

    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);

    //save info file
    saveSimInfo(step,MLUPS);


    /* ------------------------------ FREE ------------------------------ */
    hipFree(fMom);
    hipFree(dNodeType);
    hipFree(hNodeType);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    #ifdef SECOND_DIST 
    hipFree(C);
    #endif 

    
    hipFree(ghostInterface.fGhost.X_0);
    hipFree(ghostInterface.fGhost.X_1);
    hipFree(ghostInterface.fGhost.Y_0);
    hipFree(ghostInterface.fGhost.Y_1);
    hipFree(ghostInterface.fGhost.Z_0);
    hipFree(ghostInterface.fGhost.Z_1);

    hipFree(ghostInterface.gGhost.X_0);
    hipFree(ghostInterface.gGhost.X_1);
    hipFree(ghostInterface.gGhost.Y_0);
    hipFree(ghostInterface.gGhost.Y_1);
    hipFree(ghostInterface.gGhost.Z_0);
    hipFree(ghostInterface.gGhost.Z_1);

    #ifdef SECOND_DIST 
    hipFree(ghostInterface.g_fGhost.X_0);
    hipFree(ghostInterface.g_fGhost.X_1);
    hipFree(ghostInterface.g_fGhost.Y_0);
    hipFree(ghostInterface.g_fGhost.Y_1);
    hipFree(ghostInterface.g_fGhost.Z_0);
    hipFree(ghostInterface.g_fGhost.Z_1);

    hipFree(ghostInterface.g_gGhost.X_0);
    hipFree(ghostInterface.g_gGhost.X_1);
    hipFree(ghostInterface.g_gGhost.Y_0);
    hipFree(ghostInterface.g_gGhost.Y_1);
    hipFree(ghostInterface.g_gGhost.Z_0);
    hipFree(ghostInterface.g_gGhost.Z_1);
    #endif 

    if(LOAD_CHECKPOINT){
        hipFree(ghostInterface.h_fGhost.X_0);
        hipFree(ghostInterface.h_fGhost.X_1);
        hipFree(ghostInterface.h_fGhost.Y_0);
        hipFree(ghostInterface.h_fGhost.Y_1);
        hipFree(ghostInterface.h_fGhost.Z_0);
        hipFree(ghostInterface.h_fGhost.Z_1);
        #ifdef SECOND_DIST 
        hipFree(ghostInterface.g_h_fGhost.X_0);
        hipFree(ghostInterface.g_h_fGhost.X_1);
        hipFree(ghostInterface.g_h_fGhost.Y_0);
        hipFree(ghostInterface.g_h_fGhost.Y_1);
        hipFree(ghostInterface.g_h_fGhost.Z_0);
        hipFree(ghostInterface.g_h_fGhost.Z_1);
        #endif 
    }

    
    #if MEAN_FLOW
        hipFree(m_fMom);
        hipFree(m_rho);
        hipFree(m_ux);
        hipFree(m_uy);
        hipFree(m_uz);
        #ifdef SECOND_DIST
        hipFree(m_c);
        #endif
    #endif //MEAN_FLOW



    #ifdef DENSITY_CORRECTION
        hipFree(d_mean_rho);
        free(h_mean_rho);
    #endif
    #ifdef PARTICLE_TRACER
    hipFree(h_particlePos);
    hipFree(d_particlePos);
    #endif

    #if SAVE_BC
    hipFree(nodeTypeSave);
    #endif


    return 0;



}
