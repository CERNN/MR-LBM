#include "main.cuh"

using namespace std;

int main() {
    checkCudaErrors(hipSetDevice(GPU_INDEX));

    dfloat* fMom;
    ghostInterfaceData ghostInterface;

    unsigned int* dNodeType;
    unsigned int* hNodeType;
    #if SAVE_BC
    dfloat* nodeTypeSave;
    #endif

    #ifdef DENSITY_CORRECTION
    dfloat* h_mean_rho;
    dfloat* d_mean_rho;
    #endif

    #ifdef PARTICLE_TRACER
    dfloat3* h_particlePos;
    dfloat3* d_particlePos;
    #endif

    #if MEAN_FLOW
        dfloat* m_fMom;
        dfloat* m_rho;
        dfloat* m_ux;
        dfloat* m_uy;
        dfloat* m_uz;
        #ifdef SECOND_DIST
        dfloat* m_c;
        #endif
    #endif //MEAN_FLOW

    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        dfloat* h_BC_Fx;
        dfloat* h_BC_Fy;
        dfloat* h_BC_Fz;
        #endif

        dfloat* d_BC_Fx;
        dfloat* d_BC_Fy;
        dfloat* d_BC_Fz;
    #endif //_BC_FORCES


    /* ------------------------- ALLOCATION FOR CPU ------------------------- */
    dfloat* h_fMom;
    dfloat* rho;
    dfloat* ux;
    dfloat* uy;
    dfloat* uz;

    int step = 0;

    #ifdef NON_NEWTONIAN_FLUID
    dfloat* omega;
    #endif

    #ifdef SECOND_DIST
    dfloat* C;
    #endif 


    float** randomNumbers = nullptr; // useful for turbulence

    checkCudaErrors(hipHostMalloc((void**)&(h_fMom), MEM_SIZE_MOM));
    checkCudaErrors(hipHostMalloc((void**)&(rho), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(ux), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uy), MEM_SIZE_SCALAR));
    checkCudaErrors(hipHostMalloc((void**)&(uz), MEM_SIZE_SCALAR));
    #ifdef NON_NEWTONIAN_FLUID
    checkCudaErrors(hipHostMalloc((void**)&(omega), MEM_SIZE_SCALAR));
    #endif
    #ifdef SECOND_DIST
    checkCudaErrors(hipHostMalloc((void**)&(C), MEM_SIZE_SCALAR));
    #endif 
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipHostMalloc((void**)&(h_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif
    #if MEAN_FLOW
        checkCudaErrors(hipHostMalloc((void**)&(m_fMom), MEM_SIZE_MOM));
        checkCudaErrors(hipHostMalloc((void**)&(m_rho), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_ux), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(m_uz), MEM_SIZE_SCALAR));
        #ifdef SECOND_DIST
        checkCudaErrors(hipHostMalloc((void**)&(m_c), MEM_SIZE_SCALAR));
        #endif
    #endif //MEAN_FLOW
    #ifdef BC_FORCES
        #ifdef SAVE_BC_FORCES
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fx), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fy), MEM_SIZE_SCALAR));
        checkCudaErrors(hipHostMalloc((void**)&(h_BC_Fz), MEM_SIZE_SCALAR));
        #endif
    #endif //_BC_FORCES
    randomNumbers = (float**)malloc(sizeof(float*));


    // Setup saving folder
    folderSetup();

    /* -------------- ALLOCATION AND CONFIGURATION FOR EACH GPU ------------- */

    hipMalloc((void**)&fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS);  
    hipMalloc((void**)&dNodeType, sizeof(int) * NUMBER_LBM_NODES);
    interfaceMalloc(ghostInterface);

    #ifdef DENSITY_CORRECTION
        checkCudaErrors(hipHostMalloc((void**)&(h_mean_rho), sizeof(dfloat)));
        hipMalloc((void**)&d_mean_rho, sizeof(dfloat));  
    #endif
    #ifdef PARTICLE_TRACER
    checkCudaErrors(hipMalloc((void**)&(d_particlePos), sizeof(dfloat3)*NUM_PARTICLES));
    #endif

    #ifdef BC_FORCES
        hipMalloc((void**)&d_BC_Fx, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fy, MEM_SIZE_SCALAR);    
        hipMalloc((void**)&d_BC_Fz, MEM_SIZE_SCALAR);            
    #endif //_BC_FORCES
    //printf("Allocated memory \n"); if(console_flush){fflush(stdout);}
    

    hipStream_t streamsLBM[1];
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());
    #ifdef PARTICLE_TRACER
    hipStream_t streamsPart[1];
    checkCudaErrors(hipStreamCreate(&streamsPart[0]));
    #endif

    if(RANDOM_NUMBERS)
    {   
        //printf("Initializing random numbers\n");if(console_flush){fflush(stdout);}
        checkCudaErrors(hipMallocManaged((void**)&randomNumbers[0], 
            sizeof(float)*NUMBER_LBM_NODES));
        initializationRandomNumbers(randomNumbers[0], CURAND_SEED);
        checkCudaErrors(hipDeviceSynchronize());
        getLastCudaError("random numbers transfer error");
        //printf("random numbers initialized \n");if(console_flush){fflush(stdout);}
    }

    /* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
    dim3 threadBlock(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
    dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y, NUM_BLOCK_Z);

    /* ------------------------- LBM INITIALIZATION ------------------------- */
    if(LOAD_CHECKPOINT){
        printf("Loading checkpoint");
        step = INI_STEP;
        loadSimCheckpoint(h_fMom, ghostInterface, &step);

        checkCudaErrors(hipMemcpy(fMom, h_fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyHostToDevice));
        interfaceCudaMemcpy(ghostInterface,ghostInterface.fGhost,ghostInterface.h_fGhost,hipMemcpyHostToDevice,QF);
        #ifdef SECOND_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.g_fGhost,ghostInterface.g_h_fGhost,hipMemcpyHostToDevice,GF);
        #endif 
       

    }else{
        if(LOAD_FIELD){
        }else{
            gpuInitialization_mom << <gridBlock, threadBlock >> >(fMom, randomNumbers[0]);
        }
        //printf("Moments initialized \n");if(console_flush){fflush(stdout);}
        gpuInitialization_pop << <gridBlock, threadBlock >> >(fMom,ghostInterface);
    }

    #if MEAN_FLOW
        //initialize mean moments
        checkCudaErrors(hipMemcpy(m_fMom,fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToDevice));
    #endif //MEAN_FLOW
    checkCudaErrors(hipHostMalloc((void**)&(hNodeType), sizeof(unsigned int) * NUMBER_LBM_NODES));
    #if SAVE_BC
    checkCudaErrors(hipHostMalloc((void**)&(nodeTypeSave), sizeof(dfloat) * NUMBER_LBM_NODES));
    #endif 

    #ifndef VOXEL_FILENAME
    //gpuInitialization_nodeType << <gridBlock, threadBlock >> >(dNodeType);
    //checkCudaErrors(hipDeviceSynchronize());
        hostInitialization_nodeType(hNodeType);
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  
        checkCudaErrors(hipDeviceSynchronize());
    #endif
    #ifdef VOXEL_FILENAME
        hostInitialization_nodeType_bulk(hNodeType); //initialize the domain with  BULK
        read_xyz_file(VOXEL_FILENAME,hNodeType); //overwrite the domain with the voxels information + add missing defintion 
        hostInitialization_nodeType(hNodeType); //initialize the domain with BC
        checkCudaErrors(hipMemcpy(dNodeType, hNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyHostToDevice));  // copy inform\ation to device
        checkCudaErrors(hipDeviceSynchronize());
        define_voxel_bc << <gridBlock, threadBlock >> >(dNodeType); //update information of BC condition nearby the voxels
        checkCudaErrors(hipMemcpy(hNodeType, dNodeType, sizeof(unsigned int) * NUMBER_LBM_NODES, hipMemcpyDeviceToHost)); 
    #endif

    #ifdef BC_FORCES
    gpuInitialization_force << <gridBlock, threadBlock >> >(d_BC_Fx,d_BC_Fy,d_BC_Fz);
    #endif //_BC_FORCES

    //printf("Interface Populations initialized \n");if(console_flush){fflush(stdout);}
    interfaceCudaMemcpy(ghostInterface,ghostInterface.gGhost,ghostInterface.fGhost,hipMemcpyDeviceToDevice,QF);
    #ifdef SECOND_DIST 
    interfaceCudaMemcpy(ghostInterface,ghostInterface.g_gGhost,ghostInterface.g_fGhost,hipMemcpyDeviceToDevice,GF);
    #endif 

    #ifdef DENSITY_CORRECTION
        h_mean_rho[0] = RHO_0;
        checkCudaErrors(hipMemcpy(d_mean_rho, h_mean_rho, sizeof(dfloat), hipMemcpyHostToDevice)); 
    #endif
    checkCudaErrors(hipDeviceSynchronize());

    #ifdef PARTICLE_TRACER
        initializeParticles(h_particlePos,d_particlePos);
    #endif

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    // Free random numbers
    if (RANDOM_NUMBERS) {
        checkCudaErrors(hipSetDevice(GPU_INDEX));
        hipFree(randomNumbers[0]);
        free(randomNumbers);
    }

    checkCudaErrors(hipSetDevice(GPU_INDEX));
    hipEvent_t start, stop, start_step, stop_step;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventCreate(&start_step));
    checkCudaErrors(hipEventCreate(&stop_step));

    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipEventRecord(start_step, 0));
    /* ------------------------------ LBM LOOP ------------------------------ */
    #ifdef DYNAMIC_SHARED_MEMORY
    hipFuncSetAttribute(reinterpret_cast<const void*>(gpuMomCollisionStream), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEMORY_SIZE); // DOESNT WORK: DYNAMICALLY SHARED MEMORY HAS WORSE PERFORMANCE
    #endif
    for (step=INI_STEP; step<N_STEPS;step++){

        int aux = step-INI_STEP;
        bool checkpoint = false;
        #ifdef DENSITY_CORRECTION
        mean_rho(fMom,step,d_mean_rho);
        #endif 
        bool save =false;
        bool reportSave = false;
        bool macrSave = false;
        if(aux != 0){
            if(REPORT_SAVE){
                reportSave = !(step % REPORT_SAVE);
                //reportSave = true;
            }                
            if(MACR_SAVE){
                macrSave = !(step % MACR_SAVE);
                //macrSave = true;
            }
            if(MACR_SAVE || REPORT_SAVE)
                save = (reportSave || macrSave);
            if(CHECKPOINT_SAVE)
                checkpoint = !(aux % CHECKPOINT_SAVE);
        }
       
        gpuMomCollisionStream << <gridBlock, threadBlock 
        #ifdef DYNAMIC_SHARED_MEMORY
        , SHARED_MEMORY_SIZE
        #endif
        >> > (fMom,dNodeType,ghostInterface,
        #ifdef DENSITY_CORRECTION
        d_mean_rho,
        #endif
        #ifdef BC_FORCES
        d_BC_Fx,d_BC_Fy,d_BC_Fz,
        #endif 
        step,
        save); 

        #ifdef PARTICLE_TRACER
            checkCudaErrors(hipDeviceSynchronize());
            updateParticlePos(d_particlePos, h_particlePos, fMom, streamsPart[0],step);
        #endif

        if(checkpoint){
            printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            // throwing a warning for being used without being initialized. But does not matter since we are overwriting it;
            checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
            interfaceCudaMemcpy(ghostInterface,ghostInterface.h_fGhost,ghostInterface.gGhost,hipMemcpyDeviceToHost,QF);       
            #ifdef SECOND_DIST 
            interfaceCudaMemcpy(ghostInterface,ghostInterface.g_h_fGhost,ghostInterface.g_fGhost,hipMemcpyDeviceToHost,GF);
            #endif             
            saveSimCheckpoint(fMom, ghostInterface, &step);
        }
       
        //swap interface pointers
        swapGhostInterfaces(ghostInterface);
        
        //save macroscopics

        //if(save){
            //if (N_STEPS - step < 4*((int)turn_over_time)){
            if(reportSave){
                printf("\n--------------------------- Saving report %06d ---------------------------\n", step);
                #if TREATFIELD
                treatData(h_fMom,fMom,
                #if MEAN_FLOW
                m_fMom,
                #endif //MEAN_FLOW
                step);
                //totalKineticEnergy(fMom,step);
                #endif //TREATFIELD
            
                #if TREATPOINT
                    probeExport(fMom,
                    #ifdef NON_NEWTONIAN_FLUID
                    omega,
                    #endif
                    step);
                #endif
                #if TREATLINE
                velocityProfile(fMom,1,step);
                velocityProfile(fMom,2,step);
                velocityProfile(fMom,3,step);
                velocityProfile(fMom,4,step);
                velocityProfile(fMom,5,step);
                velocityProfile(fMom,6,step);
                #endif
            }
            if(macrSave){
                #if defined BC_FORCES && defined SAVE_BC_FORCES
                checkCudaErrors(hipDeviceSynchronize()); 
                checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
                #endif
                //if (!(step%((int)turn_over_time/10))){
                //if((step>N_STEPS-80*(int)(MACR_SAVE))){ 
                //    if((step%((int)(turn_over_time/2))) == 0){
                        checkCudaErrors(hipDeviceSynchronize()); 
                        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
                        linearMacr(h_fMom,rho,ux,uy,uz,
                        #ifdef NON_NEWTONIAN_FLUID
                        omega,
                        #endif
                        #ifdef SECOND_DIST 
                        C,
                        #endif 
                        #if SAVE_BC
                        nodeTypeSave,
                        hNodeType,
                        #endif
                        #if defined BC_FORCES && defined SAVE_BC_FORCES
                        h_BC_Fx,
                        h_BC_Fy,
                        h_BC_Fz,
                        #endif
                        step); 

                        printf("\n--------------------------- Saving macro %06d ---------------------------\n", step);
                        if(console_flush){fflush(stdout);}
                        //if(step > N_STEPS - 14000){
                        if(!ONLY_FINAL_MACRO){
                        saveMacr(rho,ux,uy,uz,
                        #ifdef NON_NEWTONIAN_FLUID
                        omega,
                        #endif
                        #ifdef SECOND_DIST 
                        C,
                        #endif 
                        #if SAVE_BC
                        nodeTypeSave,
                        #endif
                        #if defined BC_FORCES && defined SAVE_BC_FORCES
                        h_BC_Fx,
                        h_BC_Fy,
                        h_BC_Fz,
                        #endif
                        step);
                    // }
                      //  }
                    //}
                }

                #ifdef BC_FORCES
                    totalBcDrag(d_BC_Fx, d_BC_Fy, d_BC_Fz, step);
                #endif
            }

        //}

    } // end of the loop
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));

    #if defined BC_FORCES && defined SAVE_BC_FORCES
    checkCudaErrors(hipDeviceSynchronize()); 
    checkCudaErrors(hipMemcpy(h_BC_Fx, d_BC_Fx, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fy, d_BC_Fy, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_BC_Fz, d_BC_Fz, MEM_SIZE_SCALAR, hipMemcpyDeviceToHost));
    #endif

    linearMacr(h_fMom,rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    hNodeType,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step); 

    if(console_flush){fflush(stdout);}
    
    saveMacr(rho,ux,uy,uz,
    #ifdef NON_NEWTONIAN_FLUID
    omega,
    #endif
    #ifdef SECOND_DIST 
    C,
    #endif 
    #if SAVE_BC
    nodeTypeSave,
    #endif
    #if defined BC_FORCES && defined SAVE_BC_FORCES
    h_BC_Fx,
    h_BC_Fy,
    h_BC_Fz,
    #endif
    step);

    #ifdef PARTICLE_TRACER
        checkCudaErrors(hipMemcpy(h_particlePos, d_particlePos, sizeof(dfloat3)*NUM_PARTICLES, hipMemcpyDeviceToHost)); 
        saveParticleInfo(h_particlePos,step);
    #endif
    if(CHECKPOINT_SAVE){
        printf("\n--------------------------- Saving checkpoint %06d ---------------------------\n", step);fflush(stdout);
            
        checkCudaErrors(hipMemcpy(h_fMom, fMom, sizeof(dfloat) * NUMBER_LBM_NODES*NUMBER_MOMENTS, hipMemcpyDeviceToHost));
        interfaceCudaMemcpy(ghostInterface,ghostInterface.h_fGhost,ghostInterface.gGhost,hipMemcpyDeviceToHost,QF);    
        #ifdef SECOND_DIST 
        interfaceCudaMemcpy(ghostInterface,ghostInterface.g_h_fGhost,ghostInterface.g_fGhost,hipMemcpyDeviceToHost,GF);
        #endif      
        saveSimCheckpoint(fMom,ghostInterface,&step);
    }
    checkCudaErrors(hipDeviceSynchronize());
    #if MEAN_FLOW
            linearMacr(m_fMom,m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            hNodeType,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX); 

            saveMacr(m_rho,m_ux,m_uy,m_uz,
            #ifdef NON_NEWTONIAN_FLUID
            omega,
            #endif
            #ifdef SECOND_DIST 
            m_c,
            #endif 
            #if SAVE_BC
            nodeTypeSave,
            #endif
            #if defined BC_FORCES && defined SAVE_BC_FORCES
            h_BC_Fx,
            h_BC_Fy,
            h_BC_Fz,
            #endif
            INT_MAX);
    #endif //MEAN_FLOW

    /* ------------------------------ POST ------------------------------ */
    //Calculate MLUPS
    checkCudaErrors(hipSetDevice(GPU_INDEX));
    checkCudaErrors(hipEventRecord(stop_step, 0));
    checkCudaErrors(hipEventSynchronize(stop_step));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&(elapsedTime), start_step, stop_step));
    elapsedTime *= 0.001;
    size_t nodesUpdatedSync = (step) * NUMBER_LBM_NODES;
    dfloat MLUPS = (nodesUpdatedSync / 1e6) / elapsedTime;

    printf("MLUPS: %f\n",MLUPS);

    //save info file
    saveSimInfo(step,MLUPS);


    /* ------------------------------ FREE ------------------------------ */
    hipFree(fMom);
    hipFree(dNodeType);
    hipFree(hNodeType);

    hipFree(h_fMom);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(uz);

    #ifdef SECOND_DIST 
    hipFree(C);
    #endif 

    interfaceFree(ghostInterface);

    #if MEAN_FLOW
        hipFree(m_fMom);
        hipFree(m_rho);
        hipFree(m_ux);
        hipFree(m_uy);
        hipFree(m_uz);
        #ifdef SECOND_DIST
        hipFree(m_c);
        #endif
    #endif //MEAN_FLOW



    #ifdef DENSITY_CORRECTION
        hipFree(d_mean_rho);
        free(h_mean_rho);
    #endif
    #ifdef PARTICLE_TRACER
    hipFree(h_particlePos);
    hipFree(d_particlePos);
    #endif

    #if SAVE_BC
    hipFree(nodeTypeSave);
    #endif
    return 0;
}